#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//						                                                        //
//Copyright (C) 2018 Bosserelle                                                 //
// This code contains an adaptation of the St Venant equation from Basilisk		//
// See																			//
// http://basilisk.fr/src/saint-venant.h and									//
// S. Popinet. Quadtree-adaptive tsunami modelling. Ocean Dynamics,				//
// doi: 61(9) : 1261 - 1285, 2011												//
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////


#include "Poly.h"


/*! \fn int isLeft(T P0x, T P0y, T P1x, T P1y, T P2x, T P2y)
*
* \brief isLeft(): tests if a point is Left|On|Right of an infinite line.
*
* ## Description
* a Point is defined by its coordinates {int x, y;}
* ===================================================================
*
* isLeft(): tests if a point is Left|On|Right of an infinite line.
*	Input:  three points P0, P1, and P2
*	Return: >0 for P2 left of the line through P0 and P1
*			=0 for P2  on the line
*			<0 for P2  right of the line
*	See: Algorithm 1 "Area of Triangles and Polygons"
* 
* ## Where does this come from:
* Copyright 2000 softSurfer, 2012 Dan Sunday
* ### Original Licence
* This code may be freely used and modified for any purpose
* providing that this copyright notice is included with it.
* SoftSurfer makes no warranty for this code, and cannot be held
* liable for any real or imagined damage resulting from its use.
* Users of this code must verify correctness for their application.
* Code modified to fit the use in DisperGPU
*
*/

template <class T> T isLeft(T P0x, T P0y, T P1x, T P1y, T P2x, T P2y)
{
	return ((P1x - P0x) * (P2y - P0y)
		- (P2x - P0x) * (P1y - P0y));
}
//===================================================================

/*! \fn int cn_PnPoly(T Px, T Py, F* Vx, F* Vy, int n)
* \brief cn_PnPoly(): crossing number test for a point in a polygon
*
* ## Description
* cn_PnPoly(): crossing number test for a point in a polygon
*      Input:   P = a point,
*               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
*      Return:  0 = outside, 1 = inside
*
* ## Where does this come from:
* Copyright 2000 softSurfer, 2012 Dan Sunday
* ### Original Licence
* This code may be freely used and modified for any purpose
* providing that this copyright notice is included with it.
* SoftSurfer makes no warranty for this code, and cannot be held
* liable for any real or imagined damage resulting from its use.
* Users of this code must verify correctness for their application.
* Code modified to fit the use in DisperGPU
*
* This code is patterned after [Franklin, 2000]
*/
template <class T, class F> int cn_PnPoly(T Px, T Py, F* Vx, F* Vy, int n)
{
	int    cn = 0;    // the  crossing number counter

	// loop through all edges of the polygon
	for (int i = 0; i < n; i++) {    // edge from V[i]  to V[i+1]
		if (((Vy[i] <= Py) && (Vy[i + 1] > Py))     // an upward crossing
			|| ((Vy[i] > Py) && (Vy[i + 1] <= Py))) { // a downward crossing
			// compute  the actual edge-ray intersect x-coordinate
			T vt = (T)(Py - Vy[i]) / (Vy[i + 1] - Vy[i]);
			if (Px < Vx[i] + vt * (Vx[i + 1] - Vx[i])) // P.x < intersect
				++cn;   // a valid crossing of y=P.y right of P.x
		}
	}
	return (cn & 1);    // 0 if even (out), and 1 if  odd (in)

}
//===================================================================

/*! \fn int wn_PnPoly(T Px, T Py, T* Vx, T* Vy, unsigned int n)
*
* \brief winding number test for a point in a polygon
*
* ## Description
* wn_PnPoly(): winding number test for a point in a polygon
*      Input:   P = a point,
*               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
*      Return:  wn = the winding number (=0 only when P is outside)
*
* ## Where does this come from:
* Copyright 2000 softSurfer, 2012 Dan Sunday
* ### Original Licence
* This code may be freely used and modified for any purpose
* providing that this copyright notice is included with it.
* SoftSurfer makes no warranty for this code, and cannot be held
* liable for any real or imagined damage resulting from its use.
* Users of this code must verify correctness for their application.
* Code modified to fit the use in DisperGPU
*/
template <class T> int wn_PnPoly(T Px, T Py, T* Vx, T* Vy, unsigned int n)
{
	int    wn = 0;    // the  winding number counter

	// loop through all edges of the polygon
	for (int i = 0; i < n; i++) {   // edge from V[i] to  V[i+1]
		if (Vy[i] <= Py) {          // start y <= P.y
			if (Vy[i + 1] > Py)      // an upward crossing
				if (isLeft(Vx[i], Vy[i], Vx[i + 1], Vy[i + 1], Px, Py) > 0)  // P left of  edge
					++wn;            // have  a valid up intersect
		}
		else {                        // start y > P.y (no test needed)
			if (Vy[i + 1] <= Py)     // a downward crossing
				if (isLeft(Vx[i], Vy[i], Vx[i + 1], Vy[i + 1], Px, Py) < 0)  // P right of  edge
					--wn;            // have  a valid down intersect
		}
	}
	return wn;
}

/*! \fn int wn_PnPoly(T Px, T Py, Polygon Poly)
*
* \brief winding number test for a point in a polygon
*
* ## Description
* wn_PnPoly(): winding number test for a point in a polygon
*      Input:   P = a point,
*               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
*      Return:  wn = the winding number (=0 only when P is outside)
*
* ## Where does this come from:
* Copyright 2000 softSurfer, 2012 Dan Sunday
* ### Original Licence
* This code may be freely used and modified for any purpose
* providing that this copyright notice is included with it.
* SoftSurfer makes no warranty for this code, and cannot be held
* liable for any real or imagined damage resulting from its use.
* Users of this code must verify correctness for their application.
* Code modified to fit the use in DisperGPU
*/
template <class T> int wn_PnPoly(T Px, T Py, Polygon Poly)
{
	int    wn = 0;    // the  winding number counter

	// loop through all edges of the polygon
	for (int i = 0; i < (Poly.vertices.size() - 1); i++) {   // edge from V[i] to  V[i+1]
		if (Poly.vertices[i].y <= Py) {          // start y <= P.y
			if (Poly.vertices[i + 1].y > Py)      // an upward crossing
				if (isLeft(T(Poly.vertices[i].x), T(Poly.vertices[i].y), T(Poly.vertices[i + 1].x), T(Poly.vertices[i + 1].y), Px, Py) > 0)  // P left of  edge
					++wn;            // have  a valid up intersect
		}
		else {                        // start y > P.y (no test needed)
			if (Poly.vertices[i + 1].y <= Py)     // a downward crossing
				if (isLeft(T(Poly.vertices[i].x), T(Poly.vertices[i].y), T(Poly.vertices[i + 1].x), T(Poly.vertices[i + 1].y), Px, Py) < 0)  // P right of  edge
					--wn;            // have  a valid down intersect
		}
	}
	return wn;
}
template int wn_PnPoly<float>(float Px, float Py, Polygon Poly);
template int wn_PnPoly<double>(double Px, double Py, Polygon Poly);
//===================================================================


/*! \fn Polygon CounterCWPoly(Polygon Poly)
* 
* \brief check polygon handedness and reverse if necessary. 
* 
* ## Description
* check polygon handedness and enforce left-handesness (Counter-clockwise). 
* This function is used to ensure the right polygon handedness for the winding number inpoly (using the isleft())
*
*/
Polygon CounterCWPoly(Polygon Poly)
{
	double sum = 0.0;
	Polygon Rev;
	

	for (int i = 0; i < (Poly.vertices.size() - 1); i++)
	{
		//
		sum = sum + (Poly.vertices[i + 1].x - Poly.vertices[i].x) * (Poly.vertices[i + 1].y - Poly.vertices[i].y);
	}

	std::string res = sum > 0.0 ? "ClockWise" : "CCW";

	log(" Polygon is " + res );


	// sum<0.0 -> counterclockwise Polygon; sum>0.0 -> clockwise
	if (sum > 0.0)
	{
		log(" Reversing Polygon handedness");
		for (int i = Poly.vertices.size(); i > 0; i--)
		{
			//
			
			
			Rev.vertices.push_back(Poly.vertices[i]);
		}
		Rev.vertices.push_back(Rev.vertices[0]);
		
	}
	return sum > 0.0 ? Rev : Poly;

}

/*! \fn Vertex VertAdd(Vertex A, Vertex B)
* \brief Vertex Add.
*/
Vertex VertAdd(Vertex A, Vertex B)
{
	Vertex v;
	v.x = A.x + B.x; 
	v.y = A.y + B.y;

	return v;
}

/*! \fn Vertex VertSub(Vertex A, Vertex B)
* \brief Vertex Substract
*/
Vertex VertSub(Vertex A, Vertex B)
{
	Vertex v;
	v.x = A.x - B.x;
	v.y = A.y - B.y;

	return v;
}

/*! \fn double dotprod(Vertex A, Vertex B)
* \brief Vertex dot product
*/
double dotprod(Vertex A, Vertex B)
{
	double a = 0.0;
	a = A.x * B.x + A.x + B.y + A.y * B.x + A.y * B.y;
	return a;
}

/*! \fn double xprod(Vertex A, Vertex B)
* \brief Vertex cross-product
*/
double xprod(Vertex A, Vertex B)
{
	double a = 0.0;
	a = A.x*B.y-A.y*B.x;
	return a;
}

/*! \fn bool SegmentIntersect(Polygon P, Polygon Q)
* \brief Intersection between segments
* 
* ## Description
*  Check whether 2 polygon segment intersect. Polygon P and Q are only 2 vertex long each.
* i.e. they represent a segment each.
* 
* ## Where does this come from:
* https://stackoverflow.com/questions/563198/how-do-you-detect-where-two-line-segments-intersect
* Best answer from Gareth Rees
*/
bool SegmentIntersect(Polygon P, Polygon Q)
{
	//
	Vertex r, s, p, q, qmp;
	double rxs, qmpxr, eps, t, u;
	bool intersect = false;

	eps = 1e-9;

	p = P.vertices[0];
	q = Q.vertices[0];
	r = VertSub(P.vertices[1], P.vertices[0]);
	s = VertSub(Q.vertices[1], Q.vertices[0]);
	
	qmp= VertSub(q, p);

	rxs = xprod(r, s);

	qmpxr = xprod(qmp, r);

	


	if (abs(rxs) <= eps && abs(qmpxr) <= eps)
	{
		// colinear
		double t0, t1, rdr, sdr;
		sdr= dotprod(s, r);
		rdr = dotprod(r, r);

		t0 = dotprod(qmp, r) / rdr;
		t1 = t0 + dotprod(s, r) / rdr;

		if (sdr < 0.0)
		{
			intersect = (t0 >= 0.0 && t1 <= 1);
		}
		else
		{
			intersect = (t1 >= 0.0 && t0 <= 1);
		}


	}
	else if (abs(rxs) <= eps && abs(qmpxr) > eps)
	{
		// parallele lines and non intersecting
		intersect = false;
	}
	else if (abs(rxs) > eps)
	{
		t = xprod(qmp, s) / rxs;
		u = qmpxr / rxs;

		if (t >= 0.0 && t <= 1.0 && u <= 1.0 && u >= 0.0)
		{
			intersect = true;
		}

	}
	else
	{
		intersect = false;
	}

		
	return intersect;
}

/*! \fn bool PolygonIntersect(Polygon P, Polygon Q)
* \brief Intersection between 2 polygons
*
* ## Description
*  Check whether 2 polygons intersect. The function checks whether each segment of Polygon P intersect any segment of Poly Q.
* if an intersect is detected theh loops are broken and true is returned.
*
*/
bool PolygonIntersect(Polygon P, Polygon Q)
{
	bool intersect=false;
	for (int i = 0; i < (P.vertices.size() - 1); i++)
	{
		for (int j = 0; j < (Q.vertices.size() - 1); j++)
		{
			// build segments
			Polygon Pseg, Qseg;
			Pseg.vertices = { P.vertices[i], P.vertices[i + 1] };
			Qseg.vertices = { Q.vertices[j], Q.vertices[j + 1] };

			intersect = SegmentIntersect(Pseg, Qseg);

			if (intersect)
			{
				i = P.vertices.size();
				j = Q.vertices.size();
				break;
			}

		}
		
		
	}

	return intersect;

}

/*! \fn bool blockinpoly(T xo, T yo, T dx, int blkwidth, Polygon Poly)
*
* \brief check whether a block is inside or intersectin a polygon
*
* ## Description
* Check whether a block is inside or intersectin a polygon
* 
*
*/
template <class T> bool blockinpoly(T xo, T yo, T dx, int blkwidth, Polygon Poly)
{
	bool insidepoly = false;


	
	

	//bool test = test_wninpoly();


	
	//printf("wn_inpolytest=%s\n", test ? "true" : "false");

	//test = test_intersectpoly();
	//printf("test_intersectpoly=%s\n", test ? "true" : "false");

	//test = test_SegmentIntersect();
	//printf("test_SegmentIntersect=%s\n", test ? "true" : "false");

	// First check if it isinmside the bounding box
	insidepoly = OBBdetect(xo, xo + dx * blkwidth, yo, yo + dx * blkwidth, T(Poly.xmin), T(Poly.xmax), T(Poly.ymin), T(Poly.ymax));

	if (insidepoly)
	{
		//printf("xo=%f, yo=%f, dx=%f, blkwidth=%d\n", xo, yo, dx, blkwidth);
		// being in the bounding box doesn't say much

		// Is there any corner of the block inside the polygon?
		int wnBL,wnBR,wnTL,wnTR;
		insidepoly = false;
		
		wnBL = wn_PnPoly(xo, yo, Poly);
		wnBR = wn_PnPoly(xo + blkwidth*dx, yo, Poly);
		wnTL = wn_PnPoly(xo, yo + blkwidth * dx, Poly);
		wnTR = wn_PnPoly(xo + blkwidth * dx, yo + blkwidth * dx, Poly);

		insidepoly = (wnBL != 0 || wnBR != 0 || wnTL != 0 || wnTR != 0);

		if (!insidepoly)
		{
			// maybe a thin arn of the polygon intersect the block
			Polygon Polyblock;
			Vertex vxBL, vxBR, vxTL, vxTR;
			vxBL.x = xo; vxBL.y = yo;
			vxBR.x = xo + blkwidth * dx; vxBR.y = yo;
			vxTL.x = xo; vxTL.y = yo + blkwidth * dx;
			vxTR.x = xo + blkwidth * dx; vxTR.y = yo + blkwidth * dx;

			Polyblock.vertices.push_back(vxBL);
			Polyblock.vertices.push_back(vxBR);
			Polyblock.vertices.push_back(vxTR);
			Polyblock.vertices.push_back(vxTL);
			Polyblock.vertices.push_back(vxBL);

			insidepoly = PolygonIntersect(Polyblock, Poly);
		}

	}

	return insidepoly;
}
template bool blockinpoly<float>(float xo, float yo, float dx, int blkwidth, Polygon Poly);
template bool blockinpoly<double>(double xo, double yo, double dx, int blkwidth, Polygon Poly);
//template <class T> Poly<T> ReadPoly();

/*! \fn bool test_wninpoly()
*
* \brief Test winding number inpoly function
*
*
*/
bool test_wninpoly()
{
	int in, out;
	bool success = false;
	Polygon Polyblock;
	Vertex vxBL, vxBR, vxTL, vxTR;
	vxBL.x = 0.0; vxBL.y = 0.0;
	vxBR.x = 1.0; vxBR.y = 0.0;
	vxTL.x = 0.0; vxTL.y = 1.0;
	vxTR.x = 1.0; vxTR.y = 1.0;

	Polyblock.vertices.push_back(vxBL);
	Polyblock.vertices.push_back(vxBR);
	Polyblock.vertices.push_back(vxTR);
	Polyblock.vertices.push_back(vxTL);
	Polyblock.vertices.push_back(vxBL);

	in = wn_PnPoly(0.2, 0.3, Polyblock);
	out = wn_PnPoly(1.2, 0.3, Polyblock);

	success = (out == 0 && in != 0);
	return success;
}

/*! \fn bool test_SegmentIntersect()
*
* \brief Test segment intersect function
*
*
*/
bool test_SegmentIntersect()
{
	bool in, out, success;
	Vertex a, b, c, d, e, f;
	Polygon P, Q, R;

	a.x = -1.0; a.y = -1.0;
	b.x = 1.0; b.y = 1.0;

	c.x = -1.0; c.y = 1.0;
	d.x = 1.0; d.y = -1.0;

	double eps = 0.0001;

	e.x = a.x + eps; e.y = a.y ;
	f.x = b.x + eps; f.y = b.y;

	P.vertices.push_back(a);
	P.vertices.push_back(b);

	Q.vertices.push_back(c);
	Q.vertices.push_back(d);

	R.vertices.push_back(e);
	R.vertices.push_back(f);

	in = SegmentIntersect(P, Q);
	out = SegmentIntersect(P, R);
	success = (in && !out);
	return success;
}

/*! \fn bool test_intersectpoly()
*
* \brief Test polygon intersect function
*
*
*/
bool test_intersectpoly()
{
	bool success = false;
	bool in = false;
	bool out = false;
	Polygon Polyblock;

	Polygon PolyTriA, PolyTriB;
	Vertex vxBL, vxBR, vxTL, vxTR, TriA, TriB, TriC;
	vxBL.x = 0.0; vxBL.y = 0.0;
	vxBR.x = 1.0; vxBR.y = 0.0;
	vxTL.x = 0.0; vxTL.y = 1.0;
	vxTR.x = 1.0; vxTR.y = 1.0;

	Polyblock.vertices.push_back(vxBL);
	Polyblock.vertices.push_back(vxBR);
	Polyblock.vertices.push_back(vxTR);
	Polyblock.vertices.push_back(vxTL);
	Polyblock.vertices.push_back(vxBL);

	TriA.x = -1.0; TriA.y = 1.0;

	TriB.x = -1.0; TriB.y = -1.0;

	TriC.x = 0.8; TriC.y = -0.8;

	PolyTriA.vertices.push_back(TriA);
	PolyTriA.vertices.push_back(TriB);
	PolyTriA.vertices.push_back(TriC);
	PolyTriA.vertices.push_back(TriA);

	in = PolygonIntersect(Polyblock, PolyTriA);

	TriA.x = -2.0; TriA.y = 1.0;

	TriB.x = -2.0; TriB.y = -1.0;

	TriC.x = -1.8; TriC.y = -0.8;

	PolyTriB.vertices.push_back(TriA);
	PolyTriB.vertices.push_back(TriB);
	PolyTriB.vertices.push_back(TriC);
	PolyTriB.vertices.push_back(TriA);

	out = PolygonIntersect(Polyblock, PolyTriB);

	success = (in && !out);
	return success;

}



