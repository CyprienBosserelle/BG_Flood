#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//						                                                        //
//Copyright (C) 2018 Bosserelle                                                 //
// This code contains an adaptation of the St Venant equation from Basilisk		//
// See																			//
// http://basilisk.fr/src/saint-venant.h and									//
// S. Popinet. Quadtree-adaptive tsunami modelling. Ocean Dynamics,				//
// doi: 61(9) : 1261 - 1285, 2011												//
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////


#include "Poly.h"

// Copyright 2000 softSurfer, 2012 Dan Sunday
// This code may be freely used and modified for any purpose
// providing that this copyright notice is included with it.
// SoftSurfer makes no warranty for this code, and cannot be held
// liable for any real or imagined damage resulting from its use.
// Users of this code must verify correctness for their application.
// Code modified to fit the use in DisperGPU


// a Point is defined by its coordinates {int x, y;}
//===================================================================


// isLeft(): tests if a point is Left|On|Right of an infinite line.
//    Input:  three points P0, P1, and P2
//    Return: >0 for P2 left of the line through P0 and P1
//            =0 for P2  on the line
//            <0 for P2  right of the line
//    See: Algorithm 1 "Area of Triangles and Polygons"
template <class T> T isLeft(T P0x, T P0y, T P1x, T P1y, T P2x, T P2y)
{
	return ((P1x - P0x) * (P2y - P0y)
		- (P2x - P0x) * (P1y - P0y));
}
//===================================================================


// cn_PnPoly(): crossing number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  0 = outside, 1 = inside
// This code is patterned after [Franklin, 2000]
template <class T, class F> int cn_PnPoly(T Px, T Py, F* Vx, F* Vy, int n)
{
	int    cn = 0;    // the  crossing number counter

	// loop through all edges of the polygon
	for (int i = 0; i < n; i++) {    // edge from V[i]  to V[i+1]
		if (((Vy[i] <= Py) && (Vy[i + 1] > Py))     // an upward crossing
			|| ((Vy[i] > Py) && (Vy[i + 1] <= Py))) { // a downward crossing
			// compute  the actual edge-ray intersect x-coordinate
			T vt = (T)(Py - Vy[i]) / (Vy[i + 1] - Vy[i]);
			if (Px < Vx[i] + vt * (Vx[i + 1] - Vx[i])) // P.x < intersect
				++cn;   // a valid crossing of y=P.y right of P.x
		}
	}
	return (cn & 1);    // 0 if even (out), and 1 if  odd (in)

}
//===================================================================


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
template <class T> int wn_PnPoly(T Px, T Py, T* Vx, T* Vy, unsigned int n)
{
	int    wn = 0;    // the  winding number counter

	// loop through all edges of the polygon
	for (int i = 0; i < n; i++) {   // edge from V[i] to  V[i+1]
		if (Vy[i] <= Py) {          // start y <= P.y
			if (Vy[i + 1] > Py)      // an upward crossing
				if (isLeft(Vx[i], Vy[i], Vx[i + 1], Vy[i + 1], Px, Py) > 0)  // P left of  edge
					++wn;            // have  a valid up intersect
		}
		else {                        // start y > P.y (no test needed)
			if (Vy[i + 1] <= Py)     // a downward crossing
				if (isLeft(Vx[i], Vy[i], Vx[i + 1], Vy[i + 1], Px, Py) < 0)  // P right of  edge
					--wn;            // have  a valid down intersect
		}
	}
	return wn;
}

// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
template <class T> int wn_PnPoly(T Px, T Py, Polygon Poly)
{
	int    wn = 0;    // the  winding number counter

	// loop through all edges of the polygon
	for (int i = 0; i < (Poly.vertices.size() - 1); i++) {   // edge from V[i] to  V[i+1]
		if (Poly.vertices[i].y <= Py) {          // start y <= P.y
			if (Poly.vertices[i + 1].y > Py)      // an upward crossing
				if (isLeft(T(Poly.vertices[i].x), T(Poly.vertices[i].y), T(Poly.vertices[i + 1].x), T(Poly.vertices[i + 1].y), Px, Py) > 0)  // P left of  edge
					++wn;            // have  a valid up intersect
		}
		else {                        // start y > P.y (no test needed)
			if (Poly.vertices[i + 1].y <= Py)     // a downward crossing
				if (isLeft(T(Poly.vertices[i].x), T(Poly.vertices[i].y), T(Poly.vertices[i + 1].x), T(Poly.vertices[i + 1].y), Px, Py) < 0)  // P right of  edge
					--wn;            // have  a valid down intersect
		}
	}
	return wn;
}
template int wn_PnPoly<float>(float Px, float Py, Polygon Poly);
template int wn_PnPoly<double>(double Px, double Py, Polygon Poly);
//===================================================================

Vertex VertAdd(Vertex A, Vertex B)
{
	Vertex v;
	v.x = A.x + B.x; 
	v.y = A.y + B.y;

	return v;
}

Vertex VertSub(Vertex A, Vertex B)
{
	Vertex v;
	v.x = A.x - B.x;
	v.y = A.y - B.y;

	return v;
}

double dotprod(Vertex A, Vertex B)
{
	double a = 0.0;
	a = A.x * B.x + A.x + B.y + A.y * B.x + A.y * B.y;
	return a;
}

double xprod(Vertex A, Vertex B)
{
	double a = 0.0;
	a = A.x*B.y-A.y*B.x;
	return a;
}

//===================================================================
// Intersection between segments
bool SegmentIntersect(Polygon P, Polygon Q)
{
	//
	Vertex r, s, p, q, qmp;
	double rxs, qmpxr, eps, t, u;
	bool intersect = false;

	eps = 1e-9;

	p = P.vertices[0];
	q = Q.vertices[0];
	r = VertSub(P.vertices[1], P.vertices[0]);
	s = VertSub(Q.vertices[1], Q.vertices[0]);
	
	qmp= VertSub(q, p);

	rxs = xprod(r, s);

	qmpxr = xprod(qmp, r);

	


	if (abs(rxs) <= eps && abs(qmpxr) <= eps)
	{
		// colinear
		double t0, t1, rdr, sdr;
		sdr= dotprod(s, r);
		rdr = dotprod(r, r);

		t0 = dotprod(qmp, r) / rdr;
		t1 = t0 + dotprod(s, r) / rdr;

		if (sdr < 0.0)
		{
			intersect = (t0 >= 0.0 && t1 <= 1);
		}
		else
		{
			intersect = (t1 >= 0.0 && t0 <= 1);
		}


	}
	else if (abs(rxs) <= eps && abs(qmpxr) > eps)
	{
		// parallele lines and non intersecting
		intersect = false;
	}
	else if (abs(rxs) > eps)
	{
		t = xprod(qmp, s) / rxs;
		u = qmpxr / rxs;

		if (t >= 0.0 && t <= 1.0 && u <= 1.0 && u >= 0.0)
		{
			intersect = true;
		}

	}
	else
	{
		intersect = false;
	}

		
	return intersect;
}

bool PolygonIntersect(Polygon P, Polygon Q)
{
	bool intersect=false;
	for (int i = 0; i < (P.vertices.size() - 1); i++)
	{
		for (int j = 0; j < (Q.vertices.size() - 1); j++)
		{
			// build segments
			Polygon Pseg, Qseg;
			Pseg.vertices = { P.vertices[i], P.vertices[i + 1] };
			Qseg.vertices = { Q.vertices[i], Q.vertices[i + 1] };

			intersect = SegmentIntersect(Pseg, Qseg);

			if (intersect)
			{
				i = P.vertices.size();
				j = Q.vertices.size();
				break;
			}

		}
		
	}

	return intersect;

}


template <class T> bool blockinpoly(T xo, T yo, T dx, int blkwidth, Polygon Poly)
{
	bool insidepoly = false;
	// First check if it isinmside the bounding box
	insidepoly = OBBdetect(xo, xo + dx * blkwidth, yo, yo + dx * blkwidth, T(Poly.xmin), T(Poly.xmax), T(Poly.ymin), T(Poly.ymax));

	if (insidepoly)
	{
		// being in the bounding box doesn't say much

		// Is there any corner of the block inside the polygon?
		int wnBL,wnBR,wnTL,wnTR;
		insidepoly = false;
		
		wnBL = wn_PnPoly(xo, yo, Poly);
		wnBR = wn_PnPoly(xo + blkwidth*dx, yo, Poly);
		wnTL = wn_PnPoly(xo, yo + blkwidth * dx, Poly);
		wnTR = wn_PnPoly(xo + blkwidth * dx, yo + blkwidth * dx, Poly);

		insidepoly = (wnBL > 0 || wnBR > 0 || wnTL > 0 || wnTR > 0);

		if (!insidepoly)
		{
			// maybe a thin arn of the polygon intersect the block
			Polygon Polyblock;
			Vertex vxBL, vxBR, vxTL, vxTR;
			vxBL.x = xo; vxBL.y = yo;
			vxBR.x = xo + blkwidth * dx; vxBR.y = yo;
			vxTL.x = xo; vxTL.y = yo + blkwidth * dx;
			vxTR.x = xo + blkwidth * dx; vxTR.y = yo + blkwidth * dx;

			Polyblock.vertices.push_back(vxBL);
			Polyblock.vertices.push_back(vxBR);
			Polyblock.vertices.push_back(vxTR);
			Polyblock.vertices.push_back(vxTL);
			Polyblock.vertices.push_back(vxBL);

			insidepoly = PolygonIntersect(Polyblock, Poly);
		}

	}

	return insidepoly;
}
template bool blockinpoly<float>(float xo, float yo, float dx, int blkwidth, Polygon Poly);
template bool blockinpoly<double>(double xo, double yo, double dx, int blkwidth, Polygon Poly);
//template <class T> Poly<T> ReadPoly();


