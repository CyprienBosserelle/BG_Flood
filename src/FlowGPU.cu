#include "hip/hip_runtime.h"
#include "FlowGPU.h"

template <class T> void FlowGPU(Param XParam, Loop<T>& XLoop, Forcing<float> XForcing, Model<T> XModel)
{
	//============================================
	// construct threads abnd block parameters
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	// for flux reconstruction the loop overlap the right(or top for the y direction) halo
	dim3 blockDimKX(XParam.blkwidth + XParam.halowidth, XParam.blkwidth, 1);
	dim3 blockDimKY(XParam.blkwidth, XParam.blkwidth + XParam.halowidth, 1);

	//dim3 blockDimHalo(XParam.blkwidth + XParam.halowidth*2, XParam.blkwidth + XParam.halowidth * 2, 1);
	
	//============================================
	// Build cuda threads for multitasking on the GPU
	for (int i = 0; i < XLoop.num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&XLoop.streams[i]));
	}

	if (XParam.atmpforcing)
	{
		//Update atm press forcing
		AddPatmforcingGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XForcing.Atmp, XModel);
		CUDA_CHECK(hipDeviceSynchronize());

		//Fill atmp halo
		hipStream_t atmpstreams[1];
		CUDA_CHECK(hipStreamCreate(&atmpstreams[0]));
		fillHaloGPU(XParam, XModel.blocks, atmpstreams[0], XModel.Patm);
		CUDA_CHECK(hipDeviceSynchronize());
		hipStreamDestroy(atmpstreams[0]);

		//Calc dpdx and dpdy

		gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel.blocks.active, XModel.blocks.level, (T)XParam.theta, (T)XParam.delta, XModel.Patm, XModel.datmpdx, XModel.datmpdy);

		
		CUDA_CHECK(hipDeviceSynchronize());
		gradientHaloGPU(XParam, XModel.blocks, XModel.Patm, XModel.datmpdx, XModel.datmpdy);
		//
			

		refine_linearGPU(XParam, XModel.blocks, XModel.Patm, XModel.datmpdx, XModel.datmpdy);

		gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel.blocks.active, XModel.blocks.level, (T)XParam.theta, (T)XParam.delta, XModel.Patm, XModel.datmpdx, XModel.datmpdy);
		CUDA_CHECK(hipDeviceSynchronize());

		gradientHaloGPU(XParam, XModel.blocks, XModel.Patm, XModel.datmpdx, XModel.datmpdy);


	}

		
	//============================================
	// Predictor step in reimann solver
	//============================================

	//============================================
	//  Fill the halo for gradient reconstruction
	fillHaloGPU(XParam, XModel.blocks, XModel.evolv, XModel.zb);


	//============================================
	// Reset DTmax
	reset_var <<< gridDim, blockDim, 0 >>> (XParam.halowidth,XModel.blocks.active,XLoop.hugeposval,XModel.time.dtmax);
	CUDA_CHECK(hipDeviceSynchronize());

	//============================================
	// Calculate gradient for evolving parameters for predictor step
	gradientGPUnew(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.zb);
	
	//============================================
	// Synchronise all ongoing streams
	CUDA_CHECK(hipDeviceSynchronize());

	
	
	//============================================
	// Flux and Source term reconstruction
	if (XParam.engine == 1)
	{
		// X- direction
		UpdateButtingerXGPU << < gridDim, blockDimKX, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//updateKurgXGPU <<< gridDim, blockDimKX, 0 >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceXGPU <<< gridDim, blockDimKX, 0, XLoop.streams[0] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
		// Y- direction
		UpdateButtingerYGPU << < gridDim, blockDimKY, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//updateKurgYGPU <<< gridDim, blockDimKY, 0 >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceYGPU <<< gridDim, blockDimKY, 0, XLoop.streams[1] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		// //updateKurgY << < XLoop.gridDim, XLoop.blockDim, 0, XLoop.streams[0] >> > (XParam, XLoop.epsilon, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax);

		CUDA_CHECK(hipDeviceSynchronize());
	}
	else if (XParam.engine == 2)
	{
		// X- direction
		updateKurgXGPU <<< gridDim, blockDimKX, 0 >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceXGPU <<< gridDim, blockDimKX, 0, XLoop.streams[0] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
		// Y- direction
		updateKurgYGPU <<< gridDim, blockDimKY, 0 >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceYGPU <<< gridDim, blockDimKY, 0, XLoop.streams[1] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		// //updateKurgY << < XLoop.gridDim, XLoop.blockDim, 0, XLoop.streams[0] >> > (XParam, XLoop.epsilon, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax);

		CUDA_CHECK(hipDeviceSynchronize());
	}
	else if (XParam.engine == 3)
	{
		// 
		updateKurgXATMGPU << < gridDim, blockDimKX, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdx);
		// //AddSlopeSourceXGPU <<< gridDim, blockDimKX, 0, XLoop.streams[0] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
		// Y- direction
		updateKurgYATMGPU << < gridDim, blockDimKY, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdy);
		// //AddSlopeSourceYGPU <<< gridDim, blockDimKY, 0, XLoop.streams[1] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		// //updateKurgY << < XLoop.gridDim, XLoop.blockDim, 0, XLoop.streams[0] >> > (XParam, XLoop.epsilon, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax);

		CUDA_CHECK(hipDeviceSynchronize());
	}
	//============================================
	// Fill Halo for flux from fine to coarse
	fillHaloGPU(XParam, XModel.blocks, XModel.flux);

	//============================================
	// Reduce minimum timestep
	XLoop.dt = double(CalctimestepGPU(XParam, XLoop, XModel.blocks, XModel.time));
	XLoop.dtmax = XLoop.dt;

	FlowbndFlux(XParam, XLoop.totaltime + XLoop.dt * 0.5, XModel.blocks, XForcing.left, XForcing.Atmp, XModel.evolv, XModel.flux);
	FlowbndFlux(XParam, XLoop.totaltime + XLoop.dt * 0.5, XModel.blocks, XForcing.right, XForcing.Atmp, XModel.evolv, XModel.flux);
	FlowbndFlux(XParam, XLoop.totaltime + XLoop.dt * 0.5, XModel.blocks, XForcing.top, XForcing.Atmp, XModel.evolv, XModel.flux);
	FlowbndFlux(XParam, XLoop.totaltime + XLoop.dt * 0.5, XModel.blocks, XForcing.bot, XForcing.Atmp, XModel.evolv, XModel.flux);


	

	XModel.time.dt = T(XLoop.dt);

	//============================================
	// Update advection terms (dh dhu dhv) 
	updateEVGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.evolv, XModel.flux, XModel.adv);
	CUDA_CHECK(hipDeviceSynchronize());
	
	//============================================
	// Add forcing (Rain, Wind)
	//if (!XForcing.Rain.inputfile.empty())
	//{
	//	AddrainforcingGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XForcing.Rain, XModel.adv);
	//}
	if (!XForcing.UWind.inputfile.empty())//&& !XForcing.UWind.inputfile.empty()
	{
		AddwindforcingGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XForcing.UWind, XForcing.VWind, XModel.adv);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XForcing.rivers.size() > 0)
	{
		AddRiverForcing(XParam, XLoop, XForcing.rivers, XModel);
	}

	//============================================
	//Update evolving variable by 1/2 time step
	AdvkernelGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.time.dt*T(0.5), XModel.zb, XModel.evolv, XModel.adv, XModel.evolv_o);
	CUDA_CHECK(hipDeviceSynchronize());
	
	
	//============================================
	// Corrector step in reimann solver
	//============================================

	//============================================
	//  Fill the halo for gradient reconstruction also wall boundary for masked block
	fillHaloGPU(XParam, XModel.blocks, XModel.evolv_o, XModel.zb);

	//============================================
	// Calculate gradient for evolving parameters
	gradientGPUnew(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.zb);
	CUDA_CHECK(hipDeviceSynchronize());
	
	//============================================
	// Flux and Source term reconstruction
	if (XParam.engine == 1)
	{
		// X- direction
		UpdateButtingerXGPU << < gridDim, blockDimKX, 0 >> > (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//updateKurgXGPU <<< gridDim, blockDimKX, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceXGPU <<< gridDim, blockDimKX, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
		// Y- direction
		UpdateButtingerYGPU << < gridDim, blockDimKY, 0 >> > (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//updateKurgYGPU <<< gridDim, blockDimKY, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceYGPU <<< gridDim, blockDimKY, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	else if (XParam.engine == 2)
	{
		// X- direction
		updateKurgXGPU <<< gridDim, blockDimKX, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceXGPU <<< gridDim, blockDimKX, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
		// Y- direction
		updateKurgYGPU <<< gridDim, blockDimKY, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceYGPU <<< gridDim, blockDimKY, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	else if (XParam.engine == 3)
	{
		//
		//
		updateKurgXATMGPU << < gridDim, blockDimKX, 0 >> > (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdx);
		CUDA_CHECK(hipDeviceSynchronize());

		// Y- direction
		updateKurgYATMGPU << < gridDim, blockDimKY, 0 >> > (XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdy);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	//============================================
	// Fill Halo for flux from fine to coarse
	fillHaloGPU(XParam, XModel.blocks, XModel.flux);

	FlowbndFlux(XParam, XLoop.totaltime + XLoop.dt, XModel.blocks, XForcing.left, XForcing.Atmp, XModel.evolv, XModel.flux);
	FlowbndFlux(XParam, XLoop.totaltime + XLoop.dt, XModel.blocks, XForcing.right, XForcing.Atmp, XModel.evolv, XModel.flux);
	FlowbndFlux(XParam, XLoop.totaltime + XLoop.dt, XModel.blocks, XForcing.top, XForcing.Atmp, XModel.evolv, XModel.flux);
	FlowbndFlux(XParam, XLoop.totaltime + XLoop.dt, XModel.blocks, XForcing.bot, XForcing.Atmp, XModel.evolv, XModel.flux);

	//============================================
	// Update advection terms (dh dhu dhv) 
	updateEVGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.flux, XModel.adv);
	CUDA_CHECK(hipDeviceSynchronize());
	

	//============================================
	// Add forcing (Rain, Wind)
	//if (!XForcing.Rain.inputfile.empty())
	//{
	//	AddrainforcingGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XForcing.Rain, XModel.adv);
	//}
	if (!XForcing.UWind.inputfile.empty())//&& !XForcing.UWind.inputfile.empty()
	{
		AddwindforcingGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XForcing.UWind, XForcing.VWind, XModel.adv);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XForcing.rivers.size() > 0)
	{
		AddRiverForcing(XParam, XLoop, XForcing.rivers, XModel);
	}


	//============================================
	//Update evolving variable by 1 full time step
	AdvkernelGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.time.dt, XModel.zb, XModel.evolv, XModel.adv, XModel.evolv_o);
	CUDA_CHECK(hipDeviceSynchronize());
	
	
	//============================================
	// Add bottom friction

	bottomfrictionGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv_o);
	//XiafrictionGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv, XModel.evolv_o);


	CUDA_CHECK(hipDeviceSynchronize());
	
	//============================================
	//Copy updated evolving variable back
	cleanupGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.evolv_o, XModel.evolv);
	CUDA_CHECK(hipDeviceSynchronize());

	if (!XForcing.Rain.inputfile.empty())
	{
		AddrainforcingImplicitGPU << < gridDim, blockDim, 0 >> > (XParam,XLoop, XModel.blocks, XForcing.Rain, XModel.evolv);
		CUDA_CHECK(hipDeviceSynchronize());
	}

	if (XParam.infiltration)
	{
		AddinfiltrationImplicitGPU << < gridDim, blockDim, 0 >> > (XParam, XLoop, XModel.blocks, XModel.il, XModel.cl, XModel.evolv, XModel.hgw);
		CUDA_CHECK(hipDeviceSynchronize());
	}

	if (XParam.VelThreshold > 0.0)
	{
		TheresholdVelGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv);
		CUDA_CHECK(hipDeviceSynchronize());
	}

	//============================================
	// Reset zb in prolongation halo
	if (XParam.conserveElevation)
	{
		refine_linearGPU(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);
	}
	
	for (int i = 0; i < XLoop.num_streams; i++)
	{
		hipStreamDestroy(XLoop.streams[i]);
	}

}
template void FlowGPU<float>(Param XParam, Loop<float>& XLoop, Forcing<float> XForcing, Model<float> XModel);
template void FlowGPU<double>(Param XParam, Loop<double>& XLoop, Forcing<float> XForcing, Model<double> XModel);

template <class T> void HalfStepGPU(Param XParam, Loop<T>& XLoop, Forcing<float> XForcing, Model<T> XModel)
{
	//============================================
	// construct threads abnd block parameters
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	// for flux reconstruction the loop overlap the right(or top for the y direction) halo
	dim3 blockDimKX(XParam.blkwidth + XParam.halowidth, XParam.blkwidth, 1);
	dim3 blockDimKY(XParam.blkwidth, XParam.blkwidth + XParam.halowidth, 1);

	//dim3 blockDimHalo(XParam.blkwidth + XParam.halowidth*2, XParam.blkwidth + XParam.halowidth * 2, 1);

	//============================================
	// Build cuda threads for multitasking on the GPU
	for (int i = 0; i < XLoop.num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&XLoop.streams[i]));
	}

	if (XParam.atmpforcing)
	{
		//Update atm press forcing
		AddPatmforcingGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XForcing.Atmp, XModel);
		CUDA_CHECK(hipDeviceSynchronize());

		//Fill atmp halo
		hipStream_t atmpstreams[1];
		CUDA_CHECK(hipStreamCreate(&atmpstreams[0]));
		fillHaloGPU(XParam, XModel.blocks, atmpstreams[0], XModel.Patm);
		CUDA_CHECK(hipDeviceSynchronize());
		hipStreamDestroy(atmpstreams[0]);

		//Calc dpdx and dpdy
		gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel.blocks.active, XModel.blocks.level, (T)XParam.theta, (T)XParam.delta, XModel.Patm, XModel.datmpdx, XModel.datmpdy);
		CUDA_CHECK(hipDeviceSynchronize());
		gradientHaloGPU(XParam, XModel.blocks, XModel.Patm, XModel.datmpdx, XModel.datmpdy);
		//


		refine_linearGPU(XParam, XModel.blocks, XModel.Patm, XModel.datmpdx, XModel.datmpdy);

		gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel.blocks.active, XModel.blocks.level, (T)XParam.theta, (T)XParam.delta, XModel.Patm, XModel.datmpdx, XModel.datmpdy);
		CUDA_CHECK(hipDeviceSynchronize());

		gradientHaloGPU(XParam, XModel.blocks, XModel.Patm, XModel.datmpdx, XModel.datmpdy);


	}


	//============================================
	// Predictor step in reimann solver
	//============================================

	//============================================
	//  Fill the halo for gradient reconstruction
	fillHaloGPU(XParam, XModel.blocks, XModel.evolv, XModel.zb);


	//============================================
	// Reset DTmax
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel.blocks.active, XLoop.hugeposval, XModel.time.dtmax);
	CUDA_CHECK(hipDeviceSynchronize());

	//============================================
	// Calculate gradient for evolving parameters for predictor step
	gradientGPUnew(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.zb);

	//============================================
	// Synchronise all ongoing streams
	CUDA_CHECK(hipDeviceSynchronize());



	//============================================
	// Flux and Source term reconstruction
	if (XParam.engine == 1)
	{
		// X- direction
		UpdateButtingerXGPU << < gridDim, blockDimKX, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//updateKurgXGPU <<< gridDim, blockDimKX, 0 >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceXGPU <<< gridDim, blockDimKX, 0, XLoop.streams[0] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
		// Y- direction
		UpdateButtingerYGPU << < gridDim, blockDimKY, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//updateKurgYGPU <<< gridDim, blockDimKY, 0 >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceYGPU <<< gridDim, blockDimKY, 0, XLoop.streams[1] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		// //updateKurgY << < XLoop.gridDim, XLoop.blockDim, 0, XLoop.streams[0] >> > (XParam, XLoop.epsilon, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax);

		CUDA_CHECK(hipDeviceSynchronize());
	}
	else if (XParam.engine == 2)
	{
		// X- direction
		updateKurgXGPU << < gridDim, blockDimKX, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceXGPU <<< gridDim, blockDimKX, 0, XLoop.streams[0] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
		// Y- direction
		updateKurgYGPU << < gridDim, blockDimKY, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		// //AddSlopeSourceYGPU <<< gridDim, blockDimKY, 0, XLoop.streams[1] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		// //updateKurgY << < XLoop.gridDim, XLoop.blockDim, 0, XLoop.streams[0] >> > (XParam, XLoop.epsilon, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax);

		CUDA_CHECK(hipDeviceSynchronize());
	}
	else if (XParam.engine == 3)
	{
		// 
		updateKurgXATMGPU << < gridDim, blockDimKX, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdx);
		// //AddSlopeSourceXGPU <<< gridDim, blockDimKX, 0, XLoop.streams[0] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		CUDA_CHECK(hipDeviceSynchronize());
		// Y- direction
		updateKurgYATMGPU << < gridDim, blockDimKY, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdy);
		// //AddSlopeSourceYGPU <<< gridDim, blockDimKY, 0, XLoop.streams[1] >>> (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
		// //updateKurgY << < XLoop.gridDim, XLoop.blockDim, 0, XLoop.streams[0] >> > (XParam, XLoop.epsilon, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax);

		CUDA_CHECK(hipDeviceSynchronize());
	}
	//============================================
	// Fill Halo for flux from fine to coarse
	fillHaloGPU(XParam, XModel.blocks, XModel.flux);

	//============================================
	// Reduce minimum timestep
	XLoop.dt = double(CalctimestepGPU(XParam, XLoop, XModel.blocks, XModel.time));
	XLoop.dtmax = XLoop.dt;


	XModel.time.dt = T(XLoop.dt);

	//============================================
	// Update advection terms (dh dhu dhv) 
	updateEVGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.flux, XModel.adv);
	CUDA_CHECK(hipDeviceSynchronize());

	//============================================
	// Add forcing (Rain, Wind)
	//if (!XForcing.Rain.inputfile.empty())
	//{
	//	AddrainforcingGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XForcing.Rain, XModel.adv);
	//}
	if (!XForcing.UWind.inputfile.empty())//&& !XForcing.UWind.inputfile.empty()
	{
		AddwindforcingGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XForcing.UWind, XForcing.VWind, XModel.adv);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XForcing.rivers.size() > 0)
	{
		AddRiverForcing(XParam, XLoop, XForcing.rivers, XModel);
	}

	//============================================
	//Update evolving variable by 1/2 time step
	AdvkernelGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.time.dt * T(0.5), XModel.zb, XModel.evolv, XModel.adv, XModel.evolv_o);
	CUDA_CHECK(hipDeviceSynchronize());

	//============================================
	// Add bottom friction

	bottomfrictionGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv_o);
	//XiafrictionGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv, XModel.evolv_o);


	CUDA_CHECK(hipDeviceSynchronize());

	//============================================
	//Copy updated evolving variable back
	cleanupGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv_o, XModel.evolv);
	CUDA_CHECK(hipDeviceSynchronize());

	if (!XForcing.Rain.inputfile.empty())
	{
		AddrainforcingImplicitGPU << < gridDim, blockDim, 0 >> > (XParam, XLoop, XModel.blocks, XForcing.Rain, XModel.evolv);
		CUDA_CHECK(hipDeviceSynchronize());
	}

	if (XParam.VelThreshold > 0.0)
	{
		TheresholdVelGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv);
		CUDA_CHECK(hipDeviceSynchronize());
	}

	//============================================
	// Reset zb in prolongation halo
	if (XParam.conserveElevation)
	{
		refine_linearGPU(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);
	}

	for (int i = 0; i < XLoop.num_streams; i++)
	{
		hipStreamDestroy(XLoop.streams[i]);
	}

}
template void HalfStepGPU<float>(Param XParam, Loop<float>& XLoop, Forcing<float> XForcing, Model<float> XModel);
template void HalfStepGPU<double>(Param XParam, Loop<double>& XLoop, Forcing<float> XForcing, Model<double> XModel);



template <class T> __global__ void reset_var(int halowidth, int* active, T resetval, T* Var)
{

	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int n = memloc(halowidth, blkmemwidth, ix, iy, ib);
	//int n= (ix + halowidth) + (iy + halowidth) * blkmemwidth + ib * blksize;
	Var[n] = resetval;
}
template __global__ void reset_var<float>(int halowidth, int* active, float resetval, float* Var);
template __global__ void reset_var<double>(int halowidth, int* active, double resetval, double* Var);


