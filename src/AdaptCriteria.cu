#include "hip/hip_runtime.h"
﻿


#include "AdaptCriteria.h"


template <class T> int AdaptCriteria(Param XParam, Forcing<float> XForcing, Model<T> XModel)
{
	int success = 0;
	if (XParam.AdatpCrit.compare("Threshold") == 0)
	{
		success = Thresholdcriteria(XParam, T(std::stod(XParam.Adapt_arg1)), XModel.OutputVarMap[XParam.Adapt_arg2], XModel.blocks, XModel.adapt.refine, XModel.adapt.coarsen);
	}
	if (XParam.AdatpCrit.compare("Inrange") == 0)
	{
		success = inrangecriteria(XParam, T(std::stod(XParam.Adapt_arg1)), T(std::stod(XParam.Adapt_arg2)), XModel.OutputVarMap[XParam.Adapt_arg3], XModel.blocks, XModel.adapt.refine, XModel.adapt.coarsen);
	}
	if (XParam.AdatpCrit.compare("Targetlevel") == 0)
	{
		for (int ig = 0; ig < XForcing.targetadapt.size(); ig++)
		{
			targetlevelcriteria(XParam, XForcing.targetadapt[ig], XModel.blocks, XModel.adapt.refine, XModel.adapt.coarsen);
		}
	}
	return success;
}
template int AdaptCriteria<float>(Param XParam, Forcing<float> XForcing, Model<float> XModel);
template int AdaptCriteria<double>(Param XParam, Forcing<float> XForcing, Model<double> XModel);


/*! \fn int Thresholdcriteria(Param XParam,T threshold, T* z, BlockP<T> XBlock,  bool*& refine, bool*& coarsen)
* Threshold criteria is a general form of wet dry criteria
* Simple wet/.dry refining criteria.
* if the block is wet -> refine is true
* if the block is dry -> coarsen is true
* beware the refinement sanity check is meant to be done after running this function
*/
template <class T> int Thresholdcriteria(Param XParam,T threshold, T* z, BlockP<T> XBlock, bool* refine, bool* coarsen)
{
	// Threshold criteria is a general form of wet dry criteria where esp is the threshold and h is the parameter tested
	// Below is written as a wet dry analogy where wet is vlaue above threshold and dry is below

	
	int success = 0;
	//int i;

	//Coarsen dry blocks and refine wet ones
	//CPU version


	// To start we assume all values are below the threshold
	bool iswet = false;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		refine[ib] = false; // only refine if all are wet
		coarsen[ib] = true; // always try to coarsen
		iswet = false;
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{

				int i = memloc(XParam, ix, iy, ib);
				//(ix + XParam.halowidth) + (iy + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				
				if (z[i] > threshold)
				{
					iswet = true;
				}
			}
		}


		refine[ib] = iswet;
		coarsen[ib] = !iswet;

		//printf("ib=%d; refibe[ib]=%s\n", ib, iswet ? "true" : "false");
	}
	return success;
}
template  int Thresholdcriteria<float>(Param XParam, float threshold, float* z, BlockP<float> XBlock, bool* refine, bool* coarsen);
template  int Thresholdcriteria<double>(Param XParam, double threshold, double* z, BlockP<double> XBlock, bool* refine, bool* coarsen);

/*! \fn int inrangecriteria(Param XParam, T zmin, T zmax, T* z, BlockP<T> XBlock, bool*& refine, bool*& coarsen)
* Simple in-range refining criteria.
* if any value of z (could be any variable) is zmin <= z <= zmax the block will try to refine
* otherwise, the block will try to coarsen
* beware the refinement sanity check is meant to be done after running this function
*/
template<class T>
int inrangecriteria(Param XParam, T zmin, T zmax, T* z, BlockP<T> XBlock, bool* refine, bool* coarsen)
{
	// First use a simple refining criteria: zb>zmin && zb<zmax refine otherwise corasen
	int success = 0;
	//int i;


	// To start 
	bool isinrange = false;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		refine[ib] = false; // only refine if zb is in range
		coarsen[ib] = true; // always try to coarsen otherwise
		isinrange = false;
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = (ix + XParam.halowidth) + (iy + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				if (z[i] >= zmin && z[i] <= zmax)
				{
					isinrange = true;
				}
			}
		}


		refine[ib] = isinrange;
		coarsen[ib] = !isinrange;

		//printf("ib=%d; refibe[ib]=%s\n", ib, iswet ? "true" : "false");
	}
	return success;
}
template int inrangecriteria<float>(Param XParam, float zmin, float zmax, float* z, BlockP<float> XBlock, bool* refine, bool* coarsen);
template int inrangecriteria<double>(Param XParam, double zmin, double zmax, double* z, BlockP<double> XBlock, bool* refine, bool* coarsen);

/*! \fn 
*/
template<class T>
int targetlevelcriteria(Param XParam, StaticForcingP<int> targetlevelmap, BlockP<T> XBlock, bool* refine, bool* coarsen)
{
	int targetlevel;
	bool uplevel, samelevel;
	T delta, x, y;
	int success = 0;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];

		delta = calcres(XParam.dx, XBlock.level[ib]);

		uplevel = false;
		samelevel = false;

		refine[ib] = false; // only refine if all are wet
		coarsen[ib] = true; // always try to coarsen
		
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				x = XParam.xo + XBlock.xo[ib] + T(ix) * delta;
				y = XParam.yo + XBlock.yo[ib] + T(iy) * delta;

				targetlevel = int(round(interp2BUQ(x, y, targetlevelmap)));

				if (targetlevel > XBlock.level[ib])
				{
					//printf("x=%f; y=%f; target=%d; level=%d", x, y, targetlevel, XBlock.level[ib]);
					uplevel = true;
					
				}
				if (targetlevel == XBlock.level[ib])
				{
					samelevel = true;

				}


			}
		}

		if (uplevel)
		{
			refine[ib] = true; // only refine if all are wet
			coarsen[ib] = false;
		}
		else if (samelevel)
		{
			coarsen[ib] = false;
		}

	}
	return success;
}
template int targetlevelcriteria<float>(Param XParam, StaticForcingP<int> targetlevelmap, BlockP<float> XBlock, bool* refine, bool* coarsen);
template int targetlevelcriteria<double>(Param XParam, StaticForcingP<int> targetlevelmap, BlockP<double> XBlock, bool* refine, bool* coarsen);
