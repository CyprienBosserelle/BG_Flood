#include "hip/hip_runtime.h"
﻿
#include "Testing.h"




/*! \fn bool testing(Param XParam, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
* Wrapping function for all the inbuilt test
* This function is the entry point to other function below.
*
* Test 0 is a gausian hump propagating on a flat uniorm cartesian mesh (both GPU and CPU version tested)
* Test 1 is vertical discharge on a flat uniorm cartesian mesh (GPU or CPU version)
* Test 2 Gaussian wave on Cartesian grid (same as test 0): CPU vs GPU (GPU required)
* Test 3 Test Reduction algorithm
* Test 4 Boundary condition test
* Test 5 Lake at rest test for Ardusse/kurganov reconstruction/scheme
* Test 6 Mass conservation on a slope
* Test 7 Mass conservation with rain fall on grid
* Test 8 Rain Map forcing (comparison map and Time Serie and test case with slope and non-uniform rain map)
* Test 9 Zoned output (test zoned outputs with adaptative grid)

* Test 99 Run all the test with test number < 99.

The following test are not independant, they are tools to check or debug a personnal case
* Test 998 Compare resuts between the CPU and GPU Flow functions (GPU required)
* Test 999 Run the main loop and engine in debug mode
*/
template <class T> bool Testing(Param XParam, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
{

	bool isfailed = false;
	std::string result;

	log("\nRunning internal test(s):");

	int mytest;
	mytest = XParam.test;
	if (XParam.test == 99)
	{
		mytest = 0;
	}

	while (mytest <= XParam.test)
	{
		if (mytest == 0)
		{
			bool bumptest, bumptestComp;
			bool bumptestGPU = true;
			// Test 0 is pure bump test
			log("\t ### Gaussian wave on Cartesian grid ###");
			//set gpu is -1 for cpu test

			bumptest = GaussianHumptest(0.1, -1, false);
			result = bumptest ? "successful" : "failed";
			log("\t\tCPU test: " + result);

			// If original XParam tried to use GPU we try also
			if (XParam.GPUDEVICE >= 0)
			{
				bumptestGPU = GaussianHumptest(0.1, XParam.GPUDEVICE, false);
				result = bumptestGPU ? "successful" : "failed";
				log("\t\tGPU test: " + result);

				if (!bumptestGPU)
				{
					bumptestComp = GaussianHumptest(0.1, XParam.GPUDEVICE, true);
				}
			}
			isfailed = ((bumptest == true) && (bumptestGPU == true)) ? false : true;
		}
		if (mytest == 1)
		{
			bool rivertest;
			// Test 1 is vertical discharge on a flat uniorm cartesian mesh (GPU and CU version)
			log("\t ### River Mass conservation grid ###");
			rivertest = Rivertest(0.1, -1);
			result = rivertest ? "successful" : "failed";
			log("\t\tCPU test: " + result);
			isfailed = (!rivertest || isfailed) ? true : false;

			log(" \t\t\t GPU device= " + XParam.GPUDEVICE);

			if (XParam.GPUDEVICE >= 0)
			{
				rivertest = Rivertest(0.1, XParam.GPUDEVICE);
				result = rivertest ? "successful" : "failed";
				log("\t\tGPU test: " + result);
				isfailed = (!rivertest || isfailed) ? true : false;
			}

			rivertest=RiverVolumeAdapt(XParam, T(0.4));
			result = rivertest ? "successful" : "failed";
			log("\t\tRiver Volume Adapt: " + result);
			isfailed = (!rivertest || isfailed) ? true : false;

		}
		if (mytest == 2)
		{
			if (XParam.GPUDEVICE >= 0)
			{
				bool GPUvsCPUtest;
				log("\t### Gaussian wave on Cartesian grid: CPU vs GPU ###");
				GPUvsCPUtest = GaussianHumptest(0.1, XParam.GPUDEVICE, true);
				result = GPUvsCPUtest ? "successful" : "failed";
				log("\t\tCPU vs GPU test: " + result);
				isfailed = (!GPUvsCPUtest || isfailed) ? true : false;
			}
			else
			{
				log("Specify GPU device to run test 2 (CPU vs GPU comparison)");
			}
		}
		if (mytest == 3)
		{

			bool testresults;
			bool testreduction = true;

			// Iterate this test niter times:
			int niter = 1000;
			srand(time(0));
			log("\t### Reduction Test ###");
			for (int iter = 0; iter < niter; iter++)
			{
				testresults = reductiontest(XParam, XModel, XModel_g);
				testreduction = testreduction && testresults;
			}

			result = testreduction ? "successful" : "failed";
			log("\t\tReduction test: " + result);
			isfailed = (!testreduction || isfailed) ? true : false;

		}
		if (mytest == 4)
		{
			log("\t### Boundary Test ###");
			bool testBound = testboundaries(XParam, T(0.1));
			result = testBound ? "successful" : "failed";
			isfailed = (!testBound || isfailed) ? true : false;
			log("\t\tboundaries test: " + result);
		}
		if (mytest == 5)
		{
			log("\t### Lake-at-rest Test ###");
			bool testTLAR = ThackerLakeAtRest(XParam, T(0.0));
			result = testTLAR ? "successful" : "failed";
			isfailed = (!testTLAR || isfailed) ? true : false;
			log("\t\tThaker lake-at-rest test: " + result);
			testTLAR = LakeAtRest(XParam, XModel);
			isfailed = (!testTLAR || isfailed) ? true : false;
			log("\t\tLake-at-rest test: " + result);
		}
		if (mytest == 6)
		{
			log("\t### Mass conservation Test ###");
			bool testSteepSlope = MassConserveSteepSlope(XParam.zsinit, XParam.GPUDEVICE);
			result = testSteepSlope ? "successful" : "failed";
			isfailed = (!testSteepSlope || isfailed) ? true : false;
			log("\t\tMass conservation test: " + result);
		}
		if (mytest == 7)
		{
			bool testrainGPU, testrainCPU;
			/* Test 7 is homogeneous rain on a uniform slope for cartesian mesh (GPU and CU version)
			 The input parameters are :
					- the initial water level (zs)
					- GPU option
					- the slope (%)
			*/
			log("\t### Homogeneous rain on grid Mass conservation test ###");
			testrainGPU = Raintest(0.0, 0, 10);
			result = testrainGPU ? "successful" : "failed";
			log("\t\tHomogeneous rain on grid test GPU: " + result);
			testrainCPU = Raintest(0.0, -1, 10);
			result = testrainCPU ? "successful" : "failed";
			log("\t\tHomogeneous rain on grid test CPU: " + result);
			isfailed = (!testrainCPU || !testrainGPU || isfailed) ? true : false;
		}
		if (XParam.test == 8)
		{
			bool raintest2;
			/* Test 8 is non-homogeneous rain on a non-uniform slope for cartesian mesh (GPU and CPU version)
			 It is based on a teste case from litterature (Iwagaki1955) and tests the different
			 rain inputs (time serie for 1D input or netCDF file).
			*/

			log("\t non-uniform rain forcing on slope based on Aureli2020");
			int gpu = 0;
			raintest2 = Raintestinput(gpu);
			result = raintest2 ? "successful" : "failed";
			log("\t\tNon-uniform rain forcing : " + result);
		}
		if (mytest == 9)
		{
			bool testzoneOutDef, testzoneOutUser;
			/* Test 9 is basic configuration to test the zoned outputs, with different resolutions.
			 The default (without zoned defined by user) configuration is tested.
			 Then, the creation of 3 zones is then tested(whole, zoned complexe, zoned with part of the levels).
			 The size of the created nc files is used to verified this fonctionnality.
			 Parameter: nbzones: number of zones for output defined by the user
						zsinit: initial water elevation
			*/

			log("\t### Test zoned output ###");
			int nbzones = 0;
			T zsinit = 0.01;
			testzoneOutDef = ZoneOutputTest(nbzones, zsinit);
			result = testzoneOutDef ? "successful" : "failed";
			log("\n\nDefault zoned Outputs: " + result);
			nbzones = 3; // 3 only
			testzoneOutUser = ZoneOutputTest(nbzones, zsinit);
			result = testzoneOutUser ? "successful" : "failed";
			log("\n\nUser defined zones Outputs: " + result);
			isfailed = (!testzoneOutDef || !testzoneOutUser || isfailed) ? true : false;
		}
		if (mytest == 10)
		{
			bool instab;
			log("\t### Wet/dry Instability test with Conserve Elevation ###");
			instab=TestInstability(XParam, XModel, XModel_g);
			result = instab ? "successful" : "failed";
			log("\t\tWet/dry Instability test : " + result);
		}

		if (mytest == 995)
		{
			TestFirsthalfstep(XParam, XForcing, XModel, XModel_g);
		}
		if (mytest == 996)
		{
			TestHaloSpeed(XParam,XModel,XModel_g);
		}
		if (mytest == 997)
		{
			TestGradientSpeed(XParam, XModel, XModel_g);
		}
		if (mytest == 998)
		{
			//
			bool testresults;
			log("\t### CPU vs GPU Test ###");
			testresults = CPUGPUtest(XParam, XModel, XModel_g);
			isfailed = (!testresults || isfailed) ? true : false;

			if (testresults)
			{
				exit(0);
			}
			else
			{
				exit(1);
			}
		}
		if (XParam.test == 999)
		{
			//
			DebugLoop(XParam, XForcing, XModel, XModel_g);
		}
		mytest++;
	}
	return(isfailed);
}
template bool Testing<float>(Param XParam, Forcing<float> XForcing, Model<float> XModel, Model<float> XModel_g);
template bool Testing<double>(Param XParam, Forcing<float> XForcing, Model<double> XModel, Model<double> XModel_g);


/*! \fn bool GaussianHumptest(T zsnit, int gpu, bool compare)
*
* This function tests the full hydrodynamics model and compares the results with pre-conmputed (Hard wired) values
*	The function creates it own model setup and mesh independantly to what the user might want to do
*	The setup consist of a centrally located gaussian hump radiating away
*	The test stops at an arbitrary time to compare with 8 values extracted from a identical run in basilisk
*	This function also compares the result of the GPU and CPU code (until they diverge)
*/
template <class T> bool GaussianHumptest(T zsnit, int gpu, bool compare)
{
	log("#####");
	// this is a preplica of the tutorial case for Basilisk
	Param XParam;

	T x, y, delta;
	T cc = T(0.05);// Match the 200 in chracteristic radius used in Basilisk  1/(2*cc^2)=200


	T a = T(1.0); //Gaussian wave amplitude

	// Verification data
	// This is a transect across iy=15:16:127 at ix=127 (or vice versa because the solution is symetrical)
	// These values are based on single precision output from Netcdf file so are only accurate to 10-7 
	//double ZsVerifKurganov[8] = { 0.100000000023, 0.100000063119, 0.100110376004, 0.195039970749, 0.136739044168, 0.0848024805994, 0.066275833049, 0.0637058445888 };
	//double ZsVerification[8] = { 0.100000008904, 0.187920326216, 0.152329657390, 0.117710230042, 0.0828616638138, 0.0483274739972, 0.0321501737555, 0.0307609731288 };
	double ZsVerifButtinger[8] = { 0.100000000023, 0.100000063119, 0.100093580546, 0.195088199869, 0.136767978925, 0.0850706353898, 0.0663028448129, 0.063727949607 };




	// initialise domain and required resolution
	XParam.dx = 1.0 / ((1 << 8));
	XParam.xo = -0.5;
	XParam.yo = -0.5;

	XParam.xmax = 0.5;
	XParam.ymax = 0.5;
	//level 8 is 


	XParam.initlevel = 0;
	XParam.minlevel = 0;
	XParam.maxlevel = 0;

	XParam.zsinit = zsnit;
	XParam.zsoffset = 0.0;

	//Output times for comparisons
	XParam.endtime = 1.0;
	XParam.outputtimestep = 0.1;

	XParam.smallnc = 0;

	XParam.cf = 0.0;
	XParam.frictionmodel = 0;

	// Enforece GPU/CPU
	XParam.GPUDEVICE = gpu;

	std::string outvi[18] = { "zb","h","zs","u","v","Fqux","Fqvx","Fquy","Fqvy", "Fhu", "Fhv", "dh", "dhu", "dhv", "ho", "vo", "uo", "cf" };

	std::vector<std::string> outv;

	for (int nv = 0; nv < 18; nv++)
	{
		outv.push_back(outvi[nv]);
	}

	XParam.outvars = outv;

	// create Model setup
	Model<T> XModel;
	Model<T> XModel_g;

	Forcing<float> XForcing;
	StaticForcingP<float> bathy;

	XForcing.Bathy.push_back(bathy);
	// initialise forcing bathymetry to 0
	XForcing.Bathy[0].xo = -1.0;
	XForcing.Bathy[0].yo = -1.0;

	XForcing.Bathy[0].xmax = 1.0;
	XForcing.Bathy[0].ymax = 1.0;
	XForcing.Bathy[0].nx = 3;
	XForcing.Bathy[0].ny = 3;

	XForcing.Bathy[0].dx = 1.0;

	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.Bathy[0].val);

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = 0.0f;
		}
	}

	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	// Recreate the initia;l conditions
	//InitArrayBUQ(XParam, XModel.blocks, T(0.0), XModel.zb);
	//InitArrayBUQ(XParam, XModel.blocks, zsnit, XModel.evolv.zs);
	//zs is initialised here:
	InitialConditions(XParam, XForcing, XModel);

	T xorigin = T(0.0);
	T yorigin = T(0.0);


	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XModel.blocks.active[ibl];
		delta = T(calcres(XParam.dx, XModel.blocks.level[ib]));


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				x = T(XParam.xo) + XModel.blocks.xo[ib] + ix * delta;
				y = T(XParam.yo) + XModel.blocks.yo[ib] + iy * delta;
				XModel.evolv.zs[n] = XModel.evolv.zs[n] + a * exp(T(-1.0) * ((x - xorigin) * (x - xorigin) + (y - yorigin) * (y - yorigin)) / (T(2.0) * cc * cc));
				XModel.evolv.h[n] = utils::max(XModel.evolv.zs[n] - XModel.zb[n], T(0.0));

			}
		}
	}

	SetupGPU(XParam, XModel, XForcing, XModel_g);

	Loop<T> XLoop;
	Loop<T> XLoop_g;


	XLoop.hugenegval = std::numeric_limits<T>::min();
	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();
	XLoop.totaltime = 0.0;

	//InitSave2Netcdf(XParam, XModel);
	XLoop.nextoutputtime = XParam.outputtimestep;
	XLoop.dtmax = initdt(XParam, XLoop, XModel);

	//XLoop_g = XLoop;
	XLoop_g.hugenegval = std::numeric_limits<T>::min();
	XLoop_g.hugeposval = std::numeric_limits<T>::max();
	XLoop_g.epsilon = std::numeric_limits<T>::epsilon();
	XLoop_g.totaltime = 0.0;

	//InitSave2Netcdf(XParam, XModel);
	XLoop_g.nextoutputtime = XParam.outputtimestep;
	XLoop_g.dtmax = XLoop.dtmax;


	if (XParam.GPUDEVICE >= 0 && compare)
	{
		CompareCPUvsGPU(XParam, XModel, XModel_g, outv, false);
	}
	bool modelgood = true;

	fillHaloC(XParam, XModel.blocks, XModel.zb);
	gradientC(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);
	gradientHalo(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);

	refine_linear(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);
	gradientHalo(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);

	while (XLoop.totaltime < XLoop.nextoutputtime)
	{

		if (XParam.GPUDEVICE >= 0)
		{
			FlowGPU(XParam, XLoop_g, XForcing, XModel_g);
			XLoop.dt = XLoop_g.dt;
		}
		else
		{
			FlowCPU(XParam, XLoop, XForcing, XModel);
		}
		if (XParam.GPUDEVICE >= 0 && compare)
		{
			FlowCPU(XParam, XLoop, XForcing, XModel);

			T diffdt = T(XLoop_g.dt - XLoop.dt);
			if (abs(diffdt) > T(100.0) * (XLoop.epsilon))
			{
				printf("Timestep Difference=%f\n", diffdt);

				compare = false;
			}
			CompareCPUvsGPU(XParam, XModel, XModel_g, outv, false);
		}

		//diffdh(XParam, XModel.blocks, XModel.flux.Su, diff, shuffle);
		//diffSource(XParam, XModel.blocks, XModel.flux.Fqux, XModel.flux.Su, diff);
		XLoop.totaltime = XLoop.totaltime + XLoop.dt;
		XLoop_g.totaltime = XLoop_g.totaltime + XLoop_g.dt;
		if (XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001) && XParam.outputtimestep > 0.0)
		{
			if (XParam.GPUDEVICE >= 0)
			{
				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					CUDA_CHECK(hipMemcpy(XModel.OutputVarMap[XParam.outvars[ivar]], XModel_g.OutputVarMap[XParam.outvars[ivar]], XParam.nblkmem * XParam.blksize * sizeof(T), hipMemcpyDeviceToHost));
				}
			}

			//Save2Netcdf(XParam, XModel);
			// Verify the Validity of results


			double diff;
			for (int iv = 0; iv < 8; iv++)
			{

				int ix, iy, ib, ii, jj, ibx, iby, nbx;
				jj = 127;
				ii = (iv + 1) * 16 - 1;

				// Theoretical size is 255x255
				nbx = 256 / 16;
				

				ibx = ftoi(floor(ii / XParam.blkwidth));
				iby = ftoi(floor(jj / XParam.blkwidth));

				ib = (iby)*nbx + ibx;

				ix = ii - ibx * XParam.blkwidth;
				iy = jj - iby * XParam.blkwidth;

				int n = memloc(XParam, ix, iy, ib);

				diff = abs(T(XModel.evolv.zs[n]) - ZsVerifButtinger[iv]);



				if (diff > 1e-6)//Tolerance is 1e-6 or 1e-7/1e-8??
				{

					printf("ib=%d, ix=%d, iy=%d; simulated=%f; expected=%f; diff=%e\n", ib, ix, iy, XModel.evolv.zs[n], ZsVerifButtinger[iv], diff);
					modelgood = false;
				}



			}



			//XLoop.nextoutputtime = min(XLoop.nextoutputtime + XParam.outputtimestep, XParam.endtime);

		}
	}
	log("#####");
	return modelgood;
}
template bool GaussianHumptest<float>(float zsnit, int gpu, bool compare);
template bool GaussianHumptest<double>(double zsnit, int gpu, bool compare);

/*! \fn bool Rivertest(T zsnit, int gpu)
*
* This function tests the mass conservation of the vertical injection (used for rivers)
*	The function creates it own model setup and mesh independantly to what the user might want to do
*	This starts with a initial water level (zsnit=0 is dry) and runs for 0.1s before comparing results
*	with zsnit=0.1 that is approx 20 steps
*/
template <class T> bool Rivertest(T zsnit, int gpu)
{
	log("#####");
	Param XParam;
	T delta = 0;
	T initVol = 0;
	T finalVol = 0;
	T TheoryInput = 0;

	// initialise domain and required resolution
	XParam.dx = 1.0 / ((1 << 4));
	XParam.xo = -0.5;
	XParam.yo = -0.5;

	XParam.xmax = 0.5;
	XParam.ymax = 0.5;
	//level 8 is 


	XParam.initlevel = 0;
	XParam.minlevel = 0;
	XParam.maxlevel = 0;

	XParam.zsinit = zsnit;
	XParam.zsoffset = 0.0;

	//Output times for comparisons
	XParam.endtime = 1.0;
	XParam.outputtimestep = 0.1;

	XParam.smallnc = 0;

	XParam.cf = 0.0;
	XParam.frictionmodel = 0;

	// Enforece GPU/CPU
	XParam.GPUDEVICE = gpu;

	std::vector<std::string> outv = { "zb","h","zs","u","v","Fqux","Fqvx","Fquy","Fqvy", "Fhu", "Fhv", "dh", "dhu", "dhv", "Su", "Sv","dhdx", "dhdy", "dudx", "dvdx", "dzsdx", "twet", "hUmax", "Umean"};
	XParam.outvars = outv;

	XParam.outmax = true;
	XParam.outmean = true;
	XParam.outtwet = true;

	// create Model setup
	Model<T> XModel;
	Model<T> XModel_g;

	Forcing<float> XForcing;

	StaticForcingP<float> bathy;

	XForcing.Bathy.push_back(bathy);

	// initialise forcing bathymetry to 0
	XForcing.Bathy[0].xo = -1.0;
	XForcing.Bathy[0].yo = -1.0;

	XForcing.Bathy[0].xmax = 1.0;
	XForcing.Bathy[0].ymax = 1.0;
	XForcing.Bathy[0].nx = 3;
	XForcing.Bathy[0].ny = 3;

	XForcing.Bathy[0].dx = 1.0;

	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.Bathy[0].val);

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = 0.0f;
		}
	}
	//
	//
	// 
	T Q = T(0.001);
	TheoryInput = Q * T(XParam.outputtimestep);


	//Create a temporary file with river fluxes
	std::ofstream river_file(
		"testriver.tmp", std::ios_base::out | std::ios_base::trunc);
	river_file << "0.0 " + std::to_string(Q) << std::endl;
	river_file << "3600.0 " + std::to_string(Q) << std::endl;
	river_file.close(); //destructor implicitly does it

	River thisriver;
	thisriver.Riverflowfile = "testriver.tmp";
	thisriver.xstart = -1.0 * XParam.dx * 3.0;
	thisriver.xend = XParam.dx * 3.0;
	thisriver.ystart = -1.0 * XParam.dx * 3.0;
	thisriver.yend = XParam.dx * 3.0;

	XForcing.rivers.push_back(thisriver);


	XForcing.rivers[0].flowinput = readFlowfile(XForcing.rivers[0].Riverflowfile);


	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);
	InitialAdaptation(XParam, XForcing, XModel);

	SetupGPU(XParam, XModel, XForcing, XModel_g);

	Loop<T> XLoop;

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	//InitSave2Netcdf(XParam, XModel);
	XLoop.nextoutputtime = XParam.outputtimestep;
	XLoop.dtmax = initdt(XParam, XLoop, XModel);
	initVol = T(0.0);

	fillHaloC(XParam, XModel.blocks, XModel.zb);

	// Calculate initial water volume
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XModel.blocks.active[ibl];
		delta = calcres(T(XParam.dx), XModel.blocks.level[ib]);


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				//printf("h[%d] = %f\n", n, XModel.evolv.h[n]);
				initVol = initVol + XModel.evolv.h[n] * delta * delta;
			}
		}
	}


	//InitSave2Netcdf(XParam, XModel);
	bool modelgood = true;

	while (XLoop.totaltime < XLoop.nextoutputtime)
	{

		if (XParam.GPUDEVICE >= 0)
		{
			FlowGPU(XParam, XLoop, XForcing, XModel_g);
		}
		else
		{
			printf("h[1] = %f\n", XModel.evolv.h[1]);
			FlowCPU(XParam, XLoop, XForcing, XModel);
		}
		XLoop.totaltime = XLoop.totaltime + XLoop.dt;
		//Save2Netcdf(XParam, XLoop, XModel);

		if (XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001) && XParam.outputtimestep > 0.0)
		{
			if (XParam.GPUDEVICE >= 0)
			{
				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					CUDA_CHECK(hipMemcpy(XModel.OutputVarMap[XParam.outvars[ivar]], XModel_g.OutputVarMap[XParam.outvars[ivar]], XParam.nblkmem * XParam.blksize * sizeof(T), hipMemcpyDeviceToHost));
				}
			}

			//Save2Netcdf(XParam, XLoop, XModel);
			// Verify the Validity of results
			finalVol = T(0.0);
			for (int ibl = 0; ibl < XParam.nblk; ibl++)
			{
				//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
				int ib = XModel.blocks.active[ibl];
				delta = calcres(T(XParam.dx), XModel.blocks.level[ib]);


				for (int iy = 0; iy < XParam.blkwidth; iy++)
				{
					for (int ix = 0; ix < XParam.blkwidth; ix++)
					{
						//
						int n = memloc(XParam, ix, iy, ib);
						//printf("h[%d] = %f\n", n, XModel.evolv.h[n]);
						finalVol = finalVol + XModel.evolv.h[n] * delta * delta;
					}
				}
			}
			T error = ((finalVol - initVol) - TheoryInput) / TheoryInput;
			printf("error = %g %%, initial volume=%4.4f; final Volume=%4.4f; abs. difference=%4.4f, Theoretical  input=%4.4f \n", error, initVol, finalVol, abs(finalVol - initVol), TheoryInput);


			modelgood = abs(error) < 0.05;
		}



	}

	if (!modelgood)
	{
		InitSave2Netcdf(XParam, XModel);

	}


	log("#####");
	return modelgood;
}
template bool Rivertest<float>(float zsnit, int gpu);
template bool Rivertest<double>(double zsnit, int gpu);



/*! \fn bool MassConserveSteepSlope(T zsnit, int gpu)
*
* This function tests the mass conservation of the vertical injection (used for rivers)
*	The function creates it own model setup and mesh independantly to what the user might want to do
*	This starts with a initial water level (zsnit=0 is dry) and runs for 0.1s before comparing results
*	with zsnit=0.1 that is approx 20 steps
*/
template <class T> bool MassConserveSteepSlope(T zsnit, int gpu)
{
	log("#####");
	Param XParam;
	T delta, initVol, finalVol, TheoryInput;
	// initialise domain and required resolution
	XParam.dx = 1.0 / ((1 << 8));
	XParam.xo = -0.5;
	XParam.yo = -0.5;

	XParam.xmax = 0.5;
	XParam.ymax = 0.5;
	//level 8 is 


	XParam.initlevel = 0;
	XParam.minlevel = -1;
	XParam.maxlevel = 1;

	XParam.AdaptCrit = "Threshold";
	XParam.Adapt_arg1 = "3.5";
	XParam.Adapt_arg2 = "zb";

	XParam.zsinit = zsnit;
	XParam.zsoffset = 0.0;

	//Output times for comparisons
	XParam.endtime = 1.0;
	XParam.outputtimestep = 0.04;//0.035;

	XParam.smallnc = 0;

	XParam.cf = 0.001;
	XParam.frictionmodel = 1;

	XParam.conserveElevation = false;

	// Enforece GPU/CPU
	XParam.GPUDEVICE = gpu;
	std::vector<std::string> outv = { "zb","h","zs","u","v","Fqux","Fqvx","Fquy","Fqvy", "Fhu", "Fhv", "dh", "dhu", "dhv", "Su", "Sv","dhdx", "dhdy" };


	XParam.outvars = outv;
	// create Model setup
	Model<T> XModel;
	Model<T> XModel_g;

	Forcing<float> XForcing;

	StaticForcingP<float> bathy;

	XForcing.Bathy.push_back(bathy);

	// initialise forcing bathymetry to 0
	XForcing.Bathy[0].xo = -1.0;
	XForcing.Bathy[0].yo = -1.0;

	XForcing.Bathy[0].xmax = 1.0;
	XForcing.Bathy[0].ymax = 1.0;
	XForcing.Bathy[0].nx = 3;
	XForcing.Bathy[0].ny = 3;

	XForcing.Bathy[0].dx = 1.0;

	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.Bathy[0].val);

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = T(i * 4);
		}
	}
	//
	//
	// 
	T Q = T(0.10);
	TheoryInput = Q * T(XParam.outputtimestep);


	//Create a temporary file with river fluxes
	std::ofstream river_file(
		"testriver.tmp", std::ios_base::out | std::ios_base::trunc);
	river_file << "0.0 " + std::to_string(Q) << std::endl;
	river_file << "3600.0 " + std::to_string(Q) << std::endl;
	river_file.close(); //destructor implicitly does it

	River thisriver;
	thisriver.Riverflowfile = "testriver.tmp";
	thisriver.xstart = -1.0 * XParam.dx * 3.0;
	thisriver.xend = XParam.dx * 3.0;
	thisriver.ystart = -1.0 * XParam.dx * 3.0;
	thisriver.yend = XParam.dx * 3.0;

	XForcing.rivers.push_back(thisriver);


	XForcing.rivers[0].flowinput = readFlowfile(XForcing.rivers[0].Riverflowfile);


	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);

	SetupGPU(XParam, XModel, XForcing, XModel_g);

	Loop<T> XLoop;

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;



	InitSave2Netcdf(XParam, XModel);
	XLoop.nextoutputtime = XParam.outputtimestep;
	XLoop.dtmax = 0.025;// initdt(XParam, XLoop, XModel);




	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XModel.blocks.active[ibl];
		//delta = calcres(XParam.dx, XModel.blocks.level[ib]);


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				if (XModel.zb[n] < XParam.eps)
				{
					printf("ix=%d, iy=%d, ib=%d, n=%d; zb=%f \n", ix, iy, ib, n, XModel.zb[n]);
				}
			}
		}
	}

	if (XParam.GPUDEVICE >= 0)
	{
		hipStream_t stream;
		CUDA_CHECK(hipStreamCreate(&stream));

		fillHaloGPU(XParam, XModel_g.blocks, stream, XModel_g.zb);

		hipStreamDestroy(stream);
	}
	else
	{
		fillHaloC(XParam, XModel.blocks, XModel.zb);
	}

	initVol = T(0.0);
	// Calculate initial water volume
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XModel.blocks.active[ibl];
		delta = calcres(T(XParam.dx), XModel.blocks.level[ib]);


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				initVol = initVol + XModel.evolv.h[n] * delta * delta;
			}
		}
	}


	//InitSave2Netcdf(XParam, XModel);+



	bool modelgood = true;

	while (XLoop.totaltime < XLoop.nextoutputtime)
	{

		if (XParam.GPUDEVICE >= 0)
		{
			FlowGPU(XParam, XLoop, XForcing, XModel_g);
		}
		else
		{
			FlowCPU(XParam, XLoop, XForcing, XModel);
		}
		XLoop.totaltime = XLoop.totaltime + XLoop.dt;
		//Save2Netcdf(XParam, XLoop, XModel);

		if (XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001) && XParam.outputtimestep > 0.0)
		{
			if (XParam.GPUDEVICE >= 0)
			{
				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					CUDA_CHECK(hipMemcpy(XModel.OutputVarMap[XParam.outvars[ivar]], XModel_g.OutputVarMap[XParam.outvars[ivar]], XParam.nblkmem * XParam.blksize * sizeof(T), hipMemcpyDeviceToHost));
				}
			}

			Save2Netcdf(XParam, XLoop, XModel);
			// Verify the Validity of results
			finalVol = T(0.0);
			for (int ibl = 0; ibl < XParam.nblk; ibl++)
			{
				//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
				int ib = XModel.blocks.active[ibl];
				delta = calcres(T(XParam.dx), XModel.blocks.level[ib]);


				for (int iy = 0; iy < XParam.blkwidth; iy++)
				{
					for (int ix = 0; ix < XParam.blkwidth; ix++)
					{
						//
						int n = memloc(XParam, ix, iy, ib);
						finalVol = finalVol + XModel.evolv.h[n] * delta * delta;
					}
				}
			}
			T error = (finalVol - initVol) - TheoryInput;

			modelgood = error / TheoryInput < 0.05;
		}


	}
	log("#####");
	return modelgood;
}
template bool MassConserveSteepSlope<float>(float zsnit, int gpu);
template bool MassConserveSteepSlope<double>(double zsnit, int gpu);


/*! \fn T reductiontest(Param XParam, Model<T> XModel, Model<T> XModel_g)
*
*	Test the algorithm for reducing the global time step on the user grid layout
*/
template <class T> bool reductiontest(Param XParam, Model<T> XModel, Model<T> XModel_g)
{
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	//srand(seed);
	T mininput = T(rand()) / T(RAND_MAX);
	bool test = true;

	Loop<T> XLoop;

	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	//InitSave2Netcdf(XParam, XModel);
	XLoop.nextoutputtime = mininput * T(2.0);
	XLoop.dtmax = mininput * T(2.01);

	// Fill in dtmax with random values that are larger than  mininput
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				XModel.time.dtmax[n] = mininput * T(1.1) + utils::max(T(rand()) / T(RAND_MAX), T(0.0));
			}
		}
	}

	// randomly select a block a i and a j were the maximum value will be relocated
	int ibbl = ftoi(floor(T(rand()) / T(RAND_MAX) * XParam.nblk));
	int ibb = XModel.blocks.active[ibbl];
	int ixx = ftoi(floor(T(rand()) / T(RAND_MAX) * XParam.blkwidth));
	int iyy = ftoi(floor(T(rand()) / T(RAND_MAX) * XParam.blkwidth));

	int nn = memloc(XParam, ixx, iyy, ibb);

	XModel.time.dtmax[nn] = mininput;

	T reducedt = CalctimestepCPU(XParam, XLoop, XModel.blocks, XModel.time);

	test = abs(reducedt - mininput) < T(100.0) * (XLoop.epsilon);
	bool testgpu;

	if (!test)
	{
		char buffer[256]; sprintf(buffer, "%e", abs(reducedt - mininput));
		std::string str(buffer);
		log("\t\t CPU test failed! : Expected=" + std::to_string(mininput) + ";  Reduced=" + std::to_string(reducedt)+ ";  error=" +str);
	}

	if (XParam.GPUDEVICE >= 0)
	{

		reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel_g.blocks.active, XLoop.hugeposval, XModel_g.time.dtmax);
		CUDA_CHECK(hipDeviceSynchronize());

		CopytoGPU(XParam.nblkmem, XParam.blksize, XModel.time.dtmax, XModel_g.time.dtmax);
		T reducedtgpu = CalctimestepGPU(XParam, XLoop, XModel_g.blocks, XModel_g.time);
		testgpu = abs(reducedtgpu - mininput) < T(100.0) * (XLoop.epsilon);

		if (!testgpu)
		{
			char buffer[256]; sprintf(buffer, "%e", abs(reducedtgpu - mininput));
			std::string str(buffer);
			log("\t\t GPU test failed! : Expected=" + std::to_string(mininput) + ";  Reduced=" + std::to_string(reducedtgpu) + ";  error=" + str);
		}

		if (abs(reducedtgpu - reducedt) > T(100.0) * (XLoop.epsilon))
		{
			char buffer[256]; sprintf(buffer, "%e", abs(reducedtgpu - reducedt));
			std::string str(buffer);
			log("\t\t CPU vs GPU test failed! : Expected=" + std::to_string(reducedt) + ";  Reduced=" + std::to_string(reducedtgpu) + ";  error=" + str);
		}

		test = test && testgpu;
	}


	return test;
}
template bool reductiontest<float>(Param XParam, Model<float> XModel, Model<float> XModel_g);
template bool reductiontest<double>(Param XParam, Model<double> XModel, Model<double> XModel_g);

/*! \fn CPUGPUtest(Param XParam, Model<float> XModel, Model<float> XModel_g)
*	Perform a series of test between the CPU and GPU Flow functions
*	This test only occurs if a valid GPU is specified by user
*/
template<class T> bool CPUGPUtest(Param XParam, Model<T> XModel, Model<T> XModel_g)
{
	bool test = true;

	T initdepth = T(0.1);
	T testamp = T(1.0);

	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	// for flux reconstruction the loop overlap the right(or top for the y direction) halo
	dim3 blockDimKX(XParam.blkwidth + XParam.halowidth, XParam.blkwidth, 1);
	dim3 blockDimKY(XParam.blkwidth, XParam.blkwidth + XParam.halowidth, 1);

	InitArrayBUQ(XParam, XModel.blocks, T(0.0), XModel.zb);
	InitArrayBUQ(XParam, XModel.blocks, T(initdepth), XModel.evolv.zs);
	InitArrayBUQ(XParam, XModel.blocks, T(initdepth), XModel.evolv.h);
	InitArrayBUQ(XParam, XModel.blocks, T(0.0), XModel.evolv.u);
	InitArrayBUQ(XParam, XModel.blocks, T(0.0), XModel.evolv.v);


	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel_g.blocks.active, T(0.0), XModel_g.zb);
	CUDA_CHECK(hipDeviceSynchronize());
	// Create some usefull vectors
	std::string evolvst[4] = { "h","zs","u","v" };

	std::vector<std::string> evolvVar;

	for (int nv = 0; nv < 4; nv++)
	{
		evolvVar.push_back(evolvst[nv]);
	}


	// Check fillhalo function

	// fill with all evolv array with random value
	/*
	fillrandom(XParam, XModel.blocks, XModel.evolv.zs);
	fillrandom(XParam, XModel.blocks, XModel.evolv.h);
	fillrandom(XParam, XModel.blocks, XModel.evolv.u);
	fillrandom(XParam, XModel.blocks, XModel.evolv.v);
	*/
	fillgauss(XParam, XModel.blocks, testamp, XModel.evolv.zs);
	fillgauss(XParam, XModel.blocks, testamp, XModel.evolv.h);
	fillgauss(XParam, XModel.blocks, T(0.5 * testamp), XModel.evolv.u);
	fillgauss(XParam, XModel.blocks, T(0.5 * testamp), XModel.evolv.v);

	//copy to GPU
	CopytoGPU(XParam.nblkmem, XParam.blksize, XModel.evolv, XModel_g.evolv);

	//============================================
	//  Fill the halo for gradient reconstruction
	fillHalo(XParam, XModel.blocks, XModel.evolv, XModel.zb);
	fillHaloGPU(XParam, XModel_g.blocks, XModel_g.evolv, XModel_g.zb);

	CompareCPUvsGPU(XParam, XModel, XModel_g, evolvVar, true);

	//============================================
	//perform gradient reconstruction
	//gradientCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.zb);
	//gradientGPU(XParam, XModel_g.blocks, XModel_g.evolv, XModel_g.grad, XModel.zb);


	// CPU gradients
	std::thread t0(&gradientC<T>, XParam, XModel.blocks, XModel.evolv.h, XModel.grad.dhdx, XModel.grad.dhdy);
	std::thread t1(&gradientC<T>, XParam, XModel.blocks, XModel.evolv.zs, XModel.grad.dzsdx, XModel.grad.dzsdy);
	std::thread t2(&gradientC<T>, XParam, XModel.blocks, XModel.evolv.u, XModel.grad.dudx, XModel.grad.dudy);
	std::thread t3(&gradientC<T>, XParam, XModel.blocks, XModel.evolv.v, XModel.grad.dvdx, XModel.grad.dvdy);

	t0.join();
	t1.join();
	t2.join();
	t3.join();

	//GPU gradients

	gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel_g.blocks.active, XModel_g.blocks.level, (T)XParam.theta, (T)XParam.dx, XModel_g.evolv.h, XModel_g.grad.dhdx, XModel_g.grad.dhdy);
	CUDA_CHECK(hipDeviceSynchronize());

	gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel_g.blocks.active, XModel_g.blocks.level, (T)XParam.theta, (T)XParam.dx, XModel_g.evolv.zs, XModel_g.grad.dzsdx, XModel_g.grad.dzsdy);
	CUDA_CHECK(hipDeviceSynchronize());
	gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel_g.blocks.active, XModel_g.blocks.level, (T)XParam.theta, (T)XParam.dx, XModel_g.evolv.u, XModel_g.grad.dudx, XModel_g.grad.dudy);
	CUDA_CHECK(hipDeviceSynchronize());

	gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel_g.blocks.active, XModel_g.blocks.level, (T)XParam.theta, (T)XParam.dx, XModel_g.evolv.v, XModel_g.grad.dvdx, XModel_g.grad.dvdy);
	CUDA_CHECK(hipDeviceSynchronize());

	std::string gradst[8] = { "dhdx","dzsdx","dudx","dvdx","dhdy","dzsdy","dudy","dvdy" };

	std::vector<std::string> gradVar;

	for (int nv = 0; nv < 8; nv++)
	{
		gradVar.push_back(gradst[nv]);
	}

	CompareCPUvsGPU(XParam, XModel, XModel_g, gradVar, false);

	// Gradient in Halo

	// CPU
	gradientHalo(XParam, XModel.blocks, XModel.evolv.h, XModel.grad.dhdx, XModel.grad.dhdy);
	gradientHalo(XParam, XModel.blocks, XModel.evolv.zs, XModel.grad.dzsdx, XModel.grad.dzsdy);
	gradientHalo(XParam, XModel.blocks, XModel.evolv.u, XModel.grad.dudx, XModel.grad.dudy);
	gradientHalo(XParam, XModel.blocks, XModel.evolv.v, XModel.grad.dvdx, XModel.grad.dvdy);

	// GPU
	gradientHaloGPU(XParam, XModel_g.blocks, XModel_g.evolv.h, XModel_g.grad.dhdx, XModel_g.grad.dhdy);
	gradientHaloGPU(XParam, XModel_g.blocks, XModel_g.evolv.zs, XModel_g.grad.dzsdx, XModel_g.grad.dzsdy);
	gradientHaloGPU(XParam, XModel_g.blocks, XModel_g.evolv.u, XModel_g.grad.dudx, XModel_g.grad.dudy);
	gradientHaloGPU(XParam, XModel_g.blocks, XModel_g.evolv.v, XModel_g.grad.dvdx, XModel_g.grad.dvdy);

	CompareCPUvsGPU(XParam, XModel, XModel_g, gradVar, true);

	//============================================
	// Kurganov scheme

	std::string fluxst[8] = { "Fhu","Su","Fqux","Fqvx","Fhv","Sv","Fqvy","Fquy" };

	std::vector<std::string> fluxVar;

	for (int nv = 0; nv < 8; nv++)
	{
		fluxVar.push_back(fluxst[nv]);
	}

	updateKurgXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);

	//GPU part
	updateKurgXGPU << < gridDim, blockDimKX, 0 >> > (XParam, XModel_g.blocks, XModel_g.evolv, XModel_g.grad, XModel_g.flux, XModel_g.time.dtmax, XModel_g.zb);
	CUDA_CHECK(hipDeviceSynchronize());


	// Y- direction
	updateKurgYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);

	updateKurgYGPU << < gridDim, blockDimKY, 0 >> > (XParam, XModel_g.blocks, XModel_g.evolv, XModel_g.grad, XModel_g.flux, XModel_g.time.dtmax, XModel_g.zb);
	CUDA_CHECK(hipDeviceSynchronize());

	CompareCPUvsGPU(XParam, XModel, XModel_g, fluxVar, false);


	fillHalo(XParam, XModel.blocks, XModel.flux);
	fillHaloGPU(XParam, XModel_g.blocks, XModel_g.flux);

	CompareCPUvsGPU(XParam, XModel, XModel_g, fluxVar, true);


	//============================================
	// Update step
	std::string advst[3] = { "dh","dhu","dhv" };

	std::vector<std::string> advVar;

	for (int nv = 0; nv < 3; nv++)
	{
		advVar.push_back(advst[nv]);
	}
	updateEVCPU(XParam, XModel.blocks, XModel.evolv, XModel.flux, XModel.adv);
	updateEVGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evolv, XModel_g.flux, XModel_g.adv);
	CUDA_CHECK(hipDeviceSynchronize());

	CompareCPUvsGPU(XParam, XModel, XModel_g, advVar, false);

	//============================================
	// Advance step
	std::string evost[4] = { "zso","ho","uo","vo" };

	std::vector<std::string> evoVar;

	for (int nv = 0; nv < 4; nv++)
	{
		evoVar.push_back(evost[nv]);
	}
	AdvkernelCPU(XParam, XModel.blocks, T(0.0005), XModel.zb, XModel.evolv, XModel.adv, XModel.evolv_o);
	AdvkernelGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, T(0.0005), XModel_g.zb, XModel_g.evolv, XModel_g.adv, XModel_g.evolv_o);
	CUDA_CHECK(hipDeviceSynchronize());

	CompareCPUvsGPU(XParam, XModel, XModel_g, evoVar, false);

	//============================================
	// Bottom friction

	bottomfrictionCPU(XParam, XModel.blocks, T(0.5), XModel.cf, XModel.evolv_o);

	bottomfrictionGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, T(0.5), XModel_g.cf, XModel_g.evolv_o);
	CUDA_CHECK(hipDeviceSynchronize());

	CompareCPUvsGPU(XParam, XModel, XModel_g, evoVar, false);


	//============================================
	// Repeat the full test
	Loop<T> XLoop;
	Loop<T> XLoop_g;

	XParam.endtime = utils::min(0.5 * (XParam.ymax - XParam.yo), 0.5 * (XParam.xmax - XParam.xo)) / (sqrt(XParam.g * (testamp + initdepth)));
	XParam.outputtimestep = XParam.endtime / 10.0;


	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	//InitSave2Netcdf(XParam, XModel);
	XLoop.nextoutputtime = XParam.outputtimestep;
	XLoop.dtmax = initdt(XParam, XLoop, XModel);

	XLoop_g.hugenegval = std::numeric_limits<T>::min();

	XLoop_g.hugeposval = std::numeric_limits<T>::max();
	XLoop_g.epsilon = std::numeric_limits<T>::epsilon();

	XLoop_g.totaltime = 0.0;

	//InitSave2Netcdf(XParam, XModel);
	XLoop_g.nextoutputtime = XLoop.nextoutputtime;
	XLoop_g.dtmax = XLoop.dtmax;

	std::string outvi[18] = { "zb","h","zs","u","v","Fqux","Fqvx","Fquy","Fqvy", "Fhu", "Fhv", "dh", "dhu", "dhv", "ho", "vo", "uo", "cf" };

	std::vector<std::string> outv;

	for (int nv = 0; nv < 18; nv++)
	{
		outv.push_back(outvi[nv]);
	}


	InitArrayBUQ(XParam, XModel.blocks, T(0.0), XModel.evolv.u);
	InitArrayBUQ(XParam, XModel.blocks, T(0.0), XModel.evolv.v);
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel_g.blocks.active, T(0.0), XModel_g.evolv.u);
	CUDA_CHECK(hipDeviceSynchronize());

	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel_g.blocks.active, T(0.0), XModel_g.evolv.v);
	CUDA_CHECK(hipDeviceSynchronize());

	Forcing<float> XForcing;
	while (XLoop.totaltime < XParam.endtime)
	{
		FlowGPU(XParam, XLoop_g, XForcing, XModel_g);
		FlowCPU(XParam, XLoop, XForcing, XModel);

		XLoop.totaltime = XLoop.totaltime + XLoop.dt;
		XLoop_g.totaltime = XLoop_g.totaltime + XLoop_g.dt;
		if (XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001) && XParam.outputtimestep > 0.0)
		{
			CompareCPUvsGPU(XParam, XModel, XModel_g, outv, false);
			XLoop.nextoutputtime = min(XLoop.nextoutputtime + XParam.outputtimestep, XParam.endtime);
			XLoop_g.nextoutputtime = XLoop.nextoutputtime;
		}
	}


	return test;
}

/*! \fn T ValleyBathy(T x, T y, T slope, T center)
* \brief	create V shape Valley basin
*
* This function creates a simple V shape Valley basin
*
*
*/
template <class T> T ValleyBathy(T x, T y, T slope, T center)
{


	T bathy;

	bathy = (abs(x - center) + y) * slope;


	return bathy;
}


/*! \fn T ThackerBathy(T x, T y, T L, T D)
* \brief	create a parabolic bassin
*
* This function creates a parabolic bassin. The function returns a single value of the bassin
*
* Borrowed from Buttinger et al. 2019.
*
* ### Reference
* Buttinger-Kreuzhuber, A., Horváth, Z., Noelle, S., Blöschl, G., and Waser, J.: A fast second-order shallow water scheme on two-dimensional
* structured grids over abrupt topography, Advances in water resources, 127, 89–108, 2019.
*/
template <class T> T ThackerBathy(T x, T y, T L, T D)
{


	T bathy = D * ((x * x + y * y) / (L * L) - 1.0);


	return bathy;
}

/*! \fn
* \brief	Simulate the Lake-at-rest in a parabolic bassin
* 
* This function creates a parabolic bassin filled to a given level and run the modle for a while and checks that the velocities in the lake remain very small
* thus verifying the well-balancedness of teh engine and the Lake-at-rest condition.
*
* Borrowed from Buttinger et al. 2019.
*
* ### Reference
* Buttinger-Kreuzhuber, A., Horváth, Z., Noelle, S., Blöschl, G., and Waser, J.: A fast second-order shallow water scheme on two-dimensional
* structured grids over abrupt topography, Advances in water resources, 127, 89–108, 2019.
*/
template <class T> bool ThackerLakeAtRest(Param XParam,T zsinit)
{
	bool test = true;
	// Make a Parabolic bathy
	
	auto modeltype = XParam.doubleprecision < 1 ? float() : double();
	Model<decltype(modeltype)> XModel; // For CPU pointers
	Model<decltype(modeltype)> XModel_g; // For GPU pointers

	Forcing<float> XForcing;

	StaticForcingP<float> bathy;

	XForcing.Bathy.push_back(bathy);

	// initialise forcing bathymetry to 0

	T Lo = T(2500.0);
	T Do = T(1.0);

	T x, y;



	XForcing.Bathy[0].xo = -4000.0;
	XForcing.Bathy[0].yo = -4000.0;

	XForcing.Bathy[0].xmax = 4000.0;
	XForcing.Bathy[0].ymax = 4000.0;
	XForcing.Bathy[0].nx = 64;
	XForcing.Bathy[0].ny = 64;

	XForcing.Bathy[0].dx = 126.0;

	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.Bathy[0].val);

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			x = T(XForcing.Bathy[0].xo + i * XForcing.Bathy[0].dx);
			y = T(XForcing.Bathy[0].yo + j * XForcing.Bathy[0].dx);
			XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = float(ThackerBathy(x, y, Lo, Do));
		}
	}

	// Overrule whatever may be set in the param file
	XParam.xmax = XForcing.Bathy[0].xmax;
	XParam.ymax = XForcing.Bathy[0].ymax;
	XParam.xo = XForcing.Bathy[0].xo;
	XParam.yo = XForcing.Bathy[0].yo;

	XParam.dx = XForcing.Bathy[0].dx;

	XParam.zsinit = zsinit;
	XParam.endtime = 1390.0;

	XParam.outputtimestep = XParam.endtime; 

	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);

	
	SetupGPU(XParam, XModel, XForcing, XModel_g);

	MainLoop(XParam, XForcing, XModel, XModel_g);


	// Check Lake at rest state?
	// all velocities should be very small
	T smallvel = T(1e-6);
	int i;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth); ix++)
			{
				i = memloc(XParam, ix, iy, ib);
				if (abs(XModel.evolv.u[i]) > smallvel || abs(XModel.evolv.v[i]) > smallvel)
				{
					log("Lake at rest state not acheived!");
					test = false;
				}
			}
		}
	}

	return test;
}
template bool ThackerLakeAtRest<float>(Param XParam,float zsinit);
template bool ThackerLakeAtRest<double>(Param XParam, double zsinit);



/*! \fn bool RiverVolumeAdapt(Param XParam)
* \brief	Wraping function for RiverVolumeAdapt(Param XParam, T slope, bool bottop, bool flip)
*
* The function calls it's child function with different adaptation set in XParam so needs to be rerun to account for the different scenarios:
* * uniform level
* * flow from coasrse to fine
* * flow from fine to coarse
*
* and account for different flow direction
* 
*/
template <class T> bool RiverVolumeAdapt(Param XParam, T maxslope)
{
	//T maxslope = 0.45; // tthe mass conservation is better with smaller slopes 

	bool UnitestA, UnitestB, UnitestC, UnitestD;
	bool ctofA, ctofB, ctofC, ctofD;
	bool ftocA, ftocB, ftocC, ftocD;

	std::string details;

	XParam.minlevel = 1;
	XParam.maxlevel = 1;
	XParam.initlevel = 1;
	
	
	UnitestA=RiverVolumeAdapt(XParam, maxslope, false, false);
	UnitestB=RiverVolumeAdapt(XParam, maxslope, true, false);
	UnitestC=RiverVolumeAdapt(XParam, maxslope, false, true);
	UnitestD=RiverVolumeAdapt(XParam, maxslope, true, true);

	if (UnitestA && UnitestB && UnitestC && UnitestD)
	{
		log("River Volume Conservation Test: Uniform mesh: Success");
	}
	else
	{
		log("River Volume Conservation Test: Uniform mesh: Failed");
		details = UnitestA ? "successful" : "failed";
		log("\t Uniform mesh A :"+ details);
		details = UnitestB ? "successful" : "failed";
		log("\t Uniform mesh B :" + details);
		details = UnitestC ? "successful" : "failed";
		log("\t Uniform mesh C :" + details);
		details = UnitestD ? "successful" : "failed";
		log("\t Uniform mesh D :" + details);
	}

	XParam.minlevel = 0;
	XParam.maxlevel = 1;
	XParam.initlevel = 0;

	//Fine to coarse
	// Change arg 1 and 2 if the slope is changed
	XParam.AdaptCrit = "Inrange";
	XParam.Adapt_arg1 = "28.0";
	XParam.Adapt_arg2 = "40.0";
	XParam.Adapt_arg3 = "zb";

	ftocA = RiverVolumeAdapt(XParam, maxslope, false, false);
	ftocB = RiverVolumeAdapt(XParam, maxslope, true, false);
	ftocC = RiverVolumeAdapt(XParam, maxslope, false, true);
	ftocD = RiverVolumeAdapt(XParam, maxslope, true, true);
	if (ftocA && ftocB && ftocC && ftocD)
	{
		log("River Volume Conservation Test: Flow from fine to coarse adapted mesh: Success");
	}
	else
	{
		log("River Volume Conservation Test: Flow from fine to coarse adapted mesh: Failed");
		details = ftocA ? "successful" : "failed";
		log("\t Flow from fine to coarse adapted mesh A :" + details);
		details = ftocB ? "successful" : "failed";
		log("\t Flow from fine to coarse adapted mesh B :" + details);
		details = ftocC ? "successful" : "failed";
		log("\t Flow from fine to coarse adapted mesh C :" + details);
		details = ftocD ? "successful" : "failed";
		log("\t Flow from fine to coarse adapted mesh D :" + details);
	}

	//coarse to fine
	// Change arg 1 and 2 if the slope is changed
	XParam.AdaptCrit = "Inrange";
	XParam.Adapt_arg1 = "0.0";
	XParam.Adapt_arg2 = "2.0";
	XParam.Adapt_arg3 = "zb";

	ctofA = RiverVolumeAdapt(XParam, maxslope, false, false);
	ctofB = RiverVolumeAdapt(XParam, maxslope, true, false);
	ctofC = RiverVolumeAdapt(XParam, maxslope, false, true);
	ctofD = RiverVolumeAdapt(XParam, maxslope, true, true);
	if (ctofA && ctofB && ctofC && ctofD)
	{
		log("River Volume Conservation Test: Flow from coarse to fine adapted mesh: Success");
	}
	else
	{
		log("River Volume Conservation Test: Flow from coarse to fine adapted: Failed");
		details = ctofA ? "successful" : "failed";
		log("\t Flow from coarse to fine adapted mesh A :" + details);
		details = ctofB ? "successful" : "failed";
		log("\t Flow from coarse to fine adapted mesh B :" + details);
		details = ctofC ? "successful" : "failed";
		log("\t Flow from coarse to fine adapted mesh C :" + details);
		details = ctofD ? "successful" : "failed";
		log("\t Flow from coarse to fine adapted mesh D :" + details);

	}

	return (UnitestA * UnitestB * UnitestC * UnitestD * ctofA * ctofB * ctofC * ctofD * ftocA * ftocB * ftocC * ftocD);
}


/*! \fn bool RiverVolumeAdapt(Param XParam, T slope, bool bottop, bool flip)
* \brief	Simulate a river flowing in a steep valley
* and heck the Volume conservation
*
* This function creates a dry steep valley topography to a given level and run the model for a while and checks that the Volume matches the theory.
*
* The function can test the water volume for 4 scenario each time:
* * left to right: bottop=false & flip=true;
* * right to left: bottop=false & flip=false;
* * bottom to top: bottop=true & flip=true;
* * top to bottom: bottop=true & flip=false;
*
* The function inherits the adaptation set in XParam so needs to be rerun to accnout for the different scenarios:
* * uniform level
* * flow from coasrse to fine
* * flow from fine to coarse
* This is done in the higher level wrapping function
*/
template <class T> bool RiverVolumeAdapt(Param XParam, T slope, bool bottop, bool flip)
{
	//bool test = true;
	//

	auto modeltype = XParam.doubleprecision < 1 ? float() : double();
	Model<decltype(modeltype)> XModel; // For CPU pointers
	Model<decltype(modeltype)> XModel_g; // For GPU pointers

	Forcing<float> XForcing;

	XForcing = MakValleyBathy(XParam, slope, bottop, flip);

	T x, y;
	T center = T(10.5);

	float maxtopo = std::numeric_limits<float>::min();
	float mintopo = std::numeric_limits<float>::max();

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			maxtopo = max(XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx], maxtopo);
			mintopo = min(XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx], mintopo);
		}
	}

	

	// Overrule whatever is set in the river forcing
	T Q = T(1.0);
	
	double upstream = !flip ? 24.0 : 8;
	double riverx = !bottop ? upstream : center;
	double rivery = !bottop ? center : upstream;

	//Create a temporary file with river fluxes
	std::ofstream river_file(
		"testriver.tmp", std::ios_base::out | std::ios_base::trunc);
	river_file << "0.0 " + std::to_string(Q) << std::endl;
	river_file << "3600.0 " + std::to_string(Q) << std::endl;
	river_file.close(); //destructor implicitly does it

	River thisriver;
	thisriver.Riverflowfile = "testriver.tmp";
	thisriver.xstart = riverx - 1.0;
	thisriver.xend = riverx + 1.0;
	thisriver.ystart = rivery - 1.0;
	thisriver.yend = rivery + 1.0;

	XForcing.rivers.push_back(thisriver);


	XForcing.rivers[0].flowinput = readFlowfile(XForcing.rivers[0].Riverflowfile);


	// Overrule whatever may be set in the param file
	XParam.xmax = XForcing.Bathy[0].xmax;
	XParam.ymax = XForcing.Bathy[0].ymax;
	XParam.xo = XForcing.Bathy[0].xo;
	XParam.yo = XForcing.Bathy[0].yo;

	XParam.dx = XForcing.Bathy[0].dx;

	XParam.zsinit = mintopo+0.5;// Had a small amount of water to avoid a huge first step that would surely break the setup
	XParam.endtime = 20.0;

	XParam.outputtimestep = XParam.endtime;

	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);


	SetupGPU(XParam, XModel, XForcing, XModel_g);
	T initVol = T(0.0);
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		T delta = calcres(XParam.dx, XModel.blocks.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth); ix++)
			{
				int i = memloc(XParam, ix, iy, ib);
				initVol = initVol + T(XModel.evolv.h[i]) * delta * delta;
			}
		}
	}


	MainLoop(XParam, XForcing, XModel, XModel_g);
	
	T TheoryInput = Q * XParam.endtime;


	T SimulatedVolume = T(0.0);
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		T delta = calcres(XParam.dx, XModel.blocks.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth); ix++)
			{
				int i = memloc(XParam, ix, iy, ib);
				SimulatedVolume = SimulatedVolume + XModel.evolv.h[i] * delta * delta;
			}
		}
	}

	SimulatedVolume = SimulatedVolume - initVol;

	T error = abs(SimulatedVolume - TheoryInput);

	return error / TheoryInput < 0.05;

}



/*! \fn bool testboundaries(T maxslope)
* \brief	Wraping function for Boundary(Param XParam, T slope, bool bottop, bool flip)
*
* This function test the 3 types of boundaries (0: Wall/1: Neumann/3: non-reflexive)
* and on all orientations
*
*/
template <class T> bool testboundaries(Param XParam,T maxslope)
{
	//T maxslope = 0.45; // the mass conservation is better with smaller slopes 

	bool Wall_B;// , Wall_R, Wall_L, Wall_T;
	//bool ctofA, ctofB, ctofC, ctofD;
	//bool ftocA, ftocB, ftocC, ftocD;


	std::string details;
	int Bound_type;

	
	XParam.GPUDEVICE = 0;
	maxslope = 0.0;
	//Dir = 3;
	Bound_type = -1;
	Wall_B = RiverOnBoundary(XParam, maxslope, 3, Bound_type);
	//Wall_R = RiverOnBoundary(XParam, maxslope, 0, 0);
	//Wall_L = RiverOnBoundary(XParam, maxslope, 1, 0);
	//Wall_T = RiverOnBoundary(XParam, maxslope, 2, 0);
	/*

	if (UnitestA && UnitestB && UnitestC && UnitestD)
	{
		log("River Volume Conservation Test: Uniform mesh: Success");
	}
	else
	{
		log("River Volume Conservation Test: Uniform mesh: Failed");
		details = UnitestA ? "successful" : "failed";
		log("\t Uniform mesh A :" + details);
		details = UnitestB ? "successful" : "failed";
		log("\t Uniform mesh B :" + details);
		details = UnitestC ? "successful" : "failed";
		log("\t Uniform mesh C :" + details);
		details = UnitestD ? "successful" : "failed";
		log("\t Uniform mesh D :" + details);
	}

	XParam.minlevel = 0;
	XParam.maxlevel = 1;
	XParam.initlevel = 0;

	//Fine to coarse
	// Change arg 1 and 2 if the slope is changed
	XParam.AdaptCrit = "Inrange";
	XParam.Adapt_arg1 = "28.0";
	XParam.Adapt_arg2 = "40.0";
	XParam.Adapt_arg3 = "zb";

	ftocA = RiverVolumeAdapt(XParam, maxslope, false, false);
	ftocB = RiverVolumeAdapt(XParam, maxslope, true, false);
	ftocC = RiverVolumeAdapt(XParam, maxslope, false, true);
	ftocD = RiverVolumeAdapt(XParam, maxslope, true, true);
	if (ftocA && ftocB && ftocC && ftocD)
	{
		log("River Volume Conservation Test: Flow from fine to coarse adapted mesh: Success");
	}
	else
	{
		log("River Volume Conservation Test: Flow from fine to coarse adapted mesh: Failed");
		details = ftocA ? "successful" : "failed";
		log("\t Flow from fine to coarse adapted mesh A :" + details);
		details = ftocB ? "successful" : "failed";
		log("\t Flow from fine to coarse adapted mesh B :" + details);
		details = ftocC ? "successful" : "failed";
		log("\t Flow from fine to coarse adapted mesh C :" + details);
		details = ftocD ? "successful" : "failed";
		log("\t Flow from fine to coarse adapted mesh D :" + details);
	}

	//coarse to fine
	// Change arg 1 and 2 if the slope is changed
	XParam.AdaptCrit = "Inrange";
	XParam.Adapt_arg1 = "0.0";
	XParam.Adapt_arg2 = "2.0";
	XParam.Adapt_arg3 = "zb";

	ctofA = RiverVolumeAdapt(XParam, maxslope, false, false);
	ctofB = RiverVolumeAdapt(XParam, maxslope, true, false);
	ctofC = RiverVolumeAdapt(XParam, maxslope, false, true);
	ctofD = RiverVolumeAdapt(XParam, maxslope, true, true);
	if (ctofA && ctofB && ctofC && ctofD)
	{
		log("River Volume Conservation Test: Flow from coarse to fine adapted mesh: Success");
	}
	else
	{
		log("River Volume Conservation Test: Flow from coarse to fine adapted: Failed");
		details = ctofA ? "successful" : "failed";
		log("\t Flow from coarse to fine adapted mesh A :" + details);
		details = ctofB ? "successful" : "failed";
		log("\t Flow from coarse to fine adapted mesh B :" + details);
		details = ctofC ? "successful" : "failed";
		log("\t Flow from coarse to fine adapted mesh C :" + details);
		details = ctofD ? "successful" : "failed";
		log("\t Flow from coarse to fine adapted mesh D :" + details);
	}*/

	//return (UnitestA * UnitestB * UnitestC * UnitestD * ctofA * ctofB * ctofC * ctofD * ftocA * ftocB * ftocC * ftocD);
	return(Wall_B);
}


/*! \fn bool RiverOnBoundary(T slope, bool bottop, bool flip)
* \brief	Simulate a river flowing in a (steep) valley
* and check the Volume conservation
*
* This function creates a half dry steep valley topography to a given level and run the model for a while and checks that the Volume matches the theory.
* A wall is located in the center of the valley.
*
* The function can test the water volume for 4 scenario each time:
* * flowing to the right: Dir=0;
* * flowing to the left: Dir=1;
* * flowing to the top: Dir=2;
* * flowing to the bottom: Dir=3;
*
*/
template <class T> bool RiverOnBoundary(Param XParam,T slope, int Dir, int Bound_type)
{
	//bool test = true;
	// Make a Parabolic bathy

	//Param XParam;
	XParam.GPUDEVICE = -1;

	auto modeltype = XParam.doubleprecision < 1 ? float() : double();
	Model<decltype(modeltype)> XModel; // For CPU pointers
	Model<decltype(modeltype)> XModel_g; // For GPU pointers

	Forcing<float> XForcing;

	StaticForcingP<float> bathy;

	float* dummybathy;

	//Boundary conditions
	XForcing.top.type = 0;
	XForcing.bot.type = 0;
	XForcing.right.type = 0;
	XForcing.left.type = 0;

	//Physical wall boundary condition
	bool PhysWall = 0;
	if (Bound_type == -1)
	{
		PhysWall = 1;
		Bound_type = 0;
	}

	if (Dir == 0) //To right
	{
		XForcing.right.type = Bound_type;
		XForcing.top.type = 0;
	}
	else if (Dir == 1) //To left
	{
		XForcing.left.type = Bound_type;
		XForcing.bot.type = 0;
	}
	else if (Dir == 2) //To top
	{
		XForcing.top.type = Bound_type;
		XForcing.left.type = 0;
	}
	else if (Dir == 3) //To bottom
	{
		XForcing.bot.type = Bound_type;
		XForcing.right.type = 0;
	}

	XForcing.Bathy.push_back(bathy);

	XForcing.Bathy[0].xo = 0.0;
	XForcing.Bathy[0].yo = 0.0;
	XForcing.Bathy[0].xmax = 31.0;
	XForcing.Bathy[0].ymax = 31.0;
	XForcing.Bathy[0].nx = 32;
	XForcing.Bathy[0].ny = 32;

	XForcing.Bathy[0].dx = 1.0;

	T x, y;
	T center = T(31.0);

	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.Bathy[0].val);
	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, dummybathy);


	//float maxtopo = std::numeric_limits<float>::min();
	float mintopo = 1000000000000.0f;
	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			x = T(XForcing.Bathy[0].xo + i * XForcing.Bathy[0].dx);
			y = T(XForcing.Bathy[0].yo + j * XForcing.Bathy[0].dx);


			dummybathy[i + j * XForcing.Bathy[0].nx] = float(ValleyBathy(y, x, slope, center));

			//Add physical walls
			if (PhysWall == 1)
			{
				//if (j < 3)
				//{
				//	dummybathy[i + j * XForcing.Bathy[0].nx] = 100.0;
				//}
				if (j > XForcing.Bathy[0].ny - 3)
				{
					dummybathy[i + j * XForcing.Bathy[0].nx] = 100.0;
				}
				if (i > XForcing.Bathy[0].nx - 3)
				{
					dummybathy[i + j * XForcing.Bathy[0].nx] = 100.0;
				}
				if (i < 17)
				{
					dummybathy[i + j * XForcing.Bathy[0].nx] = 1000.0;
				}
			}

			mintopo = utils::min(dummybathy[i + j * XForcing.Bathy[0].nx], mintopo);
			//maxtopo = max(dummybathy[i + j * XForcing.Bathy[0].nx], maxtopo);

		}
	}

	// Flip or rotate the bathy according to what is requested
	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			if (Dir == 1) //left wise
			{
				XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = dummybathy[i + j * XForcing.Bathy[0].nx];
			}
			else if (Dir == 0) //right wise
			{
				XForcing.Bathy[0].val[(XForcing.Bathy[0].nx - 1 - i) + j * XForcing.Bathy[0].nx] = dummybathy[i + j * XForcing.Bathy[0].nx];
			}
			else if (Dir == 3) //bottom wise
			{
				XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = dummybathy[j + i * XForcing.Bathy[0].nx];
			}
			else if (Dir == 2) //top wise
			{
				XForcing.Bathy[0].val[i + (XForcing.Bathy[0].ny - 1 - j) * XForcing.Bathy[0].nx] = dummybathy[j + i * XForcing.Bathy[0].nx];
			}
		}
	}

	free(dummybathy);

	// Overrule whatever is set in the river forcing
	T Q = T(1.0);

	double riverx = (Dir == 0 | Dir == 2)? 6.0 : 25.0; //Dir=1 =>leftward
	double rivery = (Dir == 2 | Dir == 1)? 6.0 : 25.0; //Dir=2 =>topward
	
	//Create a temporary file with river fluxes
	std::ofstream river_file(
		"testriver.tmp", std::ios_base::out | std::ios_base::trunc);
	river_file << "0.0 " + std::to_string(Q) << std::endl;
	river_file << "3600.0 " + std::to_string(Q) << std::endl;
	river_file.close(); //destructor implicitly does it

	River thisriver;
	thisriver.Riverflowfile = "testriver.tmp";
	thisriver.xstart = riverx - 1.0;
	thisriver.xend = riverx + 1.0;
	thisriver.ystart = rivery - 1.0;
	thisriver.yend = rivery + 1.0;

	XForcing.rivers.push_back(thisriver);


	XForcing.rivers[0].flowinput = readFlowfile(XForcing.rivers[0].Riverflowfile);


	// Overrule whatever may be set in the param file
	XParam.xmax = XForcing.Bathy[0].xmax;
	XParam.ymax = XForcing.Bathy[0].ymax;
	XParam.xo = XForcing.Bathy[0].xo;
	XParam.yo = XForcing.Bathy[0].yo;

	XParam.dx = XForcing.Bathy[0].dx;

	XParam.zsinit = mintopo + 0.5;// Had a small amount of water to avoid a huge first step that would surely break the setup
	//XParam.zsoffset = 0.2;
	XParam.endtime = 50.0;
	XParam.dtinit = 0.1;
	XParam.mask = 999.0;
	XParam.outishift = 0;
	XParam.outjshift = 0;


	XParam.outputtimestep = 10.0;// XParam.endtime;

	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);

	InitSave2Netcdf(XParam, XModel);

	SetupGPU(XParam, XModel, XForcing, XModel_g);
	T initVol = T(0.0);
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		T delta = calcres(XParam.dx, XModel.blocks.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth); ix++)
			{
				int i = memloc(XParam, ix, iy, ib);
				initVol = initVol + XModel.evolv.h[i] * delta * delta;
			}
		}
	}


	MainLoop(XParam, XForcing, XModel, XModel_g);

	T TheoryInput = Q * XParam.endtime;


	T SimulatedVolume = T(0.0);
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		T delta = calcres(XParam.dx, XModel.blocks.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth); ix++)
			{
				int i = memloc(XParam, ix, iy, ib);
				SimulatedVolume = SimulatedVolume + XModel.evolv.h[i] * delta * delta;
			}
		}
	}


	printf("End Volume : %f \n", SimulatedVolume);
	printf("Init Volume : %f \n", initVol);

	SimulatedVolume = SimulatedVolume - initVol;

	printf("End Volume - Init volume : %f \n", SimulatedVolume);

	T error = abs(SimulatedVolume - TheoryInput);

	printf("error : %f \n", error);
	printf("Theory input : %f \n", TheoryInput);
	printf("return : %f \n", (error/TheoryInput));


	return error / TheoryInput < 0.01;

}



/*! \fn bool LakeAtRest(Param XParam, Model<T> XModel)
*	This function simulates the first predictive step and check whether the lake at rest is preserved
*	otherwise it prints out to screen the cells (and neighbour) where the test fails
*/
template <class T> bool LakeAtRest(Param XParam, Model<T> XModel)
{
	T epsi = T(1e-5);
	int ib;

	bool test = true;


	Loop<T> XLoop = InitLoop(XParam, XModel);

	fillHaloC(XParam, XModel.blocks, XModel.zb);

	gradientC(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);
	gradientHalo(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);

	refine_linear(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);
	gradientHalo(XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);
	



	//============================================
	// Predictor step in reimann solver
	//============================================

	//============================================
	//  Fill the halo for gradient reconstruction
	fillHalo(XParam, XModel.blocks, XModel.evolv, XModel.zb);

	//============================================
	// Reset DTmax
	InitArrayBUQ(XParam, XModel.blocks, XLoop.hugeposval, XModel.time.dtmax);

	//============================================
	// Calculate gradient for evolving parameters
	gradientCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.zb);

	//============================================
	// Flux and Source term reconstruction
	// X- direction
	//updateKurgXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	UpdateButtingerXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	//AddSlopeSourceXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);

	// Y- direction
	//updateKurgYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	UpdateButtingerYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	//AddSlopeSourceYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);

	//============================================
	// Fill Halo for flux from fine to coarse
	fillHalo(XParam, XModel.blocks, XModel.flux);

	// Do we need to check also before fill halo part?

	// Check Fhu and Fhv (they should be zero)
	int i, iright;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XModel.blocks.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth); ix++)
			{
				i = memloc(XParam, ix, iy, ib);
				iright = memloc(XParam, ix + 1, iy, ib);
				//ileft = memloc(XParam, ix - 1, iy, ib);
				//itop = memloc(XParam, ix, iy + 1, ib);
				//ibot = memloc(XParam, ix, iy - 1, ib);

				if (abs(XModel.flux.Fhu[i]) > epsi)
				{
					log("Fhu is not zero. Lake at rest not preserved!!!");
					test = false;
				}

				if (abs(XModel.flux.Fhv[i]) > epsi)
				{
					log("Fhv is not zero. Lake at rest not preserved!!!");
					test = false;
				}

				T dhus = (XModel.flux.Fqux[i] - XModel.flux.Su[iright]);
				if (abs(dhus) > epsi)
				{
					test = false;

					log("dhu is not zero. Lake at rest not preserved!!!");


					printf("Fhu[i]=%f\n", XModel.flux.Fhu[i]);

					printf("Fqux[i]=%f; Su[iright]=%f; Diff=%f \n",XModel.flux.Fqux[i], XModel.flux.Su[iright], (XModel.flux.Fqux[i] - XModel.flux.Su[iright]));

					printf(" At i: (ib=%d; ix=%d; iy=%d)\n", ib,ix,iy);
					testButtingerX(XParam, ib, ix, iy, XModel);

					printf(" At iright: (ib=%d; ix=%d; iy=%d)\n", ib, ix+1, iy);
					testButtingerX(XParam, ib, ix+1, iy, XModel);

				}

			}
		}
	}


	if (!test)
	{
		copyID2var(XParam, XModel.blocks, XModel.flux.Fhu);
		copyBlockinfo2var(XParam, XModel.blocks, XModel.blocks.LeftBot, XModel.grad.dhdx);
		copyBlockinfo2var(XParam, XModel.blocks, XModel.blocks.LeftTop, XModel.grad.dhdy);
		copyBlockinfo2var(XParam, XModel.blocks, XModel.blocks.TopLeft, XModel.grad.dzsdx);
		copyBlockinfo2var(XParam, XModel.blocks, XModel.blocks.TopRight, XModel.grad.dzsdy);
		copyBlockinfo2var(XParam, XModel.blocks, XModel.blocks.RightTop, XModel.grad.dudx);
		copyBlockinfo2var(XParam, XModel.blocks, XModel.blocks.RightBot, XModel.grad.dudy);
		copyBlockinfo2var(XParam, XModel.blocks, XModel.blocks.BotRight, XModel.grad.dvdx);
		copyBlockinfo2var(XParam, XModel.blocks, XModel.blocks.BotLeft, XModel.grad.dvdy);

		creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, XModel.blocks.outZone[0]);
		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "blockID", 3, XModel.flux.Fhu, XModel.blocks.outZone[0]);

		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "LeftBot", 3, XModel.grad.dhdx, XModel.blocks.outZone[0]);
		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "LeftTop", 3, XModel.grad.dhdy, XModel.blocks.outZone[0]);

		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "TopLeft", 3, XModel.grad.dzsdx, XModel.blocks.outZone[0]);
		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "TopRight", 3, XModel.grad.dzsdy, XModel.blocks.outZone[0]);

		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "RightTop", 3, XModel.grad.dudx, XModel.blocks.outZone[0]);
		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "RightBot", 3, XModel.grad.dudy, XModel.blocks.outZone[0]);

		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "BotLeft", 3, XModel.grad.dvdx, XModel.blocks.outZone[0]);
		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "BotRight", 3, XModel.grad.dvdy, XModel.blocks.outZone[0]);
	}

	return test;
}


/*! \fn  void testButtingerX(Param XParam, int ib, int ix, int iy, Model<T> XModel)
*
* This function goes through the Buttinger scheme but instead of the normal output just prints all teh usefull values
* This function is/was used in the lake-at-rest verification
*
* See also: void testkurganovX(Param XParam, int ib, int ix, int iy, Model<T> XModel)
*/
template <class T> void testButtingerX(Param XParam, int ib, int ix, int iy, Model<T> XModel)
{
	int RB, levRB, LBRB, LB, levLB, RBLB;
	int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);
	int ileft = memloc(XParam.halowidth, XParam.blkmemwidth, ix - 1, iy, ib);

	int lev = XModel.blocks.level[ib];
	T delta = calcres(T(XParam.dx), lev);

	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;

	// neighbours for source term

	RB = XModel.blocks.RightBot[ib];
	levRB = XModel.blocks.level[RB];
	LBRB = XModel.blocks.LeftBot[RB];

	LB = XModel.blocks.LeftBot[ib];
	levLB = XModel.blocks.level[LB];
	RBLB = XModel.blocks.RightBot[LB];

	
	T cm = T(1.0);
	T fmu = T(1.0);

	T hi = XModel.evolv.h[i];

	T hn = XModel.evolv.h[ileft];


	//if (hi > eps || hn > eps)
	{
		T dx, zi, zn, hr, hl, etar, etal, zr, zl, zA, zCN, hCNr, hCNl;
		T ui, vi, uli, vli, dhdxi, dhdxil, dudxi, dudxil, dvdxi, dvdxil;

		T ga = g * T(0.5);
		// along X
		dx = delta * T(0.5);
		zi = XModel.zb[i];
		zn = XModel.zb[ileft];

		ui = XModel.evolv.u[i];
		vi = XModel.evolv.v[i];
		uli = XModel.evolv.u[ileft];
		vli = XModel.evolv.v[ileft];

		dhdxi = XModel.grad.dhdx[i];
		dhdxil = XModel.grad.dhdx[ileft];
		dudxi = XModel.grad.dudx[i];
		dudxil = XModel.grad.dudx[ileft];
		dvdxi = XModel.grad.dvdx[i];
		dvdxil = XModel.grad.dvdx[ileft];


		hr = hi - dx * dhdxi;
		hl = hn + dx * dhdxil;
		etar = XModel.evolv.zs[i] - dx * XModel.grad.dzsdx[i];
		etal = XModel.evolv.zs[ileft] + dx * XModel.grad.dzsdx[ileft];

		//define the topography term at the interfaces
		zr = zi - dx * XModel.grad.dzbdx[i];
		zl = zn + dx * XModel.grad.dzbdx[ileft];

		//define the Audusse terms
		zA = max(zr, zl);

		// Now the CN terms
		zCN = min(zA, min(etal, etar));
		hCNr = max(T(0.0), min(etar - zCN, hr));
		hCNl = max(T(0.0), min(etal - zCN, hl));

		//Velocity reconstruction
		//To avoid high velocities near dry cells, we reconstruct velocities according to Bouchut.
		T ul, ur, vl, vr, sl, sr;
		if (hi > eps) {
			ur = ui - (T(1.) + dx * dhdxi / hi) * dx * dudxi;
			vr = vi - (T(1.) + dx * dhdxi / hi) * dx * dvdxi;
		}
		else {
			ur = ui - dx * dudxi;
			vr = vi - dx * dvdxi;
		}
		if (hn > eps) {
			ul = uli + (T(1.) - dx * dhdxil / hn) * dx * dudxil;
			vl = vli + (T(1.) - dx * dhdxil / hn) * dx * dvdxil;
		}
		else {
			ul = uli + dx * dudxil;
			vl = vli + dx * dvdxil;
		}




		T fh, fu, fv, dt;


		//solver below also modifies fh and fu
		dt = hllc(g, delta, epsi, CFL, cm, fmu, hCNl, hCNr, ul, ur, fh, fu);
		//hllc(T g, T delta, T epsi, T CFL, T cm, T fm, T hm, T hp, T um, T up, T & fh, T & fq)

		

		fv = (fh > 0. ? vl : vr) * fh;


		// Topographic source term

		// In the case of adaptive refinement, care must be taken to ensure
		// well-balancing at coarse/fine faces (see [notes/balanced.tm]()). 
		if ((ix == XParam.blkwidth) && levRB < lev)//(ix==16) i.e. in the right halo
		{
			int jj = LBRB == ib ? ftoi(floor(iy * (T)0.5)) : ftoi(floor(iy * (T)0.5) + XParam.blkwidth / 2);
			int iright = memloc(XParam.halowidth, XParam.blkmemwidth, 0, jj, RB);;
			hi = XModel.evolv.h[iright];
			zi = XModel.zb[iright];
		}
		if ((ix == 0) && levLB < lev)//(ix==16) i.e. in the right halo if you 
		{
			int jj = RBLB == ib ? ftoi(floor(iy * (T)0.5)) : ftoi(floor(iy * (T)0.5) + XParam.blkwidth / 2);
			int ilc = memloc(XParam.halowidth, XParam.blkmemwidth, XParam.blkwidth - 1, jj, LB);
			//int ilc = memloc(halowidth, blkmemwidth, -1, iy, ib);
			hn = XModel.evolv.h[ilc];
			zn = XModel.zb[ilc];
		}

		sl = ga * (hi + hCNr) * (zi - zCN);
		sr = ga * (hCNl + hn) * (zn - zCN);


		printf("dt=%f; etar=%f; etal=%f; zCN=%f; zi=%f; zn=%f; zA=%f, zr=%f, zl=%f\n",dt, etar,etal,zCN,zi,zn,zA, zr,zl);


		printf("hi=%f; hn=%f,fh=%f; fu=%f; sl=%f; sr=%f; hCNl=%f; hCNr=%f; hr=%f; hl=%f; zr=%f; zl=%f;\n", hi, hn, fh, fu, sl, sr, hCNl, hCNr, hr, hl, zr, zl);

		printf("h[i]=%f; h[ileft]=%f dhdx[i]=%f, dhdx[ileft]=%f, zs[i]=%f, zs[ileft]=%f, dzsdx[i]=%f, dzsdx[ileft]=%f, dzbdx[i]=%f, dzbdx[ileft]=%f\n\n", XModel.evolv.h[i], XModel.evolv.h[ileft], XModel.grad.dhdx[i], XModel.grad.dhdx[ileft], XModel.evolv.zs[i], XModel.evolv.zs[ileft], XModel.grad.dzsdx[i], XModel.grad.dzsdx[ileft], XModel.grad.dzbdx[i], XModel.grad.dzbdx[ileft]);
	}
}


/*! \fn  void testkurganovX(Param XParam, int ib, int ix, int iy, Model<T> XModel)
*
* This function goes through the Kurganov scheme but instead of the normal output just prints all teh usefull values
* This function is/was used in the lake-at-rest verification
*/
template <class T> void testkurganovX(Param XParam, int ib, int ix, int iy, Model<T> XModel)
{
	int RB, levRB, LBRB, LB, levLB, RBLB;
	int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);
	int ileft = memloc(XParam.halowidth, XParam.blkmemwidth, ix - 1, iy, ib);

	int lev = XModel.blocks.level[ib];
	T delta = calcres(T(XParam.dx), lev);

	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;

	// neighbours for source term

	RB = XModel.blocks.RightBot[ib];
	levRB = XModel.blocks.level[RB];
	LBRB = XModel.blocks.LeftBot[RB];

	LB = XModel.blocks.LeftBot[ib];
	levLB = XModel.blocks.level[LB];
	RBLB = XModel.blocks.RightBot[LB];

	T dhdxi = XModel.grad.dhdx[i];
	T dhdxmin = XModel.grad.dhdx[ileft];
	T cm = T(1.0);
	T fmu = T(1.0);

	T hi = XModel.evolv.h[i];

	T hn = XModel.evolv.h[ileft];
	T dx, zi, zl, zn, zr, zlr, hl, up, hp, hr, um, hm, ga;

	// along X
	dx = delta * T(0.5);
	zi = XModel.evolv.zs[i] - hi;

	//printf("%f\n", zi);


	//zl = zi - dx*(dzsdx[i] - dhdx[i]);
	zl = zi - dx * (XModel.grad.dzsdx[i] - dhdxi);
	//printf("%f\n", zl);

	zn = XModel.evolv.zs[ileft] - hn;

	//printf("%f\n", zn);
	zr = zn + dx * (XModel.grad.dzsdx[ileft] - dhdxmin);


	zlr = max(zl, zr);

	//hl = hi - dx*dhdx[i];
	hl = hi - dx * dhdxi;
	up = XModel.evolv.u[i] - dx * XModel.grad.dudx[i];
	hp = max(T(0.0), hl + zl - zlr);

	hr = hn + dx * dhdxmin;
	um = XModel.evolv.u[ileft] + dx * XModel.grad.dudx[ileft];
	hm = max(T(0.0), hr + zr - zlr);

	ga = g * T(0.5);
	///// Reimann solver
	T fh, fu, fv, sl, sr, dt;

	//solver below also modifies fh and fu
	dt = KurgSolver(g, delta, epsi, CFL, cm, fmu, hp, hm, up, um, fh, fu);

	if ((ix == XParam.blkwidth) && levRB < lev)//(ix==16) i.e. in the right halo
	{
		int jj = LBRB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + XParam.blkwidth / 2;
		int iright = memloc(XParam.halowidth, XParam.blkmemwidth, 0, jj, RB);;
		hi = XModel.evolv.h[iright];
		zi = XModel.zb[iright];
	}
	if ((ix == 0) && levLB < lev)//(ix==16) i.e. in the right halo if you 
	{
		int jj = RBLB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + XParam.blkwidth / 2;
		int ilc = memloc(XParam.halowidth, XParam.blkmemwidth, XParam.blkwidth - 1, jj, LB);
		hn = XModel.evolv.h[ilc];
		zn = XModel.zb[ilc];
	}


	sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
	sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));

	////Flux update
	//Fhu[i] = fmu * fh;
	//Fqux[i] = fmu * (fu - sl);
	//Su[i] = fmu * (fu - sr);
	//Fqvx[i] = fmu * fv;

	printf("hi=%f; hn=%f,fh=%f; fu=%f; sl=%f; sr=%f; hp=%f; hm=%f; hr=%f; hl=%f; zr=%f; zl=%f;\n", hi, hn, fh, fu, sl, sr, hp, hm, hr, hl, zr, zl);

	printf("h[i]=%f; h[ileft]=%f dhdx[i]=%f, dhdx[ileft]=%f, zs[i]=%f, zs[ileft]=%f, dzsdx[i]=%f, dzsdx[ileft]=%f\n", XModel.evolv.h[i], XModel.evolv.h[ileft], XModel.grad.dhdx[i], XModel.grad.dhdx[ileft], XModel.evolv.zs[i], XModel.evolv.zs[ileft], XModel.grad.dzsdx[i], XModel.grad.dzsdx[ileft]);

}

/*! \fn bool Raintest(T zsnit, int gpu, float alpha)
*
* This function tests the mass conservation of the spacial injection (used to model rain on grid)
*	The function creates its own model setup and mesh independantly to what the user inputs.
*	This starts with a initial water level (zsnit=0.0 is dry) and runs for 0.1s before comparing results
*	with zsnit=0.1 that is approx 20 steps
*/
template <class T> bool Raintest(T zsnit, int gpu, float alpha)
{
	log("#####");
	Param XParam;
	T initVol, TheoryInput;
	TheoryInput = T(0.0);
	// initialise domain and required resolution
	XParam.dx = 1.0 / ((1 << 6)); //1<<8  = 2^8
	XParam.xo = -0.5;
	XParam.yo = -0.5;
	XParam.xmax = 0.5;
	XParam.ymax = 0.5;

	//XParam.initlevel = 0;
	//XParam.minlevel = 0;
	//XParam.maxlevel = 0;

	XParam.zsinit = zsnit;
	//XParam.zsoffset = 0.0;

	//Output times for comparisons
	XParam.endtime = 1.0;
	XParam.outputtimestep = 0.1;

	XParam.smallnc = 0;

	XParam.cf = 0.01;
	XParam.frictionmodel = 0;

	//Specification of the test
	//XParam.test = 7;
	XParam.rainforcing = true;

	// Enforce GPU/CPU
	XParam.GPUDEVICE = gpu;
	XParam.rainbnd = true;
	//output vars
	std::string outvi[16] = { "zb","h","zs","u","v","Fqux","Fqvx","Fquy","Fqvy", "Fhu", "Fhv", "dh", "dhu", "dhv", "Su", "Sv" };
	std::vector<std::string> outv;
	for (int nv = 0; nv < 15; nv++)
	{
		outv.push_back(outvi[nv]);
	}
	XParam.outvars = outv;

	// create Model setup
	Model<T> XModel;
	Model<T> XModel_g;

	Forcing<float> XForcing;

	StaticForcingP<float> bathy;

	XForcing.Bathy.push_back(bathy);

	// initialise forcing bathymetry to 0
	XForcing.Bathy[0].xo = -1.0;
	XForcing.Bathy[0].yo = -1.0;
	XForcing.Bathy[0].xmax = 1.0;
	XForcing.Bathy[0].ymax = 1.0;
	XForcing.Bathy[0].nx = 3;
	XForcing.Bathy[0].ny = 3;

	XForcing.Bathy[0].dx = 1.0;

	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.Bathy[0].val);

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = i * alpha / 100;
		}
	}

	// Add wall boundary conditions
	XForcing.right.type = 0;
	XForcing.left.type = 0;
	XForcing.top.type = 0;
	XForcing.bot.type = 0;


	//Value definition for surface rain fall
	T Q = 300; // mm/hr
	std::cout << "# Theoretical volume of water input during the simulation in m3: " << TheoryInput << ", from a rain input of: " << Q << "mm/hr." << std::endl;
	//Create a temporary file with rain fluxes
	std::ofstream rain_file(
		"testrain.tmp", std::ios_base::out | std::ios_base::trunc);
	rain_file << "0.0 " + std::to_string(Q) << std::endl;
	rain_file << "3600.0 " + std::to_string(Q) << std::endl;
	rain_file.close(); //destructor implicitly does it

	XForcing.Rain.inputfile = "testrain.tmp";
	XForcing.Rain.uniform = true;

	// Reading rain forcing from file for CPU and unifor rain
	XForcing.Rain.unidata = readINfileUNI(XForcing.Rain.inputfile);

	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);

	SetupGPU(XParam, XModel, XForcing, XModel_g);
	initVol = T(0.0);
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		T delta = calcres(XParam.dx, XModel.blocks.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth); ix++)
			{
				int i = memloc(XParam, ix, iy, ib);
				initVol = initVol + XModel.evolv.h[i] * delta * delta;
			}
		}
	}


	MainLoop(XParam, XForcing, XModel, XModel_g);

	TheoryInput = Q/ T(1000.0) / T(3600.0) * XParam.endtime;


	T SimulatedVolume = T(0.0);
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		T delta = calcres(XParam.dx, XModel.blocks.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth); ix++)
			{
				int i = memloc(XParam, ix, iy, ib);
				SimulatedVolume = SimulatedVolume + XModel.evolv.h[i] * delta * delta;
			}
		}
	}

	SimulatedVolume = SimulatedVolume - initVol;

	T error = abs(SimulatedVolume - TheoryInput);

	T modelgood= error / TheoryInput < 0.05;

	//log("#####");
	return modelgood;
}


/*! \fn bool Raintestinput(int gpu)
*
* This function tests the different inputs for rain forcing.
* This test is based on the paper Aureli2020, the 3 slopes test
* with regional rain. The experiment has been presented in Iwagaki1955.
* The first test compares a time varying rain input using a uniform time serie 
* forcing and a time varying 2D field (with same value).
* The second test check the 3D rain forcing (comparing it to expected values).
*/
bool Raintestinput(int gpu)
{
	//Results of the experiment of Aureli, interpolated to output values
	bool modelgood1, modelgood2;
	std::string result;
	//int dim_flux;
	std::vector<float> Flux1D, Flux3DUni, Flux3D, Flux_obs;
	float diff, ref, error;
	
	
	//Comparison between the 1D forcing and the 3D hommgeneous forcing
	Flux1D = Raintestmap(gpu, 1, -0.03);
	Flux3DUni = Raintestmap(gpu, 31, -0.03);
	ref = 0.0;
	diff = 0.0;
	for (int i = 0; i < Flux1D.size(); i++)
	{
		diff = diff + Flux1D[i] - Flux3DUni[i];
		ref = ref + Flux1D[i];
	}

	error = abs(diff / ref);
	printf("Error %f \n", error);

	modelgood1 = error < 0.005;
	result = modelgood1 ? "successful" : "failed";
	log("\t\tRain test input 1D vs 3D homogeneous: " + result);

	//Comparison between the 3D forcing and the observations from Iwagaki1955.

	//From Observations
	//Flux_obs = { 1.75136262,  4.31856716, 24.36350225, 32.02235696, 32.41207121,
	//   31.68632601, 29.8140878 , 47.9632521 , 68.78608061, 57.03656989 };

	//From BG_run of the testcase
	Flux_obs = { 4.003079, 12.664897, 25.376514, 33.214722, 34.987427, 34.054474,
		32.696472, 30.718161, 89.497993, 58.156021 };

	Flux3D = Raintestmap(gpu, 3, -0.03);
	ref = 0.0;
	diff = 0.0;
	for (int i = 0; i < Flux3D.size(); i++)
	{
		diff = diff + Flux_obs[i] - Flux3D[i];
		ref = ref + Flux3D[i];
	}

	error = abs(diff / ref);
	printf("Error %f \n", error);

	modelgood2 = error < 0.00005;
	result = modelgood2 ? "successful" : "failed";
	log("\t\tRain test input 3D map vs Iwagaki1955: " + result);

	return (modelgood1 * modelgood2);
}

/*! \fnstd::vector<float> Raintestmap(int gpu, int dimf, T zinit)
*
* This function return the flux at the bottom of the 3 part slope
* for different types of rain forcings using the test case based on Iwagaki1955
*/
template <class T> std::vector<float> Raintestmap(int gpu, int dimf, T zinit)
{
	log("#####");
	int k;
	T rainDuration = 10.0;
	int NX = 2502;
	int NY = 22;
	int NT;
	double* xRain;
	double* yRain;
	double* tRain;
	double* rainForcing;
	

	Param XParam;
	T delta;

	// initialise domain and required resolution
	XParam.xo = 0;
	XParam.yo = 0;
	XParam.ymax = 0.196;
	XParam.dx=(XParam.ymax - XParam.yo) / (1 << 1);
	double Xmax_exp = 28.0; //minimum Xmax position (adjust to have a "full blocks" config)
	//Calculating xmax to have full blocs with at least a full block behaving as a reservoir
	XParam.xmax = XParam.xo + (16 * XParam.dx) * std::ceil((Xmax_exp - XParam.xo) / (16 * XParam.dx)) + (16 * XParam.dx);
	//Surf = T((XParam.xmax - XParam.xo) * (XParam.ymax - XParam.yo));
	XParam.nblk = ftoi(((XParam.xmax - XParam.xo) / XParam.dx / 16) * ((XParam.ymax - XParam.yo) / XParam.dx / 16));
	XParam.rainbnd = true;
	XParam.zsinit = zinit;

	//Output times for comparisons
	XParam.endtime = 30.0;
	XParam.outputtimestep = 3.0;

	XParam.smallnc = 0;

	//Specification of the test
	XParam.test = 8;

	// Enforce GPU/CPU
	XParam.GPUDEVICE = gpu;

	//Bottom friction
	XParam.frictionmodel = -1; //Manning model
	XParam.cf = 0.009; //n in m^(-1/3)s

	std::string outvi[16] = { "zb","h","zs","u","v","Fqux","Fqvx","Fquy","Fqvy", "Fhu", "Fhv", "dh", "dhu", "dhv", "Su", "Sv" };

	std::vector<std::string> outv;

	for (int nv = 0; nv < 15; nv++)
	{
		outv.push_back(outvi[nv]);
	}

	XParam.outvars = outv;

	// create Model setup
	Model<T> XModel;
	Model<T> XModel_g;

	Forcing<float> XForcing;

	StaticForcingP<float> bathy;

	XForcing.Bathy.push_back(bathy);

	// initialise forcing bathymetry to 0
	XForcing.Bathy[0].xo = -1.0;
	XForcing.Bathy[0].yo = -1.0;
	XForcing.Bathy[0].xmax = 28.0;
	XForcing.Bathy[0].ymax = 1.0;
	XForcing.Bathy[0].dx = 0.1;
	XForcing.Bathy[0].nx = ftoi((XForcing.Bathy[0].xmax - XForcing.Bathy[0].xo) / XForcing.Bathy[0].dx + 1);
	XForcing.Bathy[0].ny = ftoi((XForcing.Bathy[0].ymax - XForcing.Bathy[0].yo) / XForcing.Bathy[0].dx + 1);


	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.Bathy[0].val);

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = -10.0;
			if (i < (9 / XForcing.Bathy[0].dx + 1))
			{
				XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = T(0.2 + (9.0 - i * XForcing.Bathy[0].dx) * 2.0 / 100.0);
			}
			else if (i < (17 / XForcing.Bathy[0].dx + 1))
			{
				XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = T(0.08 + (17.0 - i * XForcing.Bathy[0].dx) * 1.5 / 100.0);
			}
			else if (i < (25 / XForcing.Bathy[0].dx + 1))
			{
				XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = T(0.0 + (25.0 - i * XForcing.Bathy[0].dx) * 1.0 / 100.0);
			}
		}
	}

	// Add wall boundary conditions but at the bottom of the slope
	//XForcing.right.type = 0;
	XForcing.left.type = 0;
	//XForcing.top.type = 0;
	//XForcing.bot.type = 0;

	//Value definition for surface rain fall
	T r1 = T(3888.0); // mm/hr
	T r2 = T(2296.8); //mm/hr
	T r3 = T(2880.0); //mm/hr
	T Q = (r1 + r2 + r3) / 3;
	//TheoryInput = Q * XParam.outputtimestep / T(1000.0) / T(3600.0) * Surf; //m3/s
	//printf("# Theoretical volume of water input during the simulation in m3: %f , from a mean rain input of: %f mm/hr.\n", TheoryInput, Q);
	double eps = 0.0001;

	// Create the rain forcing file
	if (dimf == 1)
	{
		//Create a temporary file with rain fluxes for uniform rain
		std::ofstream rain_file(
			"testrain.tmp", std::ios_base::out | std::ios_base::trunc);
		rain_file << "0.0 " + std::to_string(Q) << std::endl;
		rain_file << std::to_string(rainDuration) + " " + std::to_string(Q) << std::endl;
		rain_file << std::to_string(rainDuration + eps) + " 0.0" << std::endl;
		rain_file << std::to_string(rainDuration + 360000) + " 0.0" << std::endl;
		rain_file.close(); //destructor implicitly does it

		XForcing.Rain.inputfile = "testrain.tmp";
		XForcing.Rain.uniform = true;

		// Reading rain forcing from file for CPU and uniform rain
		XForcing.Rain.unidata = readINfileUNI(XForcing.Rain.inputfile);
		printf("1D rain forcing read\n");
	}
	else //non-uniform forcing
	{
		XForcing.Rain.uniform = false;

		//X Y variables

		xRain = (double*)malloc(sizeof(double) * NX);
		yRain = (double*)malloc(sizeof(double) * NY);

		for (int i = 0; i < NX; i++) { xRain[i] = -0.005 + 0.01 * i; }
		for (int j = 0; j < NY; j++) { yRain[j] = -0.01 + 0.01 * j; }

		NT = 601;
		tRain = (double*)malloc(sizeof(double) * NT);
		for (int tt = 0; tt < NT; tt++) { tRain[tt] = XParam.endtime / (NT - 1) * tt; }

		rainForcing = (double*)malloc(sizeof(double) * NT * NY * NX);

		//Create a non-uniform time-variable rain forcing
		if (dimf == 3)
		{
			//Create the rain forcing:
			for (k = 0; k < NT; k++)
			{
				for (int j = 0; j < NY; j++)
				{
					for (int i = 0; i < NX; i++)
					{
						if (tRain[k] < rainDuration+eps)
						{
							if (xRain[i] < 8.0)
							{
								rainForcing[k * (NX * NY) + j * NX + i] = r1;
							}
							else if (xRain[i] < 16.0)
							{
								rainForcing[k * (NX * NY) + j * NX + i] = r2;
							}
							else
							{
								rainForcing[k * (NX * NY) + j * NX + i] = r3;
							}
						}
						else
						{
							rainForcing[k * (NX * NY) + i * NY + j] = 0.0;
						}
					}
				}
			}

			//Write the netcdf file
			create3dnc("rainTemp.nc", NX, NY, NT, xRain, yRain, tRain, rainForcing, "myrainforcing");

			printf("non-uniform forcing\n");

			//End creation of the nc file for rain forcing
		}
		//Create a uniform time-variable rain forcing using a map forcing (nc file)
		else if (dimf == 31)
		{
			//Create the rain forcing:
			for (k = 0; k < NT; k++)
			{
				for (int j = 0; j < NY; j++)
				{
					for (int i = 0; i < NX; i++)
					{
						if (tRain[k] < rainDuration + eps)
						{
							rainForcing[k * (NX * NY) + j * NX + i] = Q;
						}
						else
						{
							rainForcing[k * (NX * NY) + i * NY + j] = 0.0;
						}
					}
				}
			}

			//Write the netcdf file
			create3dnc("rainTemp.nc", NX, NY, NT, xRain, yRain, tRain, rainForcing, "myrainforcing");

			printf("non-uniform forcing 31\n");
			//End creation of the nc file for rain forcing
		}
		/*
		//2D forcing (map without time variation is not working)
		else if (dimf == 2)//dimf==2 for rain forcing 
		{

			//Create a non-uniform time-constant rain forcing 
			rainForcing = (double*)malloc(sizeof(double) * NY * NX);

			//Create the rain forcing:

			for (int j = 0; j < NY; j++)
			{
				for (int i = 0; i < NX; i++)
				{

					if (xRain[i] < 8.0)
					{
						rainForcing[j * NX + i] = r1;
					}
					else if (xRain[i] < 16.0)
					{
						rainForcing[j * NX + i] = r2;
					}
					else
					{
						rainForcing[j * NX + i] = r3;
					}

				}
			}

			create2dnc("rainTempt.nc", NX, NY, xRain, yRain, rainForcing, "myrainforcing");

			//End creation of the nc file for rain forcing
		}
		*/
		else { printf("Error in rain forcing dimension (should be in [1,3,31])\n"); }

		//Reading non-unform forcing
		bool gpgpu = 0;
		if (XParam.GPUDEVICE != -1)
		{
			gpgpu = 1;
		}

		XForcing.Rain = readfileinfo("rainTemp.nc", XForcing.Rain);
		XForcing.Rain.uniform = 0;
		XForcing.Rain.varname = "myrainforcing";
		

		InitDynforcing(gpgpu, XParam.totaltime, XForcing.Rain);

		//readDynforcing(gpgpu, XParam.totaltime, XForcing.Rain);


		free(rainForcing);
		free(xRain);
		free(yRain);
		free(tRain);
	}


	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);

	SetupGPU(XParam, XModel, XForcing, XModel_g);

	log("Initialising model main loop");

	Loop<T> XLoop = InitLoop(XParam, XModel);

	//Define some useful variables 
	Initmeanmax(XParam, XLoop, XModel, XModel_g);


	log("\t\tCompleted");
	log("Model Running...");
	std::vector<float> Flux;

	while (XLoop.totaltime < XParam.endtime)
	{

		// Calculate Forcing at this step
		updateforcing(XParam, XLoop, XForcing);

		// Core engine
		if (XParam.GPUDEVICE >= 0)
		{
			FlowGPU(XParam, XLoop, XForcing, XModel_g);
		}
		else
		{
			FlowCPU(XParam, XLoop, XForcing, XModel);
		}

		// Time keeping
		XLoop.totaltime = XLoop.totaltime + XLoop.dt;
		//printf("\tTime = %f \n", XLoop.totaltime);

		//if Toutput, calculate the flux at x=24m;


		// Getting the coordinate for the flux calculation
		int bl, ixx, ibl, ix, ib;
		T dist = T(1000000000.0);
		for (ibl = 0; ibl < XParam.nblk; ibl++)
		{
			ib = XModel.blocks.active[ibl];
			delta = calcres(T(XParam.dx), XModel.blocks.level[ib]);
			for (ix = 0; ix < XParam.blkwidth; ix++)
			{
				//n = memloc(XParam, ix, 0, ib);
				if (abs(XModel.blocks.xo[ibl] + ix * delta - 24.0) < dist)
				{
					ixx = ix;
					bl = ibl;
					dist = T(abs(XModel.blocks.xo[ibl] + ix * delta - 24.0));
				}
			}
		}

		if (XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001) && XParam.outputtimestep > 0.0)
		{
			T finalFlux = T(0.0);
			if (XParam.GPUDEVICE >= 0)
			{
				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					CUDA_CHECK(hipMemcpy(XModel.OutputVarMap[XParam.outvars[ivar]], XModel_g.OutputVarMap[XParam.outvars[ivar]], XParam.nblkmem * XParam.blksize * sizeof(T), hipMemcpyDeviceToHost));
				}
			}

			Save2Netcdf(XParam, XLoop, XModel);
			

			//Calculation of the flux at the bottom of the slope (x=24m)
			ib = XModel.blocks.active[bl];
			delta = calcres(T(XParam.dx), XModel.blocks.level[ib]);

			for (int iy = 0; iy < XParam.blkwidth; iy++)
			{
				int n = memloc(XParam, ixx, iy, ib);
				finalFlux = finalFlux + XModel.evolv.h[n] * XModel.evolv.u[n] * delta;
			}
			finalFlux = finalFlux / float(XParam.ymax - XParam.yo)*100.0f*100.0f;
			Flux.push_back(finalFlux);
			XLoop.nextoutputtime = XLoop.nextoutputtime + XParam.outputtimestep;
			printf("\tTime = %f, Flux at bottom end of slope : %f \n", XLoop.totaltime, finalFlux);
		}
	}
	/*
	for (int n = 0; n < Flux.size(); n++)
	{
		printf("Flux at %i : %f \n", n, Flux[n]);
	}
	*/

	return Flux;
}
template std::vector<float> Raintestmap<float>(int gpu, int dimf, float Zsinit);
template std::vector<float> Raintestmap<double>(int gpu, int dimf, double Zsinit);


/*! \fn bool testzoneOutDef = ZoneOutputTest(int nzones, T zsinit)
*
* This function test the zoned output for a basic configuration
*/
template <class T> bool ZoneOutputTest(int nzones, T zsinit)
//template bool ZoneOutputTest<float>(int nzones, float zsinit);
{
	log("#####");

	Param XParam;
	Forcing<float> XForcing; 

	
	if (nzones  == 3)
	{
		// read param file
		readforcing(XParam, XForcing);
		outzoneP zone;
		zone.outname = "whole.nc";
		zone.xstart = -10;
		zone.xend = 10;
		zone.ystart = -10;
		zone.yend = 10;
		XParam.outzone.push_back(zone);
		zone.outname = "zoomed.nc";
		zone.xstart =1;
		zone.xend =2;
		zone.ystart = -2;
		zone.yend = 2;
		XParam.outzone.push_back(zone);
		zone.outname = "zoomed2.nc";
		zone.xstart = -2;
		zone.xend = 2;
		zone.ystart = -4;
		zone.yend = 2;
		XParam.outzone.push_back(zone);
	}

	// initialise domain and required resolution
	XParam.dx = 1.0 / ((1 << 6)); //1<<8  = 2^8
	XParam.xo = -5;
	XParam.yo = -5;
	XParam.xmax = 5;
	XParam.ymax = 5;

	XParam.initlevel = 0;
	XParam.minlevel = -1;
	XParam.maxlevel = 1;

	XParam.zsinit = zsinit;
	//XParam.zsoffset = 0.0;

	//Output times for comparisons
	XParam.endtime = 1.0;
	XParam.outputtimestep = 0.5;

	XParam.smallnc = 0;

	XParam.cf = 0.0001;
	XParam.frictionmodel = 1;

	//Specification of the test
	//XParam.test = 7;
	XParam.rainforcing = true;

	// Enforce GPU/CPU
	//XParam.GPUDEVICE = gpu;
	//XParam.rainbnd = true;

	// create Model setup
	Model<T> XModel;
	Model<T> XModel_g;

	StaticForcingP<float> bathy;

	XForcing.Bathy.push_back(bathy);

	// initialise forcing bathymetry to a central hill
	XForcing.Bathy[0].xo = -10.0;
	XForcing.Bathy[0].yo = -10.0;
	XForcing.Bathy[0].xmax = 10.0;
	XForcing.Bathy[0].ymax = 10.0;
	XForcing.Bathy[0].nx = 501;
	XForcing.Bathy[0].ny = 501;

	XForcing.Bathy[0].dx = 0.1;

	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.Bathy[0].val);
	
	float rs, x, y, r, hm;
	rs = 20; //hill radio 
	hm = 5; //hill top
	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			x = XForcing.Bathy[0].xo + i * XForcing.Bathy[0].dx;
			y = XForcing.Bathy[0].yo + j * XForcing.Bathy[0].dx;
			r = sqrt(x * x + y * y);
			if (r < rs)
			{
				XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = hm*(1-r/rs);
			}
			if (x < -4.7 | x > 4.7 | y < -4.7 | y > 4.7)
			{
				XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = 10;
			}
		}
	}

	//Adaptation
	XParam.AdaptCrit = "Targetlevel";
	
	StaticForcingP<int> Target;
	XForcing.targetadapt.push_back(Target);

	XForcing.targetadapt[0].xo = -10;
	XForcing.targetadapt[0].yo = -10;
	XForcing.targetadapt[0].xmax = 10.0;
	XForcing.targetadapt[0].ymax = 10.0;
	XForcing.targetadapt[0].nx = 501;
	XForcing.targetadapt[0].ny = 501;

	XForcing.targetadapt[0].dx = 0.1;

	AllocateCPU(XForcing.targetadapt[0].nx, XForcing.targetadapt[0].ny, XForcing.targetadapt[0].val);

	for (int j = 0; j < XForcing.targetadapt[0].ny; j++)
	{
		for (int i = 0; i < XForcing.targetadapt[0].nx; i++)
		{
			x = XForcing.targetadapt[0].xo + i * XForcing.targetadapt[0].dx;
			y = XForcing.targetadapt[0].yo + j * XForcing.targetadapt[0].dx;
			if (x < 0.0)
			{
				XForcing.targetadapt[0].val[i + j * XForcing.targetadapt[0].nx] = -1;
			}
			else
			{
				if (y < 0.0)
				{
					XForcing.targetadapt[0].val[i + j * XForcing.targetadapt[0].nx] = 0;
				}
				else
				{
					XForcing.targetadapt[0].val[i + j * XForcing.targetadapt[0].nx] = 1;
				}
			}
		}
	}

	// Add wall boundary conditions
	XForcing.right.type = 0;
	XForcing.left.type = 0;
	XForcing.top.type = 0;
	XForcing.bot.type = 0;


	//Create a temporary file with river fluxes
	float Q = 1;
	std::ofstream river_file(
		"testriver.tmp", std::ios_base::out | std::ios_base::trunc);
	river_file << "0.0 " + std::to_string(Q) << std::endl;
	river_file << "3600.0 " + std::to_string(Q) << std::endl;
	river_file.close(); //destructor implicitly does it

	River thisriver;
	thisriver.Riverflowfile = "testriver.tmp";
	thisriver.xstart = -0.2;
	thisriver.xend = 0.2;
	thisriver.ystart = -0.2;
	thisriver.yend = 0.2;

	XForcing.rivers.push_back(thisriver);


	XForcing.rivers[0].flowinput = readFlowfile(XForcing.rivers[0].Riverflowfile);


	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);

	SetupGPU(XParam, XModel, XForcing, XModel_g);

	MainLoop(XParam, XForcing, XModel, XModel_g);

	//Test if file exist and can be open:
	int error = 1;
	std::vector<int> observedSize{ 473251462,23304761,130802886 };
	for (int o = 0; o < XModel.blocks.outZone.size(); o++)
	{
		std::ifstream fs(XModel.blocks.outZone[o].outname);
		if (fs.fail()) 
		{
			error++;
		}
		else
		{
			//Calculate the size of the file in bytes
			std::ifstream in_file(XModel.blocks.outZone[o].outname, std::ios::binary);
			in_file.seekg(0, std::ios::end);
			int file_size = in_file.tellg();
			printf("sizes : %i in bytes\n", file_size);
			error = error * (observedSize[o] / file_size);
		}
	}

	bool modelgood = (1-abs(error)) < 0.05;

	//log("#####");
	return modelgood;
}
template bool ZoneOutputTest<float>(int nzones, float zsinit);
template bool ZoneOutputTest<double>(int nzones, double zsinit);


/*! \fn bool testzoneOutDef = ZoneOutputTest(int nzones, T zsinit)
*
* This function test the spped and accuracy of a new gradient function
* gradient are only calculated for zb but assigned to different gradient variable for storage
*/
template <class T> int TestGradientSpeed(Param XParam, Model<T> XModel, Model<T> XModel_g)
{
	//
	int fastest = 1;
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	// for flux reconstruction the loop overlap the right(or top for the y direction) halo
	dim3 blockDimX2(XParam.blkwidth + XParam.halowidth*2, XParam.blkwidth + XParam.halowidth * 2, 1);



	// Allocate CUDA events that we'll use for timing
	hipEvent_t startA, startB, startC, startG, startGnew;
	hipEvent_t stopA, stopB, stopC, stopG, stopGnew;

	fillHalo(XParam, XModel.blocks, XModel.evolv, XModel.zb);

	std::thread t0(&gradientC<T>, XParam, XModel.blocks, XModel.zb, XModel.grad.dzbdx, XModel.grad.dzbdy);
	t0.join();


	Loop<T> XLoop;
	// GPU stuff


	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	XLoop.nextoutputtime = 3600.0;


	hipEventCreate(&startA);

	
	hipEventCreate(&stopA);

	// Record the start event
	hipEventRecord(startA, NULL);
	gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel_g.blocks.active, XModel_g.blocks.level, (T)XParam.theta, (T)XParam.dx, XModel_g.zb, XModel_g.grad.dzbdx, XModel_g.grad.dzbdy);
	CUDA_CHECK(hipDeviceSynchronize());

	// Record the stop event
	hipEventRecord(stopA, NULL);

	// Wait for the stop event to complete
	hipEventSynchronize(stopA);

	float msecTotalGrad = 0.0f;
	hipEventElapsedTime(&msecTotalGrad, startA, stopA);

	hipEventDestroy(startA);
	hipEventDestroy(stopA);


	hipEventCreate(&startB);


	hipEventCreate(&stopB);

	// Record the start event
	hipEventRecord(startB, NULL);
	gradientSM << < gridDim, blockDim >> > (XParam.halowidth, XModel_g.blocks.active, XModel_g.blocks.level, (T)XParam.theta, (T)XParam.dx, XModel_g.zb, XModel_g.grad.dzsdx, XModel_g.grad.dzsdy);
	CUDA_CHECK(hipDeviceSynchronize());

	// Record the stop event
	hipEventRecord(stopB, NULL);

	// Wait for the stop event to complete
	hipEventSynchronize(stopB);

	float msecTotalSM = 0.0f;
	hipEventElapsedTime(&msecTotalSM, startB, stopB);

	hipEventDestroy(startB);
	hipEventDestroy(stopB);


	hipEventCreate(&startC);


	hipEventCreate(&stopC);

	// Record the start event
	hipEventRecord(startC, NULL);
	gradientSMC << < gridDim, blockDim >> > (XParam.halowidth, XModel_g.blocks.active, XModel_g.blocks.level, (T)XParam.theta, (T)XParam.dx, XModel_g.zb, XModel_g.grad.dhdx, XModel_g.grad.dhdy);
	CUDA_CHECK(hipDeviceSynchronize());

	// Record the stop event
	hipEventRecord(stopC, NULL);

	// Wait for the stop event to complete
	hipEventSynchronize(stopC);

	float msecTotalSMB = 0.0f;
	hipEventElapsedTime(&msecTotalSMB, startC, stopC);

	hipEventDestroy(startC);
	hipEventDestroy(stopC);


	

	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dudx, XModel_g.grad.dzbdx);
	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dudy, XModel_g.grad.dzbdy);

	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dzsdx, XModel_g.grad.dzsdx);
	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dzsdy, XModel_g.grad.dzsdy);

	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dhdx, XModel_g.grad.dhdx);
	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dhdy, XModel_g.grad.dhdy);

	printf("Runtime : normal=%f, shared mem=%f, SharedmemB=%f in msec\n", msecTotalGrad, msecTotalSM, msecTotalSMB);

	/*
	creatncfileBUQ(XParam, XModel.blocks);

	std::vector<std::string> varlist = { "zb", "dzbdx", "dzbdy" };

	for (int ivar = 0; ivar < varlist.size(); ivar++)
	{
		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, varlist[ivar], 3, XModel.OutputVarMap[varlist[ivar]], XModel.blocks.outZone[0]);
	}

	diffArray(XParam, XLoop, XModel.blocks, "SMdx", false, XModel.grad.dzbdx, XModel_g.grad.dzsdx, XModel.time.arrmax, XModel.grad.dzsdx);
	

	diffArray(XParam, XLoop, XModel.blocks, "SMBdx", false, XModel.grad.dzbdx, XModel_g.grad.dhdx, XModel.time.arrmax, XModel.grad.dhdx);

	diffArray(XParam, XLoop, XModel.blocks, "SMBdy", false, XModel.grad.dzbdy, XModel_g.grad.dhdy, XModel.time.arrmax, XModel.grad.dhdy);
	diffArray(XParam, XLoop, XModel.blocks, "SMdy", false, XModel.grad.dzbdy, XModel_g.grad.dzsdy, XModel.time.arrmax, XModel.grad.dzsdy);
	*/
	T maxdiffx, maxdiffy;
	maxdiffx = T(0.0);
	maxdiffy = T(0.0);
	T maxdiffsmx, maxdiffsmy;
	maxdiffsmx = T(0.0);
	maxdiffsmy = T(0.0);
	T maxdiffsmbx, maxdiffsmby;
	maxdiffsmbx = T(0.0);
	maxdiffsmby = T(0.0);
	T diffsm, diffsmb;

	

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				diffsm = abs(XModel.grad.dzbdx[i] - XModel.grad.dudx[i]);

				maxdiffx = max(maxdiffx, diffsm);

				diffsm = abs(XModel.grad.dzbdy[i] - XModel.grad.dudy[i]);

				maxdiffx = max(maxdiffx, diffsm);

				diffsm = abs(XModel.grad.dzbdx[i] - XModel.grad.dzsdx[i]);
				
				maxdiffsmx = max(maxdiffsmx, diffsm);

				diffsm = abs(XModel.grad.dzbdy[i] - XModel.grad.dzsdy[i]);

				maxdiffsmy = max(maxdiffsmy, diffsm);

				diffsm = abs(XModel.grad.dzbdx[i] - XModel.grad.dhdx[i]);
				maxdiffsmbx = max(maxdiffsmbx, diffsm);

				diffsm =  abs(XModel.grad.dzbdy[i] - XModel.grad.dhdy[i]);
				maxdiffsmby = max(maxdiffsmby, diffsm);
				//
			}
		}
	}

	
	printf("max error : normx=%e, normy=%e, smx=%e, smy=%e,  smbx=%e, smby=%e in m\n", maxdiffx, maxdiffy, maxdiffsmx, maxdiffsmy, maxdiffsmbx, maxdiffsmby);


	gradientCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.zb);


	hipEventCreate(&startG);


	hipEventCreate(&stopG);

	hipEventRecord(startG, NULL);
	gradientGPU(XParam, XModel_g.blocks, XModel_g.evolv, XModel_g.grad, XModel_g.zb);
	hipEventRecord(stopG, NULL);

	// Wait for the stop event to complete
	hipEventSynchronize(stopG);

	float msecTotalG = 0.0f;
	hipEventElapsedTime(&msecTotalG, startG, stopG);

	hipEventDestroy(startG);
	hipEventDestroy(stopG);

	CompareCPUvsGPU(XParam, XModel, XModel_g, { "dhdx","dhdy", "dzsdx","dzsdy","dudx","dudy","dvdx","dvdy" }, true);

	//CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dzbdx, XModel_g.grad.dzbdx);
	//CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dzbdy, XModel_g.grad.dzbdy);

	//CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dzsdx, XModel_g.grad.dzsdx);
	//CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dzsdy, XModel_g.grad.dzsdy);

	//CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dhdx, XModel_g.grad.dhdx);
	//CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.grad.dhdy, XModel_g.grad.dhdy);

	hipEventCreate(&startGnew);


	hipEventCreate(&stopGnew);

	hipEventRecord(startGnew, NULL);
	gradientGPUnew(XParam, XModel_g.blocks, XModel_g.evolv, XModel_g.grad, XModel_g.zb);
	hipEventRecord(stopGnew, NULL);

	// Wait for the stop event to complete
	hipEventSynchronize(stopGnew);

	float msecTotalGnew = 0.0f;
	hipEventElapsedTime(&msecTotalGnew, startGnew, stopGnew);

	hipEventDestroy(startGnew);
	hipEventDestroy(stopGnew);

	CompareCPUvsGPU(XParam, XModel, XModel_g, { "dhdx","dhdy", "dzsdx","dzsdy","dudx","dudy","dvdx","dvdy" }, true);

	printf("Runtime : old gradient=%f, new Gradient=%f in msec\n", msecTotalG, msecTotalGnew);
	
	return fastest;

}

/*! \fn bool testzoneOutDef = ZoneOutputTest(int nzones, T zsinit)
*
* This function test the spped and accuracy of a new gradient function
* gradient are only calculated for zb but assigned to different gradient variable for storage
*/
template <class T> bool TestHaloSpeed(Param XParam, Model<T> XModel, Model<T> XModel_g)
{
	Forcing<float> XForcing;

	XForcing = MakValleyBathy(XParam, T(0.4), true, true);

	float maxtopo = std::numeric_limits<float>::min();
	float mintopo = std::numeric_limits<float>::max();

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			maxtopo = max(XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx], maxtopo);
			mintopo = min(XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx], mintopo);
		}
	}

	// Overrule whatever may be set in the param file
	XParam.xmax = XForcing.Bathy[0].xmax;
	XParam.ymax = XForcing.Bathy[0].ymax;
	XParam.xo = XForcing.Bathy[0].xo;
	XParam.yo = XForcing.Bathy[0].yo;

	XParam.dx = XForcing.Bathy[0].dx;

	XParam.zsinit = mintopo + 0.5;// Had a small amount of water to avoid a huge first step that would surely break the setup
	XParam.endtime = 20.0;

	XParam.outputtimestep = XParam.endtime;

	XParam.minlevel = 0;
	XParam.maxlevel = 1;
	XParam.initlevel = 0;

	//coarse to fine
	// Change arg 1 and 2 if the slope is changed
	XParam.AdaptCrit = "Inrange";
	XParam.Adapt_arg1 = "0.0";
	XParam.Adapt_arg2 = "2.0";
	XParam.Adapt_arg3 = "zb";

	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);

	SetupGPU(XParam, XModel, XForcing, XModel_g);

	


	// Copy zs from CPU to GPU ... again
	CopytoGPU(XParam.nblkmem, XParam.blksize, XModel.evolv.zs, XModel_g.evolv_o.zs);
	CopytoGPU(XParam.nblkmem, XParam.blksize, XModel.evolv.zs, XModel_g.evolv.zs);

	hipStream_t streams[2];
	CUDA_CHECK(hipStreamCreate(&streams[0]));
	CUDA_CHECK(hipStreamCreate(&streams[1]));


	fillHaloC(XParam, XModel.blocks, XModel.evolv.zs);
	fillHaloGPU(XParam, XModel_g.blocks, streams[0], XModel_g.evolv.zs);
	fillHaloGPUnew(XParam, XModel_g.blocks, streams[1], XModel_g.evolv_o.zs);

	CUDA_CHECK(hipDeviceSynchronize());

	hipStreamDestroy(streams[0]);
	hipStreamDestroy(streams[1]);


	//CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.evolv.u, XModel_g.evolv.zs);
	//CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, XModel.evolv.v, XModel_g.evolvo.zs);

	diffArray(XParam, XModel.blocks, "GPU_old", true, XModel.evolv.zs, XModel_g.evolv.zs, XModel.evolv.u, XModel.evolv_o.u);
	diffArray(XParam, XModel.blocks, "GPU_new", true, XModel.evolv.zs, XModel_g.evolv_o.zs, XModel.evolv.v, XModel.evolv_o.v);

	return true;
}

template <class T> int TestInstability(Param XParam, Model<T> XModel, Model<T> XModel_g)
{
	Forcing<float> XForcing;

	XForcing = MakValleyBathy(XParam, T(0.4), true, true);

	XParam.conserveElevation = true;

	float maxtopo = std::numeric_limits<float>::min();
	float mintopo = std::numeric_limits<float>::max();

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			maxtopo = max(XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx], maxtopo);
			mintopo = min(XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx], mintopo);
		}
	}

	// Overrule whatever may be set in the param file
	XParam.xmax = XForcing.Bathy[0].xmax;
	XParam.ymax = XForcing.Bathy[0].ymax;
	XParam.xo = XForcing.Bathy[0].xo;
	XParam.yo = XForcing.Bathy[0].yo;

	XParam.dx = XForcing.Bathy[0].dx;

	XParam.zsinit = mintopo + 6.9;// Had a water level so that the wet and dry affects the 
	XParam.endtime = 20.0;

	XParam.outputtimestep = XParam.endtime;

	XParam.minlevel = 0;
	XParam.maxlevel = 2;
	XParam.initlevel = 0;

	// coarse to fine
	// Change arg 1 and 2 if the slope is changed
	XParam.AdaptCrit = "Targetlevel";
	XParam.Adapt_arg1 = "";
	XParam.Adapt_arg2 = "";
	XParam.Adapt_arg3 = "";

	StaticForcingP<int> targetlevel;
	XForcing.targetadapt.push_back(targetlevel);

	XForcing.targetadapt[0].xo = 0.0;
	XForcing.targetadapt[0].yo = 0.0;

	XForcing.targetadapt[0].xmax = 31.0;
	XForcing.targetadapt[0].ymax = 31.0;
	XForcing.targetadapt[0].nx = 32;
	XForcing.targetadapt[0].ny = 32;

	XForcing.targetadapt[0].dx = 1.0;

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.targetadapt[0].val);

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			XForcing.targetadapt[0].val[i + j * XForcing.Bathy[0].nx] = 1;
		}
	}

	XForcing.targetadapt[0].val[12 + 12 * XForcing.Bathy[0].nx] = 2;


	// Setup Model(s)

	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);

	SetupGPU(XParam, XModel, XForcing, XModel_g);

	// Run first full step (i.e. 2 half steps)

	Loop<T> XLoop = InitLoop(XParam, XModel);
	
	//FlowCPU(XParam, XLoop, XForcing, XModel);
	HalfStepCPU(XParam, XLoop, XForcing, XModel);

	T maxu = std::numeric_limits<float>::min();
	T maxv = std::numeric_limits<float>::min();

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				maxu = max(maxu, abs(XModel.evolv.u[i]));
				maxv = max(maxv, abs(XModel.evolv.v[i]));
			}
		}
	}

	bool test = false;

	if (maxu > T(std::numeric_limits<T>::epsilon() * T(1000.0)) || maxv > T(std::numeric_limits<T>::epsilon() * T(1000.0)))
	{
		//test = true;
		XParam.outvars = { "zb","h","zs","u","v","Fqux","Fqvx","Fquy","Fqvy", "Fhu", "Fhv", "dh", "dhu", "dhv", "Su", "Sv","dhdx", "dhdy", "dzsdx", "dzsdy" };
		InitSave2Netcdf(XParam, XModel);

	}
	else
	{
		test = true;
	}


	return test;

}

template <class T> int TestFirsthalfstep(Param XParam, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
{
	
	// Setup Model(s)

	checkparamsanity(XParam, XForcing);

	InitMesh(XParam, XForcing, XModel);

	InitialConditions(XParam, XForcing, XModel);

	InitialAdaptation(XParam, XForcing, XModel);

	SetupGPU(XParam, XModel, XForcing, XModel_g);

	// Run first full step (i.e. 2 half steps)

	Loop<T> XLoop = InitLoop(XParam, XModel);

	//FlowCPU(XParam, XLoop, XForcing, XModel);
	HalfStepCPU(XParam, XLoop, XForcing, XModel);

	T maxu = std::numeric_limits<float>::min();
	T maxv = std::numeric_limits<float>::min();

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XModel.blocks.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				maxu = max(maxu, abs(XModel.evolv.u[i]));
				maxv = max(maxv, abs(XModel.evolv.v[i]));
			}
		}
	}

	bool test = false;

	//test = true;
	XParam.outvars = { "zb","h","zs","u","v","Fqux","Fqvx","Fquy","Fqvy", "Fhu", "Fhv", "dh", "dhu", "dhv", "Su", "Sv","dhdx", "dhdy", "dzsdx", "dzsdy" };
	InitSave2Netcdf(XParam, XModel);

}


template <class T> Forcing<float> MakValleyBathy(Param XParam, T slope, bool bottop, bool flip)
{
	//

	Forcing<float> XForcing;

	StaticForcingP<float> bathy;

	float* dummybathy;

	XForcing.Bathy.push_back(bathy);

	XForcing.Bathy[0].xo = 0.0;
	XForcing.Bathy[0].yo = 0.0;

	XForcing.Bathy[0].xmax = 31.0;
	XForcing.Bathy[0].ymax = 31.0;
	XForcing.Bathy[0].nx = 32;
	XForcing.Bathy[0].ny = 32;

	XForcing.Bathy[0].dx = 1.0;

	T x, y;
	T center = T(10.5);

	AllocateCPU(1, 1, XForcing.left.blks, XForcing.right.blks, XForcing.top.blks, XForcing.bot.blks);

	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, XForcing.Bathy[0].val);
	AllocateCPU(XForcing.Bathy[0].nx, XForcing.Bathy[0].ny, dummybathy);


	float maxtopo = std::numeric_limits<float>::min();
	float mintopo = std::numeric_limits<float>::max();
	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			x = T(XForcing.Bathy[0].xo + i * XForcing.Bathy[0].dx);
			y = T(XForcing.Bathy[0].yo + j * XForcing.Bathy[0].dx);


			dummybathy[i + j * XForcing.Bathy[0].nx] = float(ValleyBathy(y, x, slope, center));

			maxtopo = max(dummybathy[i + j * XForcing.Bathy[0].nx], maxtopo);


		}
	}

	// Make surrounding wall
	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{

		dummybathy[0 + j * XForcing.Bathy[0].nx] = maxtopo + 5.0f;
		dummybathy[1 + j * XForcing.Bathy[0].nx] = maxtopo + 5.0f;

		dummybathy[j + 0 * XForcing.Bathy[0].nx] = maxtopo + 5.0f;
		dummybathy[j + 1 * XForcing.Bathy[0].nx] = maxtopo + 5.0f;

		dummybathy[(XForcing.Bathy[0].nx - 1) + j * XForcing.Bathy[0].nx] = maxtopo + 5.0f;
		dummybathy[(XForcing.Bathy[0].nx - 2) + j * XForcing.Bathy[0].nx] = maxtopo + 5.0f;

		dummybathy[j + (XForcing.Bathy[0].ny - 1) * XForcing.Bathy[0].nx] = maxtopo + 5.0f;
		dummybathy[j + (XForcing.Bathy[0].ny - 2) * XForcing.Bathy[0].nx] = maxtopo + 5.0f;


	}

	// make a specially elevated spot 

	dummybathy[(XForcing.Bathy[0].nx - 1) + 0 * XForcing.Bathy[0].nx] = maxtopo + 10.0f;
	dummybathy[(XForcing.Bathy[0].nx - 2) + 0 * XForcing.Bathy[0].nx] = maxtopo + 10.0f;

	dummybathy[(XForcing.Bathy[0].nx - 1) + 1 * XForcing.Bathy[0].nx] = maxtopo + 10.0f;
	dummybathy[(XForcing.Bathy[0].nx - 2) + 1 * XForcing.Bathy[0].nx] = maxtopo + 10.0f;

	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			mintopo = min(dummybathy[i + j * XForcing.Bathy[0].nx], mintopo);
		}
	}

	// Flip or rotate the bathy according to what is requested
	for (int j = 0; j < XForcing.Bathy[0].ny; j++)
	{
		for (int i = 0; i < XForcing.Bathy[0].nx; i++)
		{
			if (!flip && !bottop)
			{
				XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = dummybathy[i + j * XForcing.Bathy[0].nx];
			}
			else if (flip && !bottop)
			{
				XForcing.Bathy[0].val[(XForcing.Bathy[0].nx - 1 - i) + j * XForcing.Bathy[0].nx] = dummybathy[i + j * XForcing.Bathy[0].nx];
			}
			else if (!flip && bottop)
			{
				XForcing.Bathy[0].val[i + j * XForcing.Bathy[0].nx] = dummybathy[j + i * XForcing.Bathy[0].nx];
			}
			else if (flip && bottop)
			{
				XForcing.Bathy[0].val[i + (XForcing.Bathy[0].ny - 1 - j) * XForcing.Bathy[0].nx] = dummybathy[j + i * XForcing.Bathy[0].nx];
			}
		}
	}

	free(dummybathy);

	return XForcing;

}


/*! \fn void alloc_init2Darray(float** arr, int NX, int NY)
* This function allocates and fills a 2D array with zero values
*
*
*/
void alloc_init2Darray(float** arr, int NX, int NY)
{
	int i, j;
	//Allocation
	arr = (float**)malloc(sizeof(float*) * NX);
	for (i = 0; i < NX; i++) {
		arr[i] = (float*)malloc(sizeof(float) * NY);
	}

	//arr = (int **)malloc(sizeof(int *) * NX);
	//for (i = 0; i < NX; i++) {
	//	arr[i] = (int *)malloc(sizeof(int) * NY);
	//}
	//Initialisation
	for (i = 0; i < NX; i++) {
		for (j = 0; j < NY; j++) {
			arr[i][j] = 0;
		}
	}
}

/*! \fn void init3Darray(float*** arr, int rows, int cols, int depths)
* This function fill a 3D array with zero values 
*
*
*/
void init3Darray(float*** arr, int rows, int cols, int depths)
{
	int i, j, k;
	for (i = 0; i < rows; i++) {
		for (j = 0; j < cols; j++) {
			for (k = 0; k < depths; k++)
			{
				arr[i][j][k] = 0;
			}
		}
	}
}

/*! \fn void fillrandom(Param XParam, BlockP<T> XBlock, T* z)
* This function fill an array with random values (0 - 1)
*
* 
*/
template <class T> void fillrandom(Param XParam, BlockP<T> XBlock, T* z)
{
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				z[n] = T(rand()) / T(RAND_MAX);
			}
		}
	}
}
template void fillrandom<float>(Param XParam, BlockP<float> XBlock, float* z);
template void fillrandom<double>(Param XParam, BlockP<double> XBlock, double* z);

/*! \fn void fillgauss(Param XParam, BlockP<T> XBlock, T amp, T* z)
* This function fill an array with a gaussian bump
* 
* borrowed/adapted from Basilisk test (?)
*/
template <class T> void fillgauss(Param XParam, BlockP<T> XBlock, T amp, T* z)
{
	T delta, x, y;
	T cc = T(0.05) * (XParam.xmax - XParam.xo);
	T xorigin = XParam.xo + T(0.5) * (XParam.xmax - XParam.xo);
	T yorigin = XParam.yo + T(0.5) * (XParam.ymax - XParam.yo);


	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XBlock.active[ibl];
		delta = calcres(T(XParam.dx), XBlock.level[ib]);


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				//
				int n = memloc(XParam, ix, iy, ib);
				x = T(XParam.xo + XBlock.xo[ib] + ix * delta);
				y = T(XParam.yo + XBlock.yo[ib] + iy * delta);
				z[n] = z[n] + amp * exp(T(-1.0) * T(((x - xorigin) * (x - xorigin) + (y - yorigin) * (y - yorigin)) / (2.0 * cc * cc)));


			}
		}
	}
}
template void fillgauss<float>(Param XParam, BlockP<float> XBlock, float amp, float* z);
template void fillgauss<double>(Param XParam, BlockP<double> XBlock, double amp, double* z);

/*! \fn TestingOutput(Param XParam, Model<T> XModel)
*
*	OUTDATED?
*/
template <class T>
void TestingOutput(Param XParam, Model<T> XModel)
{
	std::string outvar;

	Loop<T> XLoop;
	// GPU stuff


	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	XLoop.nextoutputtime = 0.2;

	Forcing<float> XForcing;

	//FlowCPU(XParam, XLoop, XModel);

	//log(std::to_string(XForcing.Bathy.val[50]));
	creatncfileBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, XModel.blocks.outZone[0]);
	outvar = "h";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar], XModel.blocks.outZone[0]);
	outvar = "u";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar], XModel.blocks.outZone[0]);
	outvar = "v";
	//copyID2var(XParam, XModel.blocks, XModel.OutputVarMap[outvar]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar], XModel.blocks.outZone[0]);
	outvar = "zb";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar], XModel.blocks.outZone[0]);
	outvar = "zs";
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.OutputVarMap[outvar], XModel.blocks.outZone[0]);


	FlowCPU(XParam, XLoop, XForcing, XModel);


	//outvar = "cf";
	//defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, 3, XModel.cf);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhdx", 3, XModel.grad.dhdx, XModel.blocks.outZone[0]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhdy", 3, XModel.grad.dhdy, XModel.blocks.outZone[0]);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fhv", 3, XModel.flux.Fhv, XModel.blocks.outZone[0]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fhu", 3, XModel.flux.Fhu, XModel.blocks.outZone[0]);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqux", 3, XModel.flux.Fqux, XModel.blocks.outZone[0]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fquy", 3, XModel.flux.Fquy, XModel.blocks.outZone[0]);

	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvx", 3, XModel.flux.Fqvx, XModel.blocks.outZone[0]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Fqvy", 3, XModel.flux.Fqvy, XModel.blocks.outZone[0]);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Su", 3, XModel.flux.Su, XModel.blocks.outZone[0]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "Sv", 3, XModel.flux.Sv, XModel.blocks.outZone[0]);


	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dh", 3, XModel.adv.dh, XModel.blocks.outZone[0]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhu", 3, XModel.adv.dhu, XModel.blocks.outZone[0]);
	defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, "dhv", 3, XModel.adv.dhv, XModel.blocks.outZone[0]);

	writenctimestep(XParam.outfile, XLoop.totaltime + XLoop.dt);


	outvar = "h";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar], XModel.blocks.outZone[0]);

	outvar = "zs";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar], XModel.blocks.outZone[0]);
	outvar = "u";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar], XModel.blocks.outZone[0]);
	outvar = "v";
	writencvarstepBUQ(XParam, 3, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, outvar, XModel.OutputVarMap[outvar], XModel.blocks.outZone[0]);

}

template void TestingOutput<float>(Param XParam, Model<float> XModel);
template void TestingOutput<double>(Param XParam, Model<double> XModel);

/*! \fn void copyID2var(Param XParam, BlockP<T> XBlock, T* z)
* This function copies block info to an output variable
* This function is somewhat useful when checking bugs in the mesh refinement or coarsening
* one needs to provide a pointer(z) allocated on the CPU to store the clockinfo
* This fonction only works on CPU
*
*/
template <class T> void copyID2var(Param XParam, BlockP<T> XBlock, T* z)
{
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				z[n] = T(ib);
			}
		}
	}

}

template void copyID2var<float>(Param XParam, BlockP<float> XBlock, float* z);
template void copyID2var<double>(Param XParam, BlockP<double> XBlock, double* z);


/*! \fn void copyBlockinfo2var(Param XParam, BlockP<T> XBlock, int* blkinfo, T* z)
* This function copies blick info to an output variable
* This function is somewhat useful when checking bugs in the mesh refinement or coarsening
* one needs to provide a pointer(z) allocated on the CPU to store the clockinfo
* This fonction only works on CPU
*
*/
template <class T> void copyBlockinfo2var(Param XParam, BlockP<T> XBlock, int* blkinfo, T* z)
{
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		int info = blkinfo[ib];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				z[n] = T(info);
			}
		}
	}

}
template void copyBlockinfo2var<float>(Param XParam, BlockP<float> XBlock, int* blkinfo, float* z);
template void copyBlockinfo2var<double>(Param XParam, BlockP<double> XBlock, int* blkinfo, double* z);


/*! \fn void CompareCPUvsGPU(Param XParam, Model<T> XModel, Model<T> XModel_g, std::vector<std::string> varlist, bool checkhalo)
* This function compares the Valiables in a CPU model and a GPU models
* This function is quite useful when checking both are identical enough
* one needs to provide a list (vector<string>) of variable to check
* 
*/
template <class T> void CompareCPUvsGPU(Param XParam, Model<T> XModel, Model<T> XModel_g, std::vector<std::string> varlist, bool checkhalo)
{
	Loop<T> XLoop;
	// GPU stuff


	XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.epsilon = std::numeric_limits<T>::epsilon();

	XLoop.totaltime = 0.0;

	XLoop.nextoutputtime = 3600.0;


	T* gpureceive;
	T* diff;

	//Forcing<float> XForcing;

	AllocateCPU(XParam.nblkmem, XParam.blksize, gpureceive);
	AllocateCPU(XParam.nblkmem, XParam.blksize, diff);


	//============================================
	// Compare gradients for evolving parameters

	// calculate difference
	//diffArray(XParam, XLoop, XModel.blocks, XModel.evolv.h, XModel_g.evolv.h, XModel.evolv_o.u);
	/*
	creatncfileBUQ(XParam, XModel.blocks);

	for (int ivar = 0; ivar < varlist.size(); ivar++)
	{
		defncvarBUQ(XParam, XModel.blocks.active, XModel.blocks.level, XModel.blocks.xo, XModel.blocks.yo, varlist[ivar], 3, XModel.OutputVarMap[varlist[ivar]], XModel.blocks.outZone[0]);
	}
	*/
	/*
	std::string varname = "dt";
	if (abs(dtgpu - dtcpu) < (XLoop.epsilon * 2))
	{
		log(varname + " PASS");
	}
	else
	{
		log(varname + " FAIL: " + " GPU(" + std::to_string(dtgpu) + ") - CPU("+std::to_string(dtcpu) +") =  difference: "+  std::to_string(abs(dtgpu - dtcpu)) + " Eps: " + std::to_string(XLoop.epsilon));

	}
	*/
	//Check variable
	for (int ivar = 0; ivar < varlist.size(); ivar++)
	{
		diffArray(XParam, XModel.blocks, varlist[ivar], checkhalo, XModel.OutputVarMap[varlist[ivar]], XModel_g.OutputVarMap[varlist[ivar]], gpureceive, diff);
	}



	free(gpureceive);
	free(diff);

}
template void CompareCPUvsGPU<float>(Param XParam, Model<float> XModel, Model<float> XModel_g, std::vector<std::string> varlist, bool checkhalo);
template void CompareCPUvsGPU<double>(Param XParam, Model<double> XModel, Model<double> XModel_g, std::vector<std::string> varlist, bool checkhalo);


/*! \fn void diffdh(Param XParam, BlockP<T> XBlock, T* input, T* output, T* shuffle)
* This function Calculates The difference in left and right flux terms.
* This function is quite useful when checking for Lake-at-Rest states
* This function requires a preallocated output and a shuffle (right side term) CPU pointers to save the result of teh calculation
*/
template <class T> void diffdh(Param XParam, BlockP<T> XBlock, T* input, T* output, T* shuffle)
{
	int iright, itop;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XBlock.active[ibl];


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				iright = memloc(XParam.halowidth, XParam.blkmemwidth, ix + 1, iy, ib);
				itop = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy + 1, ib);

				output[i] = input[iright] - input[i];
				shuffle[i] = input[iright];
			}
		}
	}
}

/*! \fn void diffSource(Param XParam, BlockP<T> XBlock, T* Fqux, T* Su, T* output)
* This function Calculate The source term of the equation. 
* This function is quite useful when checking for Lake-at-Rest states
* This function requires an outputCPU pointers to save the result of teh calculation
*/
template <class T> void diffSource(Param XParam, BlockP<T> XBlock, T* Fqux, T* Su, T* output)
{
	int iright, itop;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XBlock.active[ibl];


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				iright = memloc(XParam.halowidth, XParam.blkmemwidth, ix + 1, iy, ib);
				itop = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy + 1, ib);

				output[i] = Fqux[i] - Su[iright];
				//shuffle[i] = input[iright];
			}
		}
	}
}

/*! \fn void diffArray(Param XParam, Loop<T> XLoop, BlockP<T> XBlock, std::string varname, bool checkhalo, T* cpu, T* gpu, T* dummy, T* out)
* Calculate and output the difference between a CPU and a GPU array
* This function is quite usefull to spot inconsistencies between the GPU and CPU algorithmes.
* This function requires two (dummy and an output) CPU pointers to transition the GPU data on the CU RAM for comparison and saving to the disk
*/
template <class T> void diffArray(Param XParam, BlockP<T> XBlock, std::string varname, bool checkhalo, T* cpu, T* gpu, T* dummy, T* out)
{
	T diff, maxdiff, rmsdiff;
	unsigned int nit = 0;
	int ixmd, iymd, ibmd;
	//copy GPU back to the CPU (store in dummy)
	CopyGPUtoCPU(XParam.nblkmem, XParam.blksize, dummy, gpu);


	T hugeposval = std::numeric_limits<T>::max();
	T hugenegval = T(-1.0) * hugeposval;
	T epsilon = std::numeric_limits<T>::epsilon();

	rmsdiff = T(0.0);
	maxdiff = hugenegval;
	ixmd = 0;
	iymd = 0;
	ibmd = 0;

	// calculate difference
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		//printf("bl=%d\tblockxo[bl]=%f\tblockyo[bl]=%f\n", bl, blockxo[bl], blockyo[bl]);
		int ib = XBlock.active[ibl];

		int yst = checkhalo ? -1 : 0;
		int ynd = checkhalo ? XParam.blkwidth + 1 : XParam.blkwidth;

		int xst = checkhalo ? -1 : 0;
		int xnd = checkhalo ? XParam.blkwidth + 1 : XParam.blkwidth;

		for (int iy = yst; iy < ynd; iy++)
		{
			for (int ix = xst; ix < xnd; ix++)
			{
				int n = memloc(XParam, ix, iy, ib);
				diff = dummy[n] - cpu[n];

				if (abs(diff) >= maxdiff)
				{
					maxdiff = utils::max(abs(diff), maxdiff);
					ixmd = ix;
					iymd = iy;
					ibmd = ib;
				}

				rmsdiff = rmsdiff + utils::sq(diff);
				nit++;
				out[n] = diff;
			}
		}

	}


	rmsdiff = rmsdiff / nit;



	if (maxdiff <= T(10000.0) * (epsilon))
	{
		log(varname + " PASS");
	}
	else
	{
		creatncfileBUQ(XParam, XBlock);
		log(varname + " FAIL: " + " Max difference: " + std::to_string(maxdiff) + " (at: ix = " + std::to_string(ixmd) + " iy = " + std::to_string(iymd) + " ib = " + std::to_string(ibmd) + ") RMS difference: " + std::to_string(rmsdiff) + " Eps: " + std::to_string(epsilon));
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_CPU", 3, cpu, XBlock.outZone[0]);
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_GPU", 3, dummy, XBlock.outZone[0]);
		defncvarBUQ(XParam, XBlock.active, XBlock.level, XBlock.xo, XBlock.yo, varname + "_diff", 3, out, XBlock.outZone[0]);
	}




}


