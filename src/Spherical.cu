#include "hip/hip_runtime.h"
#include "Spherical.h"



/*! \fn T calcCM(T Radius, T delta, T yo, int iy)
* Scale factor for y face length (x face lengh scale is always 1 in spherical model assuming that lat long are entered)
* 
*/
template <class T> 
__host__ __device__ T calcCM(T Radius, T delta, T yo, int iy)
{
	T y = yo + (iy+0.5) * delta / Radius * T(180.0 / pi);
	// THis should be the y of the face so fo the v face you need to remove 0.5*delta

	T phi = y * T(pi / 180.0);

	T dphi = delta / (T(2.0 * Radius));// dy*0.5f*pi/180.0f;

	T cm = (sin(phi + dphi) - sin(phi - dphi)) / (2.0 * dphi);

	return cm;
}
template __host__ __device__ double calcCM(double Radius, double delta, double yo, int iy);
template __host__ __device__ float calcCM(float Radius, float delta, float yo, int iy);



template <class T> 
__host__ __device__  T calcFM(T Radius, T delta, T yo, T iy)
{
	T dy = delta / Radius * T(180.0 / pi);
	T y = yo + iy * dy;
	// THis should be the y of the face so fo the v face you need to remove 0.5*delta

	T phi = y * T(pi / 180.0);

	//T dphi = delta / (T(2.0 * Radius));// dy*0.5f*pi/180.0f;

	T fmu = cos(phi);

	return fmu;
}
template __host__ __device__ double calcFM(double Radius, double delta, double yo, double iy);
template __host__ __device__ float calcFM(float Radius, float delta, float yo, float iy);

/*! \fn  T haversin(T Radius, T lon1, T lat1, T lon2, T lat2)
* Classic haversin function 
* The function is too slow to use directly in BG_flood engine but is more usable (i.e. naive) for model setup
* 
*/
template <class T>
__host__ __device__  T haversin(T Radius, T lon1, T lat1, T lon2, T lat2)
{
	T phi1, phi2, dphi, dlbda, a, c;
	dphi = (lat2 - lat1) * T(pi / 180.0);
	dlbda = (lon2 -lon1) * T(pi / 180.0);

	phi1 = lat1 * T(pi / 180.0);
	phi2 = lat2 * T(pi / 180.0);

	T sindphid2 = sin(dphi / 2.0);
	T sindlbdad2 = sin(dlbda / 2.0);
	
	a = sindphid2 * sindphid2 + cos(phi1) * cos(phi2) * sindlbdad2 * sindlbdad2;

	c = 2.0 * atan2(sqrt(a), sqrt(1.0 - a));

	return Radius * c;

}

template <class T>
__host__ __device__  T spharea(T Radius, T lon, T lat, T dx)
{
	T lon1, lon2, lat1, lat2;
	lon1 = lon - T(0.5) * dx;
	lon2 = lon + T(0.5) * dx;

	lat1 = lat - T(0.5) * dx;
	lat2 = lat + T(0.5) * dx;

	T a, b, c;

	a = haversin(Radius, lon1, lat1, lon2, lat1);
	c = haversin(Radius, lon1, lat2, lon2, lat2);
	b = haversin(Radius, lon1, lat1, lon1, lat2);

	T Area = T(0.5) * (a * b + c * b);

	return Area;

}
template __host__ __device__  double spharea(double Radius, double lon, double lat, double dx);
template __host__ __device__  float spharea(float Radius, float lon, float lat, float dx);
