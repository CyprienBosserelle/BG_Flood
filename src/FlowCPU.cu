#include "hip/hip_runtime.h"
#include "FlowCPU.h"


template <class T> void FlowCPU(Param XParam, Loop<T>& XLoop,Forcing<float> XForcing, Model<T> XModel)
{
	//============================================
	// Predictor step in reimann solver
	//============================================

	//============================================
	//  Fill the halo for gradient reconstruction
	fillHalo(XParam, XModel.blocks, XModel.evolv, XModel.zb);

	//============================================
	// Reset DTmax
	InitArrayBUQ(XParam, XModel.blocks, XLoop.hugeposval, XModel.time.dtmax);
	
	//============================================
	// Calculate gradient for evolving parameters
	gradientCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.zb);

	//============================================
	// Flux and Source term reconstruction
	// X- direction
	if (XParam.engine == 1)
	{
		// X- direction
		UpdateButtingerXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);

		// Y- direction
		UpdateButtingerYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	}
	else if (XParam.engine == 2)
	{
		// X- direction
		updateKurgXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//AddSlopeSourceXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);

		// Y- direction
		updateKurgYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//AddSlopeSourceYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
	}
	else if (XParam.engine == 3)
	{
		// X- direction
		updateKurgXATMCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdx);
		//AddSlopeSourceXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);

		// Y- direction

		updateKurgYATMCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdy);
		//AddSlopeSourceYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);
	}
	

	//============================================
	// Fill Halo for flux from fine to coarse
	fillHalo(XParam, XModel.blocks, XModel.flux);
	
	//============================================
	// Reduce minimum timestep
	XLoop.dt = double(CalctimestepCPU(XParam,XLoop, XModel.blocks, XModel.time));
	XLoop.dtmax = XLoop.dt;
	XModel.time.dt = T(XLoop.dt);
	

	//============================================
	// Update advection terms (dh dhu dhv) 
	updateEVCPU(XParam, XModel.blocks, XModel.evolv, XModel.flux, XModel.adv);

	//============================================
	// Add forcing (Rain, Wind)
	//if (!XForcing.Rain.inputfile.empty())
	//{
	//	AddrainforcingCPU(XParam, XModel.blocks, XForcing.Rain, XModel.adv);
	//}
	if (!XForcing.UWind.inputfile.empty())//&& !XForcing.UWind.inputfile.empty()
	{
		AddwindforcingCPU(XParam, XModel.blocks, XForcing.UWind, XForcing.VWind, XModel.adv);
	}
	if (XForcing.rivers.size() > 0)
	{
		AddRiverForcing(XParam, XLoop, XForcing.rivers, XModel);
	}

	//============================================
	//Update evolving variable by 1/2 time step
	AdvkernelCPU(XParam, XModel.blocks, XModel.time.dt * T(0.5), XModel.zb, XModel.evolv, XModel.adv, XModel.evolv_o);
	
	
	//============================================
	// Corrector step in reimann solver
	//============================================

	//============================================
	//  Fill the halo for gradient reconstruction
	fillHalo(XParam, XModel.blocks, XModel.evolv_o,XModel.zb);

	//============================================
	// Calculate gradient for evolving parameters
	gradientCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.zb);

	//============================================
	// Flux and Source term reconstruction
	if (XParam.engine == 1)
	{

		// X- direction
		UpdateButtingerXCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//updateKurgXCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//AddSlopeSourceXCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);

		// Y- direction
		UpdateButtingerYCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//updateKurgYCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//AddSlopeSourceYCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);
	}
	else if (XParam.engine == 2)
	{
		// X- direction
		updateKurgXCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//AddSlopeSourceXCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);

		// Y- direction
		updateKurgYCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		//AddSlopeSourceYCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);
	}
	else if (XParam.engine == 3)
	{
		// X- direction
		//UpdateButtingerXCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		updateKurgXATMCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdx);
		//AddSlopeSourceXCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);

		// Y- direction
		//UpdateButtingerYCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
		updateKurgYATMCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb, XModel.Patm, XModel.datmpdy);
		//AddSlopeSourceYCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.grad, XModel.flux, XModel.zb);
	}

	//============================================
	// Fill Halo for flux from fine to coarse
	fillHalo(XParam, XModel.blocks, XModel.flux);

	//============================================
	// Update advection terms (dh dhu dhv) 
	updateEVCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.flux, XModel.adv);
	
	//============================================
	// Add forcing (Rain, Wind)
	//if (!XForcing.Rain.inputfile.empty())
	//{
	//	AddrainforcingCPU(XParam, XModel.blocks, XForcing.Rain, XModel.adv);
	//}
	if (!XForcing.UWind.inputfile.empty())//&& !XForcing.UWind.inputfile.empty()
	{
		AddwindforcingCPU(XParam, XModel.blocks, XForcing.UWind, XForcing.VWind, XModel.adv);
	}
	if (XForcing.rivers.size() > 0)
	{
		AddRiverForcing(XParam, XLoop, XForcing.rivers, XModel);
	}

	//============================================
	//Update evolving variable by 1 full time step
	AdvkernelCPU(XParam, XModel.blocks, XModel.time.dt, XModel.zb, XModel.evolv, XModel.adv, XModel.evolv_o);
	
	

	//============================================
	// Add bottom friction

	bottomfrictionCPU(XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv_o);
	//XiafrictionCPU(XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv, XModel.evolv_o);


	//============================================
	//Copy updated evolving variable back
	cleanupCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.evolv);

	if (!XForcing.Rain.inputfile.empty())
	{
		AddrainforcingImplicitCPU(XParam, XLoop, XModel.blocks, XForcing.Rain, XModel.evolv);
	}

	if (XParam.VelThreshold > 0.0)
	{
		TheresholdVelCPU(XParam, XModel.blocks, XModel.evolv);
		
	}


}
template void FlowCPU<float>(Param XParam, Loop<float>& XLoop, Forcing<float> XForcing, Model<float> XModel);
template void FlowCPU<double>(Param XParam, Loop<double>& XLoop, Forcing<float> XForcing, Model<double> XModel);




/*! \fn  void HalfStepCPU(Param XParam, Loop<T>& XLoop, Forcing<float> XForcing, Model<T> XModel)
* Debugging flow step
* This function was crated to debug the main engine of the model
*/
template <class T> void HalfStepCPU(Param XParam, Loop<T>& XLoop, Forcing<float> XForcing, Model<T> XModel)
{
	//============================================
	// Predictor step in reimann solver
	//============================================

	//============================================
	//  Fill the halo for gradient reconstruction
	fillHalo(XParam, XModel.blocks, XModel.evolv, XModel.zb);

	//============================================
	// Reset DTmax
	InitArrayBUQ(XParam, XModel.blocks, XLoop.hugeposval, XModel.time.dtmax);

	//============================================
	// Calculate gradient for evolving parameters
	gradientCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.zb);

	//============================================
	// Flux and Source term reconstruction
	// X- direction
	UpdateButtingerXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	//updateKurgXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	//AddSlopeSourceXCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);

	// Y- direction
	UpdateButtingerYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	//updateKurgYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	//AddSlopeSourceYCPU(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.zb);

	//============================================
	// Fill Halo for flux from fine to coarse
	fillHalo(XParam, XModel.blocks, XModel.flux);

	//============================================
	// Reduce minimum timestep
	// Make only a half max step
	//XLoop.dt = double(CalctimestepCPU(XParam, XLoop, XModel.blocks, XModel.time)) * T(0.5);
	XLoop.dt = double(timestepreductionCPU(XParam, XLoop, XModel.blocks, XModel.time)) * T(0.5);
	XLoop.dtmax = XLoop.dt;
	XModel.time.dt = T(XLoop.dt);

	//============================================
	// Update advection terms (dh dhu dhv) 
	updateEVCPU(XParam, XModel.blocks, XModel.evolv, XModel.flux, XModel.adv);

	//============================================
	// Add forcing (Rain, Wind)
	//if (!XForcing.Rain.inputfile.empty())
	//{
	//	AddrainforcingCPU(XParam, XModel.blocks, XForcing.Rain, XModel.adv);
	//}
	if (!XForcing.UWind.inputfile.empty())//&& !XForcing.UWind.inputfile.empty()
	{
		AddwindforcingCPU(XParam, XModel.blocks, XForcing.UWind, XForcing.VWind, XModel.adv);
	}
	if (XForcing.rivers.size() > 0)
	{
		AddRiverForcing(XParam, XLoop, XForcing.rivers, XModel);
	}

	//============================================
	//Update evolving variable by 1 time step
	AdvkernelCPU(XParam, XModel.blocks, XModel.time.dt , XModel.zb, XModel.evolv, XModel.adv, XModel.evolv_o);


	//============================================
	// Add bottom friction
	bottomfrictionCPU(XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv_o);
	//XiafrictionCPU(XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv, XModel.evolv_o);

	//============================================
	//Copy updated evolving variable back
	cleanupCPU(XParam, XModel.blocks, XModel.evolv_o, XModel.evolv);

	if (!XForcing.Rain.inputfile.empty())
	{
		AddrainforcingImplicitCPU(XParam, XLoop, XModel.blocks, XForcing.Rain, XModel.evolv);
	}
}
template void HalfStepCPU<float>(Param XParam, Loop<float>& XLoop, Forcing<float> XForcing, Model<float> XModel);
template void HalfStepCPU<double>(Param XParam, Loop<double>& XLoop, Forcing<float> XForcing, Model<double> XModel);

