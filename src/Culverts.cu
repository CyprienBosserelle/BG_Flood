#include "hip/hip_runtime.h"

#include "Culverts.h"


template <class T> __host__ void AddCulverts(Param XParam, double dt, std::vector<Culvert> XCulverts, Model<T> XModel)
{
	dim3 gridDimCulvert(XModel.bndblk.nblkculvert, 1, 1);
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	T Qmax, Vol1, Q;
	int cc;


	// Get the elevation/water column for each culvert edge and put it in the culvert structure (loop on concerned blocks)
	
	if (XParam.GPUDEVICE >= 0)
	{
		for (cc = 0; cc < XCulverts.size(); cc++)
		{
			GetCulvertElevGPU <<< gridDimCulvert, blockDim, 0 >>> (XParam, cc, XCulverts[cc], XModel.culvertsF, XModel.bndblk.culvert, XModel.blocks, XModel.evolv);
		}
		CUDA_CHECK(hipDeviceSynchronize());
	}
	else
	{
		GetCulvertElevCPU(XParam, XCulverts, XModel.culvertsF, XModel.bndblk.nblkculvert, XModel.bndblk.culvert, XModel.blocks, XModel.evolv);
	}


	// Calculation of the transfert of water (depending of the type of culvert)(loop on culverts)
	for (cc = 0; cc < XCulverts.size(); cc++)
	{

		//ib1 = XCulverts[cc].block1;

		//Pump system
		if (XCulverts[cc].type == 0)
		{
			Qmax = T(XCulverts[cc].Qmax);
			Vol1 = XModel.culvertsF.h1[cc];// *XCulverts[cc].dx1* XCulverts[cc].dx1;
			Q = T(Vol1 * dt);
			if (Q > Qmax)
			{
				XModel.culvertsF.dq[cc] = Qmax;
			}
			else
			{
				XModel.culvertsF.dq[cc] = Q;
			}
		}
		/*
		//One way (clapped) culvert
		if (XCulverts.type == 1)
		
		//Basic 2way culvert
		if (XCulverts.type == 2)
		*/
	}

	/*

	Application of the result to h:
	(Loop on blocks)

	*/

	if (XParam.GPUDEVICE >= 0)
	{
		for (cc = 0; cc < XCulverts.size(); cc++)
		{
			InjectCulvertGPU <<<gridDimCulvert, blockDim, 0 >>> (XParam, cc, XCulverts[cc], XModel.culvertsF, XModel.bndblk.culvert, XModel.blocks, XModel.adv);
		}
		CUDA_CHECK(hipDeviceSynchronize());
	}
	else
	{
		InjectCulvertCPU(XParam, XCulverts, XModel.culvertsF, XModel.bndblk.nblkculvert, XModel.bndblk.culvert, XModel.blocks, XModel.adv);
	}
}
template __host__ void AddCulverts<float>(Param XParam, double dt, std::vector<Culvert> XCulverts, Model<float> XModel);
template __host__ void AddCulverts<double>(Param XParam, double dt, std::vector<Culvert> XCulverts, Model<double> XModel);


template <class T> __global__ void InjectCulvertGPU(Param XParam, int cc, Culvert XCulvert, CulvertF<T> XCulvertF, int* Culvertblks, BlockP<T> XBlock, AdvanceP<T>& XAdv)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = Culvertblks[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	if (i == XCulvert.i1)
	{
		XAdv.dh[i] -= XCulvertF.dq[cc] / (XCulvert.dx1 * XCulvert.dx1);
	}
	if (i == XCulvert.i2)
	{
		XAdv.dh[i] += XCulvertF.dq[cc] / (XCulvert.dx2 * XCulvert.dx2);
	}

}
template __global__ void InjectCulvertGPU<float>(Param XParam, int cc, Culvert XCulvert, CulvertF<float> XCulvertF, int* Culvertblks, BlockP<float> XBlock, AdvanceP<float>& XAdv);
template __global__ void InjectCulvertGPU<double>(Param XParam, int cc, Culvert XCulvert, CulvertF<double> XCulvertF, int* Culvertlks, BlockP<double> XBlock, AdvanceP<double>& XAdv);


template <class T> __host__ void InjectCulvertCPU(Param XParam, std::vector<Culvert> XCulverts, CulvertF<T> XCulvertF, int nblkculvert, int* Culvertblks, BlockP<T> XBlock, AdvanceP<T> XAdv)
{

	T delta1, delta2;
	int cc;

	for (cc = 0; cc < XCulverts.size(); cc++)
	{
		delta1 = calcres(T(XParam.dx), XBlock.level[XCulverts[cc].block1]);
		delta2 = calcres(T(XParam.dx), XBlock.level[XCulverts[cc].block2]);
		XAdv.dh[XCulverts[cc].i1] -= XCulvertF.dq[cc] / (delta1 * delta1);
		XAdv.dh[XCulverts[cc].i2] += XCulvertF.dq[cc] / (delta2 * delta2);
	}

}
template __host__ void InjectCulvertCPU<float>(Param XParam, std::vector<Culvert> XCulverts, CulvertF<float> XCulvertF, int nblkculvert, int* Culvertblks, BlockP<float> XBlock, AdvanceP<float> XAdv);
template __host__ void InjectCulvertCPU<double>(Param XParam, std::vector<Culvert> XCulverts, CulvertF<double> XCulvertF, int nblkculvert, int* Culvertblks, BlockP<double> XBlock, AdvanceP<double> XAdv);



template <class T> __global__ void GetCulvertElevGPU(Param XParam, int cc, Culvert XCulvert, CulvertF<T>& XCulvertF, int* Culvertblks, BlockP<T> XBlock, EvolvingP<T> XEv)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = Culvertblks[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);


	if (i == XCulvert.i1)
	{
		XCulvertF.h1[cc] = XEv.h[i];
		XCulvertF.zs1[cc] = XEv.zs[i];
	}
	if (i == XCulvert.i2)
	{
		XCulvertF.h2[cc] = XEv.h[i];
		XCulvertF.zs2[cc] = XEv.zs[i];
	}
}
template __global__ void GetCulvertElevGPU<float>(Param XParam, int cc, Culvert XCulvert, CulvertF<float>& XCulvertF, int* Culvertblks, BlockP<float> XBlock, EvolvingP<float> XEv);
template __global__ void GetCulvertElevGPU<double>(Param XParam, int cc, Culvert XCulvert, CulvertF<double>& XCulvertF, int* Culvertlks, BlockP<double> XBlock, EvolvingP<double> XEv);


template <class T> __host__ void GetCulvertElevCPU(Param XParam, std::vector<Culvert> XCulverts, CulvertF<T>& XCulvertF, int nblkculvert, int* Culvertblks, BlockP<T> XBlock, EvolvingP<T> XEv)
{
	int cc;

	for (cc = 0; cc < XCulverts.size(); cc++)
	{
		XCulvertF.h1[cc] = XEv.h[XCulverts[cc].i1];
		XCulvertF.zs1[cc] = XEv.zs[XCulverts[cc].i1];

		XCulvertF.h2[cc] = XEv.h[XCulverts[cc].i2];
		XCulvertF.zs2[cc] = XEv.zs[XCulverts[cc].i2];
	}

}
template __host__ void GetCulvertElevCPU<float>(Param XParam, std::vector<Culvert> XCulverts, CulvertF<float>& XCulvertF, int nblkculvert, int* Culvertblks, BlockP<float> XBlock, EvolvingP<float> XEv);
template __host__ void GetCulvertElevCPU<double>(Param XParam, std::vector<Culvert> XCulverts, CulvertF<double>& XCulvertF,  int nblkculvert, int* Culvertblks, BlockP<double> XBlock, EvolvingP<double> XEv);
