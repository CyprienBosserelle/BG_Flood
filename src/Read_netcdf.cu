#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//                                                                              //
//Copyright (C) 2018 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

#include "Read_netcdf.h"





inline int nc_get_var_T(int ncid, int varid, float * &zb)
{
	int status;
	status = nc_get_var_float(ncid, varid, zb);
	return status;
}
inline int nc_get_var_T(int ncid, int varid, double * &zb)
{
	int status;
	status = nc_get_var_double(ncid, varid, zb);
	return status;
}
inline int nc_get_var_T(int ncid, int varid, int*& zb)
{
	int status;
	status = nc_get_var_int(ncid, varid, zb);
	return status;
}

inline int nc_get_vara_T(int ncid, int varid, const size_t* startp, const size_t* countp, int*& zb)
{
	int status;
	status = nc_get_vara_int(ncid, varid, startp, countp, zb);
	return status;

}
inline int nc_get_vara_T(int ncid, int varid, const size_t* startp, const size_t* countp, float * &zb)
{
	int status;
	status = nc_get_vara_float(ncid, varid, startp, countp, zb);
	return status;

}
inline int nc_get_vara_T(int ncid, int varid, const size_t* startp, const size_t* countp, double * &zb)
{
	int status;
	status = nc_get_vara_double(ncid, varid, startp, countp, zb);
	return status;

}

inline int nc_get_var1_T(int ncid, int varid, const size_t* startp, float * zsa)
{
	int status;
	status = nc_get_var1_float(ncid, varid, startp, zsa);
	return status;
}
inline int nc_get_var1_T(int ncid, int varid, const size_t* startp, double * zsa)
{
	int status;
	status = nc_get_var1_double(ncid, varid, startp, zsa);
	return status;
}



void readgridncsize(const std::string ncfilestr, const std::string varstr, int &nx, int &ny, int &nt, double &dx, double &xo, double &yo, double &to, double &xmax, double &ymax, double &tmax)
{
	//read the dimentions of grid, levels and time
	int status;
	int ncid, ndimshh, ndims;
	double *xcoord, *ycoord, *tcoord;
	int varid;

	//int ndimsp, nvarsp, nattsp, unlimdimidp;

	int dimids[NC_MAX_VAR_DIMS];   /* dimension IDs */
	char coordname[NC_MAX_NAME + 1];
	//char varname[NC_MAX_NAME + 1];
	size_t  *ddimhh;

	
	//Open NC file
	//printf("Open file\n");
	status = nc_open(ncfilestr.c_str(), NC_NOWRITE, &ncid);
	if (status != NC_NOERR) handle_ncerror(status);


	//printf(" %s...\n", hhvar);
	status = nc_inq_varid(ncid, varstr.c_str(), &varid);
	if (status != NC_NOERR)	handle_ncerror(status);



	status = nc_inq_varndims(ncid, varid, &ndimshh);
	if (status != NC_NOERR) handle_ncerror(status);
	//printf("hhVar:%d dims\n", ndimshh);

	status = nc_inq_vardimid(ncid, varid, dimids);
	if (status != NC_NOERR) handle_ncerror(status);

	ddimhh = (size_t *)malloc(ndimshh*sizeof(size_t));

	//Read dimensions nx_u ny_u
	for (int iddim = 0; iddim < ndimshh; iddim++)
	{
		status = nc_inq_dimlen(ncid, dimids[iddim], &ddimhh[iddim]);
		if (status != NC_NOERR) handle_ncerror(status);

		//printf("dim:%d=%d\n", iddim, ddimhh[iddim]);
	}

	if (ndimshh > 2)
	{
		nt = (int) ddimhh[0];
		ny = (int) ddimhh[1];
		nx = (int) ddimhh[2];

	}
	else
	{
		nt = 0;
		ny = (int) ddimhh[0];
		nx = (int) ddimhh[1];
	}

	//allocate
	xcoord = (double *)malloc(nx*ny*sizeof(double));
	ycoord = (double *)malloc(nx*ny*sizeof(double));

	//inquire variable name for x dimension
	//aka x dim of hh
	int ycovar, xcovar, tcovar;

	if (ndimshh > 2)
	{
		tcovar = dimids[0];
		ycovar = dimids[1];
		xcovar = dimids[2];
	}
	else
	{
		ycovar = dimids[0];
		xcovar = dimids[1];
	}

	//ycoord
	status = nc_inq_dimname(ncid, ycovar, coordname);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_inq_varid(ncid, coordname, &varid);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_inq_varndims(ncid, varid, &ndims);
	if (status != NC_NOERR) handle_ncerror(status);

	if (ndims < 2)
	{
		double * ytempvar;
		ytempvar = (double *)malloc(ny*sizeof(double));
		size_t start[] = { 0 };
		size_t count[] = { ny };
		status = nc_get_vara_double(ncid, varid, start, count, ytempvar);
		if (status != NC_NOERR) handle_ncerror(status);

		for (int i = 0; i<nx; i++)
		{
			for (int j = 0; j<ny; j++)
			{

				ycoord[i + j*nx] = ytempvar[j];

			}
		}
		free(ytempvar);
	}
	else
	{
		size_t start[] = { 0, 0 };
		size_t count[] = { ny, nx };
		status = nc_get_vara_double(ncid, varid, start, count, ycoord);
		if (status != NC_NOERR) handle_ncerror(status);

	}
	//xcoord
	status = nc_inq_dimname(ncid, xcovar, coordname);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_inq_varid(ncid, coordname, &varid);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_inq_varndims(ncid, varid, &ndims);
	if (status != NC_NOERR) handle_ncerror(status);

	if (ndims < 2)
	{
		double * xtempvar;
		xtempvar = (double *)malloc(nx*sizeof(double));
		size_t start[] = { 0 };
		size_t count[] = { nx };
		status = nc_get_vara_double(ncid, varid, start, count, xtempvar);
		if (status != NC_NOERR) handle_ncerror(status);

		for (int i = 0; i<nx; i++)
		{
			for (int j = 0; j<ny; j++)
			{

				xcoord[i + j*nx] = xtempvar[i];

			}
		}
		free(xtempvar);
	}
	else
	{
		size_t start[] = { 0, 0 };
		size_t count[] = { ny, nx };
		status = nc_get_vara_double(ncid, varid, start, count, xcoord);
		if (status != NC_NOERR) handle_ncerror(status);

	}

	double dxx;
	//check dx
	dxx = (xcoord[nx - 1] - xcoord[0]) / (nx - 1.0);
	//log("xo=" + std::to_string(xcoord[0])+"; xmax="+ std::to_string(xcoord[nx - 1]) +"; nx="+ std::to_string(nx) +"; dxx=" +std::to_string(dxx));
	//dyy = (float) abs(ycoord[0] - ycoord[(ny - 1)*nx]) / (ny - 1);


	//Read time dimension if any
	if (nt > 0)
	{
		//read dimension name
		status = nc_inq_dimname(ncid, tcovar, coordname);
		if (status != NC_NOERR) handle_ncerror(status);

		//inquire variable id
		status = nc_inq_varid(ncid, coordname, &varid);
		if (status != NC_NOERR) handle_ncerror(status);

		// read the dimension of time variable // yes it should be == 1
		status = nc_inq_varndims(ncid, varid, &ndims);
		if (status != NC_NOERR) handle_ncerror(status);

		//allocate temporary array and read time vector
		double * ttempvar;
		ttempvar = (double *)malloc(nt * sizeof(double));
		size_t start[] = { 0 };
		size_t count[] = { nt };
		status = nc_get_vara_double(ncid, varid, start, count, ttempvar);

		to = ttempvar[0];
		tmax= ttempvar[nt-1];

		free(ttempvar);
	}
	else
	{
		//this is a 2d file so assign dummy values
		to = 0.0;
		tmax = 0.0;
	}

	dx = dxx;

	xo = xcoord[0];
	xmax = xcoord[nx - 1];
	yo= ycoord[0];
	ymax= ycoord[(ny - 1)*nx];



	status = nc_close(ncid);

	free(ddimhh);
	free(xcoord);
	free(ycoord);


}

int readvarinfo(std::string filename, std::string Varname, size_t *&ddimU)
{
	// This function reads the dimentions for each variables
	int status, varid;
	int ncid, ndims;
	int dimids[NC_MAX_VAR_DIMS];
	//Open NC file
	//printf("Open file\n");

	status = nc_open(filename.c_str(), 0, &ncid);
	if (status != NC_NOERR) handle_ncerror(status);

	//inquire variable by name
	//printf("Reading information about %s...", Varname.c_str());
	status = nc_inq_varid(ncid, Varname.c_str(), &varid);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_inq_varndims(ncid, varid, &ndims);
	if (status != NC_NOERR) handle_ncerror(status);


	status = nc_inq_vardimid(ncid, varid, dimids);
	if (status != NC_NOERR) handle_ncerror(status);

	ddimU = (size_t *)malloc(ndims*sizeof(size_t));

	//Read dimensions nx_u ny_u
	for (int iddim = 0; iddim < ndims; iddim++)
	{
		status = nc_inq_dimlen(ncid, dimids[iddim], &ddimU[iddim]);
		if (status != NC_NOERR) handle_ncerror(status);

		//printf("dim:%d=%d\n", iddim, ddimU[iddim]);
	}


	status = nc_close(ncid);

	return ndims;
}


int readnctime(std::string filename, double * &time)
{
	int status, ncid, varid;

	std::string ncfilestr;
	std::string varstr;


	//char ncfile[]="ocean_ausnwsrstwq2.nc";
	std::vector<std::string> nameelements;

	nameelements = split(filename, '?');
	if (nameelements.size() > 1)
	{
		//variable name for bathy is not given so it is assumed to be zb
		ncfilestr = nameelements[0];
		//varstr = nameelements[1];
	}
	else
	{
		ncfilestr = filename;
		//varstr = "time";
	}

	// Warning this could be more robust by taking the unlimited dimention if time does not exist!
	std::string Varname = "time";

	status = nc_open(ncfilestr.c_str(), 0, &ncid);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_inq_varid(ncid, Varname.c_str(), &varid);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_get_var_double(ncid, varid, time);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_close(ncid);

	return status;
}

template <class T>
int readncslev1(std::string filename, std::string varstr, size_t indx, size_t indy, size_t indt, bool checkhh, double eps, T * &zsa)
{
	int status, ncid, varid,ndims,sferr,oferr,misserr,fillerr, iderr, varerr;
	double scalefac, offset, missing, fillval;

	double hha,zza;

	//bool checkhh = false;

	int wet = 1;

	size_t *start;
	//std::string Varname = "time";
	ndims = 3;

	start = (size_t *)malloc(ndims*sizeof(size_t));
	//count = (size_t *)malloc(ndims*sizeof(size_t));

	start[0] = indt;
	start[1] = indy;
	start[2] = indx;

	//std::string ncfilestr;
	//std::string varstr;


	//char ncfile[]="ocean_ausnwsrstwq2.nc";
	//std::vector<std::string> nameelements;

	/*nameelements = split(filename, '?');
	if (nameelements.size() > 1)
	{
		
		ncfilestr = nameelements[0];
		varstr = nameelements[1];
	}
	else
	{
		
		ncfilestr = filename;
		varstr = "zs";
		checkhh = true;
	}*/

	status = nc_open(filename.c_str(), 0, &ncid);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_inq_varid(ncid, varstr.c_str(), &varid);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_get_var1_T(ncid, varid, start, zsa);
	if (status != NC_NOERR) handle_ncerror(status);

	

	sferr = nc_get_att_double(ncid, varid, "scale_factor", &scalefac);
	oferr = nc_get_att_double(ncid, varid, "add_offset", &offset);

	// Check if variable is a missing value

	misserr = nc_get_att_double(ncid, varid, "_FillValue", &missing);

	fillerr = nc_get_att_double(ncid, varid, "missingvalue", &fillval);

	if (misserr == NC_NOERR)
	{
		if (zsa[0] == missing)
		{
			zsa[0] = 0.0;
			wet = 0;
		}
	}
	if (fillerr == NC_NOERR)
	{
		if (zsa[0] == fillval)
		{
			zsa[0] = 0.0;
			wet = 0;
		}
	}




	if (sferr == NC_NOERR || oferr == NC_NOERR) // data must be packed
	{
		zsa[0] = zsa[0] * scalefac + offset;
	}

	if (checkhh)
	{
		zza = zsa[0];
		iderr = nc_inq_varid(ncid, "hh", &varid);
		if (iderr == NC_NOERR)
		{
			if (typeid(T) == typeid(float))
				status = nc_get_var1_T(ncid, varid, start, zsa);
			if (typeid(T) == typeid(double))
				status = nc_get_var1_T(ncid, varid, start, zsa);
			//status = nc_get_var1_double(ncid, varid, start, zsa);
			sferr = nc_get_att_double(ncid, varid, "scale_factor", &scalefac);
			oferr = nc_get_att_double(ncid, varid, "add_offset", &offset);

			// Check if variable is a missing value

			misserr = nc_get_att_double(ncid, varid, "_FillValue", &missing);

			fillerr = nc_get_att_double(ncid, varid, "missingvalue", &fillval);

			if (misserr == NC_NOERR)
			{
				if (zsa[0] == missing)
				{
					zsa[0] = 0.0;
					wet = 0;
				}
			}
			if (fillerr == NC_NOERR)
			{
				if (zsa[0] == fillval)
				{
					zsa[0] = 0.0;
					wet = 0;
				}
			}




			if (sferr == NC_NOERR || oferr == NC_NOERR) // data must be packed
			{
				zsa[0] = zsa[0] * scalefac + offset;
			}

			hha = zsa[0];
			if (hha > eps)
			{
				zsa[0] = zza;
			}
			else
			{
				zsa[0] = 0.0;
				wet = 0;
			}
			
		}


	}



	status = nc_close(ncid);

	free(start);
	//free(count);


	return wet;
}

template int readncslev1<float>(std::string filename, std::string varstr, size_t indx, size_t indy, size_t indt, bool checkhh, double eps, float * &zsa);
template int readncslev1<double>(std::string filename, std::string varstr, size_t indx, size_t indy, size_t indt, bool checkhh, double eps, double * &zsa);


template <class T>
int readvardata(std::string filename, std::string Varname, int step, T * &vardata)
{
	// function to standardise the way to read netCDF data off a file
	// The role of this function is to offload and simplify the rest of the code


	int nx, ny, nt, status, ncid, varid, sferr, oferr, merr,ndims;
	size_t * start, * count, *ddim;
	double scalefac, offset, missing;



	ndims = readvarinfo(filename, Varname, ddim);

	start = (size_t *)malloc(ndims*sizeof(size_t));
	count = (size_t *)malloc(ndims*sizeof(size_t));

	

	//
	status = nc_open(filename.c_str(), 0, &ncid);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_inq_varid(ncid, Varname.c_str(), &varid);
	if (status != NC_NOERR) handle_ncerror(status);

	if (ndims == 1)
	{
		nx = (int)ddim[0];
		start[0] = 0;
		count[0] =  nx ;



	}
	else if (ndims == 2)
	{
		ny = (int)ddim[0];
		nx = (int)ddim[1];
		start[0] = 0;
		start[1] = 0;

		count[0] = ny;
		count[1] = nx;


	}
	else //(ndim>2)
	{
		nt = (int)ddim[0];
		ny = (int)ddim[1];
		nx = (int)ddim[2];
		start[0] = size_t(utils::min(step, nt - 1));
		start[1] = size_t(0);
		start[2] = size_t(0);

		count[0] = size_t(1);
		count[1] = size_t(ny);
		count[2] = size_t(nx);



	}
	status = nc_get_vara_T(ncid, varid, start, count, vardata);
	if (status != NC_NOERR) handle_ncerror(status);

	if (ndims > 1)
	{

		sferr = nc_get_att_double(ncid, varid, "scale_factor", &scalefac);
		oferr = nc_get_att_double(ncid, varid, "add_offset", &offset);

		merr = nc_get_att_double(ncid, varid, "missingvalue", &missing);
		if (merr != NC_NOERR)
		{
			merr = nc_get_att_double(ncid, varid, "_FillValue", &missing);
		}


		// remove fill value
		if (merr == NC_NOERR)
		{
			//T maxval = T(-99999.0);
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					bool test = missing != missing ? vardata[i + j * nx] != vardata[i + j * nx] : (vardata[i + j * nx] > T(0.9 * missing));
					if (test) // i.e. if vardata is anywhere near missing
					{
						
						vardata[i + j * nx] = T(0.0);
					}
					//maxval = utils::max(maxval, vardata[i + j * nx]);
				}
			}
			//printf("maxval = %f\n", float(maxval));
		}



		// apply scale and offset
		if (sferr == NC_NOERR || oferr == NC_NOERR) // data must be packed
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					vardata[i + j * nx] = vardata[i + j * nx] * (T)scalefac + (T)offset;
					//unpacked_value = packed_value * scale_factor + add_offset
					
				}
			}
		}
	}



	//clean up
	free(start);
	free(count);

	status = nc_close(ncid);

	return status;

}
template int readvardata<int>(std::string filename, std::string Varname, int step, int*& vardata);
template int readvardata<float>(std::string filename, std::string Varname, int step, float * &vardata);
template int readvardata<double>(std::string filename, std::string Varname, int step, double * &vardata);




std::string checkncvarname(int ncid, std::string stringA, std::string stringB, std::string stringC, std::string stringD, std::string stringE)
{
	int varid;
	int errorA, errorB,errorC,errorD,errorE;
	std::string outstring;

	//std::vector<std::string> teststr;

	//teststr.push_back((stringA))


	errorA = nc_inq_varid(ncid, stringA.c_str(), &varid);
	errorB = nc_inq_varid(ncid, stringB.c_str(), &varid);
	errorC = nc_inq_varid(ncid, stringC.c_str(), &varid);
	errorD = nc_inq_varid(ncid, stringD.c_str(), &varid);
	errorE = nc_inq_varid(ncid, stringE.c_str(), &varid);


	if (errorA == NC_NOERR)
	{
		outstring = stringA;
	}
	else if (errorB == NC_NOERR)
	{
		outstring = stringB;
	}
	else if (errorC == NC_NOERR)
	{
		outstring = stringC;
	}
	else if (errorD == NC_NOERR)
	{
		outstring = stringD;
	}
	else if (errorE == NC_NOERR)
	{
		outstring = stringE;
	}

	return outstring;


}

//By default we want to read wind info as float because it will reside in a texture. the value is converted to the apropriate type only when it is used. so there is no need to template this function 
void readWNDstep(forcingmap WNDUmap, forcingmap WNDVmap, int steptoread, float *&Uo, float *&Vo)
{
	//
	int status;
	int ncid;
	float NanValU = -9999, NanValV = -9999, NanValH = -9999;
	int uu_id, vv_id;
	// step to read should be adjusted in each variables so that it keeps using the last output and teh model keeps on going
	// right now the model will catch anexception
	printf("Reading Wind data step: %d ...", steptoread);
	//size_t startl[]={hdstep-1,lev,0,0};
	//size_t countlu[]={1,1,netau,nxiu};
	//size_t countlv[]={1,1,netav,nxiv};
	size_t startl[] = { steptoread, 0, 0 };
	size_t countlu[] = { 1, WNDUmap.ny, WNDUmap.nx };
	size_t countlv[] = { 1, WNDVmap.ny, WNDVmap.nx };

	//static ptrdiff_t stridel[]={1,1,1,1};
	static ptrdiff_t stridel[] = { 1, 1, 1 };

	std::string ncfilestrU, ncfilestrV;
	std::string Uvarstr, Vvarstr;


	//char ncfile[]="ocean_ausnwsrstwq2.nc";
	std::vector<std::string> nameelements;
	//by default we expect tab delimitation
	nameelements = split(WNDUmap.inputfile, '?');
	if (nameelements.size() > 1)
	{
		//variable name for bathy is not given so it is assumed to be zb
		ncfilestrU = nameelements[0];
		Uvarstr = nameelements[1];
	}
	else
	{
		ncfilestrU = WNDUmap.inputfile;
		Uvarstr = "uwnd";
	}

	nameelements = split(WNDVmap.inputfile, '?');
	if (nameelements.size() > 1)
	{
		//variable name for bathy is not given so it is assumed to be zb
		ncfilestrV = nameelements[0];
		Vvarstr = nameelements[1];
	}
	else
	{
		ncfilestrV = WNDVmap.inputfile;
		Vvarstr = "vwnd";
	}


	//Open NC file

	status = nc_open(ncfilestrU.c_str(), 0, &ncid);
	if (status != NC_NOERR) handle_ncerror(status);

	//status = nc_inq_varid (ncid, "u", &uu_id);
	status = nc_inq_varid(ncid, Uvarstr.c_str(), &uu_id);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_get_vara_float(ncid, uu_id, startl, countlu, Uo);
	if (status != NC_NOERR) handle_ncerror(status);

	//status = nc_get_att_float(ncid, uu_id, "_FillValue", &NanValU);
	//if (status != NC_NOERR) handle_error(status);

	status = nc_close(ncid);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_open(ncfilestrV.c_str(), 0, &ncid);
	if (status != NC_NOERR) handle_ncerror(status);
	//status = nc_inq_varid (ncid, "v", &vv_id);
	status = nc_inq_varid(ncid, Vvarstr.c_str(), &vv_id);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_get_vara_float(ncid, vv_id, startl, countlv, Vo);
	if (status != NC_NOERR) handle_ncerror(status);

	//status = nc_get_att_float(ncid, vv_id, "_FillValue", &NanValV);
	//if (status != NC_NOERR) handle_error(status);

	status = nc_close(ncid);
	if (status != NC_NOERR) handle_ncerror(status);
	printf("Done!\n");

}

//Atm pressure is same as wind we on;ly read floats and that is plenty for real world application
void readATMstep(forcingmap ATMPmap, int steptoread, float *&Po)
{
	//
	int status;
	int ncid;
	float NanValU = -9999, NanValV = -9999, NanValH = -9999;
	int uu_id, vv_id;
	// step to read should be adjusted in each variables so that it keeps using the last output and teh model keeps on going
	// right now the model will catch anexception
	printf("Reading atm pressure data. step: %d ...", steptoread);
	//size_t startl[]={hdstep-1,lev,0,0};
	//size_t countlu[]={1,1,netau,nxiu};
	//size_t countlv[]={1,1,netav,nxiv};
	size_t startl[] = { steptoread, 0, 0 };
	size_t countlu[] = { 1, ATMPmap.ny, ATMPmap.nx };
	//size_t countlv[] = { 1, WNDVmap.ny, WNDVmap.nx };

	//static ptrdiff_t stridel[]={1,1,1,1};
	static ptrdiff_t stridel[] = { 1, 1, 1 };

	std::string ncfilestr;
	std::string atmpvarstr;


	//char ncfile[]="ocean_ausnwsrstwq2.nc";
	std::vector<std::string> nameelements;
	//by default we expect tab delimitation
	nameelements = split(ATMPmap.inputfile, '?');
	if (nameelements.size() > 1)
	{
		//variable name for bathy is not given so it is assumed to be zb
		ncfilestr = nameelements[0];
		atmpvarstr = nameelements[1];
	}
	else
	{
		ncfilestr = ATMPmap.inputfile;
		atmpvarstr = "atmP";
	}


	//Open NC file

	status = nc_open(ncfilestr.c_str(), 0, &ncid);
	if (status != NC_NOERR) handle_ncerror(status);

	//status = nc_inq_varid (ncid, "u", &uu_id);
	status = nc_inq_varid(ncid, atmpvarstr.c_str(), &uu_id);
	if (status != NC_NOERR) handle_ncerror(status);

	status = nc_get_vara_float(ncid, uu_id, startl, countlu, Po);
	if (status != NC_NOERR) handle_ncerror(status);

	//status = nc_get_att_float(ncid, uu_id, "_FillValue", &NanValU);
	//if (status != NC_NOERR) handle_error(status);

	status = nc_close(ncid);

	printf("Done!\n");

}

// The following functions are simple tools to create 2D or 3D netcdf files (for testing for example)

extern "C" void read3Dnc(int nx, int ny, int ntheta, char ncfile[], float * &ee)
{
	int status;
	int ncid, ee_id;
	static size_t count[] = { nx, ny,ntheta };
	status = nc_open(ncfile, NC_NOWRITE, &ncid);
	status = nc_inq_varid(ncid, "z", &ee_id);
	status = nc_get_var_float(ncid, ee_id, ee);
	status = nc_close(ncid);

}

extern "C" void read2Dnc(int nx, int ny, char ncfile[], float * &hh)
{
	int status;
	int ncid, hh_id;
	static size_t count[] = { nx, ny };
	status = nc_open(ncfile, NC_NOWRITE, &ncid);
	status = nc_inq_varid(ncid, "hh", &hh_id);
	status = nc_get_var_float(ncid, hh_id, hh);
	status = nc_close(ncid);

}

extern "C" void readnczb(int nx, int ny, std::string ncfile, float * &zb)
{
	int status;
	int ncid, hh_id;
	static size_t count[] = { nx, ny };

	status = nc_open(ncfile.c_str(), NC_NOWRITE, &ncid);
	status = nc_inq_varid(ncid, "zb", &hh_id);
	status = nc_get_var_float(ncid, hh_id, zb);
	status = nc_close(ncid);

}
