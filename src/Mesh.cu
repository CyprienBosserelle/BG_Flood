#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//                                                                              //
//Copyright (C) 2018 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////


#include "Mesh.h"

int CalcInitnblk(Param XParam, Forcing<float> XForcing)
{

	////////////////////////////////////////////////
	// Rearrange the memory in uniform blocks
	////////////////////////////////////////////////


	//max nb of blocks is ceil(nx/16)*ceil(ny/16)
	int nblk = 0;
	int nmask = 0;
	//int mloc = 0;

	double levdx = calcres(XParam.dx, XParam.initlevel);

	int maxnbx = ceil(XParam.nx / (double)XParam.blkwidth);
	int maxnby = ceil(XParam.ny / (double)XParam.blkwidth);

	for (int nblky = 0; nblky < maxnby; nblky++)
	{
		for (int nblkx = 0; nblkx < maxnbx; nblkx++)
		{
			nmask = 0;
			for (int i = 0; i < XParam.blkwidth; i++)
			{
				for (int j = 0; j < XParam.blkwidth; j++)
				{
					double x = XParam.xo + (double(i) + XParam.blkwidth * nblkx) * levdx + 0.5 * levdx;
					double y = XParam.yo + (double(j) + XParam.blkwidth * nblky) * levdx + 0.5 * levdx;

					//if (x >= XForcing.Bathy.xo && x <= XForcing.Bathy.xmax && y >= XForcing.Bathy.yo && y <= XForcing.Bathy.ymax)
					{
						// cells that falls off this domain are assigned
						double x1, x2, y1, y2;
						double q11, q12, q21, q22, q;
						int cfi, cfip, cfj, cfjp;

						x = utils::max(utils::min(x, XForcing.Bathy[0].xmax), XForcing.Bathy[0].xo);
						y = utils::max(utils::min(y, XForcing.Bathy[0].ymax), XForcing.Bathy[0].yo);

						cfi = utils::min(utils::max((int)floor((x - XForcing.Bathy[0].xo) / XForcing.Bathy[0].dx), 0), XForcing.Bathy[0].nx - 2);
						cfip = cfi + 1;

						x1 = XForcing.Bathy[0].xo + XForcing.Bathy[0].dx * cfi;
						x2 = XForcing.Bathy[0].xo + XForcing.Bathy[0].dx * cfip;

						cfj = utils::min(utils::max((int)floor((y - XForcing.Bathy[0].yo) / XForcing.Bathy[0].dx), 0), XForcing.Bathy[0].ny - 2);
						cfjp = cfj + 1;

						y1 = XForcing.Bathy[0].yo + XForcing.Bathy[0].dx * cfj;
						y2 = XForcing.Bathy[0].yo + XForcing.Bathy[0].dx * cfjp;

						q11 = XForcing.Bathy[0].val[cfi + cfj * XForcing.Bathy[0].nx];
						q12 = XForcing.Bathy[0].val[cfi + cfjp * XForcing.Bathy[0].nx];
						q21 = XForcing.Bathy[0].val[cfip + cfj * XForcing.Bathy[0].nx];
						q22 = XForcing.Bathy[0].val[cfip + cfjp * XForcing.Bathy[0].nx];

						q = BilinearInterpolation(q11, q12, q21, q22, x1, x2, y1, y2, x, y);
						//printf("q = %f\n", q);
						//printf("mloc: %i\n", mloc);
						if (q >= XParam.mask)
							nmask++;
					}
					//else
					//{
						//computational domnain is outside of the bathy domain
					///	nmask++;
					//}

				}
			}
			if (nmask < (XParam.blkwidth* XParam.blkwidth))
				nblk++;
		}
	}

	return nblk;
}

template <class T>
void InitMesh(Param &XParam, Forcing<float> & XForcing, Model<T> &XModel)
{
	//=============================
	// Calculate an initial number of block

	log("\nInitializing mesh");
	int nblk;

	nblk= CalcInitnblk(XParam, XForcing);
		
	XParam.nblk = nblk;
	// allocate a few extra blocks for adaptation
	XParam.nblkmem = (int)ceil(nblk * XParam.membuffer); //5% buffer on the memory for adaptation 

	log("\tInitial number of blocks: " + std::to_string(nblk) + "; Will be allocating " + std::to_string(XParam.nblkmem) + " in memory.");

	//==============================
	// Allocate CPU memory for the whole model
	AllocateCPU(XParam.nblkmem, XParam.blksize, XParam, XModel);

	//==============================
	// Initialise blockinfo info
	InitBlockInfo(XParam, XForcing, XModel.blocks);

	//==============================
	// Init. adaptation info if needed
	if (XParam.maxlevel != XParam.minlevel)
	{
		
		InitBlockadapt(XParam, XModel.blocks, XModel.adapt);
	}

	//==============================
	// Reallocate array containing boundary blocks

	//==============================
	// Add mask block info (flag the block with at least one empty neighbour that is not boundary)
	FindMaskblk(XParam, XModel.blocks);
	
	
}

template void InitMesh<float>(Param &XParam, Forcing<float>& XForcing, Model<float> &XModel);
template void InitMesh<double>(Param &XParam, Forcing<float>& XForcing, Model<double> &XModel);

template <class T> void InitBlockInfo(Param &XParam, Forcing<float> &XForcing, BlockP<T>& XBlock)
{
	//============================
	// Init active and level

	// Initialise activeblk array as all inactive ( = -1 )
	// Here we cannot yet use the InitBlkBUQ function since none of the blk are active
	//InitBlkBUQ(XParam, XBlock, XParam.initlevel, XBlock.level)
	for (int ib = 0; ib < XParam.nblkmem; ib++)
	{
		XBlock.active[ib] = -1;
		XBlock.level[ib] = XParam.initlevel;
	}
	
	

	//============================
	// Init xo, yo and active blk
	InitBlockxoyo(XParam, XForcing, XBlock);

	//============================
	// Init neighbours
	InitBlockneighbours(XParam, XForcing, XBlock);
	//Calcbndblks(XParam, XForcing, XBlock);

}

template <class T> void InitBlockadapt(Param &XParam, BlockP<T> XBlock, AdaptP& XAdap)
{
		InitBlkBUQ(XParam, XBlock, XParam.initlevel, XAdap.newlevel);
		InitBlkBUQ(XParam, XBlock, false, XAdap.coarsen);
		InitBlkBUQ(XParam, XBlock, false, XAdap.refine);
		//InitBlkBUQ(XParam, XBlock, XParam.initlevel, XBlock.level);
		//InitBlkBUQ(XParam, XBlock, XParam.initlevel, XBlock.level);
		//InitArrayBUQ(XParam.nblkmem, 1, 0, XParam.initlevel, XAdap.newlevel);
		//InitArrayBUQ(XParam.nblkmem, 1, 0, false, XAdap.coarsen);
		//InitArrayBUQ(XParam.nblkmem, 1, 0, false, XAdap.refine);


		for (int ibl = 0; ibl < (XParam.nblkmem - XParam.nblk); ibl++)
		{

			XAdap.availblk[ibl] = XParam.nblk + ibl;
			XParam.navailblk++;

		}
	
}
template void InitBlockadapt<float>(Param &XParam, BlockP<float> XBlock, AdaptP& XAdap);
template void InitBlockadapt<double>(Param &XParam, BlockP<double> XBlock, AdaptP& XAdap);



template <class T> void InitBlockxoyo(Param XParam, Forcing<float> XForcing, BlockP<T> &XBlock)
{

	int nmask = 0;
	//mloc = 0;
	int blkid = 0;
	double levdx = calcres(XParam.dx, XParam.initlevel);

	
	int maxnbx = ceil(XParam.nx / (double)XParam.blkwidth);
	int maxnby = ceil(XParam.ny / (double)XParam.blkwidth);

	for (int nblky = 0; nblky < maxnby; nblky++)
	{
		for (int nblkx = 0; nblkx < maxnbx; nblkx++)
		{
			nmask = 0;
			for (int i = 0; i < XParam.blkwidth; i++)
			{
				for (int j = 0; j < XParam.blkwidth; j++)
				{
					double x = XParam.xo + (double(i) + XParam.blkwidth * nblkx)*levdx + 0.5 * levdx;
					double y = XParam.yo + (double(j) + XParam.blkwidth * nblky)*levdx + 0.5 * levdx;

					int n = memloc(XParam, i, j, blkid);

					//x = max(min(x, XParam.Bathymetry.xmax), XParam.Bathymetry.xo);
					//y = max(min(y, XParam.Bathymetry.ymax), XParam.Bathymetry.yo);
					
					{
						x = utils::max(utils::min(x, XForcing.Bathy[0].xmax), XForcing.Bathy[0].xo);
						y = utils::max(utils::min(y, XForcing.Bathy[0].ymax), XForcing.Bathy[0].yo);
						// cells that falls off this domain are assigned
						double x1, x2, y1, y2;
						double q11, q12, q21, q22, q;
						int cfi, cfip, cfj, cfjp;



						cfi = utils::min(utils::max((int)floor((x - XForcing.Bathy[0].xo) / XForcing.Bathy[0].dx), 0), XForcing.Bathy[0].nx - 2);
						cfip = cfi + 1;

						x1 = XForcing.Bathy[0].xo + XForcing.Bathy[0].dx*cfi;
						x2 = XForcing.Bathy[0].xo + XForcing.Bathy[0].dx*cfip;

						cfj = utils::min(utils::max((int)floor((y - XForcing.Bathy[0].yo) / XForcing.Bathy[0].dx), 0), XForcing.Bathy[0].ny - 2);
						cfjp = cfj + 1;

						y1 = XForcing.Bathy[0].yo + XForcing.Bathy[0].dx*cfj;
						y2 = XForcing.Bathy[0].yo + XForcing.Bathy[0].dx*cfjp;

						q11 = XForcing.Bathy[0].val[cfi + cfj*XForcing.Bathy[0].nx];
						q12 = XForcing.Bathy[0].val[cfi + cfjp*XForcing.Bathy[0].nx];
						q21 = XForcing.Bathy[0].val[cfip + cfj*XForcing.Bathy[0].nx];
						q22 = XForcing.Bathy[0].val[cfip + cfjp*XForcing.Bathy[0].nx];

						q = BilinearInterpolation(q11, q12, q21, q22, x1, x2, y1, y2, x, y);
						//printf("q = %f\t q11=%f\t, q12=%f\t, q21=%f\t, q22=%f\t, x1=%f\t, x2=%f\t, y1=%f\t, y2=%f\t, x=%f\t, y=%f\t\n", q, q11, q12, q21, q22, x1, x2, y1, y2, x, y);
						//printf("mloc: %i\n", mloc);
						if (q >= XParam.mask)
						{
							nmask++;
							XBlock.activeCell[n] = 0;
						}
						else
							XBlock.activeCell[n] = 1;
					}
					

				}
			}
			if (nmask < (XParam.blkwidth * XParam.blkwidth))
			{
				//
				XBlock.xo[blkid] = nblkx * ((T)XParam.blkwidth) * levdx + 0.5 * levdx;
				XBlock.yo[blkid] = nblky * ((T)XParam.blkwidth) * levdx + 0.5 * levdx;
				XBlock.active[blkid] = blkid;
				//printf("blkxo=%f\tblkyo=%f\n", blockxo_d[blkid], blockyo_d[blkid]);
				blkid++;
			}
		}
	}




}
template void InitBlockxoyo<float>(Param XParam, Forcing<float> XForcing, BlockP<float> &XBlock);
template void InitBlockxoyo<double>(Param XParam, Forcing<float> XForcing, BlockP<double> & XBlockP);

template <class T> void InitBlockneighbours(Param &XParam,Forcing<float> &XForcing,  BlockP<T>& XBlock)
{
	// This function will only work if the blocks are uniform
	// A separate function is used for adaptivity
	T leftxo, rightxo, topxo, botxo, leftyo, rightyo, topyo, botyo;

	//====================================
	// First setp up neighbours

	T levdx = calcres(XParam.dx, XParam.initlevel);
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{

		int bl = XBlock.active[ibl];
		T espdist = std::numeric_limits<T>::epsilon() * (T)10.0; // i.e. distances are calculated within 10x theoretical machine precision

		leftxo = XBlock.xo[bl] - ((T)XParam.blkwidth) * levdx;

		leftyo = XBlock.yo[bl];
		rightxo = XBlock.xo[bl] + ((T)XParam.blkwidth) * levdx;
		rightyo = XBlock.yo[bl];
		topxo = XBlock.xo[bl];
		topyo = XBlock.yo[bl] + ((T)XParam.blkwidth) * levdx;
		botxo = XBlock.xo[bl];
		botyo = XBlock.yo[bl] - ((T)XParam.blkwidth) * levdx;

		// by default neighbour block refer to itself. i.e. if the neighbour block is itself then there are no neighbour
		XBlock.LeftBot[bl] = bl;
		XBlock.LeftTop[bl] = bl;
		XBlock.RightBot[bl] = bl;
		XBlock.RightTop[bl] = bl;
		XBlock.TopLeft[bl] = bl;
		XBlock.TopRight[bl] = bl;
		XBlock.BotLeft[bl] = bl;
		XBlock.BotRight[bl] = bl;


		for (int iblb = 0; iblb < XParam.nblk; iblb++)
		{
			//
			int blb = XBlock.active[iblb];

			if (abs(XBlock.xo[blb] - leftxo) < espdist && abs(XBlock.yo[blb] - leftyo) < espdist)
			{
				XBlock.LeftBot[bl] = blb;
				XBlock.LeftTop[bl] = blb;
			}
			if (abs(XBlock.xo[blb] - rightxo) < espdist && abs(XBlock.yo[blb] - rightyo) < espdist)
			{
				XBlock.RightBot[bl] = blb;
				XBlock.RightTop[bl] = blb;
			}
			if (abs(XBlock.xo[blb] - topxo) < espdist && abs(XBlock.yo[blb] - topyo) < espdist)
			{
				XBlock.TopLeft[bl] = blb;
				XBlock.TopRight[bl] = blb;

			}
			if (abs(XBlock.xo[blb] - botxo) < espdist && abs(XBlock.yo[blb] - botyo) < espdist)
			{
				XBlock.BotLeft[bl] = blb;
				XBlock.BotRight[bl] = blb;
			}
		}
	}
		


	//
	

}
template void InitBlockneighbours<float>(Param &XParam,  Forcing<float>& XForcing, BlockP<float>& XBlock);
template void InitBlockneighbours<double>(Param &XParam, Forcing<float>& XForcing, BlockP<double>& XBlock);



template <class T> int CalcMaskblk(Param XParam, BlockP<T> XBlock)
{
	int nmask = 0;
	bool neighbourmask = false;
	T leftxo, leftyo, rightxo, rightyo, topxo, topyo, botxo, botyo;
	T initlevdx = calcres(XParam.dx, XParam.initlevel);

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		T levdx = calcres(XParam.dx, XBlock.level[ib]);

		leftxo = XBlock.xo[ib]; // in adaptive this shoulbe be a range 

		leftyo = XBlock.yo[ib];
		rightxo = XBlock.xo[ib] + (XParam.blkwidth - 1) * levdx;
		rightyo = XBlock.yo[ib];
		topxo = XBlock.xo[ib];
		topyo = XBlock.yo[ib] + (XParam.blkwidth - 1) * levdx;
		botxo = XBlock.xo[ib];
		botyo = XBlock.yo[ib];

		neighbourmask = false;

		if ((XBlock.LeftBot[ib] == ib || XBlock.LeftTop[ib] == ib) && leftxo > levdx)
		{
			neighbourmask = true;
		}
		if ((XBlock.BotLeft[ib] == ib || XBlock.BotRight[ib] == ib) && botyo > levdx)
		{
			neighbourmask = true;
		}
		if ((XBlock.TopLeft[ib] == ib || XBlock.TopRight[ib] == ib) && ((topyo - (XParam.ymax - XParam.yo)) < (-1.0 * levdx)))
		{
			neighbourmask = true;
		}
		if ((XBlock.RightBot[ib] == ib || XBlock.RightBot[ib] == ib) && ((rightxo - (XParam.xmax - XParam.xo)) < (-1.0 * levdx)))
		{
			neighbourmask = true;
		}

		int nadd = neighbourmask ? 1 : 0;

		nmask = nmask + nadd;

	}

	return nmask;
}
template int CalcMaskblk<float>(Param XParam, BlockP<float> XBlock);
template int CalcMaskblk<double>(Param XParam, BlockP<double> XBlock);



template <class T> void FindMaskblk(Param XParam, BlockP<T> &XBlock)
{

	XBlock.mask.nblk = CalcMaskblk(XParam, XBlock);
	if (XBlock.mask.nblk > 0)
	{
		int nmask = 0;
		bool neighbourmask = false;
		T leftxo, leftyo, rightxo, rightyo, topxo, topyo, botxo, botyo;

		// Reallocate array if necessary
		ReallocArray(XBlock.mask.nblk, 1, XBlock.mask.side);
		ReallocArray(XBlock.mask.nblk, 1, XBlock.mask.blks);


		for (int ibl = 0; ibl < XParam.nblk; ibl++)
		{
			int ib = XBlock.active[ibl];
			T levdx = calcres(XParam.dx, XBlock.level[ib]);

			leftxo = XBlock.xo[ib]; // in adaptive this shoulbe be a range 

			leftyo = XBlock.yo[ib];
			rightxo = XBlock.xo[ib] + (XParam.blkwidth - 1) * levdx;
			rightyo = XBlock.yo[ib];
			topxo = XBlock.xo[ib];
			topyo = XBlock.yo[ib] + (XParam.blkwidth - 1) * levdx;
			botxo = XBlock.xo[ib];
			botyo = XBlock.yo[ib];

			neighbourmask = false;

			if (nmask < XBlock.mask.nblk)
			{
				XBlock.mask.side[nmask] = 0b00000000;
			}


			if ((XBlock.LeftBot[ib] == ib || XBlock.LeftTop[ib] == ib) && leftxo > levdx)
			{
				XBlock.mask.blks[nmask] = ib;

				if (XBlock.LeftBot[ib] == ib)
				{
					XBlock.mask.side[nmask] = XBlock.mask.side[nmask] | 0b10000000;
				}
				if (XBlock.LeftTop[ib] == ib)
				{
					XBlock.mask.side[nmask] = XBlock.mask.side[nmask] | 0b01000000;
				}
				neighbourmask = true;
			}

			if ((XBlock.TopLeft[ib] == ib || XBlock.TopRight[ib] == ib) && ((topyo - (XParam.ymax - XParam.yo)) < (-1.0 * levdx)))
			{
				XBlock.mask.blks[nmask] = ib;
				if (XBlock.TopLeft[ib] == ib)
				{
					XBlock.mask.side[nmask] = XBlock.mask.side[nmask] | 0b00100000;
				}
				if (XBlock.TopRight[ib] == ib)
				{
					XBlock.mask.side[nmask] = XBlock.mask.side[nmask] | 0b00010000;
				}

				neighbourmask = true;
			}
			if ((XBlock.RightBot[ib] == ib || XBlock.RightBot[ib] == ib) && ((rightxo - (XParam.xmax - XParam.xo)) < (-1.0 * levdx)))
			{
				XBlock.mask.blks[nmask] = ib;
				if (XBlock.RightTop[ib] == ib)
				{
					XBlock.mask.side[nmask] = XBlock.mask.side[nmask] | 0b00001000;
				}
				if (XBlock.RightBot[ib] == ib)
				{
					XBlock.mask.side[nmask] = XBlock.mask.side[nmask] | 0b00000100;
				}
				neighbourmask = true;
			}
			if ((XBlock.BotLeft[ib] == ib || XBlock.BotRight[ib] == ib) && botyo > levdx)
			{
				XBlock.mask.blks[nmask] = ib;
				if (XBlock.BotRight[ib] == ib)
				{
					XBlock.mask.side[nmask] = XBlock.mask.side[nmask] | 0b00000010;
				}
				if (XBlock.BotLeft[ib] == ib)
				{
					XBlock.mask.side[nmask] = XBlock.mask.side[nmask] | 0b00000001;
				}
				neighbourmask = true;
			}

			int nadd = neighbourmask ? 1 : 0;

			nmask = nmask + nadd;

		}
	}
}
template void FindMaskblk<float>(Param XParam, BlockP<float> &XBlock);
template void FindMaskblk<double>(Param XParam, BlockP<double> &XBlock);
