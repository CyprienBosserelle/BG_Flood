#include "hip/hip_runtime.h"
﻿#include "ConserveElevation.h"


template <class T> void conserveElevation(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		//int ii = memloc(XParam, -1, 5, 46);

		
		conserveElevationLeft(XParam, ib, XBlock.LeftBot[ib], XBlock.LeftTop[ib], XBlock, XEv, zb);
		
		conserveElevationRight(XParam, ib, XBlock.RightBot[ib], XBlock.RightTop[ib], XBlock, XEv, zb);
		
		conserveElevationTop(XParam, ib, XBlock.TopLeft[ib], XBlock.TopRight[ib], XBlock, XEv, zb);
		
		conserveElevationBot(XParam, ib, XBlock.BotLeft[ib], XBlock.BotRight[ib], XBlock, XEv, zb);
		
	}
}
template void conserveElevation<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, float* zb);
template void conserveElevation<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, double* zb);


template <class T> void conserveElevationGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	dim3 blockDimHaloLR(1, 16, 1);
	dim3 blockDimHaloBT(16, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);


		conserveElevationLeft<<<gridDim, blockDimHaloLR, 0>>> (XParam, XBlock, XEv, zb);
		CUDA_CHECK(hipDeviceSynchronize());
		conserveElevationRight<<<gridDim, blockDimHaloLR, 0 >>> (XParam, XBlock, XEv, zb);
		CUDA_CHECK(hipDeviceSynchronize());
		conserveElevationTop<<<gridDim, blockDimHaloBT, 0 >>> (XParam, XBlock, XEv, zb);
		CUDA_CHECK(hipDeviceSynchronize());
		conserveElevationBot<<<gridDim, blockDimHaloBT, 0 >>> (XParam, XBlock, XEv, zb);
		CUDA_CHECK(hipDeviceSynchronize());
	
}
template void conserveElevationGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, float* zb);
template void conserveElevationGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, double* zb);

template <class T> __host__ __device__ void ProlongationElevation(int halowidth, int blkmemwidth, T eps, int ib, int ibn, int ihalo, int jhalo, int il, int jl, int ip, int jp, int im, int jm, T* h, T* zs, T* zb)
{
	int ll, pp, halo,mm;
	pp = memloc(halowidth, blkmemwidth, ip, jp, ibn);
	ll = memloc(halowidth, blkmemwidth, il, jl, ib);
	mm = memloc(halowidth, blkmemwidth, im, jm, ibn);

	halo = memloc(halowidth, blkmemwidth, ihalo, jhalo, ib);
	//Check if parent is dry or any of close neighbour



	if (!(h[ll] > eps && h[halo]>eps && h[pp] > eps && h[mm] > eps))
	{
		
		//h[halo] = utils::max(T(0.0), zs[pp] - zb[halo]);
		//zs[halo] = h[halo] + zb[halo];
		h[halo] = h[pp];
		zb[halo] = zb[pp];
		zs[halo] = zs[pp];

	}
	


}

template <class T> __host__ __device__ void ProlongationElevationGH(int halowidth, int blkmemwidth, T eps, int ib, int ibn, int ihalo, int jhalo, int il, int jl, int ip, int jp,int im, int jm, T* h, T* dhdx, T* dzsdx)
{
	int ll, pp, halo, mm;
	pp = memloc(halowidth, blkmemwidth, ip, jp, ibn);
	ll = memloc(halowidth, blkmemwidth, il, jl, ib);
	mm = memloc(halowidth, blkmemwidth, im, jm, ibn);

	halo = memloc(halowidth, blkmemwidth, ihalo, jhalo, ib);
	//Check if parent is dry or any of close neighbour


	
	if (!(h[ll] > eps && h[halo] > eps && h[pp] > eps && h[mm] > eps))
	{

		dhdx[halo] = T(0.0);
		dzsdx[halo] = T(0.0);
	}



}

template <class T> __host__ __device__ void conserveElevation(int halowidth,int blkmemwidth,T eps, int ib, int ibn,int ihalo, int jhalo ,int i,int j, T* h, T* zs, T * zb)
{
	int ii, ir, it, itr, jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, hwet;

	int write;

	write = memloc(halowidth, blkmemwidth, ihalo, jhalo, ib);
	//jj = j * 2;
	ii = memloc(halowidth, blkmemwidth, i, j, ibn);
	ir = memloc(halowidth, blkmemwidth, i + 1, j, ibn);
	it = memloc(halowidth, blkmemwidth, i, j + 1, ibn);
	itr = memloc(halowidth, blkmemwidth, i + 1, j + 1, ibn);

	iiwet = h[ii] > eps ? h[ii] : T(0.0);
	irwet = h[ir] > eps ? h[ir] : T(0.0);
	itwet = h[it] > eps ? h[it] : T(0.0);
	itrwet = h[itr] > eps ? h[itr] : T(0.0);

	hwet = (iiwet + irwet + itwet + itrwet);
	zswet = iiwet * (zb[ii] + h[ii]) + irwet * (zb[ir] + h[ir]) + itwet * (zb[it] + h[it]) + itrwet * (zb[itr] + h[itr]);

	//conserveElevation(zb[write], zswet, hwet);
	if (hwet > T(0.0))
	{
		zswet = zswet / hwet;
		hwet = utils::max(T(0.0), zswet - zb[write]);

	}
	else
	{
		hwet = T(0.0);

	}

	//zswet = hwet + zb;

	h[write] = hwet;
	zs[write] = hwet + zb[write];


}
template __host__ __device__ void conserveElevation<float>(int halowidth, int blkmemwidth, float eps, int ib, int ibn, int ihalo, int jhalo, int i, int j, float* h, float* zs, float* zb);
template __host__ __device__ void conserveElevation<double>(int halowidth, int blkmemwidth, double eps, int ib, int ibn, int ihalo, int jhalo, int i, int j, double* h, double* zs, double* zb);





template <class T> __host__ __device__ void conserveElevation(T zb, T& zswet, T& hwet)
{
	
	if (hwet > 0.0)
	{
		zswet = zswet / hwet;
		hwet = utils::max(T(0.0), zswet - zb);

	}
	else
	{
		hwet = T(0.0);
		
	}

	zswet = hwet + zb;
}

template <class T> void conserveElevationGradHalo(Param XParam, BlockP<T> XBlock, T* h, T* zs, T* zb, T* dhdx, T* dzsdx, T* dhdy, T* dzsdy)
{
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		conserveElevationGHLeft(XParam, ib, XBlock.LeftBot[ib], XBlock.LeftTop[ib], XBlock, h, zs, zb, dhdx, dzsdx);
		conserveElevationGHRight(XParam, ib, XBlock.RightBot[ib], XBlock.RightTop[ib], XBlock, h, zs, zb, dhdx, dzsdx);
		conserveElevationGHTop(XParam, ib, XBlock.TopLeft[ib], XBlock.TopRight[ib], XBlock, h, zs, zb, dhdy, dzsdy);
		conserveElevationGHBot(XParam, ib, XBlock.BotLeft[ib], XBlock.BotRight[ib], XBlock, h, zs, zb, dhdy, dzsdy);
	}
}
template void conserveElevationGradHalo<float>(Param XParam, BlockP<float> XBlock, float* h, float* zs, float* zb, float* dhdx, float* dzsdx, float* dhdy, float* dzsdy);
template void conserveElevationGradHalo<double>(Param XParam, BlockP<double> XBlock, double* h, double* zs, double* zb, double* dhdx, double* dzsdx, double* dhdy, double* dzsdy);

template <class T> void conserveElevationGradHaloGPU(Param XParam, BlockP<T> XBlock, T* h, T* zs, T* zb, T* dhdx, T* dzsdx, T* dhdy, T* dzsdy)
{
	dim3 blockDimHaloLR(1, 16, 1);
	dim3 blockDimHaloBT(16, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	conserveElevationGHLeft <<<gridDim, blockDimHaloLR, 0 >>> (XParam, XBlock, h, zs, zb, dhdx, dzsdx);
	CUDA_CHECK(hipDeviceSynchronize());

	conserveElevationGHRight <<<gridDim, blockDimHaloLR, 0 >>> (XParam, XBlock, h, zs, zb, dhdx, dzsdx);
	CUDA_CHECK(hipDeviceSynchronize());

	conserveElevationGHTop <<<gridDim, blockDimHaloBT, 0 >>> (XParam, XBlock, h, zs, zb, dhdy, dzsdy);
	CUDA_CHECK(hipDeviceSynchronize());

	conserveElevationGHBot <<<gridDim, blockDimHaloBT, 0 >>> (XParam, XBlock, h, zs, zb, dhdy, dzsdy);
	CUDA_CHECK(hipDeviceSynchronize());
	
}
template void conserveElevationGradHaloGPU<float>(Param XParam, BlockP<float> XBlock, float* h, float* zs, float* zb, float* dhdx, float* dzsdx, float* dhdy, float* dzsdy);
template void conserveElevationGradHaloGPU<double>(Param XParam, BlockP<double> XBlock, double* h, double* zs, double* zb, double* dhdx, double* dzsdx, double* dhdy, double* dzsdy);

template <class T> __host__ __device__ void conserveElevationGradHalo(int halowidth, int blkmemwidth, T eps, int ib, int ibn, int ihalo, int jhalo,int i, int j, T* h, T* dhdx, T* dhdy)
{
	int ii, ir, it, itr, jj;
	int write;
	write = memloc(halowidth, blkmemwidth, ihalo, jhalo, ib);

	ii = memloc(halowidth, blkmemwidth, i, j, ibn);
	ir = memloc(halowidth, blkmemwidth, i + 1, j, ibn);
	it = memloc(halowidth, blkmemwidth, i, j + 1, ibn);
	itr = memloc(halowidth, blkmemwidth, i + 1, j + 1, ibn);

	if (h[write] <= eps)
	{
		// Because of the slope limiter the average slope is not the slope of the averaged values
		// It seems that it should be the closest to zero instead... With conserve elevation This will work but maybe all prolongation need to be applied this way (?)
		dhdy[write] = utils::nearest(utils::nearest(utils::nearest(dhdy[ii], dhdy[ir]), dhdy[it]), dhdy[itr]);
		dhdx[write] = utils::nearest(utils::nearest(utils::nearest(dhdx[ii], dhdx[ir]), dhdx[it]), dhdx[itr]);
	}
}


template <class T> __host__ __device__ void conserveElevationGradHaloA(int halowidth, int blkmemwidth, int ib, int ibn, int ihalo, int jhalo, int ip, int jp, int iq, int jq, T theta, T delta, T* h, T* dhdx)
{
	//int pii, pir, pit, pitr;
	int qii, qir, qit, qitr;

	T p, q;
	T s0, s1, s2;

	int write, pii;
	write = memloc(halowidth, blkmemwidth, ihalo, jhalo, ib);
	pii = memloc(halowidth, blkmemwidth, ip, jp, ib);




	//pii = memloc(halowidth, blkmemwidth, ip, jp, ibn);
	//pir = memloc(halowidth, blkmemwidth, ip + 1, jp, ibn);
	//pit = memloc(halowidth, blkmemwidth, ip, jp + 1, ibn);
	//pitr = memloc(halowidth, blkmemwidth, ip + 1, jp + 1, ibn);

	qii = memloc(halowidth, blkmemwidth, iq, jq, ibn);
	qir = memloc(halowidth, blkmemwidth, iq + 1, jq, ibn);
	qit = memloc(halowidth, blkmemwidth, iq, jq + 1, ibn);
	qitr = memloc(halowidth, blkmemwidth, iq + 1, jq + 1, ibn);

	s1 = h[write];
	p = h[pii];
	q = T(0.25) * (h[qii] + h[qir] + h[qit] + h[qitr]);



	if (ip > ihalo || jp > jhalo)
	{
		s0 = q;
		s2 = p;
	}
	else
	{
		s2 = q;
		s0 = p;
	}

	dhdx[write] = minmod2(theta, s0, s1, s2) / delta;
	//dhdx[write] = utils::nearest(utils::nearest(utils::nearest(dhdx[ii], dhdx[ir]), dhdx[it]), dhdx[itr]);

}

template <class T> __host__ __device__ void conserveElevationGradHaloB(int halowidth, int blkmemwidth, int ib, int ibn, int ihalo, int jhalo, int ip, int jp, int iq, int jq, T theta, T delta, T eps, T* h, T* zs, T* zb, T* dhdx, T* dzsdx)
{
	//int pii, pir, pit, pitr;
	int qii, qir, qit, qitr;
	
	T hp, hq,zsp,zsq, zbq;
	T hs0, hs1, hs2,zss0, zss1, zss2;

	T hwet, zswet;
	int write, pii;
	int iiwet, irwet, itwet, itrwet;
	write = memloc(halowidth, blkmemwidth, ihalo, jhalo, ib);
	pii = memloc(halowidth, blkmemwidth, ip, jp, ib);
	
	//pii = memloc(halowidth, blkmemwidth, ip, jp, ibn);
	//pir = memloc(halowidth, blkmemwidth, ip + 1, jp, ibn);
	//pit = memloc(halowidth, blkmemwidth, ip, jp + 1, ibn);
	//pitr = memloc(halowidth, blkmemwidth, ip + 1, jp + 1, ibn);

	qii = memloc(halowidth, blkmemwidth, iq, jq, ibn);
	qir = memloc(halowidth, blkmemwidth, iq + 1, jq, ibn);
	qit = memloc(halowidth, blkmemwidth, iq, jq + 1, ibn);
	qitr = memloc(halowidth, blkmemwidth, iq + 1, jq + 1, ibn);

	

	
	zbq = T(0.25) * (zb[qii] + zb[qir] + zb[qit] + zb[qitr]);

	iiwet = h[qii] > eps ? h[qii] : T(0.0);
	irwet = h[qir] > eps ? h[qir] : T(0.0);
	itwet = h[qit] > eps ? h[qit] : T(0.0);
	itrwet = h[qitr] > eps ? h[qitr] : T(0.0);

	hwet = (iiwet + irwet + itwet + itrwet);
	zswet = iiwet * (zb[qii] + h[qii]) + irwet * (zb[qir] + h[qir]) + itwet * (zb[qit] + h[qit]) + itrwet * (zb[qitr] + h[qitr]);
	
	if (hwet > T(0.0))
	{
		zswet = zswet / hwet;
		hq = utils::max(T(0.0), zswet - zbq);
		
	}
	else
	{
		hq = T(0.0);
	}

	hs1 = h[write];
	zss1= zs[write];
	hp = h[pii];
	zsp = zs[pii];
	zsq = hq + zbq;

	if (ip > ihalo || jp > jhalo )
	{
		hs0 = hq;
		hs2 = hp;
		zss0 = zsq;
		zss2 = zsp;
	}
	else
	{
		hs2 = hq;
		hs0 = hp;
		zss2 = zsq;
		zss0 = zsp;
	}

	dhdx[write] = minmod2(theta,hs0,hs1,hs2)/ delta;
	dzsdx[write] = minmod2(theta, zss0, zss1, zss2) / delta;
	//dhdx[write] = utils::nearest(utils::nearest(utils::nearest(dhdx[ii], dhdx[ir]), dhdx[it]), dhdx[itr]);
	
}

template <class T> void conserveElevationGHLeft(Param XParam, int ib, int ibLB, int ibLT, BlockP<T> XBlock, T* h, T* zs, T* zb, T* dhdx, T* dzsdx)
{
	int ibn;
	int ihalo, jhalo, ip, jp, iq, jq;
	T delta = calcres(XParam.dx, XBlock.level[ib]);
	ihalo = -1;
	ip = 0;


	if (XBlock.level[ib] < XBlock.level[ibLB])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{
			jhalo = j;
			jp = j;
			iq = XParam.blkwidth - 4;
			jq = j * 2;
			ibn = ibLB;
			conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
			//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);
			//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibLB,  -1, j, XParam.blkwidth - 2, j * 2, h, dhdx, dhdy);
		}
	}
	if (XBlock.level[ib] < XBlock.level[ibLT])
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			jhalo = j;
			jp = j;
			iq = XParam.blkwidth - 4;
			jq = (j - (XParam.blkwidth / 2)) * 2;
			ibn = ibLT;

			conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
			//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);

			//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibLT, -1, j, XParam.blkwidth - 2, (j - (XParam.blkwidth / 2)) * 2, h, dhdx, dhdy);
		}
	}

	// Prolongation part
	int il, jl, im, jm;
	ihalo = -1;

	if (XBlock.level[ib] > XBlock.level[ibLB])
	{
		//
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//
			jhalo = j;
			ibn = ibLB;

			il = 0;
			jl = j;

			ip = XParam.blkwidth - 1;
			jp = XBlock.RightBot[ibLB] == ib ? floor(j / 2) : (floor(j / 2) + XParam.blkwidth / 2);

			im = ip;
			jm = ceil(j * T(0.5)) * 2 > j ? jp + 1 : jp - 1;

			ProlongationElevationGH(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, h, dhdx, dzsdx);
		}

	}

	

}

template <class T> __global__ void conserveElevationGHLeft(Param XParam, BlockP<T> XBlock, T* h, T*zs, T*zb, T* dhdx, T* dzsdx)
{
	unsigned int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = 0;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int LB = XBlock.LeftBot[ib];
	int LT = XBlock.LeftTop[ib];

	int ip, jp, iq, jq;

	int ihalo, jhalo, ibn;
	T delta = calcres(XParam.dx, lev);


	ihalo = -1;
	jhalo = iy;
	iq = XParam.blkwidth - 4;
	ip = 0;
	jp = iy;
	if (lev < XBlock.level[LB] && iy < (blockDim.y / 2))
	{
		ibn = LB;
		jq = iy * 2;
		conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
		//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);

		//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, h, dhdx, dhdy);
	}
	if (lev < XBlock.level[LT] && iy >= (blockDim.y / 2))
	{
		ibn = LT;
		jq = (iy - (blockDim.y / 2)) * 2;
		conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
		//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, h, dhdx, dhdy);
		//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);
	}

	// Prolongation part
	int il, jl, im, jm;
	

	if (XBlock.level[ib] > XBlock.level[LB])
	{
		//
		//
		
		ibn = LB;

		il = 0;
		jl = iy;

		ip = blockDim.y - 1;
		jp = XBlock.RightBot[LB] == ib ? int(floor(iy *T(0.5))) : int((floor(iy * T(0.5)) + blockDim.y / 2));
		im = ip;
		jm = ceil(iy * T(0.5)) * 2 > iy ? jp + 1 : jp - 1;

		ProlongationElevationGH(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, h, dhdx, dzsdx);
		

	}
}

template <class T> void conserveElevationGHRight(Param XParam, int ib, int ibRB, int ibRT, BlockP<T> XBlock, T* h, T* zs, T* zb, T* dhdx, T* dzsdx)
{
	int ibn;
	int ihalo, jhalo, ip, jp, iq, jq;
	T delta = calcres(XParam.dx, XBlock.level[ib]);
	ihalo = XParam.blkwidth;
	ip = XParam.blkwidth-1;

	if (XBlock.level[ib] < XBlock.level[ibRB])
	{
		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{
			jhalo = j;
			jp = j;
			iq = 2;
			jq = j * 2;
			ibn = ibRB;
			conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
			//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);
			//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibRB, XParam.blkwidth, j, 0, j * 2, h, dhdx, dhdy);
		}
	}
	if (XBlock.level[ib] < XBlock.level[ibRT])
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			jhalo = j;
			jp = j;
			iq = 2;
			jq = (j - (XParam.blkwidth / 2)) * 2;
			ibn = ibRT;
			conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
			//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);

			//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibRT, XParam.blkwidth, j, 0, (j - (XParam.blkwidth / 2)) * 2, h, dhdx, dhdy);
		}
	}

	// Prolongation part
	int il, jl, im, jm;
	

	if (XBlock.level[ib] > XBlock.level[ibRB])
	{
		//
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//
			jhalo = j;
			ibn = ibRB;

			il = XParam.blkwidth-2;
			jl = j;

			ip = 0;
			jp = XBlock.LeftBot[ibRB] == ib ? floor(j / 2) : (floor(j / 2) + XParam.blkwidth / 2);
			im = ip;
			jm = ceil(j * T(0.5)) * 2 > j ? jp + 1 : jp - 1;

			ProlongationElevationGH(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, h, dhdx, dzsdx);
		}

	}
}

template <class T> __global__ void conserveElevationGHRight(Param XParam, BlockP<T> XBlock, T* h, T*zs, T*zb, T* dhdx, T* dzsdx)
{
	unsigned int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = blockDim.y-1;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int RB = XBlock.RightBot[ib];
	int RT = XBlock.RightTop[ib];

	

	int ihalo, jhalo, iq, jq, ip, jp, ibn;

	T delta = calcres(XParam.dx, lev);

	ihalo = blockDim.y;
	jhalo = iy;
	iq = blockDim.y - 4;
	ip = blockDim.y-1;
	jp = iy;

	if (XBlock.level[ib] < XBlock.level[RB] && iy < (blockDim.y / 2))
	{
		ibn = RB;
		jq = iy * 2;
		conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
		//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);

		//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, h, dhdx, dhdy);
	}
	if (XBlock.level[ib] < XBlock.level[RT] && iy >= (blockDim.y / 2))
	{
		ibn = RT;
		jq = (iy - (XParam.blkwidth / 2)) * 2;
		conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
		//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);
		//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, h, dhdx, dhdy);
	}

	// Prolongation part
	int il, jl, im, jm;


	if (XBlock.level[ib] > XBlock.level[RB])
	{
		//
		
		//
		jhalo = iy;
		ibn = RB;

		il = blockDim.y - 2;
		jl = iy;

		ip = 0;
		jp = XBlock.LeftBot[RB] == ib ? int(floor(iy * T(0.5))) : int((floor(iy *T (0.5)) + blockDim.y / 2));
		
		im = ip;
		jm = ceil(iy * T(0.5)) * 2 > iy ? jp + 1 : jp - 1;

		ProlongationElevationGH(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, h, dhdx, dzsdx);
		

	}
}

template <class T> void conserveElevationGHTop(Param XParam, int ib, int ibTL, int ibTR, BlockP<T> XBlock, T* h, T*zs, T*zb, T* dhdx, T* dzsdx)
{
	int ibn;
	int ihalo, jhalo, ip, jp, iq, jq;
	T delta = calcres(XParam.dx, XBlock.level[ib]);
	jhalo = XParam.blkwidth;
	jp = XParam.blkwidth - 1;

	if (XBlock.level[ib] < XBlock.level[ibTL])
	{
		for (int i = 0; i < XParam.blkwidth / 2; i++)
		{
			ihalo = i;
			ip = i;
			jq = 2;
			iq = i * 2;
			ibn = ibTL;
			conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
			//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibTL, i, XParam.blkwidth, i * 2, 0, h, dhdx, dhdy);
		}
	}
	if (XBlock.level[ib] < XBlock.level[ibTR])
	{
		for (int i = (XParam.blkwidth / 2); i < (XParam.blkwidth); i++)
		{
			ihalo = i;
			ip = i;
			jq = 2;
			iq = (i - (XParam.blkwidth / 2)) * 2;
			ibn = ibTR;
			conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
			//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);

			//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibTR, i, XParam.blkwidth, (i - (XParam.blkwidth / 2)) * 2, 0, h, dhdx, dhdy);
		}
	}

	// Prolongation part
	int il, jl, im, jm;


	if (XBlock.level[ib] > XBlock.level[ibTL])
	{
		//
		for (int i = 0; i < XParam.blkwidth; i++)
		{
			//
			ihalo = i;
			ibn = ibTL;

			jl = XParam.blkwidth - 2;
			il = i;

			jp = 0;
			ip = XBlock.BotLeft[ibTL] == ib ? floor(i / 2) : (floor(i / 2) + XParam.blkwidth / 2);
			jm = jp;
			im = ceil(i * T(0.5)) * 2 > i ? ip + 1 : ip - 1;

			ProlongationElevationGH(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, h, dhdx, dzsdx);
		}

	}
}

template <class T> __global__ void conserveElevationGHTop(Param XParam, BlockP<T> XBlock, T* h, T*zs, T*zb, T* dhdx, T* dzsdx)
{
	unsigned int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int iy = blockDim.x - 1;
	unsigned int ix = threadIdx.x;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int TL = XBlock.TopLeft[ib];
	int TR = XBlock.TopRight[ib];
	


	int ihalo, jhalo, iq, jq, ip, jp, ibn;
	T delta = calcres(XParam.dx, lev);

	ihalo = ix;
	jhalo = iy+1;
	jp = iy;
	ip = ix;
	jq = 2;

	if (XBlock.level[ib] < XBlock.level[TL] && ix < (blockDim.x / 2))
	{
		ibn = TL;
		iq = ix * 2;
		conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
		//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);

		//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, h, dhdx, dhdy);
	}
	if (XBlock.level[ib] < XBlock.level[TR] && ix >= (blockDim.x / 2))
	{
		ibn = TR;
		iq = (ix - (blockDim.x / 2)) * 2;;
		conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
		//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);
		//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, h, dhdx, dhdy);
	}

	// Prolongation part
	int il, jl, im, jm;


	if (XBlock.level[ib] > XBlock.level[TL])
	{
		//
		//
		//ihalo = i;
		ibn = TL;

		jl = blockDim.x - 2;
		il = ix;

		jp = 0;
		ip = XBlock.BotLeft[TL] == ib ? int(floor(ix *T(0.0))) : int((floor(ix * T(0.0)) + blockDim.x / 2));
		jm = jp;
		im = ceil(ix * T(0.5)) * 2 > ix ? ip + 1 : ip - 1;

		ProlongationElevationGH(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, h, dhdx, dzsdx);
		

	}
}

template <class T> void conserveElevationGHBot(Param XParam, int ib, int ibBL, int ibBR, BlockP<T> XBlock, T* h, T* zs, T* zb, T* dhdx, T* dzsdx)
{
	int ibn;
	int ihalo, jhalo, ip, jp, iq, jq;
	T delta = calcres(XParam.dx, XBlock.level[ib]);
	jhalo = -1;
	jp = 0;

	if (XBlock.level[ib] < XBlock.level[ibBL])
	{
		for (int i = 0; i < XParam.blkwidth / 2; i++)
		{
			ihalo = i;
			ip = i;
			iq = i * 2;
			jq = XParam.blkwidth - 4;
			ibn = ibBL;
			conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
			//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);

			//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibBL, i, -1, i * 2, XParam.blkwidth - 2, h, dhdx, dhdy);
		}
	}
	if (XBlock.level[ib] < XBlock.level[ibBR])
	{
		for (int i = (XParam.blkwidth / 2); i < (XParam.blkwidth); i++)
		{
			ihalo = i;
			ip = i;
			iq = (i - (XParam.blkwidth / 2)) * 2;;
			jq = XParam.blkwidth - 4;
			ibn = ibBR;
			conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
			//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);

			//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibBR, i, -1, (i - (XParam.blkwidth / 2)) * 2, XParam.blkwidth - 2, h, dhdx, dhdy);
		}
	}
	// Prolongation part
	int il, jl, im, jm;


	if (XBlock.level[ib] > XBlock.level[ibBL])
	{
		//
		for (int i = 0; i < XParam.blkwidth; i++)
		{
			//
			ihalo = i;
			ibn = ibBL;

			jl = 0;
			il = i;

			jp = XParam.blkwidth - 1;
			ip = XBlock.TopLeft[ibBL] == ib ? floor(i / 2) : (floor(i / 2) + XParam.blkwidth / 2);
			jm = jp;
			im = ceil(i * T(0.5)) * 2 > i ? ip + 1 : ip - 1;

			ProlongationElevationGH(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, h, dhdx, dzsdx);
		}

	}
}

template <class T> __global__ void conserveElevationGHBot(Param XParam, BlockP<T> XBlock, T* h, T* zs, T* zb, T* dhdx, T* dzsdx)
{
	unsigned int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int iy = blockDim.x - 1;
	unsigned int ix = threadIdx.x;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int BL = XBlock.BotLeft[ib];
	int BR = XBlock.BotRight[ib];

	int ip, jp, iq, jq;

	int ihalo, jhalo, ibn;
	T delta = calcres(XParam.dx, lev);

	ihalo = ix;
	jhalo = -1;
	jq = XParam.blkwidth - 4;
	jp = 0;
	ip = ix;

	if (XBlock.level[ib] < XBlock.level[BL] && ix < (blockDim.x / 2))
	{
		ibn = BL;
		iq = ix * 2;
		conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
		//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);
		//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, h, dhdx, dhdy);
	}
	if (XBlock.level[ib] < XBlock.level[BR] && ix >= (blockDim.x / 2))
	{
		ibn = BR;
		iq = (ix - (blockDim.x / 2)) * 2;
		conserveElevationGradHaloB(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, T(XParam.eps), h, zs, zb, dhdx, dzsdx);
		//conserveElevationGradHaloA(XParam.halowidth, XParam.blkmemwidth, ib, ibn, ihalo, jhalo, ip, jp, iq, jq, T(XParam.theta), delta, h, dhdx);
		//conserveElevationGradHalo(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, h, dhdx, dhdy);
	}

	// Prolongation part
	int il, jl, im, jm;


	if (XBlock.level[ib] > XBlock.level[BL])
	{
		//
		
		ihalo = ix;
		ibn = BL;

		jl = 0;
		il = ix;

		jp = blockDim.x - 1;
		ip = XBlock.TopLeft[BL] == ib ? int(floor(ix * T(0.0))) : int((floor(ix * T(0.0)) + blockDim.x / 2));

		jm = jp;
		im = ceil(ix * T(0.5)) * 2 > ix ? ip + 1 : ip - 1;

		ProlongationElevationGH(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, h, dhdx, dzsdx);
		

	}
}

template <class T> void conserveElevationLeft(Param XParam,int ib, int ibLB, int ibLT, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ihalo,jhalo,ibn,ip, jp;
	
	// Restriction
	ihalo = -1;
	ip = XParam.blkwidth - 2;

	//int ii = memloc(XParam, -1, 5, 46);
	if (XBlock.level[ib] < XBlock.level[ibLB])
	{
		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{
			jhalo = j;
			jp = j * 2;
			ibn = ibLB;
			conserveElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, ip, jp, XEv.h, XEv.zs, zb);
		}

	}
	
	if (XBlock.level[ib] < XBlock.level[ibLT])
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			jhalo = j;
			jp = (j - (XParam.blkwidth / 2)) * 2;
			ibn = ibLT;
			conserveElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, ip, jp, XEv.h, XEv.zs, zb);
		}

	}
	
	// Prolongation
	int il, jl, im,jm;
	ihalo = -1;

	if (XBlock.level[ib] > XBlock.level[ibLB])
	{
		//
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//
			jhalo = j;
			ibn = ibLB;

			il = 0;
			jl = j;

			ip = XParam.blkwidth - 1;
			jp = XBlock.RightBot[ibLB] == ib ? floor(j * T(0.5)) : (floor(j * T(0.5)) + XParam.blkwidth / 2);

			im = ip;
			jm = ceil(j * T(0.5)) * 2 > j ? jp + 1 : jp - 1;

			ProlongationElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, XEv.h, XEv.zs, zb);
		}

	}
	
}

template <class T> __global__ void conserveElevationLeft(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	unsigned int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = 0;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int LB = XBlock.LeftBot[ib];
	int LT = XBlock.LeftTop[ib];

	int ii, ir, it, itr, jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, hwet;

	int ihalo , jhalo, i, j, ibn, write;

	ihalo = -1;
	jhalo = iy;
	i = XParam.blkwidth - 2;

	if (lev < XBlock.level[LB] && iy < (blockDim.y / 2))
	{
		ibn = LB;
		j = iy * 2;

		conserveElevation(XParam.halowidth, blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, XEv.h, XEv.zs, zb);
	}
	if (lev < XBlock.level[LT] && iy >= (blockDim.y / 2))
	{
		ibn = LT;
		j = (iy - (blockDim.y / 2)) * 2;

		conserveElevation(XParam.halowidth, blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, XEv.h, XEv.zs, zb);
	}

	// Prolongation
	int ip, jp, il, jl, im, jm;
	ihalo = -1;

	if (XBlock.level[ib] > XBlock.level[LB])
	{
		//
		
		jhalo = iy;
		ibn = LB;

		il = 0;
		jl = iy;

		ip = XParam.blkwidth - 1;
		jp = XBlock.RightBot[ibn] == ib ? floor(iy * T(0.5)) : (floor(iy * T(0.5)) + blockDim.y / 2);


		im = ip;
		jm = ceil(iy * T(0.5)) * 2 > iy ? jp + 1 : jp - 1;

		ProlongationElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, XEv.h, XEv.zs, zb);
		

	}
}



template <class T> void conserveElevationRight(Param XParam, int ib, int ibRB, int ibRT, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ihalo, jhalo, ibn, ip, jp;

	if (XBlock.level[ib] < XBlock.level[ibRB])
	{
		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{
			conserveElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibRB, XParam.blkwidth, j, 0, j*2, XEv.h, XEv.zs, zb);
		}

	}
	if (XBlock.level[ib] < XBlock.level[ibRT])
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			conserveElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibRT, XParam.blkwidth, j, 0, (j - (XParam.blkwidth / 2)) * 2, XEv.h, XEv.zs, zb);
		}

	}

	// Prolongation
	int il, jl, im, jm;
	ihalo = XParam.blkwidth;

	if (XBlock.level[ib] > XBlock.level[ibRB])
	{
		//
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//
			jhalo = j;
			ibn = ibRB;

			il = XParam.blkwidth-1;
			jl = j;

			ip = 0;
			jp = XBlock.LeftBot[ibn] == ib ? floor(j * T(0.5)) : (floor(j * T(0.5)) + XParam.blkwidth / 2);
			im = ip;
			jm = ceil(j * T(0.5)) * 2 > j ? jp + 1 : jp - 1;

			ProlongationElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, XEv.h, XEv.zs, zb);
		}

	}
}

template <class T> __global__ void conserveElevationRight(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	unsigned int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = blockDim.y - 1;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int RB = XBlock.RightBot[ib];
	int RT = XBlock.RightTop[ib];

	int ii, ir, it, itr, jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, hwet;

	int ihalo, jhalo, i, j, ibn, write;

	ihalo = blockDim.y;
	jhalo = iy;

	i = 0;

	if (lev < XBlock.level[RB] && iy < (blockDim.y / 2))
	{
		ibn = RB;
		j = iy * 2;

		conserveElevation(XParam.halowidth, blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, XEv.h, XEv.zs, zb);
	}
	if (lev < XBlock.level[RT] && iy >= (blockDim.y / 2))
	{
		ibn = RT;
		j = (iy - (blockDim.y / 2)) * 2;

		conserveElevation(XParam.halowidth, blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, XEv.h, XEv.zs, zb);
	}
	
	// Prolongation
	int ip, jp, il, jl, im, jm;
	//ihalo = -1;

	if (lev > XBlock.level[RB])
	{
		//

		jhalo = iy;
		ibn = RB;

		il = blockDim.y - 1;
		jl = iy;

		ip = 0;
		jp = XBlock.LeftBot[ibn] == ib ? floor(iy * T(0.5)) : (floor(iy * T(0.5)) + blockDim.y / 2);

		im = ip;
		jm = ceil(iy * T(0.5)) * 2 > iy ? jp + 1 : jp - 1;

		ProlongationElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, XEv.h, XEv.zs, zb);


	}
	
}

template <class T> void conserveElevationTop(Param XParam, int ib, int ibTL, int ibTR, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ihalo, jhalo, ibn, ip, jp;

	int write;

	if (XBlock.level[ib] < XBlock.level[ibTL])
	{
		for (int i = 0; i < XParam.blkwidth / 2; i++)
		{
			conserveElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibTL, i, XParam.blkwidth, i*2, 0, XEv.h, XEv.zs, zb);
		}

	}
	if (XBlock.level[ib] < XBlock.level[ibTR])
	{
		for (int i = (XParam.blkwidth / 2); i < (XParam.blkwidth); i++)
		{
			conserveElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibTR, i, XParam.blkwidth, (i - (XParam.blkwidth / 2)) * 2, 0, XEv.h, XEv.zs, zb);
		}

	}

	// Prolongation
	int il, jl, im, jm;
	jhalo = XParam.blkwidth;

	if (XBlock.level[ib] > XBlock.level[ibTL])
	{
		//
		for (int i = 0; i < XParam.blkwidth; i++)
		{
			//
			ihalo = i;
			ibn = ibTL;

			il = i;
			jl = XParam.blkwidth - 1;

			jp = 0;
			ip = XBlock.BotLeft[ibn] == ib ? floor(i * T(0.5)) : (floor(i * T(0.5)) + XParam.blkwidth / 2);

			jm = jp;
			im = ceil(i * T(0.5)) * 2 > i ? ip + 1 : ip - 1;

			ProlongationElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, XEv.h, XEv.zs, zb);
		}

	}
}

template <class T> __global__ void conserveElevationTop(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	unsigned int blkmemwidth = blockDim.x + XParam.halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int iy = blockDim.x - 1;
	unsigned int ix = threadIdx.x;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int TL = XBlock.TopLeft[ib];
	int TR = XBlock.TopRight[ib];

	int ii, ir, it, itr, jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, hwet;

	int ihalo, jhalo, i, j, ibn, write;

	ihalo = ix;
	jhalo = blockDim.x;
	j = 0;

	if (lev < XBlock.level[TL] && ix < (blockDim.x / 2))
	{
		ibn = TL;
		
		i = ix * 2;

		conserveElevation(XParam.halowidth, blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, XEv.h, XEv.zs, zb);
	}
	if (lev < XBlock.level[TR] && ix >= (blockDim.x / 2))
	{
		ibn = TR;
		i = (ix - (blockDim.x / 2)) * 2;

		conserveElevation(XParam.halowidth, blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, XEv.h, XEv.zs, zb);
	}

	// Prolongation
	int ip, jp, il, jl, im, jm;
	

	if (lev > XBlock.level[TL])
	{
		//

		ihalo = ix;
		ibn = TL;

		il = ix;
		jl = blockDim.x - 1;

		jp = 0;
		ip = XBlock.BotLeft[ibn] == ib ? floor(ix * T(0.5)) : (floor(ix * T(0.5)) + blockDim.x / 2);

		jm = jp;
		im = ceil(ix * T(0.5)) * 2 > ix ? ip + 1 : ip - 1;

		ProlongationElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, XEv.h, XEv.zs, zb);


	}
}

template <class T> void conserveElevationBot(Param XParam, int ib, int ibBL, int ibBR, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	int ihalo, jhalo, ibn, ip, jp;

	int write;

	if (XBlock.level[ib] < XBlock.level[ibBL])
	{
		for (int i = 0; i < XParam.blkwidth / 2; i++)
		{
			conserveElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibBL, i,-1, i * 2, XParam.blkwidth-2, XEv.h, XEv.zs, zb);
		}

	}
	if (XBlock.level[ib] < XBlock.level[ibBR])
	{
		for (int i = (XParam.blkwidth / 2); i < (XParam.blkwidth); i++)
		{
			conserveElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibBR, i, -1, (i - (XParam.blkwidth / 2)) * 2, XParam.blkwidth-2, XEv.h, XEv.zs, zb);
		}

	}


	// Prolongation
	int  il, jl, im, jm;
	jhalo = -1;

	if (XBlock.level[ib] > XBlock.level[ibBL])
	{
		//
		for (int i = 0; i < XParam.blkwidth; i++)
		{
			//
			ihalo = i;
			ibn = ibBL;

			il = i;
			jl = 0;

			jp = XParam.blkwidth - 1;
			ip = XBlock.TopLeft[ibn] == ib ? floor(i * T(0.5)) : (floor(i * T(0.5)) + XParam.blkwidth / 2);

			jm = jp;
			im = ceil(i * T(0.5)) * 2 > i ? ip + 1 : ip - 1;

			ProlongationElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, XEv.h, XEv.zs, zb);
		}

	}
}


template <class T> __global__ void conserveElevationBot(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, T* zb)
{
	unsigned int blkmemwidth = blockDim.x + XParam.halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int iy = 0;
	unsigned int ix = threadIdx.x;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int BL = XBlock.BotLeft[ib];
	int BR = XBlock.BotRight[ib];

	int ii, ir, it, itr, jj;
	T iiwet, irwet, itwet, itrwet;
	T zswet, hwet;

	int ihalo, jhalo, ibn, write;
	int i, j;

	ihalo = ix;
	jhalo = -1;
	j = blockDim.x-2;

	if (lev < XBlock.level[BL] && ix < (blockDim.x / 2))
	{
		ibn = BL;

		i = ix * 2;

		conserveElevation(XParam.halowidth, blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, XEv.h, XEv.zs, zb);
	}
	if (lev < XBlock.level[BR] && ix >= (blockDim.x / 2))
	{
		ibn = BR;
		i = (ix - (blockDim.x / 2)) * 2;

		conserveElevation(XParam.halowidth, blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, i, j, XEv.h, XEv.zs, zb);
	}

	// Prolongation
	int ip, jp, il, jl, im, jm;
	//int ip, jp, il, jl, im, jm;
	//jhalo = -1;

	if (lev > XBlock.level[BL])
	{
		//

		ihalo = ix;
		ibn = BL;

		il = ix;
		jl = 0;

		jp = blockDim.x - 1;
		ip = XBlock.TopLeft[ibn] == ib ? floor(ix *T(0.5)) : (floor(ix*T(0.5)) + blockDim.x / 2);

		jm = jp;
		im = ceil(ix * T(0.5)) * 2 > ix ? ip + 1 : ip - 1;

		ProlongationElevation(XParam.halowidth, XParam.blkmemwidth, T(XParam.eps), ib, ibn, ihalo, jhalo, il, jl, ip, jp, im, jm, XEv.h, XEv.zs, zb);


	}

}
