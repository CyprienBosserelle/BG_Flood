#include "hip/hip_runtime.h"
#include "Gradients.h"


/*! \fn void gradientGPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad,T* zb)
* Wrapping function to calculate gradien of evolving variables on GPU
* This function is the entry point to the gradient functions on the GPU
*/
template <class T> void gradientGPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad,T* zb)
{
	//const int num_streams = 4;
	/*
	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}
	*/
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 blockDimLR(1, XParam.blkwidth, 1);
	dim3 blockDimBT(XParam.blkwidth, 1, 1);
	dim3 blockDimfull(XParam.blkmemwidth, XParam.blkmemwidth, 1);
	
	dim3 gridDim(XParam.nblk, 1, 1);

	//gradient << < gridDim, blockDim, 0, streams[1] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx, XGrad.dhdy);
	gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx, XGrad.dhdy);
	CUDA_CHECK(hipDeviceSynchronize());

	//gradient << < gridDim, blockDim, 0, streams[2] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	CUDA_CHECK(hipDeviceSynchronize());

	//gradient << < gridDim, blockDim, 0, streams[3] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx, XGrad.dudy);
	gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx, XGrad.dudy);
	CUDA_CHECK(hipDeviceSynchronize());

	//gradient << < gridDim, blockDim, 0, streams[0] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx, XGrad.dvdy);
	gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx, XGrad.dvdy);
	CUDA_CHECK(hipDeviceSynchronize());


	//CUDA_CHECK(hipDeviceSynchronize());
	/*
	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}
	*/


	//fillHaloGPU(XParam, XBlock, XGrad);
	gradientHaloGPU(XParam, XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
	gradientHaloGPU(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	gradientHaloGPU(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
	gradientHaloGPU(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);


	if (XParam.conserveElevation)
	{
		conserveElevationGradHaloGPU(XParam, XBlock, XEv.h, XEv.zs, zb, XGrad.dhdx, XGrad.dzsdx, XGrad.dhdy, XGrad.dzsdy);
	}
	else
	{
		refine_linearGPU(XParam, XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
		//refine_linearGPU(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
		refine_linearGPU(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
		refine_linearGPU(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);

		RecalculateZsGPU << < gridDim, blockDimfull, 0 >> > (XParam, XBlock, XEv, zb);
		CUDA_CHECK(hipDeviceSynchronize());

		//gradient << < gridDim, blockDim, 0, streams[1] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx, XGrad.dhdy);
		gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx, XGrad.dhdy);
		CUDA_CHECK(hipDeviceSynchronize());

		//gradient << < gridDim, blockDim, 0, streams[2] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
		gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
		CUDA_CHECK(hipDeviceSynchronize());

		//gradient << < gridDim, blockDim, 0, streams[3] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx, XGrad.dudy);
		gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx, XGrad.dudy);
		CUDA_CHECK(hipDeviceSynchronize());

		//gradient << < gridDim, blockDim, 0, streams[0] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx, XGrad.dvdy);
		gradient << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx, XGrad.dvdy);
		CUDA_CHECK(hipDeviceSynchronize());



		/*
		gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, 0, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx);
		gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, XParam.blkwidth-1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx);

		gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, 0, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdy);
		gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, XParam.blkwidth-1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdy);

		gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, 0, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx);
		gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx);

		gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, 0, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdy);
		gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdy);

		gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, 0, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx);
		gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx);

		gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, 0, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudy);
		gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudy);

		gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, 0, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx);
		gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx);

		gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, 0, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdy);
		gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdy);
		CUDA_CHECK(hipDeviceSynchronize());
		*/


		gradientHaloGPU(XParam, XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
		gradientHaloGPU(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
		gradientHaloGPU(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
		gradientHaloGPU(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);

		if (XParam.engine == 1)
		{
			//  wet slope limiter
			WetsloperesetXGPU << < gridDim, blockDim, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			WetsloperesetYGPU << < gridDim, blockDim, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			// ALso do the slope limiter on the halo
			WetsloperesetHaloLeftGPU << < gridDim, blockDimLR, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			WetsloperesetHaloRightGPU << < gridDim, blockDimLR, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			WetsloperesetHaloBotGPU << < gridDim, blockDimBT, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			WetsloperesetHaloTopGPU << < gridDim, blockDimBT, 0 >> > (XParam, XBlock, XEv, XGrad, zb);

			CUDA_CHECK(hipDeviceSynchronize());
		}

	}
	//conserveElevationGradHaloGPU(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	//conserveElevationGradHaloGPU(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
	//conserveElevationGradHaloGPU(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);

}
template void gradientGPU<float>(Param XParam, BlockP<float>XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, float * zb);
template void gradientGPU<double>(Param XParam,  BlockP<double>XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, double * zb);

template <class T> void gradientGPUnew(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 blockDimLR(1, XParam.blkwidth, 1);
	dim3 blockDimBT(XParam.blkwidth, 1, 1);
	dim3 blockDimLR2(2, XParam.blkwidth, 1);
	dim3 blockDimBT2(XParam.blkwidth, 2, 1);
	dim3 blockDimfull(XParam.blkmemwidth, XParam.blkmemwidth, 1);

	dim3 gridDim(XParam.nblk, 1, 1);

	//gradient << < gridDim, blockDim, 0, streams[1] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx, XGrad.dhdy);
	gradientSM << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx, XGrad.dhdy);
	CUDA_CHECK(hipDeviceSynchronize());

	//gradient << < gridDim, blockDim, 0, streams[2] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	gradientSM << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	CUDA_CHECK(hipDeviceSynchronize());

	//gradient << < gridDim, blockDim, 0, streams[3] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx, XGrad.dudy);
	gradientSM << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx, XGrad.dudy);
	CUDA_CHECK(hipDeviceSynchronize());

	//gradient << < gridDim, blockDim, 0, streams[0] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx, XGrad.dvdy);
	gradientSM << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx, XGrad.dvdy);
	CUDA_CHECK(hipDeviceSynchronize());


	//CUDA_CHECK(hipDeviceSynchronize());
	
	


	//fillHaloGPU(XParam, XBlock, XGrad);
	gradientHaloGPU(XParam, XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
	gradientHaloGPU(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	gradientHaloGPU(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
	gradientHaloGPU(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);


	if (XParam.conserveElevation)
	{
		conserveElevationGradHaloGPU(XParam, XBlock, XEv.h, XEv.zs, zb, XGrad.dhdx, XGrad.dzsdx, XGrad.dhdy, XGrad.dzsdy);
	}
	else
	{
		refine_linearGPU(XParam, XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
		//refine_linearGPU(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
		refine_linearGPU(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
		refine_linearGPU(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);

		RecalculateZsGPU << < gridDim, blockDimfull, 0 >> > (XParam, XBlock, XEv, zb);
		CUDA_CHECK(hipDeviceSynchronize());
		
		/*
		//gradient << < gridDim, blockDim, 0, streams[1] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx, XGrad.dhdy);
		gradientSM << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx, XGrad.dhdy);
		CUDA_CHECK(hipDeviceSynchronize());

		//gradient << < gridDim, blockDim, 0, streams[2] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
		gradientSM << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
		CUDA_CHECK(hipDeviceSynchronize());

		//gradient << < gridDim, blockDim, 0, streams[3] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx, XGrad.dudy);
		gradientSM << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx, XGrad.dudy);
		CUDA_CHECK(hipDeviceSynchronize());

		//gradient << < gridDim, blockDim, 0, streams[0] >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx, XGrad.dvdy);
		gradientSM << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx, XGrad.dvdy);
		CUDA_CHECK(hipDeviceSynchronize());
		*/
		/*
		const int num_streams = 16;
		
		hipStream_t streams[num_streams];

		for (int i = 0; i < num_streams; i++)
		{
			CUDA_CHECK(hipStreamCreate(&streams[i]));
		}
		*/
		
		
		
		gradientedgeX << < gridDim, blockDimLR2, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx);
		//CUDA_CHECK(hipDeviceSynchronize());
		//gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, XParam.blkwidth-1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdx);
		//CUDA_CHECK(hipDeviceSynchronize());

		gradientedgeY << < gridDim, blockDimBT2, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdy);
		//CUDA_CHECK(hipDeviceSynchronize());
		//gradientedgeY << < gridDim, blockDimBT, 0>> > (XParam.halowidth, XParam.blkwidth-1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.h, XGrad.dhdy);
		//CUDA_CHECK(hipDeviceSynchronize());

		gradientedgeX << < gridDim, blockDimLR2, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx);
		//CUDA_CHECK(hipDeviceSynchronize());
		//gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdx);
		//CUDA_CHECK(hipDeviceSynchronize());

		gradientedgeY << < gridDim, blockDimBT2, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdy);
		//CUDA_CHECK(hipDeviceSynchronize());
		//gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.zs, XGrad.dzsdy);
		//CUDA_CHECK(hipDeviceSynchronize());

		gradientedgeX << < gridDim, blockDimLR2, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx);
		//CUDA_CHECK(hipDeviceSynchronize());
		//gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudx);
		//CUDA_CHECK(hipDeviceSynchronize());

		gradientedgeY << < gridDim, blockDimBT2, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudy);
		//CUDA_CHECK(hipDeviceSynchronize());
		//gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.u, XGrad.dudy);
		//CUDA_CHECK(hipDeviceSynchronize());

		gradientedgeX << < gridDim, blockDimLR2, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx);
		//CUDA_CHECK(hipDeviceSynchronize());
		//gradientedgeX << < gridDim, blockDimLR, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdx);
		//CUDA_CHECK(hipDeviceSynchronize());

		gradientedgeY << < gridDim, blockDimBT2, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdy);
		//CUDA_CHECK(hipDeviceSynchronize());
		//gradientedgeY << < gridDim, blockDimBT, 0 >> > (XParam.halowidth, XParam.blkwidth - 1, XBlock.active, XBlock.level, (T)XParam.theta, (T)XParam.dx, XEv.v, XGrad.dvdy);
		CUDA_CHECK(hipDeviceSynchronize());
		
		/*
		for (int i = 0; i < num_streams; i++)
		{
			hipStreamDestroy(streams[i]);
		}
		*/
		gradientHaloGPU(XParam, XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
		gradientHaloGPU(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
		gradientHaloGPU(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
		gradientHaloGPU(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);

		if (XParam.engine == 1)
		{
			//  wet slope limiter
			WetsloperesetXGPU << < gridDim, blockDim, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			WetsloperesetYGPU << < gridDim, blockDim, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			// ALso do the slope limiter on the halo
			WetsloperesetHaloLeftGPU << < gridDim, blockDimLR, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			WetsloperesetHaloRightGPU << < gridDim, blockDimLR, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			WetsloperesetHaloBotGPU << < gridDim, blockDimBT, 0 >> > (XParam, XBlock, XEv, XGrad, zb);
			CUDA_CHECK(hipDeviceSynchronize());

			WetsloperesetHaloTopGPU << < gridDim, blockDimBT, 0 >> > (XParam, XBlock, XEv, XGrad, zb);

			CUDA_CHECK(hipDeviceSynchronize());
		}

	}
	//conserveElevationGradHaloGPU(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	//conserveElevationGradHaloGPU(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
	//conserveElevationGradHaloGPU(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);

}
template void gradientGPUnew<float>(Param XParam, BlockP<float>XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, float* zb);
template void gradientGPUnew<double>(Param XParam, BlockP<double>XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, double* zb);


/*! \fn void gradient(int halowidth, int* active, int* level, T theta, T dx, T* a, T* dadx, T* dady)
* Device kernel for calculating grdients for an evolving poarameter using the minmod limiter
* 
*/
template <class T> __global__ void gradient(int halowidth, int* active, int* level, T theta, T dx, T* a, T* dadx, T* dady)
{
	//int *leftblk,int *rightblk,int* topblk, int * botblk,

	//int ix = threadIdx.x+1;
	//int iy = threadIdx.y+1;
	int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = active[ibl];

	int lev = level[ib];

	T delta = calcres(dx, lev);


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	//int iright, ileft, ibot;
	// shared array index to make the code bit more readable
	//unsigned int sx = ix + halowidth;
	//unsigned int sy = iy + halowidth;


	T a_l, a_t, a_r, a_b,a_i;

	a_i = a[i];


	a_l = a[memloc(halowidth, blkmemwidth, ix - 1, iy, ib)];
	a_t = a[memloc(halowidth, blkmemwidth, ix , iy + 1, ib)];
	a_r = a[memloc(halowidth, blkmemwidth, ix + 1, iy, ib)];
	a_b = a[memloc(halowidth, blkmemwidth, ix, iy - 1, ib)];
	//__shared__ T a_s[18][18];



	//__syncthreads();
	//__syncwarp;

	dadx[i] = minmod2(theta, a_l, a_i, a_r) / delta;
	
	dady[i] = minmod2(theta, a_b, a_i, a_t) / delta;


}
template __global__ void gradient<float>(int halowidth, int* active, int* level, float theta, float dx, float* a, float* dadx, float* dady);
template __global__ void gradient<double>(int halowidth, int* active, int* level, double theta, double dx, double* a, double* dadx, double* dady);



/*! \fn void gradientSM(int halowidth, int* active, int* level, T theta, T dx, T* a, T* dadx, T* dady)
* Depreciated shared memory version of Device kernel for calculating gradients
* Much slower than above
*/
template <class T> __global__ void gradientSM(int halowidth, int* active, int* level, T theta, T dx, T* a, T* dadx, T* dady)
{
	//int *leftblk,int *rightblk,int* topblk, int * botblk,

	//int ix = threadIdx.x+1;
	//int iy = threadIdx.y+1;
	int blkmemwidth = blockDim.x + halowidth * 2;
	int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = active[ibl];

	int lev = level[ib];

	T delta = calcres(dx, lev);


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	int iright, ileft, itop, ibot;
	// shared array index to make the code bit more readable
	int sx = ix + halowidth;
	int sy = iy + halowidth;



	__shared__ T a_s[18][18];
	


	a_s[sx][sy] = a[i];
	
	//syncthread is needed here ?
		

	// read the halo around the tile
	if (threadIdx.x == blockDim.x - 1)
	{
		iright = memloc(halowidth, blkmemwidth, ix + 1, iy, ib);
		a_s[sx + 1][sy] = a[iright];
		
	}
	

	if (threadIdx.x == 0)
	{
		ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);;
		a_s[sx - 1][sy] = a[ileft];
		
	}

	if (threadIdx.y == blockDim.y - 1)
	{
		itop = memloc(halowidth, blkmemwidth, ix, iy + 1, ib);;
		a_s[sx][sy + 1] = a[itop];
		
	}
	
	if (threadIdx.y == 0)
	{
		ibot = memloc(halowidth, blkmemwidth, ix, iy - 1, ib);
		a_s[sx][sy - 1] = a[ibot];
		
	}

	__syncthreads();



	dadx[i] = minmod2(theta, a_s[sx - 1][sy], a_s[sx][sy], a_s[sx + 1][sy]) / delta;
	
	dady[i] = minmod2(theta, a_s[sx][sy - 1], a_s[sx][sy], a_s[sx][sy + 1]) / delta;


}
template __global__ void gradientSM<float>(int halowidth, int* active, int* level, float theta, float dx, float* a, float* dadx, float* dady);
template __global__ void gradientSM<double>(int halowidth, int* active, int* level, double theta, double dx, double* a, double* dadx, double* dady);

template <class T> __global__ void gradientSMB(int halowidth, int* active, int* level, T theta, T dx, T* a, T* dadx, T* dady)
{
	//int *leftblk,int *rightblk,int* topblk, int * botblk,

	//int ix = threadIdx.x+1;
	//int iy = threadIdx.y+1;
	int blkmemwidth = 18;
	int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x-1;
	int iy = threadIdx.y-1;
	int ibl = blockIdx.x;
	int ib = active[ibl];

	int lev = level[ib];

	T delta = calcres(dx, lev);


	
	int iright, ileft, itop, ibot;
	// shared array index to make the code bit more readable
	int sx = ix + 1;
	int sy = iy + 1;

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	//int o = memloc(halowidth, blkmemwidth, sx, sy, ib);

	__shared__ T a_s[18][18];




	a_s[sx][sy] = a[i];
	__syncthreads();
	//syncthread is needed here ?

	T aleft, aright, atop, abot;
	aleft = a_s[sx - 1][sy];
	aright = a_s[sx + 1][sy];
	atop = a_s[sx][sy + 1];
	abot = a_s[sx][sy - 1];


	if (ix >= 0 && ix < 16 && iy >=0 && iy < 16)
	{

		dadx[i] = minmod2(theta, aleft, a_s[sx][sy], aright) / delta;

		dady[i] = minmod2(theta, abot, a_s[sx][sy], atop) / delta;
	}

}
template __global__ void gradientSMB<float>(int halowidth, int* active, int* level, float theta, float dx, float* a, float* dadx, float* dady);
template __global__ void gradientSMB<double>(int halowidth, int* active, int* level, double theta, double dx, double* a, double* dadx, double* dady);





/*! \fn void gradientedgeX(int halowidth, int ix, int* active, int* level, T theta, T dx, T* a, T* dadx)
* Device kernel for calculating gradients for an evolving parameter using the minmod limiter only at specific column (i.e. fixed ix)
*
*/
template <class T> __global__ void gradientedgeX(int halowidth, int* active, int* level, T theta, T dx, T* a, T* dadx)
{
	//int *leftblk,int *rightblk,int* topblk, int * botblk,

	//int ix = threadIdx.x+1;
	//int iy = threadIdx.y+1;
	int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	//unsigned int ix = threadIdx.x;
	int ix;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = active[ibl];

	if (threadIdx.x == 0)
	{
		ix = 0;
	}
	else
	{
		ix = blockDim.y - 1;
	}
		

	int lev = level[ib];

	T delta = calcres(dx, lev);


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	//int iright, ileft, ibot;
	// shared array index to make the code bit more readable
	//unsigned int sx = ix + halowidth;
	//unsigned int sy = iy + halowidth;


	T a_l, a_r, a_i;

	a_i = a[i];


	a_l = a[memloc(halowidth, blkmemwidth, ix - 1, iy, ib)];
	//a_t = a[memloc(halowidth, blkmemwidth, ix, iy + 1, ib)];
	a_r = a[memloc(halowidth, blkmemwidth, ix + 1, iy, ib)];
	//a_b = a[memloc(halowidth, blkmemwidth, ix, iy - 1, ib)];
	//__shared__ T a_s[18][18];



	//__syncthreads();
	//__syncwarp;

	dadx[i] = minmod2(theta, a_l, a_i, a_r) / delta;




}
template __global__ void gradientedgeX<float>(int halowidth,  int* active, int* level, float theta, float dx, float* a, float* dadx);
template __global__ void gradientedgeX<double>(int halowidth, int* active, int* level, double theta, double dx, double* a, double* dadx);



/*! \fn void gradientedgeY(int halowidth, int iy, int* active, int* level, T theta, T dx, T* a, T* dady)
* Device kernel for calculating gradients for an evolving parameter using the minmod limiter only at specific row (i.e. fixed iy)
*
*/
template <class T> __global__ void gradientedgeY(int halowidth, int* active, int* level, T theta, T dx, T* a, T* dady)
{
	//int *leftblk,int *rightblk,int* topblk, int * botblk,

	//int ix = threadIdx.x+1;
	//int iy = threadIdx.y+1;
	int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy;
	//unsigned int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = active[ibl];

	int lev = level[ib];

	T delta = calcres(dx, lev);

	if (threadIdx.y == 0)
	{
		iy = 0;
	}
	else
	{
		iy = blockDim.y - 1;
	}


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	//int iright, ileft, ibot;
	// shared array index to make the code bit more readable
	//unsigned int sx = ix + halowidth;
	//unsigned int sy = iy + halowidth;


	T  a_t, a_b, a_i;

	a_i = a[i];


	//a_l = a[memloc(halowidth, blkmemwidth, ix - 1, iy, ib)];
	a_t = a[memloc(halowidth, blkmemwidth, ix, iy + 1, ib)];
	//a_r = a[memloc(halowidth, blkmemwidth, ix + 1, iy, ib)];
	a_b = a[memloc(halowidth, blkmemwidth, ix, iy - 1, ib)];
	//__shared__ T a_s[18][18];



	//__syncthreads();
	//__syncwarp;

	//dadx[i] = minmod2(theta, a_l, a_i, a_r) / delta;

	dady[i] = minmod2(theta, a_b, a_i, a_t) / delta;


}
template __global__ void gradientedgeY<float>(int halowidth, int* active, int* level, float theta, float dx, float* a, float* dady);
template __global__ void gradientedgeY<double>(int halowidth, int* active, int* level, double theta, double dx, double* a, double* dady);




template <class T> void gradientC(Param XParam, BlockP<T> XBlock, T* a, T* dadx, T* dady)
{

	int i,ib;
	int xplus, xminus, yplus, yminus;

	T delta;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		delta = calcres(T(XParam.dx), XBlock.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				i = memloc(XParam, ix,iy,ib);
				
				//
				xplus = memloc(XParam, ix+1, iy, ib);
				xminus = memloc(XParam, ix-1, iy, ib);
				yplus = memloc(XParam, ix, iy+1, ib);
				yminus = memloc(XParam, ix, iy-1, ib);

				dadx[i] = minmod2(T(XParam.theta), a[xminus], a[i], a[xplus]) / delta;
				dady[i] = minmod2(T(XParam.theta), a[yminus], a[i], a[yplus]) / delta;
			}


		}


	}



}
template void gradientC<float>(Param XParam, BlockP<float> XBlock, float* a, float* dadx, float* dady);
template void gradientC<double>(Param XParam, BlockP<double> XBlock, double* a, double* dadx, double* dady);

template <class T> void gradientCPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{


	std::thread t0(&gradientC<T>, XParam, XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
	std::thread t1(&gradientC<T>, XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	std::thread t2(&gradientC<T>, XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
	std::thread t3(&gradientC<T>, XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);

	t0.join();
	t1.join();
	t2.join();
	t3.join();

	//fillHalo(XParam, XBlock, XGrad);
	
	gradientHalo(XParam, XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
	gradientHalo(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	gradientHalo(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
	gradientHalo(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);
	
	if (XParam.conserveElevation)
	{
		conserveElevationGradHalo(XParam, XBlock, XEv.h, XEv.zs, zb, XGrad.dhdx, XGrad.dzsdx, XGrad.dhdy, XGrad.dzsdy);

	}
	
	
	refine_linear(XParam,XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
	refine_linear(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
	refine_linear(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);

	RecalculateZs(XParam, XBlock, XEv, zb);
				

	gradientHalo(XParam, XBlock, XEv.h, XGrad.dhdx, XGrad.dhdy);
	gradientHalo(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	gradientHalo(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
	gradientHalo(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdy);

	if (XParam.conserveElevation)
	{
		conserveElevationGradHalo(XParam, XBlock, XEv.h, XEv.zs, zb, XGrad.dhdx, XGrad.dzsdx, XGrad.dhdy, XGrad.dzsdy);

	}
	
	if (XParam.engine == 1)
	{
		WetsloperesetCPU(XParam, XBlock, XEv, XGrad, zb);

		WetsloperesetHaloLeftCPU(XParam, XBlock, XEv, XGrad, zb);
		WetsloperesetHaloRightCPU(XParam, XBlock, XEv, XGrad, zb);
		WetsloperesetHaloBotCPU(XParam, XBlock, XEv, XGrad, zb);
		WetsloperesetHaloTopCPU(XParam, XBlock, XEv, XGrad, zb);
	}


	


	//conserveElevationGradHalo(XParam, XBlock, XEv.zs, XGrad.dzsdx, XGrad.dzsdy);
	//conserveElevationGradHalo(XParam, XBlock, XEv.u, XGrad.dudx, XGrad.dudy);
	//conserveElevationGradHalo(XParam, XBlock, XEv.v, XGrad.dvdx, XGrad.dvdyythhhhhhhhhg);


}
template void gradientCPU<float>(Param XParam, BlockP<float>XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, float * zb);
template void gradientCPU<double>(Param XParam, BlockP<double>XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, double * zb);

template <class T> void WetsloperesetCPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	int i, ib;
	int xplus, xminus, yminus;

	T delta;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		delta = calcres(T(XParam.dx), XBlock.level[ib]);
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				i = memloc(XParam, ix, iy, ib);

				//
				xplus = memloc(XParam, ix + 1, iy, ib);
				xminus = memloc(XParam, ix - 1, iy, ib);
				//yplus = memloc(XParam, ix, iy + 1, ib);
				yminus = memloc(XParam, ix, iy - 1, ib);

				T dzsdxi = XGrad.dzsdx[i];
				T dzsdyi = XGrad.dzsdy[i];




				//Do X axis
				if (utils::sq(dzsdxi) > utils::sq(XGrad.dzbdx[i]))
				{
					T leftzs, rightzs;
					leftzs = XEv.zs[i] - XEv.h[i] - delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);
					rightzs = XEv.zs[i] - XEv.h[i] + delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);

					if (leftzs > XEv.zs[xminus] || rightzs > XEv.zs[xplus])
					{
						XGrad.dzsdx[i] = XGrad.dhdx[i] + XGrad.dzbdx[i];
					}

				}

				//Do Y axis
				if (utils::sq(dzsdyi) > utils::sq(XGrad.dzbdy[i]))
				{
					T botzs, topzs;
					botzs = XEv.zs[i] - XEv.h[i] - delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);
					topzs = XEv.zs[i] - XEv.h[i] + delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);

					if (botzs > XEv.zs[yminus] || topzs > XEv.zs[yminus])
					{
						XGrad.dzsdy[i] = XGrad.dhdy[i] + XGrad.dzbdy[i];
					}

				}


			}
		}
	}
}

template <class T> __global__ void WetsloperesetXGPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	unsigned int blkmemwidth = blockDim.x + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];

	T delta = calcres(XParam.dx, lev);


	int i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);

	int iright, ileft;
	iright = memloc(XParam.halowidth, blkmemwidth, ix + 1, iy, ib);
	ileft = memloc(XParam.halowidth, blkmemwidth, ix - 1, iy, ib);

	T dzsdxi = XGrad.dzsdx[i];


	if (utils::sq(dzsdxi) > utils::sq(XGrad.dzbdx[i]))
	{
		T leftzs, rightzs;
		leftzs = XEv.zs[i] - XEv.h[i] - delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);
		rightzs = XEv.zs[i] - XEv.h[i] + delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);

		if (leftzs > XEv.zs[ileft] || rightzs > XEv.zs[iright])
		{
			XGrad.dzsdx[i] = XGrad.dhdx[i] + XGrad.dzbdx[i];
		}

	}
	

}

template <class T> __global__ void WetsloperesetYGPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	unsigned int blkmemwidth = blockDim.x + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];

	T delta = calcres(XParam.dx, lev);


	int i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);

	int itop, ibot;
	itop = memloc(XParam.halowidth, blkmemwidth, ix, iy + 1, ib);
	ibot = memloc(XParam.halowidth, blkmemwidth, ix, iy - 1, ib);

	T dzsdyi = XGrad.dzsdy[i];


	if (utils::sq(dzsdyi) > utils::sq(XGrad.dzbdy[i]))
	{
		T botzs, topzs;
		botzs = XEv.zs[i] - XEv.h[i] - delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);
		topzs = XEv.zs[i] - XEv.h[i] + delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);

		if (botzs > XEv.zs[ibot] || topzs > XEv.zs[itop])
		{
			XGrad.dzsdy[i] = XGrad.dhdy[i] + XGrad.dzbdy[i];
		}

	}


}


template <class T> __global__ void WetsloperesetHaloLeftGPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int ix = -1;
	int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];


	T delta = calcres(XParam.dx, lev);

	T zsi, zsright, zsleft;

	int i, iright;
	i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);
	iright = memloc(XParam.halowidth, blkmemwidth, ix + 1, iy, ib);

	zsi = XEv.zs[i];
	zsright = XEv.zs[iright];

	int read, jj, ii, ir, it, itr;

	if (XBlock.LeftBot[ib] == ib)//The lower half is a boundary 
	{
		if (iy < (XParam.blkwidth / 2))
		{

			read = memloc(XParam.halowidth, blkmemwidth, 0, iy, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			zsleft = XEv.zs[read];
		}

		if (XBlock.LeftTop[ib] == ib) // boundary on the top half too
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, 0, iy, ib);

				zsleft = XEv.zs[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (iy >= (XParam.blkwidth / 2))
			{

				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj, XBlock.LeftTop[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj + 1, XBlock.LeftTop[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj + 1, XBlock.LeftTop[ib]);

				zsleft = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.LeftBot[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 2), iy, XBlock.LeftBot[ib]);
		zsleft = XEv.zs[read];

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] > XBlock.level[ib])
	{

		if (iy < (XParam.blkwidth / 2))
		{

			jj = iy * 2;
			int bb = XBlock.LeftBot[ib];

			ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj, bb);
			ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj, bb);
			it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj + 1, bb);
			itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj + 1, bb);

			zsleft = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.LeftTop[ib] == ib)
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, 0, iy, ib);

				zsleft = XEv.zs[read];
			}
		}
		else
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//
				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj, XBlock.LeftTop[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj + 1, XBlock.LeftTop[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj + 1, XBlock.LeftTop[ib]);

				zsleft = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib ? ceil(iy * (T)0.5) : ceil(iy * (T)0.5) + XParam.blkwidth / 2;
		T jr = ceil(iy * (T)0.5) * 2 > iy ? T(0.25) : T(0.75);

		ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 1), jj, XBlock.LeftBot[ib]);
		ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 2), jj, XBlock.LeftBot[ib]);
		it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 1), jj - 1, XBlock.LeftBot[ib]);
		itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 2), jj - 1, XBlock.LeftBot[ib]);

		zsleft = BilinearInterpolation(XEv.zs[itr], XEv.zs[ir], XEv.zs[it], XEv.zs[ii], T(0.0), T(1.0), T(0.0), T(1.0), T(0.75), jr);
	}
	

	T dzsdxi = XGrad.dzsdx[i];


	if (utils::sq(dzsdxi) > utils::sq(XGrad.dzbdx[i]))
	{
		T leftzs, rightzs;
		leftzs = zsi - XEv.h[i] - delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);
		rightzs = zsi - XEv.h[i] + delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);

		if (leftzs > zsleft || rightzs > zsright)
		{
			XGrad.dzsdx[i] = XGrad.dhdx[i] + XGrad.dzbdx[i];
		}

	}


}

template <class T> void WetsloperesetHaloLeftCPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{


	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int ix = -1;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{

		unsigned int ib = XBlock.active[ibl];

		int lev = XBlock.level[ib];


		T delta = calcres(XParam.dx, lev);

		T zsi, zsright, zsleft;

		for (int iy = 0; iy <= XParam.blkwidth; iy++)
		{

			int i, iright;
			i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);
			iright = memloc(XParam.halowidth, blkmemwidth, ix + 1, iy, ib);

			zsi = XEv.zs[i];
			zsright = XEv.zs[iright];

			int read, jj, ii, ir, it, itr;

			if (XBlock.LeftBot[ib] == ib)//The lower half is a boundary 
			{
				if (iy < (XParam.blkwidth / 2))
				{

					read = memloc(XParam.halowidth, blkmemwidth, 0, iy, ib);// or memloc(XParam, -1, j, ib) but they should be the same

					zsleft = XEv.zs[read];
				}

				if (XBlock.LeftTop[ib] == ib) // boundary on the top half too
				{
					if (iy >= (XParam.blkwidth / 2))
					{
						//

						read = memloc(XParam.halowidth, blkmemwidth, 0, iy, ib);

						zsleft = XEv.zs[read];
					}
				}
				else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
				{

					if (iy >= (XParam.blkwidth / 2))
					{

						jj = (iy - XParam.blkwidth / 2) * 2;
						ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj, XBlock.LeftTop[ib]);
						ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj, XBlock.LeftTop[ib]);
						it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj + 1, XBlock.LeftTop[ib]);
						itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj + 1, XBlock.LeftTop[ib]);

						zsleft = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);

					}
				}
			}
			else if (XBlock.level[ib] == XBlock.level[XBlock.LeftBot[ib]]) // LeftTop block does not exist
			{

				read = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 2), iy, XBlock.LeftBot[ib]);
				zsleft = XEv.zs[read];

			}
			else if (XBlock.level[XBlock.LeftBot[ib]] > XBlock.level[ib])
			{

				if (iy < (XParam.blkwidth / 2))
				{

					jj = iy * 2;
					int bb = XBlock.LeftBot[ib];

					ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj, bb);
					ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj, bb);
					it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj + 1, bb);
					itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj + 1, bb);

					zsleft = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
				}
				//now find out aboy lefttop block
				if (XBlock.LeftTop[ib] == ib)
				{
					if (iy >= (XParam.blkwidth / 2))
					{
						//

						read = memloc(XParam.halowidth, blkmemwidth, 0, iy, ib);

						zsleft = XEv.zs[read];
					}
				}
				else
				{
					if (iy >= (XParam.blkwidth / 2))
					{
						//
						jj = (iy - XParam.blkwidth / 2) * 2;
						ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj, XBlock.LeftTop[ib]);
						ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj, XBlock.LeftTop[ib]);
						it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj + 1, XBlock.LeftTop[ib]);
						itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj + 1, XBlock.LeftTop[ib]);

						zsleft = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
					}
				}

			}
			else if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
			{
				jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib ? ftoi(ceil(iy * (T)0.5)) : ftoi(ceil(iy * (T)0.5) + XParam.blkwidth / 2);
				T jr = ceil(iy * (T)0.5) * 2 > iy ? T(0.25) : T(0.75);

				ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 1), jj, XBlock.LeftBot[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 2), jj, XBlock.LeftBot[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 1), jj - 1, XBlock.LeftBot[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 2), jj - 1, XBlock.LeftBot[ib]);

				zsleft = BilinearInterpolation(XEv.zs[itr], XEv.zs[ir], XEv.zs[it], XEv.zs[ii], T(0.0), T(1.0), T(0.0), T(1.0), T(0.75), jr);
			}


			T dzsdxi = XGrad.dzsdx[i];


			if (utils::sq(dzsdxi) > utils::sq(XGrad.dzbdx[i]))
			{
				T leftzs, rightzs;
				leftzs = zsi - XEv.h[i] - delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);
				rightzs = zsi - XEv.h[i] + delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);

				if (leftzs > zsleft || rightzs > zsright)
				{
					XGrad.dzsdx[i] = XGrad.dhdx[i] + XGrad.dzbdx[i];
				}

			}
		}

	}
}


template <class T> __global__ void WetsloperesetHaloRightGPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int ix = XParam.blkwidth;
	int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];
	int i, jj, ii, ir, it, itr;
	int read;

	int lev = XBlock.level[ib];

	T delta = calcres(XParam.dx, lev);


	i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);

	int ileft;
	
	ileft = memloc(XParam.halowidth, blkmemwidth, ix - 1, iy, ib);

	T zsi, zsleft, zsright;

	zsi = XEv.zs[i];
	zsleft = XEv.zs[ileft];
	
	T dzsdxi = XGrad.dzsdx[i];


	if (XBlock.RightBot[ib] == ib)//The lower half is a boundary 
	{
		if (iy < (XParam.blkwidth / 2))
		{

			read = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, iy, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			zsright = XEv.zs[read];;
		}

		if (XBlock.RightTop[ib] == ib) // boundary on the top half too
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, iy, ib);

				zsright = XEv.zs[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (iy >= (XParam.blkwidth / 2))
			{

				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, 3, jj, XBlock.RightTop[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, 2, jj, XBlock.RightTop[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, 3, jj + 1, XBlock.RightTop[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, 2, jj + 1, XBlock.RightTop[ib]);

				zsright = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.RightBot[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam.halowidth, blkmemwidth, 1, iy, XBlock.RightBot[ib]);
		zsright = XEv.zs[read];

	}
	else if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])
	{

		if (iy < (XParam.blkwidth / 2))
		{

			jj = iy * 2;
			int bb = XBlock.RightBot[ib];

			ii = memloc(XParam.halowidth, blkmemwidth, 3, jj, bb);
			ir = memloc(XParam.halowidth, blkmemwidth, 2, jj, bb);
			it = memloc(XParam.halowidth, blkmemwidth, 3, jj + 1, bb);
			itr = memloc(XParam.halowidth, blkmemwidth, 2, jj + 1, bb);

			zsright = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.RightTop[ib] == ib)
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, iy, ib);

				zsright = XEv.zs[read];
			}
		}
		else
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//
				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, 3, jj, XBlock.RightTop[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, 2, jj, XBlock.RightTop[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, 3, jj + 1, XBlock.RightTop[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, 2, jj + 1, XBlock.RightTop[ib]);

				zsright = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? ceil(iy * (T)0.5) : ceil(iy * (T)0.5) + XParam.blkwidth / 2;
		T jr = ceil(iy * (T)0.5) * 2 > iy ? T(0.25) : T(0.75);

		ii = memloc(XParam.halowidth, blkmemwidth, 0, jj, XBlock.RightBot[ib]);
		ir = memloc(XParam.halowidth, blkmemwidth, 1, jj, XBlock.RightBot[ib]);
		it = memloc(XParam.halowidth, blkmemwidth, 0, jj - 1, XBlock.RightBot[ib]);
		itr = memloc(XParam.halowidth, blkmemwidth, 1, jj - 1, XBlock.RightBot[ib]);

		zsright = BilinearInterpolation(XEv.zs[it], XEv.zs[ii], XEv.zs[itr], XEv.zs[ir], T(0.0), T(1.0), T(0.0), T(1.0), T(0.25), jr);
	}







	if (utils::sq(dzsdxi) > utils::sq(XGrad.dzbdx[i]))
	{
		T leftzs, rightzs;
		leftzs = zsi - XEv.h[i] - delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);
		rightzs = zsi - XEv.h[i] + delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);

		if (leftzs > zsleft || rightzs > zsright)
		{
			XGrad.dzsdx[i] = XGrad.dhdx[i] + XGrad.dzbdx[i];
		}

	}


}

template <class T> void WetsloperesetHaloRightCPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int ix = XParam.blkwidth;
	
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{


		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{


			
			unsigned int ib = XBlock.active[ibl];
			int i, jj, ii, ir, it, itr;
			int read;

			int lev = XBlock.level[ib];

			T delta = calcres(XParam.dx, lev);


			i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);

			int  ileft;

			ileft = memloc(XParam.halowidth, blkmemwidth, ix - 1, iy, ib);

			T zsi, zsleft, zsright;

			zsi = XEv.zs[i];
			zsleft = XEv.zs[ileft];

			T dzsdxi = XGrad.dzsdx[i];


			if (XBlock.RightBot[ib] == ib)//The lower half is a boundary 
			{
				if (iy < (XParam.blkwidth / 2))
				{

					read = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, iy, ib);// or memloc(XParam, -1, j, ib) but they should be the same

					zsright = XEv.zs[read];;
				}

				if (XBlock.RightTop[ib] == ib) // boundary on the top half too
				{
					if (iy >= (XParam.blkwidth / 2))
					{
						//

						read = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, iy, ib);

						zsright = XEv.zs[read];
					}
				}
				else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
				{

					if (iy >= (XParam.blkwidth / 2))
					{

						jj = (iy - XParam.blkwidth / 2) * 2;
						ii = memloc(XParam.halowidth, blkmemwidth, 3, jj, XBlock.RightTop[ib]);
						ir = memloc(XParam.halowidth, blkmemwidth, 2, jj, XBlock.RightTop[ib]);
						it = memloc(XParam.halowidth, blkmemwidth, 3, jj + 1, XBlock.RightTop[ib]);
						itr = memloc(XParam.halowidth, blkmemwidth, 2, jj + 1, XBlock.RightTop[ib]);

						zsright = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);

					}
				}
			}
			else if (XBlock.level[ib] == XBlock.level[XBlock.RightBot[ib]]) // LeftTop block does not exist
			{

				read = memloc(XParam.halowidth, blkmemwidth, 1, iy, XBlock.RightBot[ib]);
				zsright = XEv.zs[read];

			}
			else if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])
			{

				if (iy < (XParam.blkwidth / 2))
				{

					jj = iy * 2;
					int bb = XBlock.RightBot[ib];

					ii = memloc(XParam.halowidth, blkmemwidth, 3, jj, bb);
					ir = memloc(XParam.halowidth, blkmemwidth, 2, jj, bb);
					it = memloc(XParam.halowidth, blkmemwidth, 3, jj + 1, bb);
					itr = memloc(XParam.halowidth, blkmemwidth, 2, jj + 1, bb);

					zsright = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
				}
				//now find out aboy lefttop block
				if (XBlock.RightTop[ib] == ib)
				{
					if (iy >= (XParam.blkwidth / 2))
					{
						//

						read = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, iy, ib);

						zsright = XEv.zs[read];
					}
				}
				else
				{
					if (iy >= (XParam.blkwidth / 2))
					{
						//
						jj = (iy - XParam.blkwidth / 2) * 2;
						ii = memloc(XParam.halowidth, blkmemwidth, 3, jj, XBlock.RightTop[ib]);
						ir = memloc(XParam.halowidth, blkmemwidth, 2, jj, XBlock.RightTop[ib]);
						it = memloc(XParam.halowidth, blkmemwidth, 3, jj + 1, XBlock.RightTop[ib]);
						itr = memloc(XParam.halowidth, blkmemwidth, 2, jj + 1, XBlock.RightTop[ib]);

						zsright = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
					}
				}

			}
			else if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
			{
				jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? ftoi(ceil(iy * (T)0.5)) : ftoi(ceil(iy * (T)0.5) + XParam.blkwidth / 2);
				T jr = ceil(iy * (T)0.5) * 2 > iy ? T(0.25) : T(0.75);

				ii = memloc(XParam.halowidth, blkmemwidth, 0, jj, XBlock.RightBot[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, 1, jj, XBlock.RightBot[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, 0, jj - 1, XBlock.RightBot[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, 1, jj - 1, XBlock.RightBot[ib]);

				zsright = BilinearInterpolation(XEv.zs[it], XEv.zs[ii], XEv.zs[itr], XEv.zs[ir], T(0.0), T(1.0), T(0.0), T(1.0), T(0.25), jr);
			}







			if (utils::sq(dzsdxi) > utils::sq(XGrad.dzbdx[i]))
			{
				T leftzs, rightzs;
				leftzs = zsi - XEv.h[i] - delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);
				rightzs = zsi - XEv.h[i] + delta * T(0.5) * (dzsdxi - XGrad.dhdx[i]);

				if (leftzs > zsleft || rightzs > zsright)
				{
					XGrad.dzsdx[i] = XGrad.dhdx[i] + XGrad.dzbdx[i];
				}

			}
		}

	}
}


template <class T> __global__ void WetsloperesetHaloBotGPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int iy = -1;
	int ix = threadIdx.x;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i, jj, ii, ir, it, itr;

	int lev = XBlock.level[ib];

	T delta = calcres(XParam.dx, lev);


	i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);

	int itop,read;
	itop = memloc(XParam.halowidth, blkmemwidth, ix, iy + 1, ib);
	
	T zsi, zstop, zsbot;

	T dzsdyi = XGrad.dzsdy[i];

	zsi = XEv.zs[i];
	zstop = XEv.zs[itop];


	if (XBlock.BotLeft[ib] == ib)//The lower half is a boundary 
	{
		if (ix < (XParam.blkwidth / 2))
		{

			read = memloc(XParam.halowidth, blkmemwidth, ix, 0, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			zsbot = XEv.zs[read];

		}

		if (XBlock.BotRight[ib] == ib) // boundary on the top half too
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, ix, 0, ib);

				zsbot = XEv.zs[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (ix >= (XParam.blkwidth / 2))
			{

				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 4), XBlock.BotRight[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 4), XBlock.BotRight[ib]);

				zsbot = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.BotLeft[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam.halowidth, blkmemwidth, ix, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);
		zsbot = XEv.zs[read];



	}
	else if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])
	{

		if (ix < (XParam.blkwidth / 2))
		{

			jj = ix * 2;
			int bb = XBlock.BotLeft[ib];

			ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 3), bb);
			ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 4), bb);
			it = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 3), bb);
			itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 4), bb);

			zsbot = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.BotRight[ib] == ib)
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, ix, 0, ib);

				zsbot = XEv.zs[read];
			}
		}
		else
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//
				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 4), XBlock.BotRight[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 4), XBlock.BotRight[ib]);

				zsbot = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + XParam.blkwidth / 2;
		T jr = ceil(ix * (T)0.5) * 2 > ix ? T(0.25) : T(0.75);

		ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
		ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);
		it = memloc(XParam.halowidth, blkmemwidth, jj - 1, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
		itr = memloc(XParam.halowidth, blkmemwidth, jj - 1, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);

		zsbot = BilinearInterpolation(XEv.zs[itr], XEv.zs[it], XEv.zs[ir], XEv.zs[ii], T(0.0), T(1.0), T(0.0), T(1.0), jr, T(0.75));
	}


	if (utils::sq(dzsdyi) > utils::sq(XGrad.dzbdy[i]))
	{
		T botzs, topzs;
		botzs = zsi - XEv.h[i] - delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);
		topzs = zsi - XEv.h[i] + delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);

		if (botzs > zsbot || topzs > zstop)
		{
			XGrad.dzsdy[i] = XGrad.dhdy[i] + XGrad.dzbdy[i];
		}

	}


}

template <class T> void WetsloperesetHaloBotCPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int iy = -1;
	

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{

		
		unsigned int ib = XBlock.active[ibl];

		 

		int i, jj, ii, ir, it, itr;

		int lev = XBlock.level[ib];

		T delta = calcres(XParam.dx, lev);

		for (int ix = 0; ix < XParam.blkwidth; ix++)
		{


			i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);

			int itop, read;
			itop = memloc(XParam.halowidth, blkmemwidth, ix, iy + 1, ib);

			T zsi, zstop, zsbot;

			T dzsdyi = XGrad.dzsdy[i];

			zsi = XEv.zs[i];
			zstop = XEv.zs[itop];


			if (XBlock.BotLeft[ib] == ib)//The lower half is a boundary 
			{
				if (ix < (XParam.blkwidth / 2))
				{

					read = memloc(XParam.halowidth, blkmemwidth, ix, 0, ib);// or memloc(XParam, -1, j, ib) but they should be the same

					zsbot = XEv.zs[read];

				}

				if (XBlock.BotRight[ib] == ib) // boundary on the top half too
				{
					if (ix >= (XParam.blkwidth / 2))
					{
						//

						read = memloc(XParam.halowidth, blkmemwidth, ix, 0, ib);

						zsbot = XEv.zs[read];
					}
				}
				else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
				{

					if (ix >= (XParam.blkwidth / 2))
					{

						jj = (ix - XParam.blkwidth / 2) * 2;
						ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
						ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 4), XBlock.BotRight[ib]);
						it = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
						itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 4), XBlock.BotRight[ib]);

						zsbot = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);

					}
				}
			}
			else if (XBlock.level[ib] == XBlock.level[XBlock.BotLeft[ib]]) // LeftTop block does not exist
			{

				read = memloc(XParam.halowidth, blkmemwidth, ix, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);
				zsbot = XEv.zs[read];



			}
			else if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])
			{

				if (ix < (XParam.blkwidth / 2))
				{

					jj = ix * 2;
					int bb = XBlock.BotLeft[ib];

					ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 3), bb);
					ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 4), bb);
					it = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 3), bb);
					itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 4), bb);

					zsbot = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
				}
				//now find out aboy lefttop block
				if (XBlock.BotRight[ib] == ib)
				{
					if (ix >= (XParam.blkwidth / 2))
					{
						//

						read = memloc(XParam.halowidth, blkmemwidth, ix, 0, ib);

						zsbot = XEv.zs[read];
					}
				}
				else
				{
					if (ix >= (XParam.blkwidth / 2))
					{
						//
						jj = (ix - XParam.blkwidth / 2) * 2;
						ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
						ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 4), XBlock.BotRight[ib]);
						it = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
						itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 4), XBlock.BotRight[ib]);

						zsbot = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
					}
				}

			}
			else if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
			{
				jj = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? ftoi(ceil(ix * (T)0.5)) : ftoi(ceil(ix * (T)0.5) + XParam.blkwidth / 2);
				T jr = ceil(ix * (T)0.5) * 2 > ix ? T(0.25) : T(0.75);

				ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj - 1, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj - 1, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);

				zsbot = BilinearInterpolation(XEv.zs[itr], XEv.zs[it], XEv.zs[ir], XEv.zs[ii], T(0.0), T(1.0), T(0.0), T(1.0), jr, T(0.75));
			}


			if (utils::sq(dzsdyi) > utils::sq(XGrad.dzbdy[i]))
			{
				T botzs, topzs;
				botzs = zsi - XEv.h[i] - delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);
				topzs = zsi - XEv.h[i] + delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);

				if (botzs > zsbot || topzs > zstop)
				{
					XGrad.dzsdy[i] = XGrad.dhdy[i] + XGrad.dzbdy[i];
				}

			}
		}

	}

}

template <class T> __global__ void WetsloperesetHaloTopGPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int iy = XParam.blkwidth;
	int ix = threadIdx.x;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i, jj, ii, ir, it, itr;

	int lev = XBlock.level[ib];

	T delta = calcres(XParam.dx, lev);


	i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);

	int ibot, read;
	
	ibot = memloc(XParam.halowidth, blkmemwidth, ix, iy - 1, ib);

	T zsi, zstop, zsbot;

	zsi = XEv.zs[i];
	zsbot = XEv.zs[ibot];

	T dzsdyi = XGrad.dzsdy[i];


	if (XBlock.TopLeft[ib] == ib)//The lower half is a boundary 
	{
		if (ix < (XParam.blkwidth / 2))
		{

			read = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			zstop = XEv.zs[read];

		}

		if (XBlock.TopRight[ib] == ib) // boundary on the top half too
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);

				zstop = XEv.zs[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (ix >= (XParam.blkwidth / 2))
			{

				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, jj, 3, XBlock.TopRight[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, 2, XBlock.TopRight[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj + 1, 3, XBlock.TopRight[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, 2, XBlock.TopRight[ib]);

				zstop = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.TopLeft[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam.halowidth, blkmemwidth, ix, 1, XBlock.TopLeft[ib]);
		zstop = XEv.zs[read];



	}
	else if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])
	{

		if (ix < (XParam.blkwidth / 2))
		{

			jj = ix * 2;
			int bb = XBlock.TopLeft[ib];;

			ii = memloc(XParam.halowidth, blkmemwidth, jj, 3, bb);
			ir = memloc(XParam.halowidth, blkmemwidth, jj, 2, bb);
			it = memloc(XParam.halowidth, blkmemwidth, jj + 1, 3, bb);
			itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, 2, bb);

			zstop = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.TopRight[ib] == ib)
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);

				zstop = XEv.zs[read];
			}
		}
		else
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//
				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, jj, 3, XBlock.TopRight[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, 2, XBlock.TopRight[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj + 1, 3, XBlock.TopRight[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, 2, XBlock.TopRight[ib]);

				zstop = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + XParam.blkwidth / 2;
		T jr = ceil(ix * (T)0.5) * 2 > ix ? T(0.25) : T(0.75);

		ii = memloc(XParam.halowidth, blkmemwidth, jj, 0, XBlock.TopLeft[ib]);
		ir = memloc(XParam.halowidth, blkmemwidth, jj, 1, XBlock.TopLeft[ib]);
		it = memloc(XParam.halowidth, blkmemwidth, jj - 1, 0, XBlock.TopLeft[ib]);
		itr = memloc(XParam.halowidth, blkmemwidth, jj - 1, 1, XBlock.TopLeft[ib]);

		zstop = BilinearInterpolation(XEv.zs[it], XEv.zs[itr], XEv.zs[ii], XEv.zs[ir], T(0.0), T(1.0), T(0.0), T(1.0), jr, T(0.25));
	}


	if (utils::sq(dzsdyi) > utils::sq(XGrad.dzbdy[i]))
	{
		T botzs, topzs;
		botzs = zsi - XEv.h[i] - delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);
		topzs = zsi - XEv.h[i] + delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);

		if (botzs > zsbot || topzs > zstop)
		{
			XGrad.dzsdy[i] = XGrad.dhdy[i] + XGrad.dzbdy[i];
		}

	}


}



template <class T>  void WetsloperesetHaloTopCPU(Param XParam, BlockP<T>XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, T* zb)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int iy = XParam.blkwidth;
	

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		unsigned int ib = XBlock.active[ibl];

		int i, jj, ii, ir, it, itr;

		int lev = XBlock.level[ib];

		T delta = calcres(XParam.dx, lev);

		for (int ix = 0; ix < XParam.blkwidth; ix++)
		{

			i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);

			int ibot, read;

			ibot = memloc(XParam.halowidth, blkmemwidth, ix, iy - 1, ib);

			T zsi, zstop, zsbot;

			zsi = XEv.zs[i];
			zsbot = XEv.zs[ibot];

			T dzsdyi = XGrad.dzsdy[i];


			if (XBlock.TopLeft[ib] == ib)//The lower half is a boundary 
			{
				if (ix < (XParam.blkwidth / 2))
				{

					read = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);// or memloc(XParam, -1, j, ib) but they should be the same

					zstop = XEv.zs[read];

				}

				if (XBlock.TopRight[ib] == ib) // boundary on the top half too
				{
					if (ix >= (XParam.blkwidth / 2))
					{
						//

						read = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);

						zstop = XEv.zs[read];
					}
				}
				else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
				{

					if (ix >= (XParam.blkwidth / 2))
					{

						jj = (ix - XParam.blkwidth / 2) * 2;
						ii = memloc(XParam.halowidth, blkmemwidth, jj, 3, XBlock.TopRight[ib]);
						ir = memloc(XParam.halowidth, blkmemwidth, jj, 2, XBlock.TopRight[ib]);
						it = memloc(XParam.halowidth, blkmemwidth, jj + 1, 3, XBlock.TopRight[ib]);
						itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, 2, XBlock.TopRight[ib]);

						zstop = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);

					}
				}
			}
			else if (XBlock.level[ib] == XBlock.level[XBlock.TopLeft[ib]]) // LeftTop block does not exist
			{

				read = memloc(XParam.halowidth, blkmemwidth, ix, 1, XBlock.TopLeft[ib]);
				zstop = XEv.zs[read];



			}
			else if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])
			{

				if (ix < (XParam.blkwidth / 2))
				{

					jj = ix * 2;
					int bb = XBlock.TopLeft[ib];;

					ii = memloc(XParam.halowidth, blkmemwidth, jj, 3, bb);
					ir = memloc(XParam.halowidth, blkmemwidth, jj, 2, bb);
					it = memloc(XParam.halowidth, blkmemwidth, jj + 1, 3, bb);
					itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, 2, bb);

					zstop = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
				}
				//now find out aboy lefttop block
				if (XBlock.TopRight[ib] == ib)
				{
					if (ix >= (XParam.blkwidth / 2))
					{
						//

						read = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);

						zstop = XEv.zs[read];
					}
				}
				else
				{
					if (ix >= (XParam.blkwidth / 2))
					{
						//
						jj = (ix - XParam.blkwidth / 2) * 2;
						ii = memloc(XParam.halowidth, blkmemwidth, jj, 3, XBlock.TopRight[ib]);
						ir = memloc(XParam.halowidth, blkmemwidth, jj, 2, XBlock.TopRight[ib]);
						it = memloc(XParam.halowidth, blkmemwidth, jj + 1, 3, XBlock.TopRight[ib]);
						itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, 2, XBlock.TopRight[ib]);

						zstop = T(0.25) * (XEv.zs[ii] + XEv.zs[ir] + XEv.zs[it] + XEv.zs[itr]);
					}
				}

			}
			else if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
			{
				jj = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? ftoi(ceil(ix * (T)0.5)) : ftoi(ceil(ix * (T)0.5) + XParam.blkwidth / 2);
				T jr = ceil(ix * (T)0.5) * 2 > ix ? T(0.25) : T(0.75);

				ii = memloc(XParam.halowidth, blkmemwidth, jj, 0, XBlock.TopLeft[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, 1, XBlock.TopLeft[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj - 1, 0, XBlock.TopLeft[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj - 1, 1, XBlock.TopLeft[ib]);

				zstop = BilinearInterpolation(XEv.zs[it], XEv.zs[itr], XEv.zs[ii], XEv.zs[ir], T(0.0), T(1.0), T(0.0), T(1.0), jr, T(0.25));
			}


			if (utils::sq(dzsdyi) > utils::sq(XGrad.dzbdy[i]))
			{
				T botzs, topzs;
				botzs = zsi - XEv.h[i] - delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);
				topzs = zsi - XEv.h[i] + delta * T(0.5) * (dzsdyi - XGrad.dhdy[i]);

				if (botzs > zsbot || topzs > zstop)
				{
					XGrad.dzsdy[i] = XGrad.dhdy[i] + XGrad.dzbdy[i];
				}

			}
		}
	}


}


template <class T> void gradientHalo(Param XParam, BlockP<T>XBlock, T* a, T* dadx, T* dady)
{
	int ib;
	//int xplus;

	//T delta;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			gradientHaloLeft(XParam, XBlock, ib, iy, a, dadx, dady);
			gradientHaloRight(XParam, XBlock, ib, iy, a, dadx, dady);
		}
		for (int ix = 0; ix < XParam.blkwidth; ix++)
		{
			gradientHaloBot(XParam, XBlock, ib, ix, a, dadx, dady);
			gradientHaloTop(XParam, XBlock, ib, ix, a, dadx, dady);
		}
	}
}


template <class T> void gradientHaloGPU(Param XParam, BlockP<T>XBlock, T* a, T* dadx, T* dady)
{
	dim3 blockDimL(1, XParam.blkwidth, 1);
	dim3 blockDimB(XParam.blkwidth, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	
	gradientHaloLeftGPU << < gridDim, blockDimL, 0 >> > (XParam, XBlock, a, dadx, dady);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientHaloRightGPU << < gridDim, blockDimL, 0 >> > (XParam, XBlock, a, dadx, dady);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientHaloBotGPU << < gridDim, blockDimB, 0 >> > (XParam, XBlock, a, dadx, dady);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientHaloTopGPU << < gridDim, blockDimB, 0 >> > (XParam, XBlock, a, dadx, dady);
	CUDA_CHECK(hipDeviceSynchronize());
		
	
}


template <class T> void gradientHaloLeft(Param XParam, BlockP<T>XBlock, int ib, int iy, T* a, T* dadx, T* dady)
{
	int i, ix, jj, ii, ir, it, itr;
	int xplus, read;
	
	T delta, aright, aleft;

	ix = -1;

	i = memloc(XParam, ix, iy, ib);
	xplus = memloc(XParam, ix + 1, iy, ib);
	

	aright = a[xplus];
	


	delta = calcres(T(XParam.dx), XBlock.level[ib]);


	if (XBlock.LeftBot[ib] == ib)//The lower half is a boundary 
	{
		if ( iy < (XParam.blkwidth / 2))
		{

			read = memloc(XParam, 0, iy, ib);// or memloc(XParam, -1, j, ib) but they should be the same
			
			aleft = a[read];
		}

		if (XBlock.LeftTop[ib] == ib) // boundary on the top half too
		{
			if ( iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam, 0, iy, ib);
				
				aleft = a[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (iy >= (XParam.blkwidth / 2))
			{
				
				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, (XParam.blkwidth - 3), jj, XBlock.LeftTop[ib]);
				ir = memloc(XParam, (XParam.blkwidth - 4), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam, (XParam.blkwidth - 3), jj + 1, XBlock.LeftTop[ib]);
				itr = memloc(XParam, (XParam.blkwidth - 4), jj + 1, XBlock.LeftTop[ib]);

				aleft = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.LeftBot[ib]]) // LeftTop block does not exist
	{
		
			read = memloc(XParam, (XParam.blkwidth - 2), iy, XBlock.LeftBot[ib]);
			aleft = a[read];
		
	}
	else if (XBlock.level[XBlock.LeftBot[ib]] > XBlock.level[ib])
	{

		if (iy < (XParam.blkwidth / 2))
		{	

			jj = iy * 2;
			int bb = XBlock.LeftBot[ib];

			ii = memloc(XParam, (XParam.blkwidth - 3), jj, bb);
			ir = memloc(XParam, (XParam.blkwidth - 4), jj, bb);
			it = memloc(XParam, (XParam.blkwidth - 3), jj + 1, bb);
			itr = memloc(XParam, (XParam.blkwidth - 4), jj + 1, bb);

			aleft = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.LeftTop[ib] == ib)
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam, 0, iy, ib);
				
				aleft = a[read];
			}
		}
		else
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//
				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, (XParam.blkwidth - 3), jj, XBlock.LeftTop[ib]);
				ir = memloc(XParam, (XParam.blkwidth - 4), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam, (XParam.blkwidth - 3), jj + 1, XBlock.LeftTop[ib]);
				itr = memloc(XParam, (XParam.blkwidth - 4), jj + 1, XBlock.LeftTop[ib]);

				aleft = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib ? ftoi(ceil(iy * (T)0.5)) : ftoi(ceil(iy * (T)0.5) + XParam.blkwidth / 2);
		//T jr = ceil(iy * (T)0.5) * 2 > iy ? T(0.75) : T(0.25);// This is the wrong way around
		T jr = ceil(iy * (T)0.5) * 2 > iy ? T(0.25) : T(0.75); // This is right (e.g. at iy==0 use 0.75 at iy==1 use 0.25)

		ii = memloc(XParam, (XParam.blkwidth - 1), jj, XBlock.LeftBot[ib]);
		ir = memloc(XParam, (XParam.blkwidth - 2), jj, XBlock.LeftBot[ib]);
		it = memloc(XParam, (XParam.blkwidth - 1), jj - 1, XBlock.LeftBot[ib]);
		itr = memloc(XParam, (XParam.blkwidth - 2), jj - 1, XBlock.LeftBot[ib]);

		aleft = BilinearInterpolation(a[itr], a[ir], a[it], a[ii], T(0.0), T(1.0), T(0.0), T(1.0), T(0.75), jr);
	}
	




	dadx[i] = minmod2(T(XParam.theta), aleft, a[i], aright) / delta;
	//dady[i] = minmod2(T(XParam.theta), abot, a[i], atop) / delta;

}


template <class T> void gradientHaloRight(Param XParam, BlockP<T>XBlock, int ib, int iy, T* a, T* dadx, T* dady)
{
	int i, ix, jj, ii, ir, it, itr;
	int xminus, read;

	T delta, aright, aleft;

	ix = 16;

	i = memloc(XParam, ix, iy, ib);
	xminus = memloc(XParam, ix - 1, iy, ib);


	aleft = a[xminus];



	delta = calcres(T(XParam.dx), XBlock.level[ib]);


	if (XBlock.RightBot[ib] == ib)//The lower half is a boundary 
	{
		if (iy < (XParam.blkwidth / 2))
		{

			read = memloc(XParam, XParam.blkwidth -1, iy, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			aright = a[read];
		}

		if (XBlock.RightTop[ib] == ib) // boundary on the top half too
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam, XParam.blkwidth - 1, iy, ib);

				aright = a[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of righttopib is levelib+1
		{

			if (iy >= (XParam.blkwidth / 2))
			{

				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, 3, jj, XBlock.RightTop[ib]);
				ir = memloc(XParam, 2, jj, XBlock.RightTop[ib]);
				it = memloc(XParam, 3, jj + 1, XBlock.RightTop[ib]);
				itr = memloc(XParam, 2, jj + 1, XBlock.RightTop[ib]);

				aright = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.RightBot[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam, 1, iy, XBlock.RightBot[ib]);
		aright = a[read];

	}
	else if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])
	{

		if (iy < (XParam.blkwidth / 2))
		{

			jj = iy * 2;
			int bb = XBlock.RightBot[ib];

			ii = memloc(XParam, 3, jj, bb);
			ir = memloc(XParam, 2, jj, bb);
			it = memloc(XParam, 3, jj + 1, bb);
			itr = memloc(XParam, 2, jj + 1, bb);

			aright = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.RightTop[ib] == ib)
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam, XParam.blkwidth - 1, iy, ib);

				aright = a[read];
			}
		}
		else
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//
				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, 3, jj, XBlock.RightTop[ib]);
				ir = memloc(XParam, 2, jj, XBlock.RightTop[ib]);
				it = memloc(XParam, 3, jj + 1, XBlock.RightTop[ib]);
				itr = memloc(XParam, 2, jj + 1, XBlock.RightTop[ib]);

				aright = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? ftoi(ceil(iy * (T)0.5)) : ftoi(ceil(iy * (T)0.5) + XParam.blkwidth / 2);
		T jr = ceil(iy * (T)0.5) * 2 > iy ? T(0.25) : T(0.75);

		ii = memloc(XParam, 0, jj, XBlock.RightBot[ib]);
		ir = memloc(XParam, 1, jj, XBlock.RightBot[ib]);
		it = memloc(XParam, 0, jj - 1, XBlock.RightBot[ib]);
		itr = memloc(XParam, 1, jj - 1, XBlock.RightBot[ib]);

		aright = BilinearInterpolation(a[it], a[ii], a[itr], a[ir], T(0.0), T(1.0), T(0.0), T(1.0), T(0.25), jr);
	}





	dadx[i] = minmod2(T(XParam.theta), aleft, a[i], aright) / delta;
	//dady[i] = minmod2(T(XParam.theta), abot, a[i], atop) / delta;

}


template <class T> void gradientHaloBot(Param XParam, BlockP<T>XBlock, int ib, int ix, T* a, T* dadx, T* dady)
{
	int i, iy, jj, ii, ir, it, itr;
	int yplus, read;

	T delta, atop, abot;

	iy = -1;

	i = memloc(XParam, ix, iy, ib);
	yplus = memloc(XParam, ix , iy + 1, ib);
	



	atop = a[yplus];



	delta = calcres(T(XParam.dx), XBlock.level[ib]);


	if (XBlock.BotLeft[ib] == ib)//The lower half is a boundary 
	{
		if (ix < (XParam.blkwidth / 2))
		{

			read = memloc(XParam, ix, 0, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			abot = a[read];
			
		}

		if (XBlock.BotRight[ib] == ib) // boundary on the top half too
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam, ix, 0, ib);

				abot = a[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (ix >= (XParam.blkwidth / 2))
			{

				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, jj, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				ir = memloc(XParam, jj, (XParam.blkwidth - 4), XBlock.BotRight[ib]);
				it = memloc(XParam, jj + 1, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				itr = memloc(XParam, jj + 1, (XParam.blkwidth - 4), XBlock.BotRight[ib]);

				abot = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.BotLeft[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam, ix, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);
		abot = a[read];

	}
	else if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])
	{

		if (ix < (XParam.blkwidth / 2))
		{

			jj = ix * 2;
			int bb = XBlock.BotLeft[ib];

			ii = memloc(XParam, jj, (XParam.blkwidth - 3), bb);
			ir = memloc(XParam, jj, (XParam.blkwidth - 4), bb);
			it = memloc(XParam, jj + 1, (XParam.blkwidth - 3), bb);
			itr = memloc(XParam, jj + 1, (XParam.blkwidth - 4), bb);

			abot = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.BotRight[ib] == ib)
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam, ix, 0, ib);

				abot = a[read];
			}
		}
		else
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//
				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, jj, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				ir = memloc(XParam, jj, (XParam.blkwidth - 4), XBlock.BotRight[ib]);
				it = memloc(XParam, jj + 1, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				itr = memloc(XParam, jj + 1, (XParam.blkwidth - 4), XBlock.BotRight[ib]);

				abot = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? ftoi(ceil(ix * (T)0.5)) : ftoi(ceil(ix * (T)0.5) + XParam.blkwidth / 2);
		T jr = ceil(ix * (T)0.5) * 2 > ix ? T(0.25) : T(0.75);

		ii = memloc(XParam, jj, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
		ir = memloc(XParam, jj, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);
		it = memloc(XParam, jj - 1, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
		itr = memloc(XParam, jj - 1, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);

		abot = BilinearInterpolation(a[itr], a[it], a[ir], a[ii], T(0.0), T(1.0), T(0.0), T(1.0), jr, T(0.75));
	}





	//dadx[i] = minmod2(T(XParam.theta), aleft, a[i], aright) / delta;
	dady[i] = minmod2(T(XParam.theta), abot, a[i], atop) / delta;

}

template <class T> void gradientHaloTop(Param XParam, BlockP<T>XBlock, int ib, int ix, T* a, T* dadx, T* dady)
{
	int i, iy, jj, ii, ir, it, itr;
	int yminus, read;

	T delta, atop, abot;

	iy = XParam.blkwidth;

	i = memloc(XParam, ix, iy, ib);
	yminus = memloc(XParam, ix, XParam.blkwidth-1, ib);




	abot = a[yminus];



	delta = calcres(T(XParam.dx), XBlock.level[ib]);


	if (XBlock.TopLeft[ib] == ib)//The lower half is a boundary 
	{
		if (ix < (XParam.blkwidth / 2))
		{

			read = memloc(XParam, ix, XParam.blkwidth - 1, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			atop = a[read];

		}

		if (XBlock.TopRight[ib] == ib) // boundary on the top half too
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam, ix, XParam.blkwidth - 1, ib);

				atop = a[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (ix >= (XParam.blkwidth / 2))
			{

				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, jj, 3, XBlock.TopRight[ib]);
				ir = memloc(XParam, jj, 2, XBlock.TopRight[ib]);
				it = memloc(XParam, jj + 1, 3, XBlock.TopRight[ib]);
				itr = memloc(XParam, jj + 1, 2, XBlock.TopRight[ib]);

				atop = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.TopLeft[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam, ix, 1, XBlock.TopLeft[ib]);
		atop = a[read];

	}
	else if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])
	{

		if (ix < (XParam.blkwidth / 2))
		{

			jj = ix * 2;
			int bb = XBlock.TopLeft[ib];

			ii = memloc(XParam, jj, 3, bb);
			ir = memloc(XParam, jj, 2, bb);
			it = memloc(XParam, jj + 1, 3, bb);
			itr = memloc(XParam, jj + 1, 2, bb);

			atop = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.TopRight[ib] == ib)
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam, ix, XParam.blkwidth - 1, ib);

				atop = a[read];
			}
		}
		else
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//
				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, jj, 3, XBlock.TopRight[ib]);
				ir = memloc(XParam, jj, 2, XBlock.TopRight[ib]);
				it = memloc(XParam, jj + 1, 3, XBlock.TopRight[ib]);
				itr = memloc(XParam, jj + 1, 2, XBlock.TopRight[ib]);

				atop = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? ftoi(ceil(ix * (T)0.5)) : ftoi(ceil(ix * (T)0.5) + XParam.blkwidth / 2);
		T jr = ceil(ix * (T)0.5) * 2 > ix ? T(0.25) : T(0.75);

		ii = memloc(XParam, jj, 0, XBlock.TopLeft[ib]);
		ir = memloc(XParam, jj, 1, XBlock.TopLeft[ib]);
		it = memloc(XParam, jj - 1, 0, XBlock.TopLeft[ib]);
		itr = memloc(XParam, jj - 1, 1, XBlock.TopLeft[ib]);

		atop = BilinearInterpolation(a[it], a[itr], a[ii], a[ir], T(0.0), T(1.0), T(0.0), T(1.0), jr, T(0.25));
	}





	//dadx[i] = minmod2(T(XParam.theta), aleft, a[i], aright) / delta;
	dady[i] = minmod2(T(XParam.theta), abot, a[i], atop) / delta;

}



template <class T> __global__ void gradientHaloLeftGPU(Param XParam, BlockP<T>XBlock, T* a, T* dadx, T* dady)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int ix = -1;
	int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];
	int i, jj, ii, ir, it, itr;
	int xplus, read;

	T delta, aright, aleft;

	

	i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);
	xplus = memloc(XParam.halowidth, blkmemwidth, ix + 1, iy, ib);


	aright = a[xplus];



	delta = calcres(T(XParam.dx), XBlock.level[ib]);


	if (XBlock.LeftBot[ib] == ib)//The lower half is a boundary 
	{
		if (iy < (XParam.blkwidth / 2))
		{

			read = memloc(XParam.halowidth, blkmemwidth, 0, iy, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			aleft = a[read];
		}

		if (XBlock.LeftTop[ib] == ib) // boundary on the top half too
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, 0, iy, ib);

				aleft = a[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (iy >= (XParam.blkwidth / 2))
			{

				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj, XBlock.LeftTop[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj + 1, XBlock.LeftTop[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj + 1, XBlock.LeftTop[ib]);

				aleft = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.LeftBot[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 2), iy, XBlock.LeftBot[ib]);
		aleft = a[read];

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] > XBlock.level[ib])
	{

		if (iy < (XParam.blkwidth / 2))
		{

			jj = iy * 2;
			int bb = XBlock.LeftBot[ib];

			ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj, bb);
			ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj, bb);
			it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj + 1, bb);
			itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj + 1, bb);

			aleft = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.LeftTop[ib] == ib)
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, 0, iy, ib);

				aleft = a[read];
			}
		}
		else
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//
				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj, XBlock.LeftTop[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 3), jj + 1, XBlock.LeftTop[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 4), jj + 1, XBlock.LeftTop[ib]);

				aleft = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib ? ceil(iy * (T)0.5) : ceil(iy * (T)0.5) + XParam.blkwidth / 2;
		T jr = ceil(iy * (T)0.5) * 2 > iy ? T(0.25) : T(0.75);

		ii = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 1), jj, XBlock.LeftBot[ib]);
		ir = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 2), jj, XBlock.LeftBot[ib]);
		it = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 1), jj - 1, XBlock.LeftBot[ib]);
		itr = memloc(XParam.halowidth, blkmemwidth, (XParam.blkwidth - 2), jj - 1, XBlock.LeftBot[ib]);

		aleft = BilinearInterpolation(a[itr], a[ir], a[it], a[ii], T(0.0), T(1.0), T(0.0), T(1.0), T(0.75), jr);
	}





	dadx[i] = minmod2(T(XParam.theta), aleft, a[i], aright) / delta;
	//dady[i] = minmod2(T(XParam.theta), abot, a[i], atop) / delta;

}

template <class T> __global__ void gradientHaloRightGPU(Param XParam, BlockP<T>XBlock, T* a, T* dadx, T* dady)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int ix = XParam.blkwidth;
	int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];
	int i, jj, ii, ir, it, itr;
	int xminus, read;

	T delta, aright, aleft;



	i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);
	xminus = memloc(XParam.halowidth, blkmemwidth, ix - 1, iy, ib);


	aleft = a[xminus];



	delta = calcres(T(XParam.dx), XBlock.level[ib]);


	if (XBlock.RightBot[ib] == ib)//The lower half is a boundary 
	{
		if (iy < (XParam.blkwidth / 2))
		{

			read = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, iy, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			aright = a[read];;
		}

		if (XBlock.RightTop[ib] == ib) // boundary on the top half too
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, iy, ib);

				aright = a[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (iy >= (XParam.blkwidth / 2))
			{

				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, 3, jj, XBlock.RightTop[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, 2, jj, XBlock.RightTop[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, 3, jj + 1, XBlock.RightTop[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, 2, jj + 1, XBlock.RightTop[ib]);

				aright = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.RightBot[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam.halowidth, blkmemwidth, 1, iy, XBlock.RightBot[ib]);
		aright = a[read];

	}
	else if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])
	{

		if (iy < (XParam.blkwidth / 2))
		{

			jj = iy * 2;
			int bb = XBlock.RightBot[ib];

			ii = memloc(XParam.halowidth, blkmemwidth, 3, jj, bb);
			ir = memloc(XParam.halowidth, blkmemwidth, 2, jj, bb);
			it = memloc(XParam.halowidth, blkmemwidth, 3, jj + 1, bb);
			itr = memloc(XParam.halowidth, blkmemwidth, 2, jj + 1, bb);

			aright = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.RightTop[ib] == ib)
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, iy, ib);

				aright = a[read];
			}
		}
		else
		{
			if (iy >= (XParam.blkwidth / 2))
			{
				//
				jj = (iy - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, 3, jj, XBlock.RightTop[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, 2, jj, XBlock.RightTop[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, 3, jj + 1, XBlock.RightTop[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, 2, jj + 1, XBlock.RightTop[ib]);

				aright = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? ceil(iy * (T)0.5) : ceil(iy * (T)0.5) + XParam.blkwidth / 2;
		T jr = ceil(iy * (T)0.5) * 2 > iy ? T(0.25) : T(0.75);

		ii = memloc(XParam.halowidth, blkmemwidth, 0, jj, XBlock.RightBot[ib]);
		ir = memloc(XParam.halowidth, blkmemwidth, 1, jj, XBlock.RightBot[ib]);
		it = memloc(XParam.halowidth, blkmemwidth, 0, jj - 1, XBlock.RightBot[ib]);
		itr = memloc(XParam.halowidth, blkmemwidth, 1, jj - 1, XBlock.RightBot[ib]);

		aright = BilinearInterpolation(a[it], a[ii], a[itr], a[ir], T(0.0), T(1.0), T(0.0), T(1.0), T(0.25), jr);
	}





	dadx[i] = minmod2(T(XParam.theta), aleft, a[i], aright) / delta;
	//dady[i] = minmod2(T(XParam.theta), abot, a[i], atop) / delta;

}


template <class T> __global__ void gradientHaloBotGPU(Param XParam, BlockP<T>XBlock, T* a, T* dadx, T* dady)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int iy = -1;
	int ix = threadIdx.x;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];


	int i, jj, ii, ir, it, itr;
	int yplus, read;

	T delta, atop, abot;

	
	i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);
	yplus = memloc(XParam.halowidth, blkmemwidth, ix, iy + 1, ib);




	atop = a[yplus];



	delta = calcres(T(XParam.dx), XBlock.level[ib]);


	if (XBlock.BotLeft[ib] == ib)//The lower half is a boundary 
	{
		if (ix < (XParam.blkwidth / 2))
		{

			read = memloc(XParam.halowidth, blkmemwidth, ix, 0, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			abot = a[read];

		}

		if (XBlock.BotRight[ib] == ib) // boundary on the top half too
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, ix, 0, ib);

				abot = a[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (ix >= (XParam.blkwidth / 2))
			{

				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 4), XBlock.BotRight[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 4), XBlock.BotRight[ib]);

				abot = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.BotLeft[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam.halowidth, blkmemwidth, ix, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);
		abot = a[read];

		

	}
	else if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])
	{

		if (ix < (XParam.blkwidth / 2))
		{

			jj = ix * 2;
			int bb = XBlock.BotLeft[ib];

			ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 3), bb);
			ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 4), bb);
			it = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 3), bb);
			itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 4), bb);

			abot = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.BotRight[ib] == ib)
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, ix, 0, ib);

				abot = a[read];
			}
		}
		else
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//
				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 4), XBlock.BotRight[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 3), XBlock.BotRight[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, (XParam.blkwidth - 4), XBlock.BotRight[ib]);

				abot = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + XParam.blkwidth / 2;
		T jr = ceil(ix * (T)0.5) * 2 > ix ? T(0.25) : T(0.75);

		ii = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
		ir = memloc(XParam.halowidth, blkmemwidth, jj, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);
		it = memloc(XParam.halowidth, blkmemwidth, jj - 1, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
		itr = memloc(XParam.halowidth, blkmemwidth, jj - 1, (XParam.blkwidth - 2), XBlock.BotLeft[ib]);

		abot = BilinearInterpolation(a[itr], a[it], a[ir], a[ii], T(0.0), T(1.0), T(0.0), T(1.0), jr, T(0.75));
	}





	//dadx[i] = minmod2(T(XParam.theta), aleft, a[i], aright) / delta;
	dady[i] = minmod2(T(XParam.theta), abot, a[i], atop) / delta;

}


template <class T> __global__ void gradientHaloTopGPU(Param XParam, BlockP<T>XBlock, T* a, T* dadx, T* dady)
{
	unsigned int blkmemwidth = XParam.blkwidth + XParam.halowidth * 2;
	//unsigned int blksize = XParam.blkmemwidth * XParam.blkmemwidth;
	int iy = XParam.blkwidth;
	int ix = threadIdx.x;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];


	int i, jj, ii, ir, it, itr;
	int yminus, read;

	T delta, atop, abot;


	i = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);
	yminus = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);




	abot = a[yminus];



	delta = calcres(T(XParam.dx), XBlock.level[ib]);


	if (XBlock.TopLeft[ib] == ib)//The lower half is a boundary 
	{
		if (ix < (XParam.blkwidth / 2))
		{

			read = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);// or memloc(XParam, -1, j, ib) but they should be the same

			atop = a[read];

		}

		if (XBlock.TopRight[ib] == ib) // boundary on the top half too
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);

				atop = a[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			if (ix >= (XParam.blkwidth / 2))
			{

				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, jj, 3, XBlock.TopRight[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, 2, XBlock.TopRight[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj + 1, 3, XBlock.TopRight[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, 2, XBlock.TopRight[ib]);

				atop = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.TopLeft[ib]]) // LeftTop block does not exist
	{

		read = memloc(XParam.halowidth, blkmemwidth, ix, 1, XBlock.TopLeft[ib]);
		atop = a[read];



	}
	else if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])
	{

		if (ix < (XParam.blkwidth / 2))
		{

			jj = ix * 2;
			int bb = XBlock.TopLeft[ib];;

			ii = memloc(XParam.halowidth, blkmemwidth, jj, 3, bb);
			ir = memloc(XParam.halowidth, blkmemwidth, jj, 2, bb);
			it = memloc(XParam.halowidth, blkmemwidth, jj + 1, 3, bb);
			itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, 2, bb);

			atop = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.TopRight[ib] == ib)
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//

				read = memloc(XParam.halowidth, blkmemwidth, ix, XParam.blkwidth - 1, ib);

				atop = a[read];
			}
		}
		else
		{
			if (ix >= (XParam.blkwidth / 2))
			{
				//
				jj = (ix - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam.halowidth, blkmemwidth, jj, 3, XBlock.TopRight[ib]);
				ir = memloc(XParam.halowidth, blkmemwidth, jj, 2, XBlock.TopRight[ib]);
				it = memloc(XParam.halowidth, blkmemwidth, jj + 1, 3, XBlock.TopRight[ib]);
				itr = memloc(XParam.halowidth, blkmemwidth, jj + 1, 2, XBlock.TopRight[ib]);

				atop = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		jj = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + XParam.blkwidth / 2;
		T jr = ceil(ix * (T)0.5) * 2 > ix ? T(0.25) : T(0.75);

		ii = memloc(XParam.halowidth, blkmemwidth, jj, 0, XBlock.TopLeft[ib]);
		ir = memloc(XParam.halowidth, blkmemwidth, jj, 1, XBlock.TopLeft[ib]);
		it = memloc(XParam.halowidth, blkmemwidth, jj - 1, 0, XBlock.TopLeft[ib]);
		itr = memloc(XParam.halowidth, blkmemwidth, jj - 1, 1, XBlock.TopLeft[ib]);

		atop = BilinearInterpolation(a[it], a[itr], a[ii], a[ir], T(0.0), T(1.0), T(0.0), T(1.0), jr, T(0.25));
	}





	//dadx[i] = minmod2(T(XParam.theta), aleft, a[i], aright) / delta;
	dady[i] = minmod2(T(XParam.theta), abot, a[i], atop) / delta;

}
