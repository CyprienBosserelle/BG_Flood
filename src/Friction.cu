#include "hip/hip_runtime.h"
#include "Friction.h"



template <class T> __global__ void bottomfrictionGPU(Param XParam, BlockP<T> XBlock, T dt, T* cf,EvolvingP<T> XEvolv)
{
	// Shear stress equation:
	// Taub=cf*rho*U*sqrt(U^2+V^2)
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	T eps = T(XParam.eps);
	T g = T(XParam.g);

	int frictionmodel = XParam.frictionmodel;

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);


	T normu, hi, ui, vi;
	
	

	hi = XEvolv.h[i];
	ui = XEvolv.u[i];
	vi = XEvolv.v[i];
	if (hi > eps)
	{
		normu = sqrt(ui * ui + vi * vi);
			
		T cfi;
		if (frictionmodel == 0)
		{
			cfi = cf[i];
		}
		else if (frictionmodel == 1)//Smart friction formulation
		{
			cfi = smartfriction(hi, cf[i]);

		}
		else if (frictionmodel == -1)// Manning friction formulation
		{
			cfi = manningfriction(g, hi, cf[i]);

		}

		T tb = cfi * normu / hi * dt;
		XEvolv.u[i] = ui / (T(1.0) + tb);
		XEvolv.v[i] = vi / (T(1.0) + tb);
	}

	

}
template __global__ void bottomfrictionGPU<float>(Param XParam, BlockP<float> XBlock,float dt, float* cf, EvolvingP<float> XEvolv);
template __global__ void bottomfrictionGPU<double>(Param XParam, BlockP<double> XBlock,double dt, double* cf, EvolvingP<double> XEvolv);



template <class T> __host__ void bottomfrictionCPU(Param XParam, BlockP<T> XBlock,T dt, T* cf, EvolvingP<T> XEvolv)
{
	T eps = T(XParam.eps);
	T g = T(XParam.g);

	

	T hi, ui, vi,normu;

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{

				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

				hi = XEvolv.h[i];
				ui = XEvolv.u[i];
				vi = XEvolv.v[i];
				if (hi > eps)
				{
					normu = sqrt(ui * ui + vi * vi);

					T cfi;
					if (XParam.frictionmodel == 0)
					{
						cfi = cf[i];
					}
					else if (XParam.frictionmodel == 1)//Smart friction formulation
					{
						
						cfi = smartfriction(hi, cf[i]);

					}
					else if (XParam.frictionmodel == -1)// Manning friction formulation
					{
						T n = cf[i];
						cfi = manningfriction(g, hi, n);


					}

					T tb = cfi * normu / hi * dt;
					XEvolv.u[i] = ui / (T(1.0) + tb);
					XEvolv.v[i] = vi / (T(1.0) + tb);
				}
			}
		}
	}


}
template __host__ void bottomfrictionCPU<float>(Param XParam, BlockP<float> XBlock,float dt, float* cf, EvolvingP<float> XEvolv);
template __host__ void bottomfrictionCPU<double>(Param XParam, BlockP<double> XBlock,double dt, double* cf, EvolvingP<double> XEvolv);

/*!\fn void XiafrictionCPU(Param XParam, BlockP<T> XBlock, T dt, T* cf, EvolvingP<T> XEvolv)
* apply bottom friction following the procedure from Xia and Lang 2018
* https://doi.org/10.1016/j.advwatres.2018.05.004
* 
*
*/
template <class T> __host__ void XiafrictionCPU(Param XParam, BlockP<T> XBlock, T dt, T* cf, EvolvingP<T> XEvolv, EvolvingP<T> XEvolv_o)
{
	T eps = T(XParam.eps);
	T g = T(XParam.g);



	T hi, ho, ui, vi, normu;

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{

				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
				ho = XEvolv.h[i];
				hi = XEvolv_o.h[i];
				ui = XEvolv_o.u[i];
				vi = XEvolv_o.v[i];
				if (hi > eps)
				{
					normu = sqrt(ui * ui + vi * vi);

					T cfi = cf[i]; //if (XParam.frictionmodel == 0)
					if (XParam.frictionmodel == 1)//Smart friction formulation
					{

						cfi = smartfriction(hi, cf[i]);

					}
					else if (XParam.frictionmodel == -1)// Manning friction formulation
					{
						T n = cf[i];
						cfi = manningfriction(g, hi, n);


					}

					T tb = cfi * normu * hi/(ho*ho) * dt;
					if (tb >= T(1e-10))
					{
						XEvolv_o.u[i] = (ui - ui * sqrt(T(1.0) + T(4.0) * tb)) / (T(-2.0) * tb);
						XEvolv_o.v[i] = (vi - vi * sqrt(T(1.0) + T(4.0) * tb)) / (T(-2.0) * tb);
					}
					
				}
			}
		}
	}


}

template __host__ void XiafrictionCPU<float>(Param XParam, BlockP<float> XBlock,float dt, float* cf, EvolvingP<float> XEvolv, EvolvingP<float> XEvolv_o);
template __host__ void XiafrictionCPU<double>(Param XParam, BlockP<double> XBlock,double dt, double* cf, EvolvingP<double> XEvolv, EvolvingP<double> XEvolv_o);

template <class T> __global__ void XiafrictionGPU(Param XParam, BlockP<T> XBlock, T dt, T* cf, EvolvingP<T> XEvolv, EvolvingP<T> XEvolv_o)
{
	// Shear stress equation:
	// Taub=cf*rho*U*sqrt(U^2+V^2)
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	T eps = T(XParam.eps);
	T g = T(XParam.g);

	int frictionmodel = XParam.frictionmodel;

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);


	T normu,ho, hi, ui, vi;



	
	ho = XEvolv.h[i];
	hi = XEvolv_o.h[i];
	ui = XEvolv_o.u[i];
	vi = XEvolv_o.v[i];
	if (hi > eps) //SHould this be both ho and hi >eps ?
	{
		normu = sqrt(ui * ui + vi * vi);

		T cfi = cf[i]; //if (XParam.frictionmodel == 0)
		if (XParam.frictionmodel == 1)//Smart friction formulation
		{

			cfi = smartfriction(hi, cf[i]);

		}
		else if (XParam.frictionmodel == -1)// Manning friction formulation
		{
			T n = cf[i];
			cfi = manningfriction(g, hi, n);


		}

		T tb = cfi * normu * hi / (ho * ho) * dt;
		if (tb >= T(1e-10))
		{
			XEvolv_o.u[i] = (ui - ui * sqrt(T(1.0) + T(4.0) * tb)) / (T(-2.0) * tb);
			XEvolv_o.v[i] = (vi - vi * sqrt(T(1.0) + T(4.0) * tb)) / (T(-2.0) * tb);
		}

	}



}
template __global__ void XiafrictionGPU<float>(Param XParam, BlockP<float> XBlock, float dt, float* cf, EvolvingP<float> XEvolv, EvolvingP<float> XEvolv_o);
template __global__ void XiafrictionGPU<double>(Param XParam, BlockP<double> XBlock, double dt, double* cf, EvolvingP<double> XEvolv, EvolvingP<double> XEvolv_o);


template <class T> __host__ __device__ T smartfriction(T hi,T zo)
{
	T cfi;
	T ee = T(2.71828182845905);

	T Hbar = hi / zo;
	if (Hbar <= ee)
	{
		cfi = T(1.0) / (T(0.46) * Hbar);
	}
	else
	{
		cfi = T(1.0) / (T(2.5) * (log(Hbar) - T(1.0) + T(1.359) / Hbar));
	}
	cfi = cfi * cfi; //

	return cfi;
}

template <class T> __host__ __device__ T manningfriction(T g, T hi, T n)
{
	T cfi= g * n * n / cbrt(hi);
	return cfi;
}
