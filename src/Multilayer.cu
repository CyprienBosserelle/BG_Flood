#include "hip/hip_runtime.h"
#include "Multilayer.h"

//template <class T> void calcAbaro()
//{
//
//	T gmetric = (2. * fm.x[i] / (cm[i] + cm[i - 1]))
//
//	a_baro[i] (G*gmetric*(eta[i-1] - eta[i])/Delta)
//}

template <class T> __global__ void CalcfaceVal(T pdt,Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux, T* dtmax,T* zb)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	

	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);

	T CFL_H = T(0.5);

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);

	T zsi = XEv.zs[i];

	T zsn = XEv.zs[ileft];

	T zbi = zb[i];
	T zbn = zb[ileft];


	T fmu = T(1.0);
	T cm = T(1.0);//T cm = XParam.spherical ? calcCM(T(XParam.Radius), delta, ybo, iy) : T(1.0);
	T gmetric = T(1.0);// (2. * fm.x[i] / (cm[i] + cm[i - 1]));

	T ax = (G * gmetric * (zsn - zsi) / delta);

	T H = 0.;
	T um = 0.;
	T Hr = 0.;
	T Hl = 0.;

	
	//foreach_layer() {
	{
		T hi = XEv.h[i];
		T hn = XEv.h[ileft];
		Hr += hi;
		Hl += hn;
		T hl = hn > dry ? hn : 0.;
		T hr = hi > dry ? hi : 0.;

		
		
		//XFlux.hu[i] = hl > 0. || hr > 0. ? (hl * XEv.u[ileft] + hr * XEvu[i]) / (hl + hr) : 0.;
		T hui = hl > 0. || hr > 0. ? (hl * XEv.u[ileft] + hr * XEvu[i]) / (hl + hr) : 0.;

		T hff;

		if (Hl <= dry)
			hff = fmax(fmin(zbi + Hr - zbi, hi), 0.);
		else if (Hr <= dry)
			hff = fmax(fmin(zbn + Hl - zbi, hn), 0.);
		else
		{
			T un = pdt * (hui + pdt * ax) / delta;
			auto a = sign(un);
			int iu = un >= 0.0 ? ileft : i;// -(a + 1.) / 2.;
			//double dhdx = h.gradient ? h.gradient(h[i - 1], h[i], h[i + 1]) / Delta : (h[i + 1] - h[i - 1]) / (2. * Delta);
			
			hff = h[iu] + a * (1. - a * un) * dhdx[iu] * delta / 2.;
		}
		XFlux.hfu[i] = fmu * hff;

		if (fabs(hui) > um)
			um = fabs(hui);

		XFlux.hu[i] *= XFlux.hfu[i];
		XFlux.hau[i] = XFlux.hfu[i] * ax;

		H += hff;
	}

	if (H > dry) {
		T c = um / CFL + sqrt(g*H) / CFL_H;//um / CFL + sqrt(g * (hydrostatic ? H : delta * tanh(H / delta))) / CFL_H;
		if (c > 0.) {
			double dtmax[i] = delta / (c * fmu);
			//if (dt < dtmax)
			//	dtmax = dt;
		}
	}
	//pdt = dt = dtnext(dtmax);
}

template <class T> __global__ void CheckadvecMLU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];


	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);

	T CFL_H = T(0.5);

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);

	//For each layer
	{
		T hul = XFlux.hu[i];
		T hi = XEv.h[i];
		T hn = XEv.h[ileft];

		T cmn = T(1.0);//cm[-1]
		T cmi = T(1.0);//cm[]

		if (hul * dt / (Delta * cmn) > CFL * hn)
		{
			hul = CFL * hn * Delta * cmn / dt;
		}
		else if (-hul * dt / (Delta * cm) > CFL * hi)
		{
			hul = -CFL * hn * Delta * cm / dt;
		}

		if (hul != XFlux.hu[i])
		{
			/*if (l < nl - 1)
			{
				hu.x[0, 0, 1] += hu.x[] - hul;
			}*/
			XFlux.hu[i] = hul;
		}
	}


}

template <class T> __global__ void AdvecMLU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];


	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);

	//For each layer
	{
		T un = dt * XFlux.hu[i] / ((XFlux.hfu[i] + dry) * delta);
		T vn = dt * XFlux.hv[i] / ((XFlux.hfv[i] + dry) * delta);
		auto au = sign(un);
		auto av = sign(vn);

		int ixshft = un >= 0.0 ? -1, 0;
		int iyshft = vn >= 0.0 ? -1, 0;
		//int iu = un >= 0.0 ? ileft : i;//-(a + 1.) / 2.;
		int iu = memloc(halowidth, blkmemwidth, ix + ixshft, iy, ib);
		int iut = memloc(halowidth, blkmemwidth, ix + ixshft, iy + 1, ib);
		int iub = memloc(halowidth, blkmemwidth, ix + ixshft, iy - 1, ib);

		int iv = memloc(halowidth, blkmemwidth, ix, iy + iyshft, ib);
		int ivr = memloc(halowidth, blkmemwidth, ix +1, iy + iyshft, ib);
		int iul = memloc(halowidth, blkmemwidth, ix -1, iy + iyshft, ib);

		T su2 = XEv.u[iu] + au * (1. - au * un) * dudx[iu] * delta / 2.0;
		T sv2 = XEv.v[iv] + av * (1. - av * vn) * dvdy[iv] * delta / 2.0;
		if (XFlux.hfv[iu] + XFlux.hfv[iut] > dry)
		{
			T vvn = (XFlux.hv[iu] + XFlux.hv[iut]) / (XFlux.hfv[iu] + XFlux.hfv[iut]);
			T syy = dudy[iu] != 0.0 ? dudy[iu] : vn < 0.0 ? XEv.u[iut] - XEv.u[iu] : XEv.u[iu] - XEv.u[iub];
			su2 -= dt * vvn * syy / (2. * delta);
		}
		if (XFlux.hfu[iv] + XFlux.hfv[ivr] > dry)
		{
			T uun = (XFlux.hv[iv] + XFlux.hv[ivr]) / (XFlux.hfv[iv] + XFlux.hfv[ivr]);
			T syy = dvdx[iv] != 0.0 ? dvdx[iv] : uun < 0.0 ? XEv.v[ivr] - XEv.v[iv] : XEv.v[iv] - XEv.v[ivl];
			sv2 -= dt * uun * syy / (2. * delta);
		}

		XFlux.Fu[i] = su2 * XFlux.hu[i];
		XFlux.Fv[i] = sv2 * XFlux.hv[i];

	}
}
