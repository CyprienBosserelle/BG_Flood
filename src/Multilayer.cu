#include "hip/hip_runtime.h"
#include "Multilayer.h"

//template <class T> void calcAbaro()
//{
//
//	T gmetric = (2. * fm.x[i] / (cm[i] + cm[i - 1]))
//
//	a_baro[i] (G*gmetric*(eta[i-1] - eta[i])/Delta)
//}

template <class T> __global__ void CalcfaceValX(T pdt,Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux, T* dtmax,T* zb)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	

	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps);// +epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);

	T CFL_H = T(0.5);

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);

	T zsi = XEv.zs[i];

	T zsn = XEv.zs[ileft];

	T zbi = zb[i];
	T zbn = zb[ileft];


	T fmu = T(1.0);
	T cm = T(1.0);//T cm = XParam.spherical ? calcCM(T(XParam.Radius), delta, ybo, iy) : T(1.0);
	T gmetric = T(1.0);// (2. * fm.x[i] / (cm[i] + cm[i - 1]));

	T ax = (g * gmetric * (zsn - zsi) / delta);

	T H = 0.;
	T um = 0.;
	T Hr = 0.;
	T Hl = 0.;

	
	//foreach_layer() {
	{
		T hi = XEv.h[i];
		T hn = XEv.h[ileft];
		Hr += hi;
		Hl += hn;
		T hl = hn > dry ? hn : 0.;
		T hr = hi > dry ? hi : 0.;

		
		
		//XFlux.hu[i] = hl > 0. || hr > 0. ? (hl * XEv.u[ileft] + hr * XEvu[i]) / (hl + hr) : 0.;
		T hui = hl > 0. || hr > 0. ? (hl * XEv.u[ileft] + hr * XEv.u[i]) / (hl + hr) : 0.;
		
		T hff;

		if (Hl <= dry)
			hff = max(min(zbi + Hr - zbn, hi), T(0.0));
		else if (Hr <= dry)
			hff = max(min(zbn + Hl - zbi, hn), T(0.0));
		else
		{
			T un = pdt * (hui + pdt * ax) / delta;
			T a =  signof(un);
			int iu = un > 0.0 ? ileft : i;// -(a + 1.) / 2.;
			//double dhdx = h.gradient ? h.gradient(h[i - 1], h[i], h[i + 1]) / Delta : (h[i + 1] - h[i - 1]) / (2. * Delta);
			
			hff = XEv.h[iu] + a * (1. - a * un) * XGrad.dhdx[iu] * delta / 2.;
		}
		XFlux.hfu[i] = fmu * hff;

		if (fabs(hui) > um)
			um = fabs(hui);

		XFlux.hu[i] = hui* fmu * hff;
		XFlux.hau[i] = fmu * hff * ax;

		H += hff;
	}

	if (H > dry) {
		T c = um / CFL + sqrt(g*H) / CFL_H;//um / CFL + sqrt(g * (hydrostatic ? H : delta * tanh(H / delta))) / CFL_H;
		if (c > 0.) {
			dtmax[i] = min(delta / (c * fmu),dtmax[i]);
			//if (dt < dtmax)
			//	dtmax = dt;
		}
	}
	//pdt = dt = dtnext(dtmax);
}
template __global__ void CalcfaceValX<float>(float pdt, Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxMLP<float> XFlux, float* dtmax, float* zb);
template __global__ void CalcfaceValX<double>(double pdt, Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxMLP<double> XFlux, double* dtmax, double* zb);

template <class T> __global__ void CalcfaceValY(T pdt, Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux, T* dtmax, T* zb)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];


	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps);// +epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);

	T CFL_H = T(0.5);

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ibot = memloc(halowidth, blkmemwidth, ix, iy-1, ib);

	T zsi = XEv.zs[i];

	T zsn = XEv.zs[ibot];

	T zbi = zb[i];
	T zbn = zb[ibot];


	T fmu = T(1.0);
	T cm = T(1.0);//T cm = XParam.spherical ? calcCM(T(XParam.Radius), delta, ybo, iy) : T(1.0);
	T gmetric = T(1.0);// (2. * fm.x[i] / (cm[i] + cm[i - 1]));

	T ax = (g * gmetric * (zsn - zsi) / delta);

	T H = 0.;
	T um = 0.;
	T Hr = 0.;
	T Hl = 0.;


	//foreach_layer() {
	{
		T hi = XEv.h[i];
		T hn = XEv.h[ibot];
		Hr += hi;
		Hl += hn;
		T hl = hn > dry ? hn : 0.;
		T hr = hi > dry ? hi : 0.;



		//XFlux.hu[i] = hl > 0. || hr > 0. ? (hl * XEv.u[ileft] + hr * XEvu[i]) / (hl + hr) : 0.;
		T hvi = hl > 0. || hr > 0. ? (hl * XEv.v[ibot] + hr * XEv.v[i]) / (hl + hr) : 0.;

		T hff;

		if (Hl <= dry)
			hff = max(min(zbi + Hr - zbn, hi), 0.);
		else if (Hr <= dry)
			hff = max(min(zbn + Hl - zbi, hn), 0.);
		else
		{
			T vn = pdt * (hvi + pdt * ax) / delta;
			T a = signof(vn);
			int iu = vn > 0.0 ? ibot : i;// -(a + 1.) / 2.;
			//double dhdx = h.gradient ? h.gradient(h[i - 1], h[i], h[i + 1]) / Delta : (h[i + 1] - h[i - 1]) / (2. * Delta);

			hff = XEv.h[iu] + a * (1. - a * vn) * XGrad.dhdy[iu] * delta / 2.;
		}
		XFlux.hfv[i] = fmu * hff;

		if (fabs(hvi) > um)
			um = fabs(hvi);

		XFlux.hv[i] = hvi* fmu * hff;
		XFlux.hav[i] = fmu * hff * ax;

		H += hff;
	}

	if (H > dry) {
		T c = um / CFL + sqrt(g * H) / CFL_H;//um / CFL + sqrt(g * (hydrostatic ? H : delta * tanh(H / delta))) / CFL_H;
		if (c > 0.) {
			dtmax[i] = min(delta / (c * fmu), dtmax[i]);
			//if (dt < dtmax)
			//	dtmax = dt;
		}
	}
	//pdt = dt = dtnext(dtmax);
}
template __global__ void CalcfaceValY<float>(float pdt, Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxMLP<float> XFlux, float* dtmax, float* zb);
template __global__ void CalcfaceValY<double>(double pdt, Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxMLP<double> XFlux, double* dtmax, double* zb);



template <class T> __global__ void CheckadvecMLX(Param XParam, BlockP<T> XBlock,T dt, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];


	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps);// +epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);

	T CFL_H = T(0.5);

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);

	//For each layer
	{
		T hul = XFlux.hu[i];
		T hi = XEv.h[i];
		T hn = XEv.h[ileft];

		T cmn = T(1.0);//cm[-1]
		T cmi = T(1.0);//cm[]

		if (hul * dt / (delta * cmn) > CFL * hn)
		{
			hul = CFL * hn * delta * cmn / dt;
		}
		else if (-hul * dt / (delta * cmi) > CFL * hi)
		{
			hul = -CFL * hn * delta * cmi / dt;
		}

		if (hul != XFlux.hu[i])
		{
			/*if (l < nl - 1)
			{
				hu.x[0, 0, 1] += hu.x[] - hul;
			}*/
			XFlux.hu[i] = hul;
		}
	}


}
template __global__ void CheckadvecMLX<float>(Param XParam, BlockP<float> XBlock, float dt, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxMLP<float> XFlux);
template __global__ void CheckadvecMLX<double>(Param XParam, BlockP<double> XBlock, double dt, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxMLP<double> XFlux);

template <class T> __global__ void CheckadvecMLY(Param XParam, BlockP<T> XBlock,T dt, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];


	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps);// +epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);

	T CFL_H = T(0.5);

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ibot = memloc(halowidth, blkmemwidth, ix, iy-1, ib);

	//For each layer
	{
		T hvl = XFlux.hv[i];
		T hi = XEv.h[i];
		T hn = XEv.h[ibot];

		T cmn = T(1.0);//cm[-1]
		T cmi = T(1.0);//cm[]

		if (hvl * dt / (delta * cmn) > CFL * hn)
		{
			hvl = CFL * hn * delta * cmn / dt;
		}
		else if (-hvl * dt / (delta * cmi) > CFL * hi)
		{
			hvl = -CFL * hn * delta * cmi / dt;
		}

		if (hvl != XFlux.hv[i])
		{
			/*if (l < nl - 1)
			{
				hu.x[0, 0, 1] += hu.x[] - hul;
			}*/
			XFlux.hv[i] = hvl;
		}
	}


}
template __global__ void CheckadvecMLY<float>(Param XParam, BlockP<float> XBlock, float dt, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxMLP<float> XFlux);
template __global__ void CheckadvecMLY<double>(Param XParam, BlockP<double> XBlock, double dt, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxMLP<double> XFlux);



template <class T> __global__ void AdvecFluxML(Param XParam, BlockP<T> XBlock,T dt, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];


	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps);// +epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);


	//For each layer
	{
		T un = dt * XFlux.hu[i] / ((XFlux.hfu[i] + dry) * delta);
		T vn = dt * XFlux.hv[i] / ((XFlux.hfv[i] + dry) * delta);
		T au = signof(un);
		T av = signof(vn);

		int ixshft = un > 0.0 ? -1: 0;
		int iyshft = vn > 0.0 ? -1: 0;
		//int iu = un >= 0.0 ? ileft : i;//-(a + 1.) / 2.;
		int iu = memloc(halowidth, blkmemwidth, ix + ixshft, iy, ib);

		int iut, iub;
		if (ix == 0 && iy == 15)
		{
			iut = memloc(halowidth, blkmemwidth, ix, iy + 1, ib);
		}
		else
		{
			iut = memloc(halowidth, blkmemwidth, ix + ixshft, iy + 1, ib);
		}
		if (ix == 0 && iy == 0)
		{
			iub = memloc(halowidth, blkmemwidth, ix, iy - 1, ib);
		}
		else
		{
			iub = memloc(halowidth, blkmemwidth, ix + ixshft, iy - 1, ib);
		}

		int iv = memloc(halowidth, blkmemwidth, ix, iy + iyshft, ib);

		int ivr, ivl;

		if (iy == 0 && ix == 15)
		{
			ivr = memloc(halowidth, blkmemwidth, ix + 1, iy, ib);
		}
		else
		{
			ivr = memloc(halowidth, blkmemwidth, ix + 1, iy + iyshft, ib);
		}
		
		if (iy == 0 && ix == 0)
		{
			ivl = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);
		}
		else
		{
			ivl = memloc(halowidth, blkmemwidth, ix - 1, iy+iyshft, ib);
		}

		T su2 = XEv.u[iu] + au * (1. - au * un) * XGrad.dudx[iu] * delta / 2.0;
		T sv2 = XEv.v[iv] + av * (1. - av * vn) * XGrad.dvdy[iv] * delta / 2.0;
		if (XFlux.hfv[iu] + XFlux.hfv[iut] > dry)
		{
			T vvn = (XFlux.hv[iu] + XFlux.hv[iut]) / (XFlux.hfv[iu] + XFlux.hfv[iut]);
			T syy = XGrad.dudy[iu] != 0.0 ? XGrad.dudy[iu] : vvn < 0.0 ? XEv.u[iut] - XEv.u[iu] : XEv.u[iu] - XEv.u[iub];
			su2 -= dt * vvn * syy / (2. * delta);
		}
		if (XFlux.hfu[iv] + XFlux.hfu[ivr] > dry)
		{
			T uun = (XFlux.hu[iv] + XFlux.hu[ivr]) / (XFlux.hfu[iv] + XFlux.hfu[ivr]);
			T syy = XGrad.dvdx[iv] != 0.0 ? XGrad.dvdx[iv] : uun < 0.0 ? XEv.v[ivr] - XEv.v[iv] : XEv.v[iv] - XEv.v[ivl];
			sv2 -= dt * uun * syy / (2. * delta);
		}

		XFlux.Fu[i] = su2 * XFlux.hu[i];
		XFlux.Fv[i] = sv2 * XFlux.hv[i];

	}
}
template __global__ void AdvecFluxML<float>(Param XParam, BlockP<float> XBlock, float dt, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxMLP<float> XFlux);
template __global__ void AdvecFluxML<double>(Param XParam, BlockP<double> XBlock, double dt, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxMLP<double> XFlux);


template <class T> __global__ void AdvecEv(Param XParam, BlockP<T> XBlock,T dt, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];


	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps);// +epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);
	int iright = memloc(halowidth, blkmemwidth, ix + 1, iy, ib);
	int itop = memloc(halowidth, blkmemwidth, ix, iy + 1, ib);
	//For each layer
	{
		T uui = XEv.u[i];
		T vvi = XEv.v[i];
		T hi = XEv.h[i];

		uui *= hi;
		vvi *= hi;

		T cmu = T(1.0);
		T cmv = T(1.0);

		uui += dt * (XFlux.Fu[i] - XFlux.Fu[iright]) / (delta * cmu);
		vvi += dt * (XFlux.Fv[i] - XFlux.Fv[itop]) / (delta * cmv);

		T h1 = hi;
		h1 += dt * (XFlux.hu[i] - XFlux.hu[iright]) / (delta * cmu);
		h1 += dt * (XFlux.hv[i] - XFlux.hv[itop]) / (delta * cmv);

		XEv.h[i] = max(h1, T(0.0));

		if (h1 < dry)
		{
			uui = T(0.0);
			vvi = T(0.0);
		}
		else
		{
			uui /= h1;
			vvi /= h1;
		}
		XEv.u[i] = uui;
		XEv.v[i] = vvi;
	}

}
template __global__ void AdvecEv<float>(Param XParam, BlockP<float> XBlock, float dt, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxMLP<float> XFlux);
template __global__ void AdvecEv<double>(Param XParam, BlockP<double> XBlock, double dt, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxMLP<double> XFlux);



template <class T> __global__ void pressureML(Param XParam, BlockP<T> XBlock,T dt, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxMLP<T> XFlux)
{
	int halowidth = XParam.halowidth;
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];


	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps);// +epsi;
	T dry = eps;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);
	int iright = memloc(halowidth, blkmemwidth, ix + 1, iy, ib);
	int itop = memloc(halowidth, blkmemwidth, ix, iy + 1, ib);

	T cm = T(1.0);// XParam.spherical ? calcCM(T(XParam.Radius), delta, ybo, iy) : T(1.0);
	T fmu = T(1.0);
	T fmv = T(1.0);// XParam.spherical ? calcFM(T(XParam.Radius), delta, ybo, ydwn) : T(1.0);
	T fmup = T(1.0);
	T fmvp = T(1.0);// XParam.spherical ? calcFM(T(XParam.Radius), delta, ybo, yup) : T(1.0);

	T cmdinv, ga;

	cmdinv = T(1.0) / (cm * delta);
	ga = T(0.5) * g;

	//For each layer
	{

		T uui = XEv.u[i];
		T vvi = XEv.v[i];
		//
		XFlux.hu[i] += dt * XFlux.hau[i];
		XFlux.hv[i] += dt * XFlux.hav[i];
		
		uui += dt * (XFlux.hau[i] + XFlux.hau[iright]) / (XFlux.hfu[i] + XFlux.hfu[iright] + dry);
		vvi += dt * (XFlux.hav[i] + XFlux.hav[itop]) / (XFlux.hfv[i] + XFlux.hfv[itop] + dry);

		T dmdl = (fmup - fmu) * cmdinv;// absurd if not spherical!
		T dmdt = (fmvp - fmv) * cmdinv;
		T fG = vvi * dmdl - uui * dmdt;

		uui += dt * fG * vvi;
		vvi -= dt * fG * uui;

		XEv.u[i] = uui;
		XEv.v[i] = vvi;
	}
	


}
template __global__ void pressureML<float>(Param XParam, BlockP<float> XBlock, float dt, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxMLP<float> XFlux);
template __global__ void pressureML<double>(Param XParam, BlockP<double> XBlock, double dt, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxMLP<double> XFlux);



template <class T> __global__ void CleanupML()
{

}