#include "hip/hip_runtime.h"

#include "Setup_GPU.h"


template <class T> void SetupGPU(Param XParam, Model<T> XModel,Forcing<float> &XForcing, Model<T>& XModel_g)
{
	if (XParam.GPUDEVICE >= 0)
	{
		log("Setting up GPU");


		hipSetDevice(XParam.GPUDEVICE);
		//Allocate memory for the model on the GPU
		AllocateGPU(XParam.nblkmem, XParam.blksize, XParam, XModel_g);
		
		// Copy arrays from CPU to GPU
		CopytoGPU(XParam.nblkmem, XParam.blksize,XParam, XModel, XModel_g);

		//
		fillHaloGPU(XParam, XModel_g.blocks, XModel_g.evolv);

		//=============================
		// Same for Bnds


		// Allocate memory for the boundary blk
		AllocateGPU(XForcing.left.nblk, 1, XForcing.left.blks_g);
		//copy bnd blk info on GPU
		CopytoGPU(XForcing.left.nblk, 1, XForcing.left.blks, XForcing.left.blks_g);

		AllocateGPU(XForcing.right.nblk, 1, XForcing.right.blks_g);
		CopytoGPU(XForcing.right.nblk, 1, XForcing.right.blks, XForcing.right.blks_g);

		AllocateGPU(XForcing.top.nblk, 1, XForcing.top.blks_g);
		CopytoGPU(XForcing.top.nblk, 1, XForcing.top.blks, XForcing.top.blks_g);

		AllocateGPU(XForcing.bot.nblk, 1, XForcing.bot.blks_g);
		CopytoGPU(XForcing.bot.nblk, 1, XForcing.bot.blks, XForcing.bot.blks_g);

		// Also for mask
		XModel_g.blocks.mask.nblk = XModel.blocks.mask.nblk;
		AllocateGPU(XModel_g.blocks.mask.nblk, 1, XModel_g.blocks.mask.side);
		AllocateGPU(XModel_g.blocks.mask.nblk, 1, XModel_g.blocks.mask.blks);
		CopytoGPU(XModel_g.blocks.mask.nblk, 1, XModel.blocks.mask.side, XModel_g.blocks.mask.side);
		CopytoGPU(XModel_g.blocks.mask.nblk, 1, XModel.blocks.mask.blks, XModel_g.blocks.mask.blks);


		// things are quite different for Time Series output. Why is that?.
		if (XParam.TSnodesout.size() > 0)
		{

			AllocateGPU(XModel.bndblk.nblkTs, 1, XModel_g.bndblk.Tsout);
			CopytoGPU(XModel.bndblk.nblkTs, 1, XModel.bndblk.Tsout, XModel_g.bndblk.Tsout);

		}

		// River are a bit of a special case too
		if (XForcing.rivers.size() > 0)
		{
			//
			XModel_g.bndblk.nblkriver = XModel.bndblk.nblkriver;
			AllocateGPU(XModel.bndblk.nblkriver, 1, XModel_g.bndblk.river);
			CopytoGPU(XModel.bndblk.nblkriver, 1, XModel.bndblk.river, XModel_g.bndblk.river);
		}

		// Reset GPU mean and max arrays
		if (XParam.outmax)
		{
			//ResetmaxvarGPU(XParam);
		}
		if (XParam.outmean)
		{
			//ResetmeanvarGPU(XParam);
		}

		Initmaparray(XModel_g);

		InitzbgradientGPU(XParam, XModel_g);
	}
}
template void SetupGPU<float>(Param XParam, Model<float> XModel, Forcing<float>& XForcing, Model<float>& XModel_g);
template void SetupGPU<double>(Param XParam, Model<double> XModel, Forcing<float>& XForcing, Model<double>& XModel_g);


//from hip/hip_runtime_api.h
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

template <typename T>
void check(T result, char const* const func, const char* const file,
	int const line)
{
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), cudaGetErrorEnum(result), func);
		exit(EXIT_FAILURE);
	}
}

void CUDA_CHECK(hipError_t CUDerr)
{

	// This will output the proper CUDA error strings in the event
// that a CUDA host call returns an error

	checkCudaErrors(CUDerr);

	/*if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}*/
}


template <class T> void CopytoGPU(int nblk, int blksize, T * z_cpu, T* z_gpu)
{
	CUDA_CHECK(hipMemcpy(z_gpu, z_cpu, nblk * blksize * sizeof(T), hipMemcpyHostToDevice));
}
template void CopytoGPU<bool>(int nblk, int blksize, bool* z_cpu, bool* z_gpu);
template void CopytoGPU<int>(int nblk, int blksize, int* z_cpu, int* z_gpu);
template void CopytoGPU<float>(int nblk, int blksize, float* z_cpu, float* z_gpu);
template void CopytoGPU<double>(int nblk, int blksize, double* z_cpu, double* z_gpu);

template <class T> void CopyGPUtoCPU(int nblk, int blksize, T* z_cpu, T* z_gpu)
{
	CUDA_CHECK(hipMemcpy(z_cpu, z_gpu, nblk * blksize * sizeof(T), hipMemcpyDeviceToHost));
}
template void CopyGPUtoCPU<bool>(int nblk, int blksize, bool* z_cpu, bool* z_gpu);
template void CopyGPUtoCPU<int>(int nblk, int blksize, int* z_cpu, int* z_gpu);
template void CopyGPUtoCPU<float>(int nblk, int blksize, float* z_cpu, float* z_gpu);
template void CopyGPUtoCPU<double>(int nblk, int blksize, double* z_cpu, double* z_gpu);

template <class T> void CopytoGPU(int nblk, int blksize, EvolvingP<T> XEv_cpu, EvolvingP<T> XEv_gpu)
{
	CopytoGPU(nblk, blksize, XEv_cpu.h, XEv_gpu.h);
	CopytoGPU(nblk, blksize, XEv_cpu.zs, XEv_gpu.zs);
	CopytoGPU(nblk, blksize, XEv_cpu.u, XEv_gpu.u);
	CopytoGPU(nblk, blksize, XEv_cpu.v, XEv_gpu.v);
}
template void CopytoGPU<float>(int nblk, int blksize, EvolvingP<float> XEv_cpu, EvolvingP<float> XEv_gpu);
template void CopytoGPU < double >(int nblk, int blksize, EvolvingP<double> XEv_cpu, EvolvingP < double >  XEv_gpu);


template <class T> void CopytoGPU(int nblk, int blksize, GradientsP<T> XGrad_cpu, GradientsP<T> XGrad_gpu)
{
	CopytoGPU(nblk, blksize, XGrad_cpu.dhdx, XGrad_gpu.dhdx);
	CopytoGPU(nblk, blksize, XGrad_cpu.dhdy, XGrad_gpu.dhdy);
	CopytoGPU(nblk, blksize, XGrad_cpu.dudx, XGrad_gpu.dudx);
	CopytoGPU(nblk, blksize, XGrad_cpu.dudy, XGrad_gpu.dudy);
	CopytoGPU(nblk, blksize, XGrad_cpu.dvdx, XGrad_gpu.dvdx);
	CopytoGPU(nblk, blksize, XGrad_cpu.dvdy, XGrad_gpu.dvdy);
	CopytoGPU(nblk, blksize, XGrad_cpu.dzsdx, XGrad_gpu.dzsdx);
	CopytoGPU(nblk, blksize, XGrad_cpu.dzsdy, XGrad_gpu.dzsdy);
}
template void CopytoGPU(int nblk, int blksize, GradientsP<float> XGrad_cpu, GradientsP<float> XGrad_gpu);
template void CopytoGPU(int nblk, int blksize, GradientsP<double> XGrad_cpu, GradientsP<double> XGrad_gpu);

template <class T> void CopytoGPU(int nblk, int blksize, Param XParam, Model<T> XModel_cpu, Model<T> XModel_gpu)
{
	CopytoGPU(nblk, blksize, XModel_cpu.zb, XModel_gpu.zb);

	CopytoGPU(nblk, blksize, XModel_cpu.evolv, XModel_gpu.evolv);
	//CopytoGPU(nblk, blksize, XModel_cpu.evolv_o, XModel_gpu.evolv_o);

	CopytoGPU(nblk, blksize, XModel_cpu.evolv_o, XModel_gpu.evolv_o);

	CopytoGPU(nblk, blksize, XModel_cpu.cf, XModel_gpu.cf);

	CopytoGPU(nblk, blksize, XModel_cpu.zb, XModel_gpu.zb);

	

	//Block info
	CopytoGPU(nblk, 1, XModel_cpu.blocks.active, XModel_gpu.blocks.active);
	CopytoGPU(nblk, blksize, XModel_cpu.blocks.activeCell, XModel_gpu.blocks.activeCell);
	CopytoGPU(nblk, 1, XModel_cpu.blocks.level, XModel_gpu.blocks.level);

	CopytoGPU(nblk, 1, XModel_cpu.blocks.xo, XModel_gpu.blocks.xo);
	CopytoGPU(nblk, 1, XModel_cpu.blocks.yo, XModel_gpu.blocks.yo);

	CopytoGPU(nblk, 1, XModel_cpu.blocks.BotLeft, XModel_gpu.blocks.BotLeft);
	CopytoGPU(nblk, 1, XModel_cpu.blocks.BotRight, XModel_gpu.blocks.BotRight);

	CopytoGPU(nblk, 1, XModel_cpu.blocks.TopLeft, XModel_gpu.blocks.TopLeft);
	CopytoGPU(nblk, 1, XModel_cpu.blocks.TopRight, XModel_gpu.blocks.TopRight);

	CopytoGPU(nblk, 1, XModel_cpu.blocks.LeftBot, XModel_gpu.blocks.LeftBot);
	CopytoGPU(nblk, 1, XModel_cpu.blocks.LeftTop, XModel_gpu.blocks.LeftTop);

	CopytoGPU(nblk, 1, XModel_cpu.blocks.RightBot, XModel_gpu.blocks.RightBot);
	CopytoGPU(nblk, 1, XModel_cpu.blocks.RightTop, XModel_gpu.blocks.RightTop);
	

	if (XParam.outmax)
	{
		CopytoGPU(nblk, blksize, XModel_cpu.evolv, XModel_gpu.evmax);
	}
	if (XParam.outmean)
	{
		CopytoGPU(nblk, blksize, XModel_cpu.evolv, XModel_gpu.evmean);
	}

}
template void CopytoGPU<float>(int nblk, int blksize, Param XParam, Model<float> XModel_cpu, Model<float> XModel_gpu);
template void CopytoGPU<double>(int nblk, int blksize, Param XParam, Model<double> XModel_cpu, Model<double> XModel_gpu);


void AllocateTEX(int nx, int ny, TexSetP& Tex, float* input)
{


	CUDA_CHECK(hipMallocArray(&Tex.CudArr, &Tex.channelDesc, nx, ny));
	CUDA_CHECK(hipMemcpyToArray(Tex.CudArr, 0, 0, input, nx * ny * sizeof(float), hipMemcpyHostToDevice));


	memset(&Tex.texDesc, 0, sizeof(hipTextureDesc));
	Tex.texDesc.addressMode[0] = hipAddressModeClamp;
	Tex.texDesc.addressMode[1] = hipAddressModeClamp;
	Tex.texDesc.filterMode = hipFilterModeLinear;
	Tex.texDesc.normalizedCoords = false;

	memset(&Tex.resDesc, 0, sizeof(hipResourceDesc));

	Tex.resDesc.resType = hipResourceTypeArray;
	Tex.resDesc.res.array.array = Tex.CudArr;

	CUDA_CHECK(hipCreateTextureObject(&Tex.tex, &Tex.resDesc, &Tex.texDesc, NULL));
	//CUDA_CHECK(hipBindTextureToArray(Tex, zca, cCFD));


}


void AllocateBndTEX(bndparam & side)
{
	int nbndtimes = (int)side.data.size();
	int nbndvec = (int)side.data[0].wlevs.size();
	
	float* lWLS;
	lWLS = (float*)malloc(nbndtimes * nbndvec * sizeof(float));

	for (int ibndv = 0; ibndv < nbndvec; ibndv++)
	{
		for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
		{
			//
			lWLS[ibndt + ibndv * nbndtimes] = (float)side.data[ibndt].wlevs[ibndv];
		}
	}
	AllocateTEX(nbndtimes, nbndvec, side.GPU.WLS, lWLS);
	
	// In case of Nesting U and V are also prescribed

	// If uu information is available in the boundary we can assume it is a nesting type of bnd
	int nbndvecuu = (int)side.data[0].uuvel.size();
	if (nbndvecuu == nbndvec)
	{
		//
		for (int ibndv = 0; ibndv < nbndvec; ibndv++)
		{
			for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
			{
				//
				lWLS[ibndt + ibndv * nbndtimes] = (float)side.data[ibndt].uuvel[ibndv];
			}
		}
		AllocateTEX(nbndtimes, nbndvec, side.GPU.Uvel, lWLS);
		
	}
	//V velocity side
	int nbndvecvv = (int)side.data[0].vvvel.size();

	if (nbndvecvv == nbndvec)
	{
		for (int ibndv = 0; ibndv < nbndvec; ibndv++)
		{
			for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
			{
				//
				lWLS[ibndt + ibndv * nbndtimes] = (float)side.data[ibndt].vvvel[ibndv];
			}
		}
		AllocateTEX(nbndtimes, nbndvec, side.GPU.Vvel, lWLS);
	}

	free(lWLS);


}



