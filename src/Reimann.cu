#include "hip/hip_runtime.h"
#include "Reimann.h"


/*! \fn void UpdateButtingerXGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T* zb)
* \brief "Adaptive" second-order hydrostatic reconstruction. GPU version for t X-axis
*
* ## Description
* This function computes the flux term at the cell interface using the hydrostatic reconstruction from Buttinger et al (2019).
* This reconstruction is safe for steep slope with thin water depth and is well-balanced meaning that it conserve the "lake-at-rest" states.
*
* For optimising the code on CPU and GPU there are 4 versions of this function: X or Y and CPU or GPU
*
* ## Where does this come from:
* This scheme was adapted/modified from the Basilisk / B-Flood source code. I (CypB) changed the zr and zl term back to the Audusse type reconstruction
* http://basilisk.fr/sandbox/b-flood/saint-venant-topo.h
*
* Reference:
* Kirstetter, G., Delestre, O., Lagree, P.-Y., Popinet, S., and Josserand, C.: B-flood 1.0: an open-source Saint-Venant model for flash flood simulation using adaptive refinement, Geosci. Model Dev. Discuss. [preprint], https://doi.org/10.5194/gmd-2021-15, in review, 2021.*
* Buttinger-Kreuzhuber, A., Horvath, Z., Noelle, S., Bloschl, G., and Waser, J.: A fast second-order shallow water scheme on two-dimensional
* structured grids over abrupt topography, Advances in water resources, 127, 89-108, 2019.
*/
template <class T> __global__ void UpdateButtingerXGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T* zb)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int RB, LBRB, LB, RBLB, levRB, levLB;
	RB = XBlock.RightBot[ib];
	levRB = XBlock.level[RB];
	LBRB = XBlock.LeftBot[RB];

	LB = XBlock.LeftBot[ib];
	levLB = XBlock.level[LB];
	RBLB = XBlock.RightBot[LB];

	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);

	T ybo = T(XParam.yo + XBlock.yo[ib]);


	//T dhdxi = XGrad.dhdx[i];
	//T dhdxmin = XGrad.dhdx[ileft];
	T cm = XParam.spherical ? calcCM(T(XParam.Radius), delta, ybo, iy) : T(1.0);
	T fmu = T(1.0);

	T hi = XEv.h[i];

	T hn = XEv.h[ileft];


	if (hi > eps || hn > eps)
	{
		T dx, zi, zn, hr, hl, etar, etal, zr, zl, zA, zCN, hCNr, hCNl;
		T ui, vi, uli, vli, dhdxi, dhdxil, dudxi, dudxil, dvdxi,dvdxil;

		T ga = g * T(0.5);
		// along X
		dx = delta * T(0.5);
		zi = zb[i];
		zn = zb[ileft];

		ui = XEv.u[i];
		vi = XEv.v[i];
		uli = XEv.u[ileft];
		vli = XEv.v[ileft];

		dhdxi = XGrad.dhdx[i];
		dhdxil = XGrad.dhdx[ileft];
		dudxi = XGrad.dudx[i];
		dudxil = XGrad.dudx[ileft];
		dvdxi = XGrad.dvdx[i];
		dvdxil = XGrad.dvdx[ileft];


		hr = hi - dx * dhdxi;
		hl = hn + dx * dhdxil;
		etar = XEv.zs[i] - dx * XGrad.dzsdx[i];
		etal = XEv.zs[ileft] + dx * XGrad.dzsdx[ileft];

		//define the topography term at the interfaces
		zr = etar - hr;// zi - dx * XGrad.dzbdx[i];
		zl = etal - hl;// zn + dx * XGrad.dzbdx[ileft];

		//define the Audusse terms
		zA = max(zr, zl);

		// Now the CN terms
		zCN = min(zA, min(etal, etar));
		hCNr = max(T(0.0), min(etar - zCN, hr));
		hCNl = max(T(0.0), min(etal - zCN, hl));
		
		//Velocity reconstruction
		//To avoid high velocities near dry cells, we reconstruct velocities according to Bouchut.
		T ul, ur, vl, vr,sl,sr;
		if (hi > eps) {
			ur = ui - (1. + dx * dhdxi / hi) * dx * dudxi;
			vr = vi - (1. + dx * dhdxi / hi) * dx * dvdxi;
		}
		else {
			ur = ui - dx * dudxi;
			vr = vi - dx * dvdxi;
		}
		if (hn > eps) {
			ul = uli + (1. - dx * dhdxil / hn) * dx * dudxil;
			vl = vli + (1. - dx * dhdxil / hn) * dx * dvdxil;
		}
		else {
			ul = uli + dx * dudxil;
			vl = vli + dx * dvdxil;
		}

	


		T fh, fu, fv, dt;


		//solver below also modifies fh and fu
		dt = hllc(g, delta, epsi, CFL, cm, fmu, hCNl, hCNr, ul, ur, fh, fu);
		//hllc(T g, T delta, T epsi, T CFL, T cm, T fm, T hm, T hp, T um, T up, T & fh, T & fq)

		if (dt < dtmax[i])
		{
			dtmax[i] = dt;
		}
		

		fv = (fh > 0. ? vl : vr) * fh;

	
		// Topographic source term

		// In the case of adaptive refinement, care must be taken to ensure
		// well-balancing at coarse/fine faces (see [notes/balanced.tm]()). 
		if ((ix == blockDim.y) && levRB < lev)//(ix==16) i.e. in the right halo
		{
			int jj = LBRB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + blockDim.y / 2;
			int iright = memloc(halowidth, blkmemwidth, 0, jj, RB);;
			hi = XEv.h[iright];
			zi = zb[iright];
		}
		if ((ix == 0) && levLB < lev)//(ix==16) i.e. in the right halo
		{
			int jj = RBLB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + blockDim.y / 2;
			int ilc = memloc(halowidth, blkmemwidth, blockDim.y - 1, jj, LB);
			hn = XEv.h[ilc];
			zn = zb[ilc];
		}

		sl = ga * (hi + hCNr) * (zi - zCN);
		sr = ga * (hCNl + hn) * (zn - zCN);

		////Flux update

		XFlux.Fhu[i] = fmu * fh;
		XFlux.Fqux[i] = fmu * (fu - sl);
		XFlux.Su[i] = fmu * (fu - sr);
		XFlux.Fqvx[i] = fmu * fv;
	}
	else
	{
		dtmax[i] = T(1.0) / epsi;
		XFlux.Fhu[i] = T(0.0);
		XFlux.Fqux[i] = T(0.0);
		XFlux.Su[i] = T(0.0);
		XFlux.Fqvx[i] = T(0.0);
	}

}
template __global__ void UpdateButtingerXGPU(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax, float* zb);
template __global__ void UpdateButtingerXGPU(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax, double* zb);


/*! \fn void UpdateButtingerXCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T* zb)
* \brief "Adaptive" second-order hydrostatic reconstruction. CPU version for the X-axis
*
* ## Description
* This function computes the flux term at the cell interface using the hydrostatic reconstruction from Buttinger et al (2019).
* This reconstruction is safe for steep slope with thin water depth and is well-balanced meaning that it conserve the "lake-at-rest" states.
*
* For optimising the code on CPU and GPU there are 4 versions of this function: X or Y and CPU or GPU
*
* ## Where does this come from:
* This scheme was adapted/modified from the Basilisk / B-Flood source code. I (CypB) changed the zr and zl term back to the Audusse type reconstruction
* http://basilisk.fr/sandbox/b-flood/saint-venant-topo.h
*
* Reference:
* Kirstetter, G., Delestre, O., Lagree, P.-Y., Popinet, S., and Josserand, C.: B-flood 1.0: an open-source Saint-Venant model for flash flood simulation using adaptive refinement, Geosci. Model Dev. Discuss. [preprint], https://doi.org/10.5194/gmd-2021-15, in review, 2021.*
* Buttinger-Kreuzhuber, A., Horvath, Z., Noelle, S., Bloschl, G., and Waser, J.: A fast second-order shallow water scheme on two-dimensional
* structured grids over abrupt topography, Advances in water resources, 127, 89-108, 2019.
*/
template <class T> __host__ void UpdateButtingerXCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T* zb)
{


	T delta;
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	int RB, LBRB, LB, RBLB, levRB, levLB;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		int lev = XBlock.level[ib];
		delta = calcres(T(XParam.delta), lev);

		// neighbours for source term

		RB = XBlock.RightBot[ib];
		levRB = XBlock.level[RB];
		LBRB = XBlock.LeftBot[RB];

		LB = XBlock.LeftBot[ib];
		levLB = XBlock.level[LB];
		RBLB = XBlock.RightBot[LB];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth + XParam.halowidth); ix++)
			{

				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
				int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);

				T ybo = T(XParam.yo + XBlock.yo[ib]);


				//T dhdxi = XGrad.dhdx[i];
				//T dhdxmin = XGrad.dhdx[ileft];
				T cm = XParam.spherical ? calcCM(T(XParam.Radius), delta, ybo, iy) : T(1.0);
				T fmu = T(1.0);

				T hi = XEv.h[i];

				T hn = XEv.h[ileft];


				if (hi > eps || hn > eps)
				{
					T dx, zi, zn, hr, hl, etar, etal, zr, zl, zA, zCN, hCNr, hCNl;
					T ui, vi, uli, vli, dhdxi, dhdxil, dudxi, dudxil, dvdxi, dvdxil;

					T ga = g * T(0.5);
					// along X
					dx = delta * T(0.5);
					zi = zb[i];
					zn = zb[ileft];

					ui = XEv.u[i];
					vi = XEv.v[i];
					uli = XEv.u[ileft];
					vli = XEv.v[ileft];

					dhdxi = XGrad.dhdx[i];
					dhdxil = XGrad.dhdx[ileft];
					dudxi = XGrad.dudx[i];
					dudxil = XGrad.dudx[ileft];
					dvdxi = XGrad.dvdx[i];
					dvdxil = XGrad.dvdx[ileft];


					hr = hi - dx * dhdxi;
					hl = hn + dx * dhdxil;
					etar = XEv.zs[i] - dx * XGrad.dzsdx[i];
					etal = XEv.zs[ileft] + dx * XGrad.dzsdx[ileft];

					//define the topography term at the interfaces
					zr = etar - hr;// zi - dx * XGrad.dzbdx[i];
					zl = etal - hl;// zn + dx * XGrad.dzbdx[ileft];

					//define the Audusse terms
					zA = max(zr, zl);

					// Now the CN terms
					zCN = min(zA, min(etal, etar));
					hCNr = max(T(0.0), min(etar - zCN, hr));
					hCNl = max(T(0.0), min(etal - zCN, hl));

					//Velocity reconstruction
					//To avoid high velocities near dry cells, we reconstruct velocities according to Bouchut.
					T ul, ur, vl, vr, sl, sr;
					if (hi > eps) {
						ur = ui - (1. + dx * dhdxi / hi) * dx * dudxi;
						vr = vi - (1. + dx * dhdxi / hi) * dx * dvdxi;
					}
					else {
						ur = ui - dx * dudxi;
						vr = vi - dx * dvdxi;
					}
					if (hn > eps) {
						ul = uli + (1. - dx * dhdxil / hn) * dx * dudxil;
						vl = vli + (1. - dx * dhdxil / hn) * dx * dvdxil;
					}
					else {
						ul = uli + dx * dudxil;
						vl = vli + dx * dvdxil;
					}




					T fh, fu, fv, dt;


					//solver below also modifies fh and fu
					dt = hllc(g, delta, epsi, CFL, cm, fmu, hCNl, hCNr, ul, ur, fh, fu);
					//hllc(T g, T delta, T epsi, T CFL, T cm, T fm, T hm, T hp, T um, T up, T & fh, T & fq)

					if (dt < dtmax[i])
					{
						dtmax[i] = dt;
					}
					

					fv = (fh > 0. ? vl : vr) * fh;


					// Topographic source term

					// In the case of adaptive refinement, care must be taken to ensure
					// well-balancing at coarse/fine faces (see [notes/balanced.tm]()). 
					if ((ix == XParam.blkwidth) && levRB < lev)//(ix==16) i.e. in the right halo
					{
						int jj = LBRB == ib ? ftoi(floor(iy * (T)0.5)) : ftoi(floor(iy * (T)0.5) + XParam.blkwidth / 2);
						int iright = memloc(halowidth, blkmemwidth, 0, jj, RB);;
						hi = XEv.h[iright];
						zi = zb[iright];
					}
					if ((ix == 0) && levLB < lev)//(ix==16) i.e. in the right halo if you 
					{
						int jj = RBLB == ib ? ftoi(floor(iy * (T)0.5)) : ftoi(floor(iy * (T)0.5) + XParam.blkwidth / 2);
						int ilc = memloc(halowidth, blkmemwidth, XParam.blkwidth- 1, jj, LB);
						
						hn = XEv.h[ilc];
						zn = zb[ilc];
					}

					sl = ga * (hi + hCNr) * (zi - zCN);
					sr = ga * (hCNl + hn) * (zn - zCN);

					////Flux update

					XFlux.Fhu[i] = fmu * fh;
					XFlux.Fqux[i] = fmu * (fu - sl);
					XFlux.Su[i] = fmu * (fu - sr);
					XFlux.Fqvx[i] = fmu * fv;
				}
				else
				{
					dtmax[i] = T(1.0) / epsi;
					XFlux.Fhu[i] = T(0.0);
					XFlux.Fqux[i] = T(0.0);
					XFlux.Su[i] = T(0.0);
					XFlux.Fqvx[i] = T(0.0);
				}
			}
		}
	}
}
template __host__ void UpdateButtingerXCPU(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax, float* zb);
template __host__ void UpdateButtingerXCPU(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax, double* zb);


/*! \fn void UpdateButtingerYGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T* zb)
* \brief "Adaptive" second-order hydrostatic reconstruction. GPU version for the Y-axis
*
* ## Description
* This function computes the flux term at the cell interface using the hydrostatic reconstruction from Buttinger et al (2019).
* This reconstruction is safe for steep slope with thin water depth and is well-balanced meaning that it conserve the "lake-at-rest" states.
*
* For optimising the code on CPU and GPU there are 4 versions of this function: X or Y and CPU or GPU
*
* ## Where does this come from:
* This scheme was adapted/modified from the Basilisk / B-Flood source code. I (CypB) changed the zr and zl term back to the Audusse type reconstruction
* http://basilisk.fr/sandbox/b-flood/saint-venant-topo.h
*
* Reference:
* Kirstetter, G., Delestre, O., Lagree, P.-Y., Popinet, S., and Josserand, C.: B-flood 1.0: an open-source Saint-Venant model for flash flood simulation using adaptive refinement, Geosci. Model Dev. Discuss. [preprint], https://doi.org/10.5194/gmd-2021-15, in review, 2021.*
* Buttinger-Kreuzhuber, A., Horvath, Z., Noelle, S., Bloschl, G., and Waser, J.: A fast second-order shallow water scheme on two-dimensional
* structured grids over abrupt topography, Advances in water resources, 127, 89-108, 2019.
*/
template <class T> __global__ void UpdateButtingerYGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T* zb)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int TL, BLTL, BL, TLBL, levTL, levBL;
	TL = XBlock.TopLeft[ib];
	levTL = XBlock.level[TL];
	BLTL = XBlock.BotLeft[TL];

	BL = XBlock.BotLeft[ib];
	levBL = XBlock.level[BL];
	TLBL = XBlock.TopLeft[BL];

	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;
	T delta = calcres(T(XParam.delta), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ibot = memloc(halowidth, blkmemwidth, ix, iy - 1, ib);


	T ybo = T(XParam.yo + XBlock.yo[ib]);

	//T dhdyi = XGrad.dhdy[i];
	//T dhdymin = XGrad.dhdy[ibot];
	T cm = XParam.spherical ? calcCM(T(XParam.Radius), delta, ybo, iy) : T(1.0);
	T fmv = XParam.spherical ? calcFM(T(XParam.Radius), delta, ybo, T(iy)) : T(1.0);

	T hi = XEv.h[i];

	T hn = XEv.h[ibot];


	if (hi > eps || hn > eps)
	{
		T dx, zi, zn, hr, hl, etar, etal, zr, zl, zA, zCN, hCNr, hCNl;
		T ui, vi, uli, vli, dhdyi, dhdyil, dudyi, dudyil, dvdyi, dvdyil;

		T ga = g * T(0.5);
		// along X
		dx = delta * T(0.5);
		zi = zb[i];
		zn = zb[ibot];

		ui = XEv.u[i];
		vi = XEv.v[i];
		uli = XEv.u[ibot];
		vli = XEv.v[ibot];

		dhdyi = XGrad.dhdy[i];
		dhdyil = XGrad.dhdy[ibot];
		dudyi = XGrad.dudy[i];
		dudyil = XGrad.dudy[ibot];
		dvdyi = XGrad.dvdy[i];
		dvdyil = XGrad.dvdy[ibot];


		hr = hi - dx * dhdyi;
		hl = hn + dx * dhdyil;
		etar = XEv.zs[i] - dx * XGrad.dzsdy[i];
		etal = XEv.zs[ibot] + dx * XGrad.dzsdy[ibot];

		//define the topography term at the interfaces
		zr = etar - hr;// zi - dx * XGrad.dzbdy[i];
		zl = etal - hl;// zn + dx * XGrad.dzbdy[ibot];

		//define the Audusse terms
		zA = max(zr, zl);

		// Now the CN terms
		zCN = min(zA, min(etal, etar));
		hCNr = max(T(0.0), min(etar - zCN, hr));
		hCNl = max(T(0.0), min(etal - zCN, hl));

		//Velocity reconstruction
		//To avoid high velocities near dry cells, we reconstruct velocities according to Bouchut.
		T ul, ur, vl, vr, sl, sr;
		if (hi > eps) {
			ur = ui - (1. + dx * dhdyi / hi) * dx * dudyi;
			vr = vi - (1. + dx * dhdyi / hi) * dx * dvdyi;
		}
		else {
			ur = ui - dx * dudyi;
			vr = vi - dx * dvdyi;
		}
		if (hn > eps) {
			ul = uli + (1. - dx * dhdyil / hn) * dx * dudyil;
			vl = vli + (1. - dx * dhdyil / hn) * dx * dvdyil;
		}
		else {
			ul = uli + dx * dudyil;
			vl = vli + dx * dvdyil;
		}




		T fh, fu, fv, dt;


		//solver below also modifies fh and fu
		dt = hllc(g, delta, epsi, CFL, cm, fmv, hCNl, hCNr, vl, vr, fh, fu);
		//hllc(T g, T delta, T epsi, T CFL, T cm, T fm, T hm, T hp, T um, T up, T & fh, T & fq)

		if (dt < dtmax[i])
		{
			dtmax[i] = dt;
		}
		

		fv = (fh > 0. ? ul : ur) * fh;


		// Topographic source term

		// In the case of adaptive refinement, care must be taken to ensure
		// well-balancing at coarse/fine faces (see [notes/balanced.tm]()). 
		if ((iy == blockDim.x) && levTL < lev)//(ix==16) i.e. in the right halo
		{
			int jj = BLTL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + blockDim.x / 2;
			int itop = memloc(halowidth, blkmemwidth, jj, 0, TL);;
			hi = XEv.h[itop];
			zi = zb[itop];
		}
		if ((iy == 0) && levBL < lev)//(ix==16) i.e. in the right halo
		{
			int jj = TLBL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + blockDim.x / 2;
			int ibc = memloc(halowidth, blkmemwidth, jj, blockDim.x - 1, BL);
			hn = XEv.h[ibc];
			zn = zb[ibc];
		}

		sl = ga * (hi + hCNr) * (zi - zCN);
		sr = ga * (hCNl + hn) * (zn - zCN);

		////Flux update

		XFlux.Fhv[i] = fmv * fh;
		XFlux.Fqvy[i] = fmv * (fu - sl);
		XFlux.Sv[i] = fmv * (fu - sr);
		XFlux.Fquy[i] = fmv * fv;
	}
	else
	{
		dtmax[i] = T(1.0) / epsi;
		XFlux.Fhv[i] = T(0.0);
		XFlux.Fqvy[i] = T(0.0);
		XFlux.Sv[i] = T(0.0);
		XFlux.Fquy[i] = T(0.0);
	}

}
template __global__ void UpdateButtingerYGPU(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax, float* zb);
template __global__ void UpdateButtingerYGPU(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax, double* zb);

/*! \fn void UpdateButtingerYCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T* zb)
* \brief "Adaptive" second-order hydrostatic reconstruction. CPU version for the Y-axis
*
* ## Description
* This function computes the flux term at the cell interface using the hydrostatic reconstruction from Buttinger et al (2019).
* This reconstruction is safe for steep slope with thin water depth and is well-balanced meaning that it conserve the "lake-at-rest" states.
*
* For optimising the code on CPU and GPU there are 4 versions of this function: X or Y and CPU or GPU
*
* ## Where does this come from:
* This scheme was adapted/modified from the Basilisk / B-Flood source code. I (CypB) changed the zr and zl term back to the Audusse type reconstruction
* http://basilisk.fr/sandbox/b-flood/saint-venant-topo.h
*
* Reference:
* Kirstetter, G., Delestre, O., Lagree, P.-Y., Popinet, S., and Josserand, C.: B-flood 1.0: an open-source Saint-Venant model for flash flood simulation using adaptive refinement, Geosci. Model Dev. Discuss. [preprint], https://doi.org/10.5194/gmd-2021-15, in review, 2021.*
* Buttinger-Kreuzhuber, A., Horvath, Z., Noelle, S., Bloschl, G., and Waser, J.: A fast second-order shallow water scheme on two-dimensional
* structured grids over abrupt topography, Advances in water resources, 127, 89-108, 2019.
*/
template <class T> __host__ void UpdateButtingerYCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T* zb)
{

	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;
	T delta;
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);


	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	int TL, BLTL, BL, TLBL, levTL, levBL, lev;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];




		TL = XBlock.TopLeft[ib];
		levTL = XBlock.level[TL];
		BLTL = XBlock.BotLeft[TL];

		BL = XBlock.BotLeft[ib];
		levBL = XBlock.level[BL];
		TLBL = XBlock.TopLeft[BL];

		lev = XBlock.level[ib];

		delta = calcres(T(XParam.delta), lev);

		for (int iy = 0; iy < (XParam.blkwidth + XParam.halowidth); iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
				int ibot = memloc(halowidth, blkmemwidth, ix, iy - 1, ib);

				T ybo = T(XParam.yo + XBlock.yo[ib]);


				//T dhdyi = XGrad.dhdy[i];
				//T dhdymin = XGrad.dhdy[ibot];
				T cm = XParam.spherical ? calcCM(T(XParam.Radius), delta, ybo, iy) : T(1.0);
				T fmv = XParam.spherical ? calcFM(T(XParam.Radius), delta, ybo, T(iy)) : T(1.0);

				T hi = XEv.h[i];

				T hn = XEv.h[ibot];


				if (hi > eps || hn > eps)
				{
					T dx, zi, zn, hr, hl, etar, etal, zr, zl, zA, zCN, hCNr, hCNl;
					T ui, vi, uli, vli, dhdyi, dhdyil, dudyi, dudyil, dvdyi, dvdyil;

					T ga = g * T(0.5);
					// along X
					dx = delta * T(0.5);
					zi = zb[i];
					zn = zb[ibot];

					ui = XEv.u[i];
					vi = XEv.v[i];
					uli = XEv.u[ibot];
					vli = XEv.v[ibot];

					dhdyi = XGrad.dhdy[i];
					dhdyil = XGrad.dhdy[ibot];
					dudyi = XGrad.dudy[i];
					dudyil = XGrad.dudy[ibot];
					dvdyi = XGrad.dvdy[i];
					dvdyil = XGrad.dvdy[ibot];


					hr = hi - dx * dhdyi;
					hl = hn + dx * dhdyil;
					etar = XEv.zs[i] - dx * XGrad.dzsdy[i];
					etal = XEv.zs[ibot] + dx * XGrad.dzsdy[ibot];

					//define the topography term at the interfaces
					zr = etar - hr;// zi - dx * XGrad.dzbdy[i];
					zl = etal - hl;// zn + dx * XGrad.dzbdy[ibot];

					//define the Audusse terms
					zA = max(zr, zl);

					// Now the CN terms
					zCN = min(zA, min(etal, etar));
					hCNr = max(T(0.0), min(etar - zCN, hr));
					hCNl = max(T(0.0), min(etal - zCN, hl));

					//Velocity reconstruction
					//To avoid high velocities near dry cells, we reconstruct velocities according to Bouchut.
					T ul, ur, vl, vr, sl, sr;
					if (hi > eps) {
						ur = ui - (1. + dx * dhdyi / hi) * dx * dudyi;
						vr = vi - (1. + dx * dhdyi / hi) * dx * dvdyi;
					}
					else {
						ur = ui - dx * dudyi;
						vr = vi - dx * dvdyi;
					}
					if (hn > eps) {
						ul = uli + (1. - dx * dhdyil / hn) * dx * dudyil;
						vl = vli + (1. - dx * dhdyil / hn) * dx * dvdyil;
					}
					else {
						ul = uli + dx * dudyil;
						vl = vli + dx * dvdyil;
					}




					T fh, fu, fv, dt;


					//solver below also modifies fh and fu
					dt = hllc(g, delta, epsi, CFL, cm, fmv, hCNl, hCNr, vl, vr, fh, fu);
					//hllc(T g, T delta, T epsi, T CFL, T cm, T fm, T hm, T hp, T um, T up, T & fh, T & fq)

					if (dt < dtmax[i])
					{
						dtmax[i] = dt;
					}
					

					fv = (fh > 0. ? ul : ur) * fh;


					// Topographic source term

					// In the case of adaptive refinement, care must be taken to ensure
					// well-balancing at coarse/fine faces (see [notes/balanced.tm]()). 
					if ((iy == XParam.blkwidth) && levTL < lev)//(ix==16) i.e. in the top halo
					{
						int jj = BLTL == ib ? ftoi(floor(ix * (T)0.5)) : ftoi(floor(ix * (T)0.5) + XParam.blkwidth / 2);
						int itop = memloc(halowidth, blkmemwidth, jj, 0, TL);
						hi = XEv.h[itop];
						zi = zb[itop];
					}
					if ((iy == 0) && levBL < lev)//(ix==16) i.e. in the bot halo
					{
						int jj = TLBL == ib ? ftoi(floor(ix * (T)0.5)) : ftoi(floor(ix * (T)0.5) + XParam.blkwidth / 2);
						int ibc = memloc(halowidth, blkmemwidth, jj, XParam.blkwidth - 1, BL);
						// Warning I think the above is wrong and should be as below to be consistent with halo flux scheme:
						//int ibc = memloc(halowidth, blkmemwidth, jj, XParam.blkwidth, BL);
						hn = XEv.h[ibc];
						zn = zb[ibc];
					}

					sl = ga * (hi + hCNr) * (zi - zCN);
					sr = ga * (hCNl + hn) * (zn - zCN);

					////Flux update

					XFlux.Fhv[i] = fmv * fh;
					XFlux.Fqvy[i] = fmv * (fu - sl);
					XFlux.Sv[i] = fmv * (fu - sr);
					XFlux.Fquy[i] = fmv * fv;
				}
				else
				{
					dtmax[i] = T(1.0) / epsi;
					XFlux.Fhv[i] = T(0.0);
					XFlux.Fqvy[i] = T(0.0);
					XFlux.Sv[i] = T(0.0);
					XFlux.Fquy[i] = T(0.0);
				}
			}
		}
	}
}
template __host__ void UpdateButtingerYCPU(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax, float* zb);
template __host__ void UpdateButtingerYCPU(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax, double* zb);




/*! \fn T hllc(T g, T delta, T epsi, T CFL, T cm, T fm, T hm, T hp, T um, T up, T &fh, T &fq)
* \brief Calculate the  Harten-Lax-van Leer-contact (HLLC) flux.
*
* ## Description
* This an implementation of the HLLC solver. 
* 
*
* ## Where does this come from:
* This scheme was adapted/modified from the Basilisk source code.
* http://basilisk.fr/src/riemann.h
* 
* Reference:
* (Basilisk reference the scheme from Kurganov reference below)
* Kurganov, A., & Levy, D. (2002). Central-upwind schemes for the
*    Saint-Venant system. Mathematical Modelling and Numerical
*    Analysis, 36(3), 397-425.
*
*/
template <class T> __host__ __device__ T hllc(T g, T delta, T epsi, T CFL, T cm, T fm, T hm, T hp, T um, T up, T &fh, T &fq)
{
	T cp, cmo , dt, ustar, cstar, SL, SR, fhm, fum,fhp, fup,dlt;
	cmo = sqrt(g * hm);
	cp = sqrt(g * hp);
	ustar = (um + up) / T(2.) + cmo - cp;
	cstar = (cmo + cp) / T(2.) + (um - up) / T(4.);
	SL = hm == T(0.) ? up - T(2.) * cp : min(um - cmo, ustar - cstar);
	SR = hp == T(0.) ? um + T(2.) * cmo : max(up + cp, ustar + cstar);

	if (T(0.) <= SL) {
		fh = um * hm;
		fq = hm * (um * um + g * hm / T(2.));
	}
	else if (T(0.) >= SR) {
		fh = up * hp;
		fq = hp * (up * up + g * hp / T(2.));
	}
	else {
		fhm = um * hm;
		fum = hm * (um * um + g * hm / T(2.));
		fhp = up * hp;
		fup = hp * (up * up + g * hp / T(2.));
		fh = (SR * fhm - SL * fhp + SL * SR * (hp - hm)) / (SR - SL);
		fq = (SR * fum - SL * fup + SL * SR * (hp * up - hm * um)) / (SR - SL);
	}

	double a = max(fabs(SL), fabs(SR));
	if (a > epsi) {
		dlt = delta * cm / fm;
		dt = CFL * dlt / a;
		
	}
	else
	{
		dt = T(1.0) / epsi;
	}
	return dt;
}

