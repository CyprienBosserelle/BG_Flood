#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//                                                                              //
//Copyright (C) 2018 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////


#include "InitEvolv.h"

template <class T> void initevolv(Param XParam, BlockP<T> XBlock,Forcing<float> XForcing, EvolvingP<T> &XEv,T* &zb)
{
	//move this to a subroutine
	int hotstartsucess = 0;
	if (!XParam.hotstartfile.empty())
	{
		// hotstart
		log("\tHotstart file used : " + XParam.hotstartfile);

		hotstartsucess = readhotstartfile(XParam, XBlock, XEv, zb);

		//add offset if present
		if (!std::isnan(XParam.zsoffset)) // apply specified zsoffset
		{
			printf("\t\tadd offset to zs and hh... ");
			//
			AddZSoffset(XParam, XBlock, XEv, zb);

		}


		if (hotstartsucess == 0)
		{
			printf("\t\tFailed...  ");
			write_text_to_log_file("\tHotstart failed switching to cold start");
		}
	}


	
	if (XParam.hotstartfile.empty() || hotstartsucess == 0)
	{
		//printf("Cold start  ");
		//log("Cold start");
		//Cold start
		// 2 options:
		//		(1) if zsinit is set, then apply zsinit everywhere
		//		(2) zsinit is not set so interpolate from boundaries. (if no boundaries were specified set zsinit to zeros and apply case (1))

		//Param defaultParam;
		//!leftWLbnd.empty()

		//case 0 (i.e. zsinint not specified by user and no boundaries were specified)
		
		if (std::isnan(XParam.zsinit) && (!XForcing.left.on && !XForcing.right.on && !XForcing.top.on && !XForcing.bot.on)) //zsinit is default
		{
			XParam.zsinit = 0.0; // better default value than nan
		}
		
		//case 1 cold start
		
		if (!std::isnan(XParam.zsinit)) // apply specified zsinit
		{
			log("\tCold start");
			int coldstartsucess = 0;
			coldstartsucess = coldstart(XParam, XBlock, zb, XEv);
			
		}
		// case 2 warm start
		else // lukewarm start i.e. inv. dist interpolation of zs at bnds // Argggh!
		{
			log("\tWarm start");
			warmstart(XParam, XForcing, XBlock, zb, XEv);
			
		}// end else
		
	}
}
template void initevolv<float>(Param XParam, BlockP<float> XBlock, Forcing<float> XForcing, EvolvingP<float> &XEv, float* &zb);
template void initevolv<double>(Param XParam, BlockP< double > XBlock, Forcing<float> XForcing, EvolvingP< double > &XEv, double* &zb);


template <class T>
int coldstart(Param XParam, BlockP<T> XBlock, T* zb, EvolvingP<T> & XEv)
{
	T zzini = std::isnan(XParam.zsinit)? T(0.0): T(XParam.zsinit);
	T zzoffset = std::isnan(XParam.zsoffset) ? T(0.0) : T(XParam.zsoffset);
	

	
	int coldstartsucess = 0;
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			for (int i = 0; i < XParam.blkwidth; i++)
			{
				int n = (i + XParam.halowidth) + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				
				XEv.u[n] = T(0.0);
				XEv.v[n] = T(0.0);
				//zb[n] = 0.0f;
				XEv.zs[n] = utils::max(zzini + zzoffset, zb[n]);
				
				//if (i >= 64 && i < 82)
				//{
				//	zs[n] = max(zsbnd+0.2f, zb[i + j*nx]);
				//}
				XEv.h[n] = utils::max(XEv.zs[n] - zb[n], T(0.0));//0.0 or XParam.eps ??
			}
		}
	}
	
	coldstartsucess = 1;
	return coldstartsucess = 1;
}


template <class T>
void warmstart(Param XParam,Forcing<float> XForcing, BlockP<T> XBlock, T* zb, EvolvingP<T>& XEv)
{
	// This function read water level boundary if they have been setup and calculate the distance to the boundary 
	// toward the end the water level value is calculated as an inverse distance to the available boundaries.
	// While this may look convoluted its working quite simply.
	// look for each boundary side and calculate the closest water level value and the distance to that value

	double zsleft = 0.0;
	double zsright = 0.0;
	double zstop = 0.0;
	double zsbot = 0.0;
	T zsbnd = 0.0;

	double distleft, distright, disttop, distbot;

	double lefthere = 0.0;
	double righthere = 0.0;
	double tophere = 0.0;
	double bothere = 0.0;

	double xi, yi, jj, ii;
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			for (int i = 0; i < XParam.blkwidth; i++)
			{
				int n = (i + XParam.halowidth) + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;

				double levdx = calcres(XParam.dx, XBlock.level[ib]);
				xi = XParam.xo + XBlock.xo[ib] + i * levdx;
				yi = XParam.yo + XBlock.yo[ib] + j * levdx;

				disttop = max((XParam.ymax - yi) / levdx, 0.1);//max((double)(ny - 1) - j, 0.1);// WTF is that 0.1? // distleft cannot be 0 //theoretical minumun is 0.5?
				distbot = max((yi - XParam.yo) / levdx, 0.1);
				distleft = max((xi - XParam.xo) / levdx, 0.1);//max((double)i, 0.1);
				distright = max((XParam.xmax - xi) / levdx, 0.1);//max((double)(nx - 1) - i, 0.1);

				jj = (yi - XParam.yo) / (XParam.ymax - XParam.yo);
				ii = (xi - XParam.xo) / (XParam.xmax - XParam.xo);

				if (XForcing.left.on)
				{
					lefthere = 1.0;
					int SLstepinbnd = 1;



					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = XForcing.left.data[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = XForcing.left.data[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int k = 0; k < XForcing.left.data[SLstepinbnd].wlevs.size(); k++)
					{
						zsbndvec.push_back(interptime(XForcing.left.data[SLstepinbnd].wlevs[k], XForcing.left.data[SLstepinbnd - 1].wlevs[k], XForcing.left.data[SLstepinbnd].time - XForcing.left.data[SLstepinbnd - 1].time, XParam.totaltime - XForcing.left.data[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsleft = zsbndvec[0];
					}
					else
					{
						int iprev = utils::min(utils::max((int)floor(jj * (zsbndvec.size() - 1)), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsleft = interptime(zsbndvec[inext], zsbndvec[iprev], 1.0, (double)(jj * (zsbndvec.size() - 1) - iprev));
					}

				}

				if (XForcing.right.on)
				{
					int SLstepinbnd = 1;
					righthere = 1.0;


					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = XForcing.right.data[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = XForcing.right.data[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int k = 0; k < XForcing.right.data[SLstepinbnd].wlevs.size(); k++)
					{
						zsbndvec.push_back(interptime(XForcing.right.data[SLstepinbnd].wlevs[k], XForcing.right.data[SLstepinbnd - 1].wlevs[k], XForcing.right.data[SLstepinbnd].time - XForcing.right.data[SLstepinbnd - 1].time, XParam.totaltime - XForcing.right.data[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsright = zsbndvec[0];
					}
					else
					{
						int iprev = utils::min(utils::max((int)floor(jj * (zsbndvec.size() - 1)), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsright = interptime(zsbndvec[inext], zsbndvec[iprev], 1.0, (double)(jj * (zsbndvec.size() - 1) - iprev));
					}


				}
				if (XForcing.bot.on)
				{
					int SLstepinbnd = 1;
					bothere = 1.0;




					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = XForcing.bot.data[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = XForcing.bot.data[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int k = 0; k < XForcing.bot.data[SLstepinbnd].wlevs.size(); k++)
					{
						zsbndvec.push_back(interptime(XForcing.bot.data[SLstepinbnd].wlevs[k], XForcing.bot.data[SLstepinbnd - 1].wlevs[k], XForcing.bot.data[SLstepinbnd].time - XForcing.bot.data[SLstepinbnd - 1].time, XParam.totaltime - XForcing.bot.data[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsbot = zsbndvec[0];
					}
					else
					{
						int iprev = utils::min(utils::max((int)floor(ii * (zsbndvec.size() - 1)), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsbot = interptime(zsbndvec[inext], zsbndvec[iprev], 1.0, (double)(ii * (zsbndvec.size() - 1) - iprev));
					}

				}
				if (XForcing.top.on)
				{
					int SLstepinbnd = 1;
					tophere = 1.0;




					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = XForcing.top.data[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = XForcing.top.data[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int k= 0; k < XForcing.top.data[SLstepinbnd].wlevs.size(); k++)
					{
						zsbndvec.push_back(interptime(XForcing.top.data[SLstepinbnd].wlevs[k], XForcing.top.data[SLstepinbnd - 1].wlevs[k], XForcing.top.data[SLstepinbnd].time - XForcing.top.data[SLstepinbnd - 1].time, XParam.totaltime - XForcing.top.data[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zstop = zsbndvec[0];
					}
					else
					{
						int iprev = utils::min(utils::max((int)floor(ii * (zsbndvec.size() - 1)), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zstop = interptime(zsbndvec[inext], zsbndvec[iprev], 1.0, (double)(ii * (zsbndvec.size() - 1) - iprev));
					}

				}


				zsbnd = ((zsleft / distleft) * lefthere + (zsright / distright) * righthere + (zstop / disttop) * tophere + (zsbot / distbot) * bothere) / ((1.0 / distleft) * lefthere + (1.0 / distright) * righthere + (1.0 / disttop) * tophere + (1.0 / distbot) * bothere);



				XEv.zs[n] = utils::max(zsbnd, zb[n]);
				XEv.h[n] = utils::max(XEv.zs[n] - zb[n], T(0.0));
				XEv.u[n] = T(0.0);
				XEv.v[n] = T(0.0);



			}
		}
	}
}


template <class T>
int AddZSoffset(Param XParam, BlockP<T> XBlock, EvolvingP<T> &XEv, T*zb)
{
	int success = 1;
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			for (int i = 0; i < XParam.blkwidth; i++)
			{
				int n = memloc(XParam, i, j, ib);

				if (XEv.h[n] > XParam.eps)
				{

					XEv.zs[n] = max(XEv.zs[n] + T(XParam.zsoffset), zb[n]);

					XEv.h[n] = utils::max(XEv.zs[n] - zb[n], T(0.0));
				}
			}

		}
	}

	return success;
}


template <class T>
int readhotstartfile(Param XParam, BlockP<T> XBlock, EvolvingP<T>& XEv, T*& zb)
{
	int status;
	int ncid, varid, ndims;
	//int dimids[NC_MAX_VAR_DIMS];   // dimension IDs 
	int ib;
	double scalefac = 1.0;
	double offset = 0.0;
	
	std::string zbname, zsname, hname, uname, vname, xname, yname;
	// Open the file for read access
	//netCDF::NcFile dataFile(XParam.hotstartfile, NcFile::read);


	//Open NC file
	printf("Open file...");
	status = nc_open(XParam.hotstartfile.c_str(), NC_NOWRITE, &ncid);
	if (status != NC_NOERR) handle_ncerror(status);
	zbname = checkncvarname(ncid, "zb", "z", "ZB", "Z", "zb_P0");
	zsname = checkncvarname(ncid, "zs", "eta", "ZS", "ETA", "zs_P0");
	hname = checkncvarname(ncid, "h", "hh", "hhh", "hhhh", "h_P0");
	uname = checkncvarname(ncid, "u", "uu", "uvel", "UVEL", "u_P0");
	vname = checkncvarname(ncid, "v", "vv", "vvel", "VVEL", "v_P0");

	//by default we assume that the x axis is called "xx" but that is not sure "x" shoudl be accepted and so does "lon" for spherical grid
	// The folowing section figure out which one is in the file and if none exits with the netcdf error
	// default name is "xx"
	//xname = checkncvarname(ncid, "x", "xx","lon","Lon");
	//yname = checkncvarname(ncid, "y", "yy", "lat", "Lat");

	status = nc_close(ncid);


	// First we should read x and y coordinates
	// Just as with other variables we expect the file follow the output naming convention of "xx" and "yy" both as a dimension and a variable
	StaticForcingP<float> zbhotstart, zshotstart, hhotstart, uhotstart, vhotstart;

	// Read hotstart block info if it exist
	// By default reuse mesh-layout
	// for now we pretend hotstart are just unifomr maesh layout



	//if hotstart has zb variable overright the previous ne
	//printf("Found variables: ");
	if (!zbname.empty())
	{
		//zb is set
		zbhotstart = readfileinfo(XParam.hotstartfile + "?" + zbname, zbhotstart);

		readstaticforcing(XParam.hotstep, zbhotstart);
		interp2BUQ(XParam, XBlock, zbhotstart, zb);

		//because we set the edges around empty blocks we need the set the edges for zs too
		// otherwise we create some gitantic waves at the edges of empty blocks
		setedges(XParam, XBlock, zb);



	}
	// second check if zs or hh are in teh file


	//zs Section
	if (!zsname.empty())
	{
		log(" zs... ");

		zshotstart = readfileinfo(XParam.hotstartfile + "?" + zsname, zshotstart);
		//readforcingmaphead(zshotstart);
		readstaticforcing(XParam.hotstep, zshotstart);

		interp2BUQ(XParam, XBlock, zshotstart, XEv.zs);

		setedges(XParam, XBlock, XEv.zs);

		//setedges(XParam.nblk, leftblk, rightblk, topblk, botblk, zs);

		//check sanity
		for (int ibl = 0; ibl < XParam.nblk; ibl++)
		{
			ib = XBlock.active[ibl];
			for (int j = 0; j < XParam.blkwidth; j++)
			{
				for (int i = 0; i < XParam.blkwidth; i++)
				{
					int n = (i + XParam.halowidth) + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
					XEv.zs[n] = utils::max(XEv.zs[n], zb[n]);
					//unpacked_value = packed_value * scale_factor + add_offset
				}
			}
		}


	}
	else
	{
		//Variable not found
		//It's ok if hh is specified
		log("zs not found in hotstart file. Looking for hh... ");

	}

	//hh section
	if (!hname.empty())
	{
		log("h... ");
		hhotstart = readfileinfo(XParam.hotstartfile + "?" + hname, hhotstart);
		//readforcingmaphead(zshotstart);
		readstaticforcing(XParam.hotstep, hhotstart);

		interp2BUQ(XParam, XBlock, hhotstart, XEv.h);

		setedges(XParam, XBlock, XEv.h);

		//if zs was not specified
		if (zsname.empty())
		{
			for (int ibl = 0; ibl < XParam.nblk; ibl++)
			{
				ib = XBlock.active[ibl];
				for (int j = 0; j < XParam.blkwidth; j++)
				{
					for (int i = 0; i < XParam.blkwidth; i++)
					{
						int n = (i + XParam.halowidth) + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
						XEv.zs[n] = zb[n] + XEv.h[n];
						//unpacked_value = packed_value * scale_factor + add_offset
					}
				}
			}

		}
		


	}
	else
	{
		//if both zs and h were not specified
		if (zsname.empty() && hname.empty())
		{
			//Variable not found
			//It's ok if hh is specified
			log("neither zs nor hh were found in hotstart file. this is not a valid hotstart file. using a cold start instead");
			return 0;
		}
		else
		{
			//zs was specified but not h
			for (int ibl = 0; ibl < XParam.nblk; ibl++)
			{
				ib = XBlock.active[ibl];
				for (int j = 0; j < XParam.blkwidth; j++)
				{
					for (int i = 0; i < XParam.blkwidth; i++)
					{
						int n = memloc(XParam, i, j, ib);


						XEv.h[n] = utils::max(XEv.zs[n] - zb[n], T(0.0));
					}

				}
			}

		}
	}

	//u Section

	if (!uname.empty())
	{
		log("u... ");
		uhotstart = readfileinfo(XParam.hotstartfile + "?" + uname, uhotstart);
		//readforcingmaphead(zshotstart);
		readstaticforcing(XParam.hotstep, uhotstart);

		interp2BUQ(XParam, XBlock, uhotstart, XEv.u);

		setedges(XParam, XBlock, XEv.u);

	}
	else
	{
		InitArrayBUQ(XParam, XBlock, (T)0.0, XEv.u);
	}

	//vv section

	if (!vname.empty())
	{
		log("v... ");
		vhotstart = readfileinfo(XParam.hotstartfile + "?" + vname, vhotstart);
		//readforcingmaphead(zshotstart);
		readstaticforcing(XParam.hotstep, vhotstart);

		interp2BUQ(XParam, XBlock, vhotstart, XEv.v);

		setedges(XParam, XBlock, XEv.v);


	}
	else
	{
		InitArrayBUQ(XParam,XBlock, (T)0.0, XEv.v);
	}
	//status = nc_get_var_float(ncid, hh_id, zb);
	status = nc_close(ncid);



	return 1;

}
template int readhotstartfile<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float>& XEv, float*& zb);
template int readhotstartfile<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double>& XEv, double*& zb);
//template int readhotstartfile<float>(Param XParam, int * leftblk, int *rightblk, int * topblk, int* botblk, double * blockxo, double * blockyo, float * &zs, float * &zb, float * &hh, float *&uu, float * &vv);

//template int readhotstartfile<double>(Param XParam, int * leftblk, int *rightblk, int * topblk, int* botblk, double * blockxo, double * blockyo, double * &zs, double * &zb, double * &hh, double *&uu, double * &vv);
