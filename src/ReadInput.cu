#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//                                                                              //
//Copyright (C) 2018 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

#include "ReadInput.h"

// Collection of functions to read input to the model


/*! \fn T readfileinfo(std::string input,T outinfo)
* convert file name into name and extension
* This is used for various input classes
*
* template inputmap readfileinfo<inputmap>(std::string input, inputmap outinfo);
* template forcingmap readfileinfo<forcingmap>(std::string input, forcingmap outinfo);
* template StaticForcingP<float> readfileinfo<StaticForcingP<float>>(std::string input, StaticForcingP<float> outinfo);
* template DynForcingP<float> readfileinfo<DynForcingP<float>>(std::string input, DynForcingP<float> outinfo);
* template deformmap<float> readfileinfo<deformmap<float>>(std::string input, deformmap<float> outinfo);
*/
template <class T> T readfileinfo(std::string input, T outinfo)
{
	// Outinfo is based on an inputmap (or it's sub classes)

	//filename include the file extension

	std::vector<std::string> extvec = split(input, '.');

	//outinfo.inputfile = extvec.front();

	std::vector<std::string> nameelements, filename;
	//
	nameelements = split(extvec.back(), '?');

	filename = split(input, '?');
	if (nameelements.size() > 1)
	{
		//variable name for bathy is not given so it is assumed to be zb
		outinfo.extension = nameelements[0];
		outinfo.varname = nameelements.back();

	}
	else
	{
		outinfo.extension = extvec.back();
		outinfo.varname = "z";
	}

	//Reconstruct filename with extension but without varname
	//outinfo.inputfile = extvec.front() + "." + outinfo.extension;
	outinfo.inputfile = filename.front();

	return outinfo;
}

template inputmap readfileinfo<inputmap>(std::string input, inputmap outinfo);
template forcingmap readfileinfo<forcingmap>(std::string input, forcingmap outinfo);
template StaticForcingP<float> readfileinfo<StaticForcingP<float>>(std::string input, StaticForcingP<float> outinfo);
template DynForcingP<float> readfileinfo<DynForcingP<float>>(std::string input, DynForcingP<float> outinfo);
template deformmap<float> readfileinfo<deformmap<float>>(std::string input, deformmap<float> outinfo);




/*! \fn void Readparamfile(Param &XParam, Forcing<float> & XForcing)
* Open the BG_param.txt file and read the parameters
* save the parameter in the Param class and or Forcing class.
*/
void Readparamfile(Param& XParam, Forcing<float>& XForcing, std::string Paramfile)
{
	//
	log("\nReading parameter file: " + Paramfile + " ...");
	//std::ifstream fs("BG_param.txt");
	std::ifstream fs(Paramfile);

	if (fs.fail()) {
		//std::cerr << "BG_param.txt file could not be opened" << std::endl;
		log("ERROR: BG_param.txt file could not be opened...use this log file to create a file named BG_param.txt");
		SaveParamtolog(XParam);

		exit(1);

	}
	else
	{
		// Read and interpret each line of the BG_param.txt
		std::string line;
		while (std::getline(fs, line))
		{

			//Get param or skip empty lines
			if (!line.empty() && line.substr(0, 1).compare("#") != 0)
			{
				XParam = readparamstr(line, XParam);
				XForcing = readparamstr(line, XForcing);

				//std::cout << line << std::endl;
			}

		}
		fs.close();


	}

}





/*! \fn Param readparamstr(std::string line, Param param)
* Read BG_param.txt line and convert parameter to the righ parameter in the class
* retrun an updated Param class
*/
Param readparamstr(std::string line, Param param)
{


	std::string parameterstr, parametervalue;
	std::vector<std::string> paramvec;
	///////////////////////////////////////////////////////
	// General parameters
	//

	parameterstr = "test";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.test = std::stoi(parametervalue);
	}

	paramvec = { "GPUDEVICE","gpu" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.GPUDEVICE = std::stoi(parametervalue);
	}

	parameterstr = "doubleprecision";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.doubleprecision = std::stoi(parametervalue);
	}

	parameterstr = "engine";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		std::vector<std::string> buttingerstr = { "b","butt","buttinger","1" };
		std::size_t found;
		bool foo = false;
		for (int ii = 0; ii < buttingerstr.size(); ii++)
		{
			found = case_insensitive_compare(parametervalue, buttingerstr[ii]);// it needs to strictly compare
			if (found == 0)
			{
				param.engine = 1;
				foo = true;
			}

		}
		if (!foo)
		{
			param.engine = 2;
		}
	}
	///////////////////////////////////////////////////////
	// Adaptation
	//
	parameterstr = "maxlevel";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.maxlevel = std::stoi(parametervalue);
	}

	parameterstr = "minlevel";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.minlevel = std::stoi(parametervalue);
	}

	parameterstr = "initlevel";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.initlevel = std::stoi(parametervalue);
	}

	paramvec = { "adaptmaxiteration","maxiterationadapt" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.adaptmaxiteration = std::stoi(parametervalue);
	}

	parameterstr = "conserveElevation";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.conserveElevation = readparambool(parametervalue, param.conserveElevation);
	}

	paramvec = { "wetdryfix","reminstab","fixinstab" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{

		param.wetdryfix = readparambool(parametervalue, param.wetdryfix);

	}


	parameterstr = "membuffer";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.membuffer = std::stod(parametervalue);
	}

	///////////////////////////////////////////////////////
	// Flow parameters
	//
	parameterstr = "eps";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.eps = std::stod(parametervalue);
	}

	paramvec = { "cf","roughness","cfmap" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		if (std::any_of(parametervalue.begin(), parametervalue.end(), ::isalpha) == false) //(std::isdigit(parametervalue[0]) == true)
		{
			param.cf = std::stod(parametervalue);
		}
	}

	paramvec = { "il","Rain_il","initialloss" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		if (std::any_of(parametervalue.begin(), parametervalue.end(), ::isalpha) == false) //(std::isdigit(parametervalue[0]) == true)
		{
			param.il = std::stod(parametervalue);
			param.infiltration = true;
		}
	}

	paramvec = { "cl","Rain_cl","continuousloss" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		if (std::any_of(parametervalue.begin(), parametervalue.end(), ::isalpha) == false) //(std::isdigit(parametervalue[0]) == true)
		{
			param.cl = std::stod(parametervalue);
			param.infiltration = true;
		}
	}

	paramvec = { "VelThreshold","vthresh","vmax","velmax" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.VelThreshold = std::stod(parametervalue);
	}

	parameterstr = "Cd";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.Cd = std::stod(parametervalue);
	}

	parameterstr = "Pa2m";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.Pa2m = std::stod(parametervalue);
	}

	parameterstr = "Paref";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.Paref = std::stod(parametervalue);
	}

	parameterstr = "mask";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.mask = std::stod(parametervalue);
	}

	///////////////////////////////////////////////////////
	// Timekeeping parameters
	//
	parameterstr = "dt";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.dt = std::stod(parametervalue);

	}

	parameterstr = "dtmin";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.dtmin = std::stod(parametervalue);

	}
	parameterstr = "bndtaper";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.bndtaper = std::stod(parametervalue);

	}

	parameterstr = "CFL";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.CFL = std::stod(parametervalue);

	}
	parameterstr = "theta";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.theta = std::stod(parametervalue);

	}

	paramvec = { "outputtimestep","outtimestep","outputstep" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.outputtimestep = std::stod(parametervalue);

	}

	paramvec = { "endtime", "stoptime", "end", "stop","end_time","stop_time" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.endtime = readinputtimetxt(parametervalue, param.reftime);

	}

	paramvec = { "totaltime","inittime","starttime", "start_time", "init_time", "start", "init" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		//param.totaltime = std::stod(parametervalue);
		param.totaltime = readinputtimetxt(parametervalue, param.reftime);

	}

	parameterstr = "dtinit";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.dtinit = std::stod(parametervalue);

	}

	paramvec = { "reftime","referencetime","timeref" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		if (param.reftime.empty())
		{
			param.reftime = parametervalue;
		}

	}

	///////////////////////////////////////////////////////
	// Input and output files
	//

	parameterstr = "outfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.outfile = parametervalue;

	}


	// Below is a bit more complex than usual because more than 1 node can be outputed as a timeseries
	paramvec = { "TSnodesout","TSOutput" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		TSoutnode node;
		std::vector<std::string> nodeitems = split(parametervalue, ',');
		if (nodeitems.size() >= 3)
		{
			node.outname = nodeitems[0];
			node.x = std::stod(nodeitems[1]);
			node.y = std::stod(nodeitems[2]);

			param.TSnodesout.push_back(node);
		}
		else
		{
			std::cerr << "Node input failed there should be 3 arguments (comma separated) when inputing a outout node: TSOutput = filename, xvalue, yvalue; see log file for details" << std::endl;

			log("Node input failed there should be 3 arguments (comma separated) when inputing a outout node: TSOutput = filename, xvalue, yvalue; see log file for details. Input was: " + parametervalue);

		}

	}



	//outvars
	parameterstr = "outvars";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		std::vector<std::string> vars = split(parametervalue, ',');
		for (int nv = 0; nv < vars.size(); nv++)
		{
			//Verify that the variable name makes sense?
			//Need to add more here

			std::vector<std::string> SupportedVarNames = { "zb","zs","u","v","h","hmean","zsmean","umean","vmean","hUmean","Umean","hmax","zsmax","umax","vmax","hUmax","Umax","twet","dhdx","dhdy","dzsdx","dzsdy","dzbdx","dzbdy","dudx","dudy","dvdx","dvdy","Fhu","Fhv","Fqux","Fqvy","Fquy","Fqvx","Su","Sv","dh","dhu","dhv","cf","Patm","datmpdx","datmpdy","il","cl","hgw" };

			std::string vvar = trim(vars[nv], " ");
			for (int isup = 0; isup < SupportedVarNames.size(); isup++)
			{

				//std::cout << "..." << vvar << "..." << std::endl;
				if (vvar.compare(SupportedVarNames[isup]) == 0)
				{
					param.outvars.push_back(vvar);
					break;
				}

			}

			param.outmean = (vvar.compare("hmean") == 0) ? true : param.outmean;
			param.outmean = (vvar.compare("zsmean") == 0) ? true : param.outmean;
			param.outmean = (vvar.compare("umean") == 0) ? true : param.outmean;
			param.outmean = (vvar.compare("vmean") == 0) ? true : param.outmean;
			param.outmean = (vvar.compare("Umean") == 0) ? true : param.outmean;
			param.outmean = (vvar.compare("hUmean") == 0) ? true : param.outmean;

			param.outmax = (vvar.compare("hmax") == 0) ? true : param.outmax;
			param.outmax = (vvar.compare("zsmax") == 0) ? true : param.outmax;
			param.outmax = (vvar.compare("umax") == 0) ? true : param.outmax;
			param.outmax = (vvar.compare("vmax") == 0) ? true : param.outmax;
			param.outmax = (vvar.compare("Umax") == 0) ? true : param.outmax;
			param.outmax = (vvar.compare("hUmax") == 0) ? true : param.outmax;

			param.outtwet = (vvar.compare("twet") == 0) ? true : param.outtwet;

			//param.outvort = (vvar.compare("vort") == 0) ? true : param.outvort;
			//param.outU = (vvar.compare("U") == 0) ? true : param.outU;
		}



	}


	// Same as for TSnodesout, the same key word can be used for different zones Output
	parameterstr = "outzone";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		outzoneP zone;
		std::vector<std::string> zoneitems = split(parametervalue, ',');
		if (zoneitems.size() >= 5)
		{
			zone.outname = zoneitems[0];
			zone.xstart = std::stod(zoneitems[1]);
			zone.xend = std::stod(zoneitems[2]);
			zone.ystart = std::stod(zoneitems[3]);
			zone.yend = std::stod(zoneitems[4]);
		}
		if (zoneitems.size() > 5)
		{
			std::vector<std::string> Toutputpar_vect = split_full(zoneitems[5], ':');
			if (Toutputpar_vect.size() == 3)
			{
				double init, tstep, end;
				double tiny = 0.000001;
				if (!Toutputpar_vect[0].empty()) {
					init = std::stod(Toutputpar_vect[0]);
				}
				if (!Toutputpar_vect[1].empty()) {
					tstep = std::max(std::stod(Toutputpar_vect[1]),tiny);
				}
				if (!Toutputpar_vect[2].empty()) {
					end = std::stod(Toutputpar_vect[2]);
				}

				int nstep = (end - init) / tstep + 1;

				for (int k = 0; k < nstep; k++)
				{
					zone.Toutput.val.push_back(std::min(init + tstep * k,end));
				}

			}
			else if (Toutputpar_vect.size() > 1)
			{
				//Failed: Toutput must be exactly 3 values, separated by ":" for a vector shape, in virst position. "t_init:t_step:t_end" (with possible empty values as "t_init:t_setps: " to use the last time steps as t_end;
				std::cerr << "Failed: Toutput must be exactly 3 values, separated by ':' for a vector shape, in first position. 't_init : t_step : t_end' (with possible empty values as 't_init : t_setps : ' to use the last time steps as t_end; see log file for details" << std::endl;

				log("Failed: Toutput must be exactly 3 values, separated by ':' for a vector shape, in virst position. 't_init : t_step : t_end' (with possible empty values as 't_init : t_setps : ' to use the last time steps as t_end;");
				log(parametervalue);
			}
			else { //only values
				zone.Toutput.val.push_back(std::stod(Toutputpar_vect[0]));
			}
			if (zoneitems.size() > 6) //vector + values
			{
				for (int ii = 6; ii < zoneitems.size(); ii++)
				{
					zone.Toutput.val.push_back(std::stod(zoneitems[ii]));
				}
			}
		}
		else if (zoneitems.size() == 5)//No time input in the zone area
		{
			zone.Toutput = param.Toutput;
		}
		else
		{
			std::cerr << "Zone input failed there should be at least 5 arguments (comma separated) when inputing a outout zone: outzone = filename, xstart, xend, ystart, yend; see log file for details" << std::endl;
			log("Node input failed there should be at least 5 arguments (comma separated) when inputing a outout zone: outzone = filename, xstart, xend, ystart, yend; see log file for details (with possibly some time inputs after). Input was: " + parametervalue);
		}
		param.outzone.push_back(zone);
	}

	parameterstr = "resetmax";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		if (std::stoi(parametervalue) == 1)
		{
			param.resetmax = true;
		}
	}

	// WARNING FOR DEBUGGING PURPOSE ONLY
	// For debugging one can shift the output by 1 or -1 in the i and j direction.
	// this will save the value in the halo to the output file allowing debugging of values there.
	parameterstr = "outishift";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.outishift = std::stoi(parametervalue);
	}
	parameterstr = "outjshift";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.outjshift = std::stoi(parametervalue);
	}

	////////////////////////////////////////////////////////////////


	parameterstr = "nx";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.nx = std::stoi(parametervalue);
	}

	parameterstr = "ny";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.ny = std::stoi(parametervalue);
	}

	parameterstr = "dx";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.dx = std::stod(parametervalue);
	}

	parameterstr = "grdalpha";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.grdalpha = std::stod(parametervalue);
	}

	paramvec = { "xo","xmin" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.xo = std::stod(parametervalue);
	}

	paramvec = { "yo","ymin" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.yo = std::stod(parametervalue);
	}

	parameterstr = "xmax";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.xmax = std::stod(parametervalue);
	}

	parameterstr = "ymax";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.ymax = std::stod(parametervalue);
	}

	parameterstr = "g";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.g = std::stod(parametervalue);

	}

	parameterstr = "rho";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.rho = std::stod(parametervalue);
	}

	parameterstr = "smallnc";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.smallnc = std::stoi(parametervalue);
	}
	parameterstr = "scalefactor";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.scalefactor = std::stof(parametervalue);
	}
	parameterstr = "addoffset";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.addoffset = std::stof(parametervalue);
	}
	parameterstr = "posdown";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.posdown = std::stoi(parametervalue);
	}

#ifdef USE_CATALYST
	parameterstr = "use_catalyst";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.use_catalyst = std::stoi(parametervalue);
	}
	parameterstr = "catalyst_python_pipeline";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.catalyst_python_pipeline = std::stoi(parametervalue);
	}
	parameterstr = "vtk_output_frequency";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.vtk_output_frequency = std::stoi(parametervalue);
	}
	parameterstr = "vtk_output_time_interval";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.vtk_output_time_interval = std::stod(parametervalue);
	}
	parameterstr = "vtk_outputfile_root";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.vtk_outputfile_root = parametervalue;
	}
	parameterstr = "python_pipeline";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.python_pipeline = parametervalue;
	}
#endif

	paramvec = { "zsinit", "initzs" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.zsinit = std::stod(parametervalue);
	}

	parameterstr = "zsoffset";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.zsoffset = std::stod(parametervalue);
	}
	paramvec = { "rainbnd", "rainonbnd" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.rainbnd = readparambool(parametervalue, param.rainbnd);

	}


	parameterstr = "hotstartfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.hotstartfile = parametervalue;

	}

	parameterstr = "hotstep";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.hotstep = std::stoi(parametervalue);
	}


	paramvec = { "spherical", "geo" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.spherical = readparambool(parametervalue, param.spherical);
	}

	parameterstr = "Radius";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.Radius = std::stod(parametervalue);
	}

	parameterstr = "frictionmodel";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		param.frictionmodel = std::stoi(parametervalue);
	}

	parameterstr = "Adaptation";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		std::vector<std::string> adaptpar = split(parametervalue, ',');

		if (!adaptpar.empty())
		{
			param.AdaptCrit = adaptpar[0];
			if (adaptpar.size() > 1)
				param.Adapt_arg1 = adaptpar[1];
			if (adaptpar.size() > 2)
				param.Adapt_arg2 = adaptpar[2];
			if (adaptpar.size() > 3)
				param.Adapt_arg3 = adaptpar[3];
			if (adaptpar.size() > 4)
				param.Adapt_arg4 = adaptpar[4];
			if (adaptpar.size() > 5)
				param.Adapt_arg5 = adaptpar[5];
		}
	}

	paramvec = { "crs", "spatialref", "spatial_ref", "wtk", "crsinfo","crs_info" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.crs_ref = parametervalue;
	}

	//Read Flexible Toutput variable
	parameterstr = "Toutput";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		std::vector<std::string> Toutputpar = split(parametervalue, ',');

		if (!Toutputpar.empty())
		{
			std::vector<std::string> Toutputpar_vect = split_full(Toutputpar[0], ':');
			if (Toutputpar_vect.size() == 3)
			{
				/*
				if (!Toutputpar_vect[0].empty()) {
					param.Toutput.init = std::stod(Toutputpar_vect[0]);
				}
				if (!Toutputpar_vect[1].empty()) {
					param.Toutput.tstep = std::stod(Toutputpar_vect[1]);
				}
				if (!Toutputpar_vect[2].empty()) {
					param.Toutput.end = std::stod(Toutputpar_vect[2]);
				}
				*/
				double init, tstep, end;
				double tiny = 0.000001;
				if (!Toutputpar_vect[0].empty()) {
					init = std::stod(Toutputpar_vect[0]);
				}
				if (!Toutputpar_vect[1].empty()) {
					tstep = std::max(std::stod(Toutputpar_vect[1]), tiny);
				}
				if (!Toutputpar_vect[2].empty()) {
					end = std::stod(Toutputpar_vect[2]);
				}

				int nstep = (end - init) / tstep + 1;

				for (int k = 0; k < nstep; k++)
				{
					param.Toutput.val.push_back(std::min(init + tstep * k, end));
				}

			}
			else if (Toutputpar_vect.size() > 1)
			{
				//Failed: Toutput must be exactly 3 values, separated by ":" for a vector shape, in virst position. "t_init:t_step:t_end" (with possible empty values as "t_init:t_setps: " to use the last time steps as t_end;
				std::cerr << "Failed: Toutput must be exactly 3 values, separated by ':' for a vector shape, in virst position. 't_init : t_step : t_end' (with possible empty values as 't_init : t_setps : ' to use the last time steps as t_end; see log file for details" << std::endl;

				log("Failed: Toutput must be exactly 3 values, separated by ':' for a vector shape, in virst position. 't_init : t_step : t_end' (with possible empty values as 't_init : t_setps : ' to use the last time steps as t_end;");
				log(parametervalue);
			}
			else {
				param.Toutput.val.push_back(std::stod(Toutputpar_vect[0]));
			}
			if (Toutputpar.size() > 1)
			{
				for (int ii = 1; ii < Toutputpar.size(); ii++)
				{
					param.Toutput.val.push_back(std::stod(Toutputpar[ii]));
				}
			}
		}
	}

	paramvec = { "savebyblk", "writebyblk","saveperblk", "writeperblk","savebyblock", "writebyblock","saveperblock", "writeperblock" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		param.savebyblk = readparambool(parametervalue, param.savebyblk);
	}

	return param;
}



/*! \fn Forcing<T> readparamstr(std::string line, Forcing<T> forcing)
* Read BG_param.txt line and convert parameter to the righ parameter in the class
* return an updated Param class
*/
template <class T>
Forcing<T> readparamstr(std::string line, Forcing<T> forcing)
{
	std::string parameterstr, parametervalue;
	std::vector<std::string> paramvec;

	paramvec = { "Bathy","bathyfile","bathymetry","depfile","depthfile","topofile","topo","DEM" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		StaticForcingP<float> infobathy;
		forcing.Bathy.push_back(readfileinfo(parametervalue, infobathy));
		//std::cerr << "Bathymetry file found!" << std::endl;
	}



	paramvec = { "AOI","aoipoly" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		forcing.AOI.file = parametervalue;
		forcing.AOI.active = true;
	}

	/*parameterstr = "bathyfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		StaticForcingP<float> infobathy;
		forcing.Bathy.push_back(readfileinfo(parametervalue, infobathy));
		//forcing.Bathy = readfileinfo(parametervalue, forcing.Bathy);
		//std::cerr << "Bathymetry file found!" << std::endl;
	}

	parameterstr = "bathymetry";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		StaticForcingP<float> infobathy;
		forcing.Bathy.push_back(readfileinfo(parametervalue, infobathy));
		//forcing.Bathy = readfileinfo(parametervalue, forcing.Bathy);
		//std::cerr << "Bathymetry file found!" << std::endl;
	}

	//
	parameterstr = "depfile";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		StaticForcingP<float> infobathy;
		forcing.Bathy.push_back(readfileinfo(parametervalue, infobathy));
		//forcing.Bathy = readfileinfo(parametervalue, forcing.Bathy);
	}*/


	// Boundaries

	paramvec = { "left","leftbndfile","leftbnd" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		//forcing.left = readbndline(parametervalue);
		forcing.bndseg.push_back(readbndlineside(parametervalue, "left"));



	}

	paramvec = { "right","rightbndfile","rightbnd" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		//forcing.right = readbndline(parametervalue);
		forcing.bndseg.push_back(readbndlineside(parametervalue, "right"));

	}

	paramvec = { "top","topbndfile","topbnd" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		//forcing.top = readbndline(parametervalue);
		forcing.bndseg.push_back(readbndlineside(parametervalue, "top"));
	}

	paramvec = { "bot","botbndfile","botbnd","bottom" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		//forcing.bot = readbndline(parametervalue);
		forcing.bndseg.push_back(readbndlineside(parametervalue, "bot"));
	}

	paramvec = { "bnd","bndseg" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		//forcing.bot = readbndline(parametervalue);
		forcing.bndseg.push_back(readbndline(parametervalue));
	}


	//Tsunami deformation input files
	parameterstr = "deform";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{

		deformmap<float> thisdeform;
		std::vector<std::string> items = split(parametervalue, ',');
		//Need sanity check here
		thisdeform = readfileinfo(items[0], thisdeform);
		//thisdeform.inputfile = items[0];
		if (items.size() > 1)
		{
			thisdeform.startime = std::stod(items[1]);

		}
		if (items.size() > 2)
		{
			thisdeform.duration = std::stod(items[2]);

		}

		forcing.deform.push_back(thisdeform);

	}

	//Tsunami deformation input files
	parameterstr = "cavity";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{

		deformmap<float> thisdeform;

		thisdeform.iscavity = true;
		std::vector<std::string> items = split(parametervalue, ',');
		//Need sanity check here
		thisdeform = readfileinfo(items[0], thisdeform);
		//thisdeform.inputfile = items[0];
		if (items.size() > 1)
		{
			thisdeform.startime = std::stod(items[1]);

		}
		if (items.size() > 2)
		{
			thisdeform.duration = std::stod(items[2]);

		}

		forcing.deform.push_back(thisdeform);

	}

	//River
	paramvec = { "rivers","river" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		std::vector<std::string> vars = split(parametervalue, ',');
		if (vars.size() == 5)
		{
			River thisriver;
			thisriver.Riverflowfile = trim(vars[0], " ");
			thisriver.xstart = std::stod(vars[1]);
			thisriver.xend = std::stod(vars[2]);
			thisriver.ystart = std::stod(vars[3]);
			thisriver.yend = std::stod(vars[4]);

			forcing.rivers.push_back(thisriver);
		}
		else
		{
			//Failed there should be 5 arguments (comma separated) when inputing a river: filename, xstart,xend,ystart,yend;
			std::cerr << "River input failed there should be 5 arguments (comma separated) when inputing a river: river = filename, xstart,xend,ystart,yend; see log file for details" << std::endl;

			log("River input below failed there should be 5 arguments (comma separated) when inputing a river: river = filename, xstart,xend,ystart,yend;");
			log(parametervalue);
		}
	}

	// friction coefficient (mapped or constant)
	// if it is a constant no-need to do anything below but if it is a file it overwrites any other value
	paramvec = { "cf","roughness","cfmap" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		if (std::any_of(parametervalue.begin(), parametervalue.end(), ::isalpha)) //(std::isdigit(parametervalue[0]) == false)
		{
			//forcing.cf = readfileinfo(parametervalue, forcing.cf);
			StaticForcingP<float> infoRoughness;
			forcing.cf.push_back(readfileinfo(parametervalue, infoRoughness));
		}
	}


	//if (!parametervalue.empty())
	//{
	//
		//std::cerr << "Bathymetry file found!" << std::endl;
	//}

	// Rain losses, initial and continuous loss
	paramvec = { "il","Rain_il","initialloss" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		if (std::any_of(parametervalue.begin(), parametervalue.end(), ::isalpha)) //(std::isdigit(parametervalue[0]) == false)
		{
			forcing.il = readfileinfo(parametervalue, forcing.il);
		}
	}
	paramvec = { "cl","Rain_cl","continuousloss" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		if (std::any_of(parametervalue.begin(), parametervalue.end(), ::isalpha)) //(std::isdigit(parametervalue[0]) == false)
		{
			forcing.cl = readfileinfo(parametervalue, forcing.cl);
		}
	}

	// wind forcing
	paramvec = { "Wind","windfiles" }; //## forcing.Wind
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{

		std::vector<std::string> vars = split(parametervalue, ',');
		if (vars.size() == 2)
		{
			// If 2 parameters (files) are given then 1st file is U wind and second is V wind.
			// This is for variable winds no rotation of the data is performed

			forcing.UWind = readfileinfo(trim(vars[0], " "), forcing.UWind);
			forcing.VWind = readfileinfo(trim(vars[1], " "), forcing.VWind);
		}
		else if (vars.size() == 1)
		{
			// if 1 parameter(file) is given then a 3 column file is expected showing time windspeed and direction
			// wind direction is rotated (later) to the grid direction (via grdalpha)
			forcing.UWind = readfileinfo(parametervalue, forcing.UWind);
			forcing.UWind.uniform = 1;

			//apply the same for Vwind? seem unecessary but need to be careful later in the code
		}
		else
		{
			//Failed there should be 5 arguments (comma separated) when inputing a river: filename, xstart,xend,ystart,yend;
			//std::cerr << "Wind input failed there should be 2 arguments (comma separated) when inputing a wind: windfiles = windfile.nc?uwind, windfile.nc?vwind; see log file for details" << std::endl;

			log("Wind input failed there should be 2 arguments(comma separated) when inputing a wind : windfiles = windfile.nc ? uwind, windfile.nc ? vwind; see log file for details");
			log(parametervalue);
		}

	}

	// atmospheric pressure forcing
	paramvec = { "Atmp","atmpfile" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		// needs to be a netcdf file 
		forcing.Atmp = readfileinfo(parametervalue, forcing.Atmp);

	}

	// rain forcing
	paramvec = { "Rain","rainfile" };
	parametervalue = findparameter(paramvec, line);
	if (!parametervalue.empty())
	{
		// netcdf file == Variable spatially
		// txt file (other than .nc) == spatially cst (txt file with 2 col time and mmm/h )
		forcing.Rain = readfileinfo(parametervalue, forcing.Rain);

		//set the expected type of input

		if (forcing.Rain.extension.compare("nc") == 0)
		{
			forcing.Rain.uniform = 0;
		}
		else
		{
			forcing.Rain.uniform = 1;
		}

	}

	parameterstr = "Adaptation";
	parametervalue = findparameter(parameterstr, line);
	if (!parametervalue.empty())
	{
		std::vector<std::string> adaptpar = split(parametervalue, ',');
		// special case for 'Targetlevel' adaptation
		if (!adaptpar.empty())
		{
			//if (adaptpar[0].compare("Targetlevel") == 0)
			if (case_insensitive_compare(adaptpar[0], std::string("Targetlevel")) == 0)
			{
				for (int ng = 1; ng < adaptpar.size(); ng++)
				{
					StaticForcingP<int> infogrid;
					forcing.targetadapt.push_back(readfileinfo(adaptpar[ng], infogrid));
				}
			}
		}

	}

	return forcing;
}


/*! \fn void checkparamsanity(Param & XParam, Forcing<float> & XForcing)
* Check the Sanity of both Param and Forcing class
* If required some parameter are infered
*/
void checkparamsanity(Param& XParam, Forcing<float>& XForcing)
{
	Param DefaultParams;

	double tiny = 0.0000001;

	// Sanity check for model levels
	int minlev = XParam.minlevel;
	int maxlev = XParam.maxlevel;

	if (minlev == -99999)
	{
		minlev = XParam.initlevel;
	}
	if (maxlev == -99999)
	{
		maxlev = XParam.initlevel;
	}

	XParam.maxlevel = utils::max(maxlev, minlev);
	XParam.minlevel = utils::min(maxlev, minlev);

	XParam.initlevel = utils::min(utils::max(XParam.minlevel, XParam.initlevel), XParam.maxlevel);

	//force double for Rain on grid cases
	if (!XForcing.Rain.inputfile.empty())
	{
		XParam.doubleprecision = 1;
	}

	XParam.blkmemwidth = XParam.blkwidth + 2 * XParam.halowidth;
	XParam.blksize = utils::sq(XParam.blkmemwidth);

	///////////////////////////////////////////
	//  Read Bathy Information
	///////////////////////////////////////////

	//this sets xo yo  etc...

	// Any of xo,yo,xmax,ymax or dx not defined is assigned the value from bathy file
	//default value is nan in default param file

	//inputmap Bathymetry;
	//Bathymetry.inputfile = XForcing.Bathy.inputfile;
	//XForcing.Bathy = readforcinghead(XForcing.Bathy);



	if (std::isnan(XParam.xo))
		XParam.xo = XForcing.Bathy[0].xo - (0.5 * XForcing.Bathy[0].dx);
	if (std::isnan(XParam.xmax))
		XParam.xmax = XForcing.Bathy[0].xmax + (0.5 * XForcing.Bathy[0].dx);
	if (std::isnan(XParam.yo))
		XParam.yo = XForcing.Bathy[0].yo - (0.5 * XForcing.Bathy[0].dx);
	if (std::isnan(XParam.ymax))
		XParam.ymax = XForcing.Bathy[0].ymax + (0.5 * XForcing.Bathy[0].dx);

	if (std::isnan(XParam.dx))
		XParam.dx = XForcing.Bathy[0].dx;

	if (std::isnan(XParam.grdalpha))
		XParam.grdalpha = XForcing.Bathy[0].grdalpha; // here the default bathy grdalpha is 0.0 as defined by inputmap/Bathymetry class


	//Check Bathy input type
	if (XForcing.Bathy[0].extension.compare("dep") == 0 || XForcing.Bathy[0].extension.compare("bot") == 0)
	{
		if (std::isnan(XParam.dx))
		{
			//std::cerr << "FATAL ERROR: nx or ny or dx were not specified. These parameters are required when using ." << bathyext << " file" << std::endl;
			log("FATAL ERROR: nx or ny or dx were not specified. These parameters are required when using ." + XForcing.Bathy[0].extension + " file");
			exit(1);
		}
	}

	double levdx = calcres(XParam.dx, XParam.initlevel);// true grid resolution as in dx/2^(initlevel)
	//printf("levdx=%f;1 << XParam.initlevel=%f\n", levdx, calcres(1.0, XParam.initlevel));

	// First estimate nx and ny
	XParam.nx = ftoi((XParam.xmax - XParam.xo) / (levdx));
	XParam.ny = ftoi((XParam.ymax - XParam.yo) / (levdx)); //+1?
	//if desire size in one direction is under the bathy resolution or dx requested
	if (XParam.nx == 0) { XParam.nx = 1; }
	if (XParam.ny == 0) { XParam.ny = 1; }


	// Adjust xmax and ymax so that nx and ny are a factor of XParam.blkwidth [16]
	XParam.xmax = XParam.xo + (ceil(XParam.nx / ((double)XParam.blkwidth)) * ((double)XParam.blkwidth)) * levdx;
	XParam.ymax = XParam.yo + (ceil(XParam.ny / ((double)XParam.blkwidth)) * ((double)XParam.blkwidth)) * levdx;

	// Update nx and ny 
	XParam.nx = ftoi((XParam.xmax - XParam.xo) / (levdx));
	XParam.ny = ftoi((XParam.ymax - XParam.yo) / (levdx)); //+1?

	log("\nAdjusted model domain (xo/xmax/yo/ymax): ");
	log("\t" + std::to_string(XParam.xo) + "/" + std::to_string(XParam.xmax) + "/" + std::to_string(XParam.yo) + "/" + std::to_string(XParam.ymax));
	log("\t Initial resolution (level " + std::to_string(XParam.initlevel) + ") = " + std::to_string(levdx));

	if (XParam.spherical == false)
	{
		XParam.delta = XParam.dx;
		XParam.grdalpha = XParam.grdalpha * pi / 180.0; // grid rotation

	}
	else
	{
		//Geo grid

		XParam.delta = XParam.dx * XParam.Radius * pi / 180.0;
		//XParam.engine = 2;

		//printf("Using spherical coordinate; delta=%f rad\n", XParam.delta);
		log("Using spherical coordinate; delta=" + std::to_string(XParam.delta));
		if (XParam.grdalpha != 0.0)
		{
			//printf("grid rotation in spherical coordinate is not supported yet. grdalpha=%f rad\n", XParam.grdalpha);
			log("grid rotation in spherical coordinate is not supported yet. grdalpha=" + std::to_string(XParam.grdalpha * 180.0 / pi));
		}
	}

	// Read/setup bdn segment polygon. Note this can't be part of the "readforcing" step because xmin, xmax ymin ymax are not known then
	for (int iseg = 0; iseg < XForcing.bndseg.size(); iseg++)
	{
		XForcing.bndseg[iseg].poly = readbndpolysegment(XForcing.bndseg[iseg], XParam);
		if (XForcing.bndseg[iseg].type == 2)
		{
			XForcing.bndseg[iseg].type = 3;
		}


		XForcing.bndseg[iseg].left.isright = -1;
		XForcing.bndseg[iseg].left.istop = 0;

		XForcing.bndseg[iseg].right.isright = 1;
		XForcing.bndseg[iseg].right.istop = 0;

		XForcing.bndseg[iseg].top.isright = 0;
		XForcing.bndseg[iseg].top.istop = 1;

		XForcing.bndseg[iseg].bot.isright = 0;
		XForcing.bndseg[iseg].bot.istop = -1;
	}

	bndsegment remainderblk;

	remainderblk.left.isright = -1;
	remainderblk.left.istop = 0;

	remainderblk.right.isright = 1;
	remainderblk.right.istop = 0;

	remainderblk.top.isright = 0;
	remainderblk.top.istop = 1;

	remainderblk.bot.isright = 0;
	remainderblk.bot.istop = -1;
	remainderblk.type = XParam.aoibnd;

	XForcing.bndseg.push_back(remainderblk);
	for (int iseg = 0; iseg < XForcing.bndseg.size(); iseg++)
	{

		AllocateCPU(1, 1, XForcing.bndseg[iseg].left.blk);
		AllocateCPU(1, 1, XForcing.bndseg[iseg].right.blk);
		AllocateCPU(1, 1, XForcing.bndseg[iseg].top.blk);
		AllocateCPU(1, 1, XForcing.bndseg[iseg].bot.blk);

		AllocateCPU(1, 1, XForcing.bndseg[iseg].left.qmean);
		AllocateCPU(1, 1, XForcing.bndseg[iseg].right.qmean);
		AllocateCPU(1, 1, XForcing.bndseg[iseg].top.qmean);
		AllocateCPU(1, 1, XForcing.bndseg[iseg].bot.qmean);
	}





	//setup extra infor about boundaries
	// This is not needed anymore
	XForcing.left.side = 3;
	XForcing.left.isright = -1;
	XForcing.left.istop = 0;

	XForcing.right.side = 1;
	XForcing.right.isright = 1;
	XForcing.right.istop = 0;

	XForcing.top.side = 0;
	XForcing.top.isright = 0;
	XForcing.top.istop = 1;

	XForcing.bot.side = 2;
	XForcing.bot.isright = 0;
	XForcing.bot.istop = -1;


	//

	XForcing.Atmp.clampedge = float(XParam.Paref);

	if (!XForcing.Atmp.inputfile.empty())
	{
		XParam.atmpforcing = true;
		XParam.engine = 3;
	}


	// Make sure the nriver in param (used for preallocation of memory) and number of rivers in XForcing are consistent
	XParam.nrivers = int(XForcing.rivers.size());



	// Check whether endtime was specified by the user
	//No; i.e. endtimne =0.0
	//so the following conditions are useless



	if (abs(XParam.endtime - DefaultParams.endtime) <= tiny)
	{
		//No; i.e. endtimne =0.0
		XParam.endtime = 1.0 / tiny; //==huge
	}

	XParam.endtime = setendtime(XParam, XForcing);


	// Assign a value for reftime if not yet set. 
	//It is needed in the Netcdf file generation
	if (XParam.reftime.empty())
	{
		XParam.reftime = "2000-01-01T00:00:00";
	}

	log("Reference time: " + XParam.reftime);
	log("Model Initial time: " + std::to_string(XParam.totaltime));

	log("Model end time: " + std::to_string(XParam.endtime));

	// Check that outputtimestep is not zero, so at least the first and final time step are saved
	// If only the model stepup is needed than just run with endtime=0.0
	if (abs(XParam.outputtimestep - DefaultParams.outputtimestep) <= tiny)
	{
		XParam.outputtimestep = XParam.endtime;
		//otherwise there is really no point running the model
	}
	if (XParam.outputtimestep > XParam.endtime)
	{
		XParam.outputtimestep = XParam.endtime;
		//otherwise, no final output
	}

	//Initialisation of the main time output vector
	//Initialise default values for Toutput (output times for map outputs)
	InitialiseToutput(XParam.Toutput, XParam);


	// Initialisation of the time output vector for the zones outputs
	if (XParam.outzone.size() > 0)
	{
		for (int ii = 0; ii < XParam.outzone.size(); ii++)
		{
			{
				InitialiseToutput(XParam.outzone[ii].Toutput, XParam);
			}
		}
	}



	if (XParam.outvars.empty() && XParam.outputtimestep > 0.0)
	{
		//a nc file was specified but no output variable were specified
		std::vector<std::string> SupportedVarNames = { "zb", "zs", "u", "v", "h" };
		for (int isup = 0; isup < SupportedVarNames.size(); isup++)
		{
			XParam.outvars.push_back(SupportedVarNames[isup]);

		}

	}


	// Check whether a cuda compatible GPU is present
	if (XParam.GPUDEVICE >= 0)
	{
		// Init GPU
		int nDevices;
		hipGetDeviceCount(&nDevices);
		hipDeviceProp_t prop;

		if (XParam.GPUDEVICE > (nDevices - 1))
		{
			//  if no GPU device are present then use the CPU (GPUDEVICE = -1)
			XParam.GPUDEVICE = (nDevices - 1);
		}
		hipGetDeviceProperties(&prop, XParam.GPUDEVICE);
		//printf("There are %d GPU devices on this machine\n", nDevices);
		log("There are " + std::to_string(nDevices) + " GPU devices on this machine");

		if (XParam.GPUDEVICE >= 0)
		{

			log("Using Device: " + std::string(prop.name));
		}
		else
		{
			log("No GPU device were detected on this machine... Using CPU instead");
		}

	}


	if (XParam.minlevel != XParam.maxlevel)
	{
		if (XParam.AdaptCrit.empty())
		{
			XParam.AdaptCrit = "Threshold";
			XParam.Adapt_arg1 = "0.0";
			XParam.Adapt_arg2 = "h";
		}
	}

	//Check that we have both initial loss and continuous loss if one is given
	if (!XForcing.il.inputfile.empty())
	{
		if (XForcing.cl.inputfile.empty())
		{
			log("Error: File identified for initial loss but no data entered for continuous loss.\n Please, enter a ");
		}
	}
	if (!XForcing.cl.inputfile.empty())
	{
		if (XForcing.il.inputfile.empty())
		{
			log("Error: File identified for continuous loss but no data entered for initial loss");
		}
	}

	//Check that the Initial Loss/ Continuing Loss model is used if il, cl or hgw output are asked by user.
	if (!XParam.infiltration) // (XForcing.il.inputfile.empty() && XForcing.cl.inputfile.empty() && (XParam.il == 0.0) && (XParam.cl == 0.0))
	{
		std::vector<std::string> namestr = { "il","cl","hgw" };
		for (int ii = 0; ii < namestr.size(); ii++)
		{
			std::vector<std::string>::iterator itr = std::find(XParam.outvars.begin(), XParam.outvars.end(), namestr[ii]);
			if (itr != XParam.outvars.end())
			{
				log("The output variable associated to the ILCL model \"" + namestr[ii] + "\" is requested but the model is not used. The variable is removed from the outputs.");
				XParam.outvars.erase(itr);
			}
		}
	}

	//Check that the atmospheric forcing is used if datmpdx, datmpdy output are asked by user.
	if (XForcing.Atmp.inputfile.empty())
	{
		std::vector<std::string> namestr = { "datmpdx", "datmpdy" };
		for (int ii = 0; ii < namestr.size(); ii++)
		{
			std::vector<std::string>::iterator itr = std::find(XParam.outvars.begin(), XParam.outvars.end(), namestr[ii]);
			if (itr != XParam.outvars.end())
			{
				log("The output variable associated to the atmosheric forcing \"" + namestr[ii] + "\" is requested but the model is not used. The variable is removed from the outputs.");
				XParam.outvars.erase(itr);
			}
		}

	}

}

//Initialise default values for Toutput (output times for map outputs)
void InitialiseToutput(T_output& Toutput_loc, Param XParam)
{
	if (std::isnan(Toutput_loc.init))
	{
		Toutput_loc.init = XParam.totaltime;
	}
	if (std::isnan(Toutput_loc.end))
	{
		Toutput_loc.end = XParam.endtime;
	}
	if (std::isnan(Toutput_loc.tstep))
	{
		Toutput_loc.tstep = XParam.outputtimestep;
	}
}

/*! \fn double setendtime(Param XParam,Forcing<float> XForcing)
* Calculate/modify endtime based on maximum time in forcing
*
*/
double setendtime(Param XParam, Forcing<float> XForcing)
{
	//endtime cannot be bigger than the smallest time set in a boundary
	SLTS tempSLTS;
	double endtime = XParam.endtime;
	if (XForcing.left.on)
	{
		tempSLTS = XForcing.left.data.back();
		endtime = utils::min(endtime, tempSLTS.time);

	}
	if (XForcing.right.on)
	{
		tempSLTS = XForcing.right.data.back();
		endtime = utils::min(endtime, tempSLTS.time);
	}
	if (XForcing.top.on)
	{
		tempSLTS = XForcing.top.data.back();
		endtime = utils::min(endtime, tempSLTS.time);
	}
	if (XForcing.bot.on)
	{
		tempSLTS = XForcing.bot.data.back();
		endtime = utils::min(endtime, tempSLTS.time);
	}

	if (endtime < XParam.endtime)
	{
		log("\nWARNING: Boundary definition too short, endtime of the simulation reduced to : " + std::to_string(endtime));
	}

	return endtime;
}

/*! \fn std::string findparameter(std::string parameterstr, std::string line)
* separate parameter from value
*
*/
std::string findparameter(std::vector<std::string> parameterstr, std::string line)
{
	std::size_t found;
	std::string parameternumber, left, right;
	std::vector<std::string> splittedstr;

	// first look for an equal sign
	// No equal sign mean not a valid line so skip
	splittedstr = split(line, '=');
	if (splittedstr.size() > 1)
	{
		left = trim(splittedstr[0], " ");
		right = splittedstr[1]; // if there are more than one equal sign in the line the second one is ignored
		for (int ieq = 2; ieq < splittedstr.size(); ieq++)
		{
			right = right + "=" + splittedstr[ieq];
		}
		for (int ii = 0; ii < parameterstr.size(); ii++)
		{
			found = case_insensitive_compare(left, parameterstr[ii]);// it needs to strictly compare
			if (found == 0)
				break;
		}
		if (found == 0) // found the parameter
		{
			//std::cout <<"found LonMin at : "<< found << std::endl;
			//Numberstart = found + parameterstr.length();
			splittedstr = split(right, ';');
			if (splittedstr.size() >= 1)
			{
				parameternumber = splittedstr[0];
			}
			//std::cout << parameternumber << std::endl;

		}
	}
	return trim(parameternumber, " ");
	//return parameternumber;
}


std::string findparameter(std::string parameterstr, std::string line)
{
	std::vector<std::string> parametervec;

	parametervec.push_back(parameterstr);
	return findparameter(parametervec, line);
}


/*! \fn void split(const std::string &s, char delim, std::vector<std::string> &elems)
* split string based in character
*
*/
void split(const std::string& s, char delim, std::vector<std::string>& elems) {
	std::stringstream ss;
	ss.str(s);
	std::string item;
	while (std::getline(ss, item, delim)) {
		if (!item.empty())//skip empty tokens
		{
			elems.push_back(item);
		}

	}
}

/*! \fn std::vector<std::string> split(const std::string &s, char delim)
* split string based in character
*
*/
std::vector<std::string> split(const std::string& s, char delim) {
	std::vector<std::string> elems;
	split(s, delim, elems);
	return elems;
}



/*! \fn void split_full(const std::string &s, char delim, std::vector<std::string> &elems)
* split string based in character, conserving empty item
*
*/
void split_full(const std::string& s, char delim, std::vector<std::string>& elems) {
	std::stringstream ss;
	ss.str(s);
	std::string item;
	while (std::getline(ss, item, delim)) {
		std::string::iterator end_pos = std::remove(item.begin(), item.end(), ' ');
		item.erase(end_pos, item.end());
		elems.push_back(item);
	}
	if (s[s.length() - 1] == delim)
	{
		std::string item;
		elems.push_back(item);
	}
}

/*! \fn std::vector<std::string> split_full(const std::string &s, char delim)
* split string based in character, conserving empty items
*
*/
std::vector<std::string> split_full(const std::string& s, char delim) {
	std::vector<std::string> elems;
	split_full(s, delim, elems);
	return elems;
}


std::vector<std::string> split(const std::string s, const std::string delim)
{
	size_t ide = 0;
	int loc = 0;
	std::vector<std::string> output;
	std::string rem = s;


	while (ide < std::string::npos || output.size() == 0)
	{

		ide = rem.find(delim);
		if (ide == 0 || ide == std::string::npos)
		{
			output.push_back(rem);
			ide = std::string::npos;
		}
		else
		{
			output.push_back(rem.substr(loc, ide));
		}

		if (ide < (rem.length() - delim.length()))
		{
			loc = int(ide + delim.length());
			rem = rem.substr(loc);
		}
	}

	return output;



}


/*! \fn std::string trim(const std::string& str, const std::string& whitespace)
* remove leading and trailing space in a string
*
*/
std::string trim(const std::string& str, const std::string& whitespace)
{
	const auto strBegin = str.find_first_not_of(whitespace);
	if (strBegin == std::string::npos)
		return ""; // no content

	const auto strEnd = str.find_last_not_of(whitespace);
	const auto strRange = strEnd - strBegin + 1;

	return str.substr(strBegin, strRange);
}

/*! \fn std::size_t case_insensitive_compare(const std::string& str, const std::string& str)
* case non-sensitive string comparison (return 0 if the same, as for the "compare" function)
*
*/
std::size_t case_insensitive_compare(std::string s1, std::string s2)
{
	//Convert s1 and s2 to lower case strings
	std::transform(s1.begin(), s1.end(), s1.begin(), ::tolower);
	std::transform(s2.begin(), s2.end(), s2.begin(), ::tolower);
	//if (s1.compare(s2) == 0)
	return s1.compare(s2);
}

std::size_t case_insensitive_compare(std::string s1, std::vector<std::string> vecstr)
{
	std::size_t found;
	//Convert s1 and s2 to lower case strings
	for (int ii = 0; ii < vecstr.size(); ii++)
	{
		found = case_insensitive_compare(s1, vecstr[ii]);// it needs to strictly compare
		if (found == 0)
		{
			break;
		}
	}
	return found;
}


bndsegment readbndlineside(std::string parametervalue, std::string side)
{
	bndsegment bnd;


	std::vector<std::string> items = split(parametervalue, ',');

	if (items.size() == 1)
	{
		bnd.type = std::stoi(items[0]);

	}
	else if (items.size() >= 2)
	{
		const char* cstr = items[1].c_str();

		if (isdigit(cstr[0]))
		{
			//?
			bnd.type = std::stoi(items[1]);
			bnd.inputfile = items[0];
			bnd.on = true;



		}
		else
		{
			bnd.type = std::stoi(items[0]);
			bnd.inputfile = items[1];
			bnd.on = true;
		}

	}
	bnd.polyfile = side;
	if (bnd.on)
	{
		bnd.WLmap = readfileinfo(bnd.inputfile, bnd.WLmap);

		//set the expected type of input

		if (bnd.WLmap.extension.compare("nc") == 0)
		{
			bnd.WLmap.uniform = 0;
			bnd.uniform = 0;
		}
		else
		{
			bnd.WLmap.uniform = 1;
			bnd.uniform = 1;
		}
	}
	return bnd;
}


bndsegment readbndline(std::string parametervalue)
{
	//bndseg = area.txt, waterlevelforcing, 1;
	bndsegment bnd;
	std::vector<std::string> items = split(parametervalue, ',');
	if (items.size() == 1)
	{
		bnd.type = std::stoi(items[0]);

	}
	else if (items.size() >= 2)
	{
		const char* cstr = items[1].c_str();
		if (items[1].length() > 2)
		{
			bnd.polyfile = items[0];
			bnd.type = std::stoi(items[2]);
			bnd.inputfile = items[1];
			bnd.on = true;

		}
		else
		{
			bnd.polyfile = items[0];
			bnd.type = std::max(std::stoi(items[1]), 1); // only 2 param implies that it is either a wall or Neumann bnd

		}
	}


	//set the expected type of input

	if (bnd.on)
	{
		bnd.WLmap = readfileinfo(bnd.inputfile, bnd.WLmap);

		//set the expected type of input

		if (bnd.WLmap.extension.compare("nc") == 0)
		{
			bnd.WLmap.uniform = 0;
			bnd.uniform = 0;
		}
		else
		{
			bnd.WLmap.uniform = 1;
			bnd.uniform = 1;
		}
	}
	return bnd;
}



bool readparambool(std::string paramstr, bool defaultval)
{
	bool out = defaultval;
	std::vector<std::string> truestr = { "1","true","yes", "on" };
	std::vector<std::string> falsestr = { "-1","false","no","off" };

	if (case_insensitive_compare(paramstr, truestr) == 0)
	{
		out = true;
	}
	if (case_insensitive_compare(paramstr, falsestr) == 0)
	{
		out = false;
	}

	return out;
}




//inline bool fileexists(const std::string& name) {
//	struct stat buffer;
//	return (stat(name.c_str(), &buffer) == 0);
//}


