#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////////////
// BG_Flood Main function						                                //
// Copyright (C) 2018 Bosserelle                                                //
// This code contains an adaptation of the St Venant equation from Basilisk		//
// See																			//
// http://basilisk.fr/src/saint-venant.h and									//
// S. Popinet. Quadtree-adaptive tsunami modelling. Ocean Dynamics,				//
// doi: 61(9) : 1261 - 1285, 2011												//
//                                                                              //
// This program is free software: you can redistribute it and/or modify         //
// it under the terms of the GNU General Public License as published by         //
// the Free Software Foundation.                                                //
//                                                                              //
// This program is distributed in the hope that it will be useful,              //
// but WITHOUT ANY WARRANTY; without even the implied warranty of               //
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                //
// GNU General Public License for more details.                                 //
//                                                                              //
// You should have received a copy of the GNU General Public License            //
// along with this program.  If not, see <http://www.gnu.org/licenses/>.        //
//////////////////////////////////////////////////////////////////////////////////

// includes, system

#include "BG_Flood.h"


/*! \fn int main(int argc, char **argv)
* Main function 
* This function is the entry point to the software
* The main function setups all the init of the model and then calls the mainloop to actually run the model
*
*	There are 3 main class storing information about the model: XParam (class Param), XModel (class Model) and XForcing (class Forcing)
*	Leading X stands for eXecution and is to avoid confusion between the class variable and the class declaration
*	When running with the GPU there is also XModel_g
*	which is the same as XModel but with GPU specific pointers
*
*
* This function does:
* * Reads the inputs to the model
* * Allocate memory on GPU and CPU
* * Prepare and initialise memory and arrays on CPU and GPU
* * Setup initial condition
* * Adapt grid if require
* * Prepare output file
* * Run main loop
* * Clean up and close
*/
int main(int argc, char* argv[])
{
	//===========================================
	// Read model argument (filename). If one is not given use the default name
	std::string ParamFile;

	if (argc > 1)
	{
		ParamFile = argv[1];
	}
	else
	{
		ParamFile = "BG_param.txt";
	}

	//std::cout << ParamFile << '\n';

	//===========================================
	//  Define the main parameter controling the model (XModels class at produced later) 
	Param XParam;
	Forcing<float> XForcing; // for reading and storing forcing data (CPU only) // by default we read only float precision!
	// Start timer to keep track of time
	XParam.startcputime = clock();

	
	// Create/overwrite existing 
	create_logfile();

	//============================================
	// Read Operational file
	// Also check XParam sanity

	Readparamfile(XParam, XForcing, ParamFile);


	//============================================
	// Create external forcing and model pointers
	// Before this is done we need to check
	// if the model will be double or float precision

	Model<double> XModel_d; // For CPU double pointers
	Model<double> XModel_gd; // For GPU double pointers

	Model<float> XModel_f; // For CPU float pointers
	Model<float> XModel_gf; // For GPU float pointers

	if (XParam.doubleprecision < 1)
	{
		// Call the Float precision run
		mainwork(XParam, XForcing, XModel_f, XModel_gf);
	}
	else
	{
		mainwork(XParam, XForcing, XModel_d, XModel_gd);
	}

}

template < class T > int mainwork(Param XParam, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
{
	//============================================
	// Read the forcing data (Including bathymetry)
	readforcing(XParam, XForcing);

	//=============================================
	// Verify Compatibility of forcing and model Parameters
	checkparamsanity(XParam, XForcing);

	//============================================
	// Prepare initial mesh layout
	InitMesh(XParam, XForcing, XModel);

	//============================================
	// Prepare initial conditions on CPU
	InitialConditions(XParam, XForcing, XModel);
	printf("XCulvertsF h1=%f\n", XModel.culvertsF.h1[0]);

	//============================================
	// Initial adaptation
	InitialAdaptation(XParam, XForcing, XModel);

	//============================================
	// Setup GPU (bypassed within the function if no suitable GPU is available)
	SetupGPU(XParam, XModel,XForcing, XModel_g);
	printf("XCulvertsF h1=%f\n", XModel.culvertsF.h1[0]);
	printf("XCulvertsF h1=%f\n", XModel_g.culvertsF.h1[0]);



	//
	log("\nModel setup complete");
	log("#################################");
	//===========================================
	//   End of Initialisation time
	//===========================================
	XParam.setupcputime = clock();
	bool isfailed = false;

	if (XParam.test < 0)
	{
		//============================================
		// MainLoop
		printf("XCulvertsF h1=%f\n", XModel.culvertsF.h1[0]);
		MainLoop(XParam, XForcing, XModel, XModel_g);
	}
	else
	{
		//============================================
		// Testing
		//Gaussianhump(XParam, XModel, XModel_g);
		isfailed = Testing(XParam, XForcing, XModel, XModel_g);
	}

		

	//===========================================
	//   End of Model
	//===========================================
	XParam.endcputime = clock();

	//===========================================
	//   Log the timer
	//===========================================
	log("#################################");
	log("End Computation");
	log("#################################");
	log("Total runtime= " + std::to_string((XParam.endcputime - XParam.startcputime) / CLOCKS_PER_SEC) + " seconds");
	log("Model Setup time= " + std::to_string((XParam.setupcputime - XParam.startcputime) / CLOCKS_PER_SEC) + " seconds");
	log("Model runtime= " + std::to_string((XParam.endcputime - XParam.setupcputime) / CLOCKS_PER_SEC) + " seconds");


	if (XParam.GPUDEVICE >= 0)
	{
		size_t free_byte;

		size_t total_byte;

		CUDA_CHECK(hipMemGetInfo(&free_byte, &total_byte));

		XParam.GPU_totalmem_byte = (total_byte - free_byte) - XParam.GPU_initmem_byte;
		log("Model final memory usage= " + std::to_string((XParam.GPU_totalmem_byte) / 1024.0 / 1024.0) + " MB");

	}


	//============================================
	// Cleanup and free memory
	//
	if (XParam.test < 0)
	{
		exit(0);
	}
	else 
	{
		exit(isfailed);
	}
	
}
