#include "hip/hip_runtime.h"

#include "Meanmax.h"


template <class T> void Calcmeanmax(Param XParam, Loop<T>& XLoop, Model<T> XModel, Model<T> XModel_g)
{
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	
	if (XParam.outmean)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			addavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmean.h, XModel_g.evolv.h);
			addavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmean.zs, XModel_g.evolv.zs);
			addavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmean.u, XModel_g.evolv.u);
			addavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmean.v, XModel_g.evolv.v);
			addUandhU_GPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evolv.h, XModel_g.evolv.u, XModel_g.evolv.v, XModel_g.evmean.U, XModel_g.evmean.hU);

			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{

			addavg_varCPU(XParam, XModel.blocks, XModel.evmean.h, XModel.evolv.h);
			addavg_varCPU(XParam, XModel.blocks, XModel.evmean.zs, XModel.evolv.zs);
			addavg_varCPU(XParam, XModel.blocks, XModel.evmean.u, XModel.evolv.u);
			addavg_varCPU(XParam, XModel.blocks, XModel.evmean.v, XModel.evolv.v);
			addUandhU_CPU(XParam, XModel.blocks, XModel.evolv.h, XModel.evolv.u, XModel.evolv.v, XModel.evmean.U, XModel.evmean.hU);

		}


		XLoop.nstep++;

		if (XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001))
		{
			// devide by number of steps
			if (XParam.GPUDEVICE >= 0)
			{
				divavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, T(XLoop.nstep), XModel_g.evmean.h);
				divavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, T(XLoop.nstep), XModel_g.evmean.zs);
				divavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, T(XLoop.nstep), XModel_g.evmean.u);
				divavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, T(XLoop.nstep), XModel_g.evmean.v);
				divavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, T(XLoop.nstep), XModel_g.evmean.U);
				divavg_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, T(XLoop.nstep), XModel_g.evmean.hU);
				CUDA_CHECK(hipDeviceSynchronize());
			}
			else
			{
				divavg_varCPU(XParam, XModel.blocks, T(XLoop.nstep), XModel.evmean.h);
				divavg_varCPU(XParam, XModel.blocks, T(XLoop.nstep), XModel.evmean.zs);
				divavg_varCPU(XParam, XModel.blocks, T(XLoop.nstep), XModel.evmean.u);
				divavg_varCPU(XParam, XModel.blocks, T(XLoop.nstep), XModel.evmean.v);
				divavg_varCPU(XParam, XModel.blocks, T(XLoop.nstep), XModel.evmean.U);
				divavg_varCPU(XParam, XModel.blocks, T(XLoop.nstep), XModel.evmean.hU);
			}

			//XLoop.nstep will be reset after a save to the disk which occurs in a different function
		}

	}
	if (XParam.outmax)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			max_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmax.h, XModel_g.evolv.h);
			max_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmax.zs, XModel_g.evolv.zs);
			max_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmax.u, XModel_g.evolv.u);
			max_varGPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmax.v, XModel_g.evolv.v);
			max_Norm_GPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmax.U, XModel_g.evolv.u, XModel_g.evolv.v);
			max_hU_GPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.evmax.hU, XModel_g.evolv.h, XModel_g.evolv.u, XModel_g.evolv.v);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			max_varCPU(XParam, XModel.blocks, XModel.evmax.h, XModel.evolv.h);
			max_varCPU(XParam, XModel.blocks, XModel.evmax.zs, XModel.evolv.zs);
			max_varCPU(XParam, XModel.blocks, XModel.evmax.u, XModel.evolv.u);
			max_varCPU(XParam, XModel.blocks, XModel.evmax.v, XModel.evolv.v);
			max_Norm_CPU(XParam, XModel.blocks, XModel.evmax.U, XModel.evolv.u, XModel.evolv.v);
			max_hU_CPU(XParam, XModel.blocks, XModel.evmax.hU, XModel.evolv.h, XModel.evolv.u, XModel.evolv.v);
		}
	}
	if (XParam.outtwet)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			// Add value GPU
			addwettime_GPU << < gridDim, blockDim, 0 >> > (XParam, XModel_g.blocks, XModel_g.wettime, XModel_g.evolv.h, T(0.1), T(XLoop.dt));

		}
		else
		{
			// Add value CPU
			addwettime_CPU(XParam, XModel.blocks, XModel.wettime, XModel.evolv.h, T(0.1), T(XLoop.dt));
		}
	}
}
template void Calcmeanmax<float>(Param XParam, Loop<float>& XLoop, Model<float> XModel, Model<float> XModel_g);
template void Calcmeanmax<double>(Param XParam, Loop<double>& XLoop, Model<double> XModel, Model<double> XModel_g);


template <class T> void resetmeanmax(Param XParam, Loop<T>& XLoop, Model<T> XModel, Model<T> XModel_g)
{
	// Reset mean and or max only at output steps
	//XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001)
	if (XLoop.nstepout == 0) //This implis an output was just produced so need to reset
	{
		//Define some useful variables 
		if (XParam.outmean)
		{
			if (XParam.GPUDEVICE >= 0)
			{
				resetmeanGPU(XParam, XLoop, XModel_g.blocks, XModel_g.evmean);
			}
			else
			{
				resetmeanCPU(XParam, XLoop, XModel.blocks, XModel.evmean);
			}
			XLoop.nstep = 0;
		}

		//Reset Max 
		if (XParam.outmax && XParam.resetmax)
		{
			if (XParam.GPUDEVICE >= 0)
			{
				resetmaxGPU(XParam, XLoop, XModel_g.blocks, XModel_g.evmax);
			}
			else
			{
				resetmaxCPU(XParam, XLoop, XModel.blocks, XModel.evmax);

			}
		}

		//Reset Wet duration
		if (XParam.outtwet && XParam.resetmax)
		{
			if (XParam.GPUDEVICE >= 0)
			{
				resetvalGPU(XParam, XModel_g.blocks, XModel_g.wettime, T(0.0));
			}
			else
			{
				resetvalCPU(XParam, XModel.blocks, XModel.wettime, T(0.0));
			}
		}
	}
}
template void resetmeanmax<float>(Param XParam, Loop<float>& XLoop, Model<float> XModel, Model<float> XModel_g);
template void resetmeanmax<double>(Param XParam, Loop<double>& XLoop, Model<double> XModel, Model<double> XModel_g);

template <class T> void Initmeanmax(Param XParam, Loop<T> XLoop, Model<T> XModel, Model<T> XModel_g)
{
	//at the initial step overide the reset max to initialise the max variable (if needed)
	//this override is not preserved so wont affect the rest of the loop
	XParam.resetmax = true;
	XLoop.nextoutputtime = XLoop.totaltime;
	XLoop.dt = T(1.0);
	resetmeanmax(XParam, XLoop, XModel, XModel_g);
}
template void Initmeanmax<float>(Param XParam, Loop<float> XLoop, Model<float> XModel, Model<float> XModel_g);
template void Initmeanmax<double>(Param XParam, Loop<double> XLoop, Model<double> XModel, Model<double> XModel_g);

template <class T> void resetmaxGPU(Param XParam, Loop<T> XLoop, BlockP<T> XBlock, EvolvingP_M<T>& XEv)
{
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XLoop.hugenegval, XEv.h);
	CUDA_CHECK(hipDeviceSynchronize());
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XLoop.hugenegval, XEv.zs);
	CUDA_CHECK(hipDeviceSynchronize());
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XLoop.hugenegval, XEv.u);
	CUDA_CHECK(hipDeviceSynchronize());
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XLoop.hugenegval, XEv.v);
	CUDA_CHECK(hipDeviceSynchronize());
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XLoop.hugenegval, XEv.U);
	CUDA_CHECK(hipDeviceSynchronize());
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, XLoop.hugenegval, XEv.hU);
	CUDA_CHECK(hipDeviceSynchronize());

}


template <class T> void resetmaxCPU(Param XParam, Loop<T> XLoop, BlockP<T> XBlock, EvolvingP_M<T>& XEv)
{

	InitArrayBUQ(XParam, XBlock, XLoop.hugenegval, XEv.h);
	InitArrayBUQ(XParam, XBlock, XLoop.hugenegval, XEv.zs);
	InitArrayBUQ(XParam, XBlock, XLoop.hugenegval, XEv.u);
	InitArrayBUQ(XParam, XBlock, XLoop.hugenegval, XEv.v);
	InitArrayBUQ(XParam, XBlock, XLoop.hugenegval, XEv.U);
	InitArrayBUQ(XParam, XBlock, XLoop.hugenegval, XEv.hU);

}


template <class T> void resetmeanCPU(Param XParam, Loop<T> XLoop, BlockP<T> XBlock, EvolvingP_M<T>& XEv)
{

	InitArrayBUQ(XParam, XBlock, T(0.0), XEv.h);
	InitArrayBUQ(XParam, XBlock, T(0.0), XEv.zs);
	InitArrayBUQ(XParam, XBlock, T(0.0), XEv.u);
	InitArrayBUQ(XParam, XBlock, T(0.0), XEv.v);
	InitArrayBUQ(XParam, XBlock, T(0.0), XEv.U);
	InitArrayBUQ(XParam, XBlock, T(0.0), XEv.hU);

}
template void resetmeanCPU<float>(Param XParam, Loop<float> XLoop, BlockP<float> XBlock, EvolvingP_M<float>& XEv);
template void resetmeanCPU<double>(Param XParam, Loop<double> XLoop, BlockP<double> XBlock, EvolvingP_M<double>& XEv);

template <class T> void resetmeanGPU(Param XParam, Loop<T> XLoop, BlockP<T> XBlock, EvolvingP_M<T>& XEv)
{
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	//
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, T(0.0), XEv.h);
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, T(0.0), XEv.zs);
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, T(0.0), XEv.u);
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, T(0.0), XEv.v);
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, T(0.0), XEv.U);
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, T(0.0), XEv.hU);
	CUDA_CHECK(hipDeviceSynchronize());


}
template void resetmeanGPU<float>(Param XParam, Loop<float> XLoop, BlockP<float> XBlock, EvolvingP_M<float>& XEv);
template void resetmeanGPU<double>(Param XParam, Loop<double> XLoop, BlockP<double> XBlock, EvolvingP_M<double>& XEv);


template <class T> void resetvalCPU(Param XParam, BlockP<T> XBlock, T*& var, T val)
{

	InitArrayBUQ(XParam, XBlock, val, var);

}
template void resetvalCPU<float>(Param XParam, BlockP<float> XBlock, float*& var, float val);
template void resetvalCPU<double>(Param XParam, BlockP<double> XBlock, double*& var, double val);

template <class T> void resetvalGPU(Param XParam, BlockP<T> XBlock, T*& var, T val)
{
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	reset_var << < gridDim, blockDim, 0 >> > (XParam.halowidth, XBlock.active, val, var);
	CUDA_CHECK(hipDeviceSynchronize());

}
template void resetvalGPU<float>(Param XParam, BlockP<float> XBlock, float*& var, float val);
template void resetvalGPU<double>(Param XParam, BlockP<double> XBlock, double*& var, double val);



template <class T> __global__ void addavg_varGPU(Param XParam, BlockP<T> XBlock, T* Varmean, T* Var)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);


	Varmean[i] = Varmean[i] + Var[i];

}


template <class T> __host__ void addavg_varCPU(Param XParam, BlockP<T> XBlock, T* Varmean, T* Var)
{
	int ib, n;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				Varmean[i] = Varmean[i] + Var[i];
			}
		}
	}

}

template <class T> __global__ void divavg_varGPU(Param XParam, BlockP<T> XBlock, T ntdiv, T* Varmean)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	Varmean[i] = Varmean[i] / ntdiv;

}

template <class T> __host__ void divavg_varCPU(Param XParam, BlockP<T> XBlock, T ntdiv, T* Varmean)
{
	int ib, n;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				Varmean[i] = Varmean[i] / ntdiv;
			}
		}
	}

}

template <class T> __global__ void addUandhU_GPU(Param XParam, BlockP<T> XBlock, T * h, T * u, T * v, T* U, T* hU)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	U[i] = sqrt((u[i] * u[i]) + (v[i] * v[i]));
	hU[i] = h[i] * U[i];

}

template <class T> __host__ void addUandhU_CPU(Param XParam, BlockP<T> XBlock, T* h, T* u, T* v, T* U, T* hU)
{
	int ib, n;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				U[i] = sqrt((u[i] * u[i]) + (v[i] * v[i]));
				hU[i] = h[i] * U[i];
			}
		}
	}

}

template <class T> __global__ void max_varGPU(Param XParam, BlockP<T> XBlock, T* Varmax, T* Var)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	Varmax[i] = max(Varmax[i], Var[i]);

}

template <class T> __global__ void max_Norm_GPU(Param XParam, BlockP<T> XBlock, T* Varmax, T* Var1, T* Var2)
{
	T Var_norm;
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	
	Var_norm = sqrt((Var1[i] * Var1[i]) + (Var2[i] * Var2[i]));
	Varmax[i] = max(Varmax[i], Var_norm);

}

template <class T> __global__ void max_hU_GPU(Param XParam, BlockP<T> XBlock, T* Varmax, T* h, T* u, T* v)
{
	T Var_hU;
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	Var_hU = h[i] * sqrt((u[i]*u[i])+(v[i]*v[i]));
	Varmax[i] = max(Varmax[i], Var_hU);

}

template <class T> __host__ void max_varCPU(Param XParam, BlockP<T> XBlock, T* Varmax, T* Var)
{
	int ib, n;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				Varmax[i] = utils::max(Varmax[i], Var[i]);
			}
		}
	}

}

template <class T> __host__ void max_Norm_CPU(Param XParam, BlockP<T> XBlock, T* Varmax, T* Var1, T* Var2)
{
	int ib, n;
	T Var_norm;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);
				Var_norm = sqrt((Var1[i] * Var1[i]) + (Var2[i] * Var2[i]));
				Varmax[i] = utils::max(Varmax[i], Var_norm);
			}
		}
	}

}

template <class T> __host__ void max_hU_CPU(Param XParam, BlockP<T> XBlock, T* Varmax, T* h, T* u, T* v)
{
	int ib, n;
	T Var_hU;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);
				Var_hU = h[i] * sqrt((u[i] * u[i]) + (v[i] * v[i]));
				Varmax[i] = utils::max(Varmax[i], Var_hU);
			}
		}
	}

}

template <class T> __global__ void addwettime_GPU(Param XParam, BlockP<T> XBlock, T* wett, T* h, T thresold, T time)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	if (h[i] > thresold)
	{
		wett[i] = wett[i] + time;
	}

}


template <class T> __host__ void addwettime_CPU(Param XParam, BlockP<T> XBlock, T* wett, T* h, T thresold, T time)
{
	int ib, n;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				if (h[i] > thresold)
				{
					wett[i] = wett[i] + time;
				}
			}
		}
	}

}