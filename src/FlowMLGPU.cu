#include "hip/hip_runtime.h"
#include "FlowMLGPU.h"

template <class T> void FlowMLGPU(Param XParam, Loop<T>& XLoop, Forcing<float> XForcing, Model<T> XModel)
{
	
	//============================================
	//  Fill the halo for gradient reconstruction & Recalculate zs
	fillHaloGPU(XParam, XModel.blocks, XModel.evolv, XModel.zb);
	

	//============================================
	// Calculate gradient for evolving parameters for predictor step
	gradientGPUnew(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.zb);

	//============================================
	// Synchronise all ongoing streams
	CUDA_CHECK(hipDeviceSynchronize());

	// Set max timestep
	reset_var <<< gridDim, blockDim, 0 >>> (XParam.halowidth, XModel.blocks.active, XLoop.hugeposval, XModel.time.dtmax);
	CUDA_CHECK(hipDeviceSynchronize());

	// Compute face value
	CalcfaceValX << < gridDim, blockDim, 0 >> > (XLoop.dt, XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	CUDA_CHECK(hipDeviceSynchronize());

	CalcfaceValY << < gridDim, blockDim, 0 >> > (XLoop.dt, XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux, XModel.time.dtmax, XModel.zb);
	CUDA_CHECK(hipDeviceSynchronize());

	// Timestep reduction
	XLoop.dt = double(CalctimestepGPU(XParam, XLoop, XModel.blocks, XModel.time));
	XLoop.dtmax = XLoop.dt;

	// Check hu/hv
	CheckadvecMLY << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux);
	CUDA_CHECK(hipDeviceSynchronize());

	CheckadvecMLX << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux);
	CUDA_CHECK(hipDeviceSynchronize());


	
	// Acceleration
	// Pressure
	pressureML << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux);
	CUDA_CHECK(hipDeviceSynchronize());

	// Advection
	AdvecFluxML << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux);
	CUDA_CHECK(hipDeviceSynchronize());

	AdvecEv << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.flux);
	CUDA_CHECK(hipDeviceSynchronize());

}
