#include "hip/hip_runtime.h"
#include "FlowMLGPU.h"

template <class T> void FlowMLGPU(Param XParam, Loop<T>& XLoop, Forcing<float> XForcing, Model<T> XModel)
{
	//============================================
	// construct threads abnd block parameters
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XParam.nblk, 1, 1);
	// for flux reconstruction the loop overlap the right(or top for the y direction) halo
	dim3 blockDimKX(XParam.blkwidth + XParam.halowidth, XParam.blkwidth, 1);
	dim3 blockDimKY(XParam.blkwidth, XParam.blkwidth + XParam.halowidth, 1);

	// Fill halo for Fu and Fv
	dim3 blockDimHaloLR(2, XParam.blkwidth, 1);
	//dim3 blockDimHaloBT(16, 1, 1);
	dim3 gridDimHaloLR(ceil(XParam.nblk / 2), 1, 1);

	dim3 blockDimHaloBT(XParam.blkwidth, 2, 1);
	dim3 gridDimHaloBT(ceil(XParam.nblk / 2), 1, 1);


	// fill halo for zs,h,u and v 

	//============================================
	//  Fill the halo for gradient reconstruction & Recalculate zs
	fillHaloGPU(XParam, XModel.blocks, XModel.evolv, XModel.zb);
	
	// calculate grad for dhdx dhdy only
	
	//============================================
	// Calculate gradient for evolving parameters for predictor step
	gradientGPUnew(XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.zb);
	//gradientSMC << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel.blocks.active, XModel.blocks.level, (T)XParam.theta, (T)XParam.delta, XModel.evolv.h, XModel.grad.dhdx, XModel.grad.dhdy);
	//CUDA_CHECK(hipDeviceSynchronize());


	//============================================
	// Synchronise all ongoing streams
	CUDA_CHECK(hipDeviceSynchronize());

	// Set max timestep
	reset_var <<< gridDim, blockDim, 0 >>> (XParam.halowidth, XModel.blocks.active, XLoop.hugeposval, XModel.time.dtmax);
	CUDA_CHECK(hipDeviceSynchronize());

	// Compute face value
	CalcfaceValX << < gridDim, blockDim, 0 >> > (T(XLoop.dtmax), XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.fluxml, XModel.time.dtmax, XModel.zb);
	CUDA_CHECK(hipDeviceSynchronize());

	CalcfaceValY << < gridDim, blockDim, 0 >> > (T(XLoop.dtmax), XParam, XModel.blocks, XModel.evolv, XModel.grad, XModel.fluxml, XModel.time.dtmax, XModel.zb);
	CUDA_CHECK(hipDeviceSynchronize());

	// Timestep reduction
	XLoop.dt = double(CalctimestepGPU(XParam, XLoop, XModel.blocks, XModel.time));
	XLoop.dtmax = XLoop.dt;

	// Check hu/hv
	CheckadvecMLY << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, T(XLoop.dt), XModel.evolv, XModel.grad, XModel.fluxml);
	CUDA_CHECK(hipDeviceSynchronize());

	CheckadvecMLX << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, T(XLoop.dt), XModel.evolv, XModel.grad, XModel.fluxml);
	CUDA_CHECK(hipDeviceSynchronize());

	//Fill flux Halo for ha and hf
	fillHaloGPU(XParam, XModel.blocks, XModel.fluxml.hfu);
	fillHaloGPU(XParam, XModel.blocks, XModel.fluxml.hfv);
	fillHaloGPU(XParam, XModel.blocks, XModel.fluxml.hau);
	fillHaloGPU(XParam, XModel.blocks, XModel.fluxml.hav);

	//HaloFluxGPULRnew << < gridDimHaloLR, blockDimHaloLR, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hfu);
	//CUDA_CHECK(hipDeviceSynchronize());

	//HaloFluxGPUBTnew << <gridDimHaloBT, blockDimHaloBT, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hfv);
	//CUDA_CHECK(hipDeviceSynchronize());

	//HaloFluxGPULRnew << < gridDimHaloLR, blockDimHaloLR, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hau);
	//CUDA_CHECK(hipDeviceSynchronize());

	//HaloFluxGPUBTnew << <gridDimHaloBT, blockDimHaloBT, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hav);
	//CUDA_CHECK(hipDeviceSynchronize());

	// Acceleration
	// Pressure
	pressureML << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, T(XLoop.dt), XModel.evolv, XModel.grad, XModel.fluxml);
	CUDA_CHECK(hipDeviceSynchronize());

	// Fill halo u and v calc grd for u and v and fill halo for hu and hv
	// 

	fillHaloGPU(XParam, XModel.blocks, XModel.evolv.u);
	fillHaloGPU(XParam, XModel.blocks, XModel.evolv.v);

	fillHaloGPU(XParam, XModel.blocks, XModel.fluxml.hu);
	fillHaloGPU(XParam, XModel.blocks, XModel.fluxml.hv);
	//HaloFluxGPULRnew << < gridDimHaloLR, blockDimHaloLR, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hu);
	//CUDA_CHECK(hipDeviceSynchronize());

	//HaloFluxGPUBTnew << <gridDimHaloBT, blockDimHaloBT, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hv);
	//CUDA_CHECK(hipDeviceSynchronize());

	//HaloFluxGPULRnew << < gridDimHaloLR, blockDimHaloLR, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hv);
	//CUDA_CHECK(hipDeviceSynchronize());

	//HaloFluxGPUBTnew << <gridDimHaloBT, blockDimHaloBT, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hu);
	//CUDA_CHECK(hipDeviceSynchronize());

	gradientSMC << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel.blocks.active, XModel.blocks.level, (T)XParam.theta, (T)XParam.delta, XModel.evolv.u, XModel.grad.dudx, XModel.grad.dudy);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientSMC << < gridDim, blockDim, 0 >> > (XParam.halowidth, XModel.blocks.active, XModel.blocks.level, (T)XParam.theta, (T)XParam.delta, XModel.evolv.v, XModel.grad.dvdx, XModel.grad.dvdy);
	CUDA_CHECK(hipDeviceSynchronize());


	fillCornersGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.fluxml.hu);
	CUDA_CHECK(hipDeviceSynchronize());

	/*fillCornersGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hv);
	CUDA_CHECK(hipDeviceSynchronize());

	fillCornersGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hfu);
	CUDA_CHECK(hipDeviceSynchronize());

	fillCornersGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.fluxml.hfv);
	CUDA_CHECK(hipDeviceSynchronize());

	fillCornersGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv.u);
	CUDA_CHECK(hipDeviceSynchronize());

	fillCornersGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv.v);
	CUDA_CHECK(hipDeviceSynchronize());*/

	//hv hfv u hu hfu v
	
	// Advection
	AdvecFluxML << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, T(XLoop.dt), XModel.evolv, XModel.grad, XModel.fluxml);
	CUDA_CHECK(hipDeviceSynchronize());

	fillHaloGPU(XParam, XModel.blocks, XModel.fluxml.Fu);
	fillHaloGPU(XParam, XModel.blocks, XModel.fluxml.Fv);

	//HaloFluxGPULRnew << < gridDimHaloLR, blockDimHaloLR, 0 >> > (XParam, XModel.blocks, XModel.fluxml.Fu);
	//CUDA_CHECK(hipDeviceSynchronize());

	//HaloFluxGPUBTnew << <gridDimHaloBT, blockDimHaloBT, 0 >> > (XParam, XModel.blocks, XModel.fluxml.Fv);
	//CUDA_CHECK(hipDeviceSynchronize());

	AdvecEv << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, T(XLoop.dt), XModel.evolv, XModel.grad, XModel.fluxml);
	CUDA_CHECK(hipDeviceSynchronize());


	if (XForcing.rivers.size() > 0)
	{
		//Add River ML
	}


	bottomfrictionGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv);
	//XiafrictionGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.time.dt, XModel.cf, XModel.evolv, XModel.evolv_o);


	CUDA_CHECK(hipDeviceSynchronize());

	if (XForcing.rivers.size() > 0)
	{
		//Add River ML
	}


	if (!XForcing.Rain.inputfile.empty())
	{
		AddrainforcingImplicitGPU << < gridDim, blockDim, 0 >> > (XParam, XLoop, XModel.blocks, XForcing.Rain, XModel.evolv);
		CUDA_CHECK(hipDeviceSynchronize());
	}

	if (XParam.infiltration)
	{
		AddinfiltrationImplicitGPU << < gridDim, blockDim, 0 >> > (XParam, XLoop, XModel.blocks, XModel.il, XModel.cl, XModel.evolv, XModel.hgw);
		CUDA_CHECK(hipDeviceSynchronize());
	}

	if (XParam.VelThreshold > 0.0)
	{
		TheresholdVelGPU << < gridDim, blockDim, 0 >> > (XParam, XModel.blocks, XModel.evolv);
		CUDA_CHECK(hipDeviceSynchronize());
	}

}
template void FlowMLGPU<float>(Param XParam, Loop<float>& XLoop, Forcing<float> XForcing, Model<float> XModel);
template void FlowMLGPU<double>(Param XParam, Loop<double>& XLoop, Forcing<float> XForcing, Model<double> XModel);

