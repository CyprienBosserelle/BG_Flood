#include "hip/hip_runtime.h"
﻿#include "Halo.h"



template <class T> void fillHaloD(Param XParam, int ib, BlockP<T> XBlock, T* z)
{
	

	fillLeft(XParam, ib, XBlock, z);
	fillRight(XParam, ib, XBlock, z);
	fillTop(XParam, ib, XBlock, z);
	fillBot(XParam, ib, XBlock, z);
	//fill bot
	//fill top
	

}
template void fillHaloD<double>(Param XParam, int ib, BlockP<double> XBlock, double* z);
template void fillHaloD<float>(Param XParam, int ib, BlockP<float> XBlock, float* z);

template <class T> void fillHaloC(Param XParam, BlockP<T> XBlock, T* z)
{
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillHaloD(XParam, ib, XBlock, z);
	}
}
template void fillHaloC<float>(Param XParam, BlockP<float> XBlock, float* z);
template void fillHaloC<double>(Param XParam, BlockP<double> XBlock, double* z);

template <class T> void RecalculateZs(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, T* zb)
{
	int ib,n,left, right, top,bot;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		/*
		//We only need to recalculate zs on the halo side 
		for (int n = -1; n <= (XParam.blkwidth); n++)
		{
			left = memloc(XParam.halowidth, XParam.blkmemwidth, -1, n, ib);
			right = memloc(XParam.halowidth, XParam.blkmemwidth, XParam.blkwidth, n, ib);
			top = memloc(XParam.halowidth, XParam.blkmemwidth, n, XParam.blkwidth, ib);
			bot = memloc(XParam.halowidth, XParam.blkmemwidth, n, -1, ib);

			Xev.zs[left] = zb[left] + Xev.h[left];
			Xev.zs[right] = zb[right] + Xev.h[right];
			Xev.zs[top] = zb[top] + Xev.h[top];
			Xev.zs[bot] = zb[bot] + Xev.h[bot];

			//printf("n=%d; zsold=%f; zsnew=%f (zb=%f + h=%f)\n",n, Xev.zs[n], zb[n] + Xev.h[n], zb[n] , Xev.h[n]);
		}
		*/
		
		// Recalculate zs everywhere maybe we only need to do that on the halo ?
		for (int j = -1; j < (XParam.blkwidth+1); j++)
		{
			for (int i = -1; i < (XParam.blkwidth+1); i++)
			{
				n = memloc(XParam.halowidth,XParam.blkmemwidth, i, j, ib);
				Xev.zs[n] = zb[n] + Xev.h[n];
			}
		}
		
	}
}


template <class T> __global__ void RecalculateZsGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, T* zb)
{
	unsigned int blkmemwidth = XParam.blkmemwidth;
	
	int ix = threadIdx.x -1;
	int iy = threadIdx.y -1;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];
	
	int  n;
	
	//ib = XBlock.active[ibl];
		
	n = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);
	Xev.zs[n] = zb[n] + Xev.h[n];
	/*
	if(zb[n] < XParam.eps)
	{
		printf("ix=%d, iy=%d, ib=%d, n=%d; zsold=%f; zsnew=%f (zb=%f + h=%f)\n",ix,iy,ib, n, Xev.zs[n], zb[n] + Xev.h[n], zb[n], Xev.h[n]);
	}
	*/
	
}

template <class T> void fillHaloF(Param XParam, bool doProlongation, BlockP<T> XBlock, T* z)
{
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillLeftFlux(XParam, doProlongation, ib, XBlock, z);
		fillBotFlux(XParam, doProlongation, ib, XBlock, z);
		fillRightFlux(XParam, doProlongation, ib, XBlock, z);
		fillTopFlux(XParam, doProlongation, ib, XBlock, z);
	
	}
}
template void fillHaloF<float>(Param XParam, bool doProlongation, BlockP<float> XBlock, float* z);
template void fillHaloF<double>(Param XParam, bool doProlongation, BlockP<double> XBlock, double* z);


template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
{

	dim3 blockDimHaloLR(1, 16, 1);
	dim3 blockDimHaloBT(16, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	fillLeft << <gridDim, blockDimHaloLR, 0 , stream >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, z);
	//fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, z);
	//CUDA_CHECK(hipDeviceSynchronize());
	fillRight << <gridDim, blockDimHaloLR, 0, stream >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	//fillRight << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	//CUDA_CHECK(hipDeviceSynchronize());
	fillBot << <gridDim, blockDimHaloBT, 0, stream >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, z);
	//fillBot << <gridDim, blockDimHaloBT, 0>> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, z);
	//CUDA_CHECK(hipDeviceSynchronize());
	fillTop << <gridDim, blockDimHaloBT, 0, stream >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	//fillTop << <gridDim, blockDimHaloBT, 0>> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	//CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipStreamSynchronize(stream));

}
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, hipStream_t stream, double* z);
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, hipStream_t stream, float* z);


template <class T> void fillHaloTopRightC(Param XParam, BlockP<T> XBlock, T* z)
{
	// for flux term and actually most terms, only top and right neighbours are needed!

	//fillLeft(XParam, ib, XBlock, z);
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillRightFlux(XParam,false, ib, XBlock, z);
		fillTopFlux(XParam,false, ib, XBlock, z);
	}
	//fillBot(XParam, ib, XBlock, z);
	//fill bot
	//fill top


}
template void fillHaloTopRightC<double>(Param XParam, BlockP<double> XBlock, double* z);
template void fillHaloTopRightC<float>(Param XParam, BlockP<float> XBlock, float* z);


template <class T> void fillHaloTopRightGPU(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
{

	dim3 blockDimHaloLR(1, 16, 1);
	dim3 blockDimHaloBT(16, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	//fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, a);
	fillRightFlux << <gridDim, blockDimHaloLR, 0, stream >> > (XParam.halowidth,false, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	//fillBot << <gridDim, blockDimHaloBT, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, a);
	fillTopFlux << <gridDim, blockDimHaloBT, 0, stream >> > (XParam.halowidth,false, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);

	CUDA_CHECK(hipStreamSynchronize(stream));

}
template void fillHaloTopRightGPU<double>(Param XParam, BlockP<double> XBlock, hipStream_t stream, double* z);
template void fillHaloTopRightGPU<float>(Param XParam, BlockP<float> XBlock, hipStream_t stream, float* z);


template <class T> void fillHalo(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, T*zb)
{
	
		
		std::thread t0(fillHaloC<T>,XParam, XBlock, Xev.h);
		std::thread t1(fillHaloC<T>,XParam, XBlock, Xev.zs);
		//std::thread t2(fillHaloF<T>,XParam,true, XBlock, Xev.u);
		//std::thread t3(fillHaloF<T>,XParam,true, XBlock, Xev.v);

		std::thread t2(fillHaloC<T>, XParam, XBlock, Xev.u);
		std::thread t3(fillHaloC<T>, XParam, XBlock, Xev.v);

		t0.join();
		t1.join();
		t2.join();
		t3.join();

		if (XParam.conserveElevation)
		{
			conserveElevation(XParam, XBlock, Xev, zb);
		}

		RecalculateZs(XParam, XBlock, Xev, zb);

		maskbnd(XParam, XBlock, Xev, zb);
	
}
template void fillHalo<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev, float *zb);
template void fillHalo<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev,double * zb);

template <class T> void fillHalo(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev)
{

	std::thread t0(fillHaloC<T>, XParam, XBlock, Xev.h);
	std::thread t1(fillHaloC<T>, XParam, XBlock, Xev.zs);
	std::thread t2(fillHaloF<T>, XParam, true, XBlock, Xev.u);
	std::thread t3(fillHaloF<T>, XParam, true, XBlock, Xev.v);

	t0.join();
	t1.join();
	t2.join();
	t3.join();

	
	//maskbnd(XParam, XBlock, Xev, zb);

}
template void fillHalo<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev);
template void fillHalo<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev);


template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev)
{
	const int num_streams = 4;

	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}


	fillHaloGPU(XParam, XBlock, streams[0], Xev.h);
	fillHaloGPU(XParam, XBlock, streams[1], Xev.zs);
	fillHaloGPU(XParam, XBlock, streams[2], Xev.u);
	fillHaloGPU(XParam, XBlock, streams[3], Xev.v);

	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}
}
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev);
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev);

template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev,T * zb)
{
	const int num_streams = 4;
	dim3 blockDimHalo(XParam.blkwidth, 1, 1);

	dim3 gridDim(XBlock.mask.nblk, 1, 1);
	
	dim3 blockDimfull(XParam.blkmemwidth, XParam.blkmemwidth, 1);
	dim3 gridDimfull(XParam.nblk, 1, 1);
	
	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}
	

	fillHaloGPU(XParam, XBlock, streams[0], Xev.h);
	fillHaloGPU(XParam, XBlock, streams[1], Xev.zs);
	fillHaloGPU(XParam, XBlock, streams[2], Xev.u);
	fillHaloGPU(XParam, XBlock, streams[3], Xev.v);
	CUDA_CHECK(hipDeviceSynchronize());

	if (XParam.conserveElevation)
	{
		conserveElevationGPU(XParam, XBlock, Xev, zb);
	}

	RecalculateZsGPU << < gridDimfull, blockDimfull, 0 >> > (XParam, XBlock, Xev, zb);
	CUDA_CHECK(hipDeviceSynchronize());


	maskbndGPUleft << <gridDim, blockDimHalo, 0 , streams[0] >> > (XParam, XBlock,  Xev, zb);
	maskbndGPUtop << <gridDim, blockDimHalo, 0, streams[1] >> > (XParam, XBlock, Xev, zb);
	maskbndGPUright << <gridDim, blockDimHalo, 0, streams[2] >> > (XParam, XBlock, Xev, zb);
	maskbndGPUtop << <gridDim, blockDimHalo, 0, streams[3] >> > (XParam, XBlock, Xev, zb);
	CUDA_CHECK(hipDeviceSynchronize());
	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}

	
}
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev,float *zb);
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev,double* zb);

template <class T> void fillHalo(Param XParam, BlockP<T> XBlock, GradientsP<T> Grad)
{
	
	/*
	std::thread t0(fillHaloF<T>,XParam, true, XBlock, Grad.dhdx);
	std::thread t1(fillHaloF<T>,XParam, true, XBlock, Grad.dudx);
	std::thread t2(fillHaloF<T>,XParam, true, XBlock, Grad.dvdx);
	std::thread t3(fillHaloF<T>,XParam, true, XBlock, Grad.dzsdx);

	std::thread t4(fillHaloF<T>,XParam, true, XBlock, Grad.dhdy);
	std::thread t5(fillHaloF<T>,XParam, true, XBlock, Grad.dudy);
	std::thread t6(fillHaloF<T>,XParam, true, XBlock, Grad.dvdy);
	std::thread t7(fillHaloF<T>,XParam, true, XBlock, Grad.dzsdy);

	*/
	std::thread t0(fillHaloC<T>, XParam, XBlock, Grad.dhdx);
	std::thread t1(fillHaloC<T>, XParam, XBlock, Grad.dudx);
	std::thread t2(fillHaloC<T>, XParam, XBlock, Grad.dvdx);
	std::thread t3(fillHaloC<T>, XParam, XBlock, Grad.dzsdx);

	std::thread t4(fillHaloC<T>, XParam, XBlock, Grad.dhdy);
	std::thread t5(fillHaloC<T>, XParam, XBlock, Grad.dudy);
	std::thread t6(fillHaloC<T>, XParam, XBlock, Grad.dvdy);
	std::thread t7(fillHaloC<T>, XParam, XBlock, Grad.dzsdy);

	t0.join();
	t1.join();
	t2.join();
	t3.join();
	t4.join();
	t5.join();
	t6.join();
	t7.join();

	
}
template void fillHalo<float>(Param XParam, BlockP<float> XBlock, GradientsP<float> Grad);
template void fillHalo<double>(Param XParam, BlockP<double> XBlock, GradientsP<double> Grad);

template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, GradientsP<T> Grad)
{
	const int num_streams = 8;

	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}
		
	fillHaloGPU(XParam, XBlock, streams[0], Grad.dhdx);
	fillHaloGPU(XParam, XBlock, streams[2], Grad.dudx);
	fillHaloGPU(XParam, XBlock, streams[3], Grad.dvdx);
	fillHaloGPU(XParam, XBlock, streams[4], Grad.dzsdx);

	fillHaloGPU(XParam, XBlock, streams[5], Grad.dhdy);
	fillHaloGPU(XParam, XBlock, streams[6], Grad.dudy);
	fillHaloGPU(XParam, XBlock, streams[7], Grad.dvdy);
	fillHaloGPU(XParam, XBlock, streams[1], Grad.dzsdy);

	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}
	
}
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, GradientsP<float> Grad);
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, GradientsP<double> Grad);


template <class T> void fillHalo(Param XParam, BlockP<T> XBlock, FluxP<T> Flux)
{
	
		
	std::thread t0(fillHaloTopRightC<T>,XParam, XBlock, Flux.Fhu);
	std::thread t1(fillHaloTopRightC<T>,XParam, XBlock, Flux.Fhv);
	std::thread t2(fillHaloTopRightC<T>,XParam, XBlock, Flux.Fqux);
	std::thread t3(fillHaloTopRightC<T>, XParam, XBlock, Flux.Fquy);

	std::thread t4(fillHaloTopRightC<T>, XParam, XBlock, Flux.Fqvx);
	std::thread t5(fillHaloTopRightC<T>, XParam, XBlock, Flux.Fqvy);
	std::thread t6(fillHaloTopRightC<T>, XParam, XBlock, Flux.Su);
	std::thread t7(fillHaloTopRightC<T>, XParam, XBlock, Flux.Sv);

	t0.join();
	t1.join();
	t2.join();
	t3.join();
	t4.join();
	t5.join();
	t6.join();
	t7.join();
	
}
template void fillHalo<float>(Param XParam, BlockP<float> XBlock, FluxP<float> Flux);
template void fillHalo<double>(Param XParam, BlockP<double> XBlock, FluxP<double> Flux);

template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, FluxP<T> Flux)
{
	const int num_streams = 8;

	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}

	fillHaloTopRightGPU(XParam, XBlock, streams[0], Flux.Fhu);
	fillHaloTopRightGPU(XParam, XBlock, streams[1], Flux.Fhv);
	fillHaloTopRightGPU(XParam, XBlock, streams[2], Flux.Fqux);
	fillHaloTopRightGPU(XParam, XBlock, streams[3], Flux.Fquy);

	fillHaloTopRightGPU(XParam, XBlock, streams[4], Flux.Fqvx);
	fillHaloTopRightGPU(XParam, XBlock, streams[5], Flux.Fqvy);
	fillHaloTopRightGPU(XParam, XBlock, streams[6], Flux.Su);
	fillHaloTopRightGPU(XParam, XBlock, streams[7], Flux.Sv);

	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}
	
}
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, FluxP<float> Flux);
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, FluxP<double> Flux);




template <class T> void fillLeft(Param XParam, int ib, BlockP<T> XBlock, T* &z)
{
	int jj,bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.LeftBot[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.LeftTop[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, -1, j, ib);
				jj = (j - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, XBlock.LeftTop[ib]);
				ir = memloc(XParam, (XParam.blkwidth - 2), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, XBlock.LeftTop[ib]);
				itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, XBlock.LeftTop[ib]);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[ XBlock.LeftBot[ib] ]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, -1, j, ib);
			read = memloc(XParam, (XParam.blkwidth - 1), j, XBlock.LeftBot[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.LeftBot[ib] ]> XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, -1, j, ib);

			jj = j * 2;
			bb = XBlock.LeftBot[ib];

			ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
			ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
			it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
			itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

			z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.LeftTop[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.LeftTop[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
				ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
				itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, -1, j, ib);

			T w1, w2, w3;
			

			int jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib?ceil(j * (T)0.5): ceil(j * (T)0.5)+ XParam.blkwidth/2;
			w1 = T(1.0 / 3.0);
			w2 = ceil(j * (T)0.5) * 2 > j ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(j * (T)0.5) * 2 > j ? T(0.5) : T(1.0 / 6.0);
						
			ii= memloc(XParam, 0, j, ib);
			ir= memloc(XParam, XParam.blkwidth-1, jj, XBlock.LeftBot[ib]);
			it = memloc(XParam, XParam.blkwidth-1, jj - 1, XBlock.LeftBot[ib]);
			//2 scenarios here ib is the rightbot neighbour of the leftbot block or ib is the righttop neighbour
			if (XBlock.RightBot[XBlock.LeftBot[ib]] == ib)
			{
				if (j == 0)
				{
					if (XBlock.BotRight[XBlock.LeftBot[ib]] == XBlock.LeftBot[ib]) // no botom of leftbot block
					{
						w3 = (T)0.5 * (1.0 - w1);
						w2 = w3;
						it = ir;

					}
					else if (XBlock.level[XBlock.BotRight[XBlock.LeftBot[ib]]] < XBlock.level[XBlock.LeftBot[ib]]) // exists but is coarser
					{
						w1 = T(4.0 / 10.0);
						w2 = T(5.0 / 10.0);
						w3 = T(1.0 / 10.0);
						it = memloc(XParam, XParam.blkwidth-1, XParam.blkwidth - 1, XBlock.BotRight[XBlock.LeftBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotRight[XBlock.LeftBot[ib]]] == XBlock.level[XBlock.LeftBot[ib]]) // exists with same level
					{
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.BotRight[XBlock.LeftBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotRight[XBlock.LeftBot[ib]]] > XBlock.level[XBlock.LeftBot[ib]]) // exists with higher level
					{
						w1 = T(1.0 / 4.0);
						w2 = T(1.0 / 2.0);
						w3 = T(1.0 / 4.0);
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.BotRight[XBlock.LeftBot[ib]]);
					}
					
					
				}
									
				
			}
			else//righttopleftif == ib
			{
				if (j == (XParam.blkwidth - 1))
				{
					if (XBlock.TopRight[XBlock.LeftTop[ib]] == XBlock.LeftTop[ib]) // no botom of leftbot block
					{
						w3 = 0.5*(1.0-w1);
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.TopRight[XBlock.LeftTop[ib]]] < XBlock.level[XBlock.LeftTop[ib]]) // exists but is coarser
					{
						w1 = 4.0 / 10.0;
						w2 = 1.0 / 10.0;
						w3 = 5.0 / 10.0;
						ir = memloc(XParam, XParam.blkwidth - 1,0, XBlock.TopRight[XBlock.LeftTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopRight[XBlock.LeftTop[ib]]] == XBlock.level[XBlock.LeftTop[ib]]) // exists with same level
					{
						ir = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.TopRight[XBlock.LeftTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopRight[XBlock.LeftTop[ib]]] > XBlock.level[XBlock.LeftTop[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.TopRight[XBlock.LeftTop[ib]]);
					}
				}
				//
			}


			z[write] = w1 * z[ii] + w2 * z[ir] + w3 * z[it];

			

		}
	}
	


}


template <class T> __global__ void fillLeft(int halowidth, int* active, int * level, int* leftbot, int * lefttop, int * rightbot, int* botright,int * topright, T * a)
{
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = 0;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int lev = level[ib];
	int LB = leftbot[ib];
	int LT = lefttop[ib];

	int RBLB = rightbot[LB];
	int BRLB = botright[LB];
	int TRLT = topright[LT];

	int levBRLB = level[BRLB];
	int levTRLT = level[TRLT];
	int levLB = level[LB];
	int levLT = level[LT];
	int write = memloc(halowidth, blkmemwidth, -1, iy, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;

	if (LB == ib)
	{
		if (iy < (blockDim.y / 2))
		{
			read = memloc(halowidth, blkmemwidth, 0, iy, ib);
			a_read = a[read];
		}
		else
		{
			if (LT == ib)
			{
				read = memloc(halowidth, blkmemwidth, 0, iy, ib);
				a_read = a[read];

			}
			else
			{
				
				jj = (iy - (blockDim.y / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj, LT);
				ir = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj, LT);
				it = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj + 1, LT);
				itr = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj + 1, LT);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
		
	}
	else if (levLB == lev )
	{
		read = memloc(halowidth, blkmemwidth, (blockDim.y - 1), iy, LB);
		a_read = a[read];
	}
	else if (levLB > lev)
	{
		if (iy < (blockDim.y / 2))
		{
			jj = iy * 2;
			ii = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj, LB);
			ir = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj, LB);
			it = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj + 1, LB);
			itr = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj + 1, LB);
			a_read= T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		else
		{
			if (LT == ib)
			{
				read = memloc(halowidth, blkmemwidth, 0, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;
								
				ii = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj, LT);
				ir = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj, LT);
				it = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj + 1, LT);
				itr = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj + 1, LT);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levLB < lev)
	{
		jj = RBLB==ib? ceil(iy * (T)0.5): ceil(iy * (T)0.5) + blockDim.y / 2;
		w1 = (T)1.0 / (T)3.0;
		w2 = ceil(iy * (T)0.5) * 2 > iy ? T(1.0 / 6.0) : T(0.5);
		w3 = ceil(iy * (T)0.5) * 2 > iy ? T(0.5) : T(1.0 / 6.0);

		ii = memloc(halowidth, blkmemwidth, 0, iy, ib);
		ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, jj, LB);
		it = memloc(halowidth, blkmemwidth, blockDim.y - 1, jj - 1, LB);
		if (RBLB == ib)
		{
			if (iy == 0)
			{
				if (BRLB == LB)
				{
					w3 = (T)0.5 * (1.0 - w1);
					w2 = w3;
					it = ir;
				}
				else if (levBRLB < levLB)
				{
					w1 = T(4.0 / 10.0);
					w2 = T(5.0 / 10.0);
					w3 = T(1.0 / 10.0);
					it = memloc(halowidth, blkmemwidth, blockDim.y - 1, blockDim.y - 1, BRLB);

				}
				else if (levBRLB == levLB)
				{
					it = memloc(halowidth, blkmemwidth, blockDim.y - 1, blockDim.y - 1, BRLB);
				}
				else if (levBRLB > levLB)
				{
					w1 = T(1.0 / 4.0);
					w2 = T(1.0 / 2.0);
					w3 = T(1.0 / 4.0);
					it = memloc(halowidth, blkmemwidth, blockDim.y - 1, blockDim.y - 1, BRLB);
				}
			}
		}
		else
		{
			if (iy == (blockDim.y - 1))
			{
				if (TRLT == LT)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					ir = it;
				}
				else if (levTRLT < levLT)
				{
					w1 = 4.0 / 10.0;
					w2 = 1.0 / 10.0;
					w3 = 5.0 / 10.0;
					ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, 0, TRLT);
				}
				else if (levTRLT == levLT)
				{
					ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, 0, TRLT);
				}
				else if (levTRLT > levLT)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, 0, TRLT);
					
				}
			}
		}
		a_read = w1 * a[ii] + w2 * a[ir] + w3 * a[it];
	}

	a[write] = a_read;
}
template __global__ void fillLeft<float>(int halowidth, int* active, int* level, int* leftbot, int* lefttop, int* rightbot, int* botright, int* topright, float* a);
template __global__ void fillLeft<double>(int halowidth, int* active, int* level, int* leftbot, int* lefttop, int* rightbot, int* botright, int* topright, double* a);


template <class T> void fillLeftFlux(Param XParam, bool doProlongation, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.LeftBot[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.LeftTop[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, -1, j, ib);
				jj = (j - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, XBlock.LeftTop[ib]);
				//ir = memloc(XParam, (XParam.blkwidth - 2), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, XBlock.LeftTop[ib]);
				//itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, XBlock.LeftTop[ib]);

				z[write] = T(0.5) * (z[ii]  + z[it]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.LeftBot[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, -1, j, ib);
			read = memloc(XParam, (XParam.blkwidth - 1), j, XBlock.LeftBot[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.LeftBot[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, -1, j, ib);

			jj = j * 2;
			bb = XBlock.LeftBot[ib];

			ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
			//ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
			it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
			//itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

			z[write] = T(0.5) * (z[ii] + z[it]);
		}
		//now find out aboy lefttop block
		if (XBlock.LeftTop[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.LeftTop[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
				//ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
				//itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

				z[write] = T(0.5) * (z[ii] + z[it]);
			}
		}

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, -1, j, ib);

			T w1, w2, w3;


			int jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib ? ceil(j * (T)0.5) : ceil(j * (T)0.5) + XParam.blkwidth / 2;
			

			ii = memloc(XParam, XParam.blkwidth - 1, jj, XBlock.LeftBot[ib]);
			if (doProlongation)
				z[write] = z[ii];


			
		}
	}



}



template <class T> void fillRight(Param XParam, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.RightBot[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, XParam.blkwidth-1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.RightTop[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, XParam.blkwidth - 1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];

				
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, XParam.blkwidth, j, ib);
				jj = (j - 8) * 2;
				ii = memloc(XParam, 0, jj, XBlock.RightTop[ib]);
				ir = memloc(XParam, 1, jj, XBlock.RightTop[ib]);
				it = memloc(XParam, 0, jj + 1, XBlock.RightTop[ib]);
				itr = memloc(XParam, 1, jj + 1, XBlock.RightTop[ib]);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.RightBot[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, XParam.blkwidth, j, ib);
			read = memloc(XParam, 0, j, XBlock.RightBot[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, XParam.blkwidth, j, ib);

			jj = j * 2;
			bb = XBlock.RightBot[ib];

			ii = memloc(XParam, 0, jj, bb);
			ir = memloc(XParam, 1, jj, bb);
			it = memloc(XParam, 0, jj + 1, bb);
			itr = memloc(XParam, 1, jj + 1, bb);

			z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.RightTop[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, XParam.blkwidth-1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.RightTop[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, 0, jj, bb);
				ir = memloc(XParam, 1, jj, bb);
				it = memloc(XParam, 0, jj + 1, bb);
				itr = memloc(XParam, 1, jj + 1, bb);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, XParam.blkwidth, j, ib);

			T w1, w2, w3;
			

			int jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? ceil(j * (T)0.5) : ceil(j * (T)0.5) + XParam.blkwidth / 2;
			w1 = 1.0 / 3.0;
			w2 = ceil(j * (T)0.5) * 2 > j ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(j * (T)0.5) * 2 > j ? T(0.5) : T(1.0 / 6.0);

			ii = memloc(XParam, XParam.blkwidth-1, j, ib);
			ir = memloc(XParam, 0, jj, XBlock.RightBot[ib]);
			it = memloc(XParam, 0, jj - 1, XBlock.RightBot[ib]);
			//2 scenarios here ib is the leftbot neighbour of the rightbot block or ib is the lefttop neighbour
			if (XBlock.LeftBot[XBlock.RightBot[ib]] == ib)
			{
				if (j == 0)
				{
					if (XBlock.BotLeft[XBlock.RightBot[ib]] == XBlock.RightBot[ib]) // no botom of leftbot block
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						it = ir;

					}
					else if (XBlock.level[XBlock.BotLeft[XBlock.RightBot[ib]]] < XBlock.level[XBlock.RightBot[ib]]) // exists but is coarser
					{
						w1 = 4.0 / 10.0;
						w2 = 5.0 / 10.0;
						w3 = 1.0 / 10.0;
						it = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.BotLeft[XBlock.RightBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotLeft[XBlock.RightBot[ib]]] == XBlock.level[XBlock.RightBot[ib]]) // exists with same level
					{
						it = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.BotLeft[XBlock.RightBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotLeft[XBlock.RightBot[ib]]] > XBlock.level[XBlock.RightBot[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						it = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.BotLeft[XBlock.RightBot[ib]]);
					}


				}


			}
			else//
			{
				if (j == (XParam.blkwidth - 1))
				{
					if (XBlock.TopLeft[XBlock.RightTop[ib]] == XBlock.RightTop[ib]) // no botom of leftbot block
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.TopLeft[XBlock.RightTop[ib]]] < XBlock.level[XBlock.RightTop[ib]]) // exists but is coarser
					{
						w1 = 4.0 / 10.0;
						w2 = 1.0 / 10.0;
						w3 = 5.0 / 10.0;
						ir = memloc(XParam, 0, 0, XBlock.TopLeft[XBlock.RightTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopLeft[XBlock.RightTop[ib]]] == XBlock.level[XBlock.RightTop[ib]]) // exists with same level
					{
						ir = memloc(XParam, 0, 0, XBlock.TopLeft[XBlock.RightTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopLeft[XBlock.RightTop[ib]]] > XBlock.level[XBlock.RightTop[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(XParam, 0, 0, XBlock.TopLeft[XBlock.RightTop[ib]]);
					}
				}
				//
			}


			z[write] = w1 * z[ii] + w2 * z[ir] + w3 * z[it];
		}
	}



}



template <class T> __global__ void fillRight(int halowidth, int* active, int* level, int * rightbot,int* righttop,int * leftbot,int*botleft,int* topleft, T* a)
{
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = blockDim.y - 1;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int RB = rightbot[ib];
	int RT = righttop[ib];
	int LB = leftbot[ib];
	int BL = botleft[ib];
	int LBRB = leftbot[RB];
	int TLRT = topleft[RT];
	int BLRB = botleft[RB];


	int lev = level[ib];
	int levRB = level[RB];
	int levRT = level[RT];
	int levBLRB = level[BLRB];
	int levTLRT = level[TLRT];

	int write = memloc(halowidth, blkmemwidth, blockDim.y, iy, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;


	if (RB == ib)
	{
		if (iy < (blockDim.y / 2))
		{
			read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
			a_read = a[read];
		}
		else
		{
			if (RT == ib)
			{
				read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
				ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
				it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
				itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levRB == lev)
	{
		read = memloc(halowidth, blkmemwidth, 0, iy, RB);
		a_read = a[read];
	}
	else if (levRB > lev)
	{
		if (iy < (blockDim.y / 2))
		{
			jj = iy * 2;


			ii = memloc(halowidth, blkmemwidth, 0, jj, RB);
			ir = memloc(halowidth, blkmemwidth, 1, jj, RB);
			it = memloc(halowidth, blkmemwidth, 0, jj + 1, RB);
			itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RB);

			a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		else
		{
			if (RT == ib)
			{
				read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;
				
				ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
				ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
				it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
				itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levRB < lev)
	{
		//
		jj = LBRB == ib ? ceil(iy * (T)0.5) : ceil(iy * (T)0.5) + blockDim.y / 2;
		w1 = 1.0 / 3.0;
		w2 = ceil(iy * (T)0.5) * 2 > iy ? T(1.0 / 6.0) : T(0.5);
		w3 = ceil(iy * (T)0.5) * 2 > iy ? T(0.5) : T(1.0 / 6.0);
		ii = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
		ir = memloc(halowidth, blkmemwidth, 0, jj, RB);
		it = memloc(halowidth, blkmemwidth, 0, jj - 1, RB);
		if (LBRB == ib)
		{
			if (iy == 0)
			{
				if (BLRB == RB)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					it = ir;
				}
				else if (levBLRB < levRB)
				{
					w1 = 4.0 / 10.0;
					w2 = 5.0 / 10.0;
					w3 = 1.0 / 10.0;
					it = memloc(halowidth, blkmemwidth, 0, blockDim.y - 1, BLRB);
				}
				else if (levBLRB == levRB)
				{
					it = memloc(halowidth, blkmemwidth, 0, blockDim.y - 1, BLRB);
				}
				else if (levBLRB > levRB)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					it = memloc(halowidth, blkmemwidth, 0, blockDim.y - 1, BLRB);
				}
			}
		}
		else
		{
			if (iy == (blockDim.y - 1))
			{
				if (TLRT == RT)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					ir = it;
				}
				else if (levTLRT < levRT)
				{
					w1 = 4.0 / 10.0;
					w2 = 1.0 / 10.0;
					w3 = 5.0 / 10.0;
					ir = memloc(halowidth, blkmemwidth, 0, 0, TLRT);
				}
				else if (levTLRT == levRT)
				{
					ir = memloc(halowidth, blkmemwidth, 0, 0, TLRT);
				}
				else if (levTLRT > levRT)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					ir = memloc(halowidth, blkmemwidth, 0, 0, TLRT);
				}
			}
		}

		a_read= w1 * a[ii] + w2 * a[ir] + w3 * a[it];
	}
	a[write] = a_read;
}

template __global__ void fillRight<float>(int halowidth, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, float* a);
template __global__ void fillRight<double>(int halowidth, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, double* a);

template <class T> void fillRightFlux(Param XParam, bool doProlongation, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.RightBot[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, XParam.blkwidth - 1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.RightTop[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, XParam.blkwidth - 1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, XParam.blkwidth, j, ib);
				jj = (j - 8) * 2;
				ii = memloc(XParam, 0, jj, XBlock.RightTop[ib]);
				//ir = memloc(XParam, 1, jj, XBlock.RightTop[ib]);
				it = memloc(XParam, 0, jj + 1, XBlock.RightTop[ib]);
				//itr = memloc(XParam, 1, jj + 1, XBlock.RightTop[ib]);

				z[write] = T(0.5) * (z[ii] + z[it]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.RightBot[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, XParam.blkwidth, j, ib);
			read = memloc(XParam, 0, j, XBlock.RightBot[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, XParam.blkwidth, j, ib);

			jj = j * 2;
			bb = XBlock.RightBot[ib];

			ii = memloc(XParam, 0, jj, bb);
			//ir = memloc(XParam, 1, jj, bb);
			it = memloc(XParam, 0, jj + 1, bb);
			//itr = memloc(XParam, 1, jj + 1, bb);

			//z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			z[write] = T(0.5) * (z[ii] + z[it]);
		}
		//now find out aboy lefttop block
		if (XBlock.RightTop[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, XParam.blkwidth - 1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.RightTop[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, 0, jj, bb);
				//ir = memloc(XParam, 1, jj, bb);
				it = memloc(XParam, 0, jj + 1, bb);
				//itr = memloc(XParam, 1, jj + 1, bb);
				z[write] = T(0.5) * (z[ii] + z[it]);
				//z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, XParam.blkwidth, j, ib);


			int jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? floor(j * (T)0.5) : floor(j * (T)0.5) + XParam.blkwidth / 2;

			ii = memloc(XParam, 0, jj, XBlock.RightBot[ib]);
			if (doProlongation)
				z[write] = z[ii];
		}
	}



}

template void fillRightFlux<float>(Param XParam, bool doProlongation, int ib, BlockP<float> XBlock, float*& z);
template void fillRightFlux<double>(Param XParam, bool doProlongation, int ib, BlockP<double> XBlock, double*& z);



template <class T> __global__ void fillRightFlux(int halowidth, bool doProlongation, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, T* a)
{
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = blockDim.y - 1;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int RB = rightbot[ib];
	int RT = righttop[ib];
	int LB = leftbot[ib];
	int BL = botleft[ib];
	int LBRB = leftbot[RB];
	int TLRT = topleft[RT];
	int BLRB = botleft[RB];


	int lev = level[ib];
	int levRB = level[RB];
	int levRT = level[RT];
	int levBLRB = level[BLRB];
	int levTLRT = level[TLRT];

	int write = memloc(halowidth, blkmemwidth, blockDim.y, iy, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;


	if (RB == ib)
	{
		if (iy < (blockDim.y / 2))
		{
			read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
			a_read = a[read];
		}
		else
		{
			if (RT == ib)
			{
				read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
				//ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
				it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
				//itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

				a_read = T(0.5) * (a[ii] + a[it]);
			}
		}
	}
	else if (levRB == lev)
	{
		read = memloc(halowidth, blkmemwidth, 0, iy, RB);
		a_read = a[read];
	}
	else if (levRB > lev)
	{
		if (iy < (blockDim.y / 2))
		{
			jj = iy * 2;


			ii = memloc(halowidth, blkmemwidth, 0, jj, RB);
			//ir = memloc(halowidth, blkmemwidth, 1, jj, RB);
			it = memloc(halowidth, blkmemwidth, 0, jj + 1, RB);
			//itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RB);

			a_read = T(0.5) * (a[ii] + a[it]);
		}
		else
		{
			if (RT == ib)
			{
				read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;

				ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
				//ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
				it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
				//itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

				a_read = T(0.5) * (a[ii] + a[it] );
			}
		}
	}
	else if (levRB < lev)
	{
		//
		jj = LBRB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + blockDim.y / 2;
		
		
		ir = memloc(halowidth, blkmemwidth, 0, jj, RB);
		
		if (doProlongation)
			a_read = a[ir];
		
		else
			a_read = a[write];
	}
	a[write] = a_read;
}
template __global__ void fillRightFlux<float>(int halowidth, bool doProlongation, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, float* a);
template __global__ void fillRightFlux<double>(int halowidth, bool doProlongation, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, double* a);



template <class T> void fillBot(Param XParam, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.BotLeft[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.BotRight[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam,j, -1, ib);
				jj = (j - 8) * 2;
				ii = memloc(XParam,jj, (XParam.blkwidth - 1), XBlock.BotRight[ib]);
				ir = memloc(XParam,jj, (XParam.blkwidth - 2), XBlock.BotRight[ib]);
				it = memloc(XParam,jj+1, (XParam.blkwidth - 1), XBlock.BotRight[ib]);
				itr = memloc(XParam,jj+1, (XParam.blkwidth - 2), XBlock.BotRight[ib]);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.BotLeft[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam,j, -1, ib);
			read = memloc(XParam, j, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, j, -1, ib);

			jj = j * 2;
			bb = XBlock.BotLeft[ib];

			ii = memloc(XParam, jj, (XParam.blkwidth - 1), bb);
			ir = memloc(XParam, jj, (XParam.blkwidth - 2), bb);
			it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), bb);
			itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), bb);

			z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
		}
		//now find out aboy botright block
		if (XBlock.BotRight[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.BotRight[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, jj, (XParam.blkwidth - 1), bb);
				ir = memloc(XParam, jj, (XParam.blkwidth - 2), bb);
				it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), bb);
				itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), bb);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, j, -1, ib);

			T w1, w2, w3;
			

			int jj = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? ceil(j * (T)0.5) : ceil(j * (T)0.5) + XParam.blkwidth / 2;
			w1 = 1.0 / 3.0;
			w2 = ceil(j * (T)0.5) * 2 > j ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(j * (T)0.5) * 2 > j ? T(0.5) : T(1.0 / 6.0);

			ii = memloc(XParam, j, 0, ib);
			ir = memloc(XParam, jj, XParam.blkwidth - 1, XBlock.BotLeft[ib]);
			it = memloc(XParam, jj -1, XParam.blkwidth - 1, XBlock.BotLeft[ib]);
			//2 scenarios here ib is the rightbot neighbour of the leftbot block or ib is the righttop neighbour
			if (XBlock.TopLeft[XBlock.BotLeft[ib]] == ib)
			{
				if (j == 0)
				{
					if (XBlock.LeftTop[XBlock.BotLeft[ib]] == XBlock.BotLeft[ib]) // no botom of leftbot block
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						it = ir;

					}
					else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] < XBlock.level[XBlock.BotLeft[ib]]) // exists but is coarser
					{
						w1 = 4.0 / 10.0;
						w2 = 5.0 / 10.0;
						w3 = 1.0 / 10.0;
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
					}
					else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] == XBlock.level[XBlock.BotLeft[ib]]) // exists with same level
					{
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
					}
					else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] > XBlock.level[XBlock.BotLeft[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
					}


				}


			}
			else//righttopleftif == ib
			{
				if (j == (XParam.blkwidth - 1))
				{
					if (XBlock.RightTop[XBlock.BotRight[ib]] == XBlock.BotRight[ib]) // no botom of leftbot block
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.RightTop[XBlock.BotRight[ib]]] < XBlock.level[XBlock.BotRight[ib]]) // exists but is coarser
					{
						w1 = 4.0 / 10.0;
						w2 = 1.0 / 10.0;
						w3 = 5.0 / 10.0;
						ir = memloc(XParam, 0,XParam.blkwidth - 1, XBlock.RightTop[XBlock.BotRight[ib]]);
					}
					else if (XBlock.level[XBlock.RightTop[XBlock.BotRight[ib]]] == XBlock.level[XBlock.BotRight[ib]]) // exists with same level
					{
						ir = memloc(XParam,0, XParam.blkwidth - 1, XBlock.RightTop[XBlock.BotRight[ib]]);
					}
					else if (XBlock.level[XBlock.RightTop[XBlock.BotRight[ib]]] > XBlock.level[XBlock.BotRight[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(XParam,0,XParam.blkwidth - 1, XBlock.RightTop[XBlock.BotRight[ib]]);
					}
				}
				//
			}


			z[write] = w1 * z[ii] + w2 * z[ir] + w3 * z[it];
		}
	}



}


template <class T> __global__ void fillBot(int halowidth, int* active, int* level, int* botleft, int* botright, int* topleft, int* lefttop, int* righttop, T* a)
{
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = 0;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int BL = botleft[ib];
	int BR = botright[ib];
	int TLBL = topleft[BL];
	int LTBL = lefttop[BL];
	int RTBR = righttop[BR];

	int lev = level[ib];
	int levBL = level[BL];
	int levBR = level[BR];
	int levLTBL = level[LTBL];
	int levRTBR = level[RTBR];

	int write = memloc(halowidth, blkmemwidth, ix, -1, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;
	if (BL == ib)
	{
		if (ix < (blockDim.x / 2))
		{
			read = memloc(halowidth, blkmemwidth, ix, 0, ib);
			a_read = a[read];
		}
		else
		{
			if (BR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, 0, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x/2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 1), BR);
				ir = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 2), BR);
				it = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 1), BR);
				itr = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 2), BR);
				a_read = T(0.25)* (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levBL == lev)
	{
		read = memloc(halowidth, blkmemwidth, ix, (blockDim.x - 1), BL);
		a_read = a[read];
	}
	else if (levBL > lev)
	{
		if (ix < (blockDim.x / 2))
		{
			jj = ix * 2;
			ii = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 1), BL);
			ir = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 2), BL);
			it = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 1), BL);
			itr = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 2), BL);
			a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		else
		{
			if (BR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, 0, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x/2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 1), BR);
				ir = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 2), BR);
				it = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 1), BR);
				itr = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 2), BR);
				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levBL < lev)
	{
		jj = TLBL == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + blockDim.x / 2;
		w1 = 1.0 / 3.0;
		w2 = ceil(ix * (T)0.5) * 2 > ix ? T(1.0 / 6.0) : T(0.5);
		w3 = ceil(ix * (T)0.5) * 2 > ix ? T(0.5) : T(1.0 / 6.0);

		ii = memloc(halowidth, blkmemwidth, ix, 0, ib);
		ir = memloc(halowidth, blkmemwidth, jj, blockDim.x - 1, BL);
		it = memloc(halowidth, blkmemwidth, jj - 1, blockDim.x - 1, BL);

		if (TLBL == ib)
		{
			if (ix == 0)
			{
				if (LTBL == BL)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					it = ir;
				}
				else if (levLTBL < levBL)
				{
					w1 = 4.0 / 10.0;
					w2 = 5.0 / 10.0;
					w3 = 1.0 / 10.0;
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, LTBL);
				}
				else if (levLTBL == levBL)
				{
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, LTBL);
				}
				else if (levLTBL > levBL)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, LTBL);
				}
			}
		}
		else
		{
			if (ix == (blockDim.x - 1))
			{
				if (RTBR == BR)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					ir = it;
				}
				else if (levRTBR < levBR)
				{
					w1 = 4.0 / 10.0;
					w2 = 1.0 / 10.0;
					w3 = 5.0 / 10.0;
					ir = memloc(halowidth, blkmemwidth,0, blockDim.x - 1, RTBR);
				}
				else if (levRTBR == levBR)
				{
					ir = memloc(halowidth, blkmemwidth,0, blockDim.x - 1, RTBR);
				}
				else if (levRTBR > levBR)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					ir = memloc(halowidth, blkmemwidth, 0, blockDim.x - 1, RTBR);
				}
			}
		}
		a_read = w1 * a[ii] + w2 * a[ir] + w3 * a[it];
	}
	a[write] = a_read;

}
template __global__ void fillBot<float>(int halowidth, int* active, int* level, int* botleft, int* botright, int* topleft, int* lefttop, int* righttop, float* a);
template __global__ void fillBot<double>(int halowidth, int* active, int* level, int* botleft, int* botright, int* topleft, int* lefttop, int* righttop, double* a);

template <class T> void fillBotFlux(Param XParam, bool doProlongation, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.BotLeft[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.BotRight[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, j, -1, ib);
				jj = (j - 8) * 2;
				ii = memloc(XParam, jj, (XParam.blkwidth - 1), XBlock.BotRight[ib]);
				//ir = memloc(XParam, jj, (XParam.blkwidth - 2), XBlock.BotRight[ib]);
				it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), XBlock.BotRight[ib]);
				//itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), XBlock.BotRight[ib]);

				z[write] = T(0.5) * (z[ii] + z[it] );

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.BotLeft[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, j, -1, ib);
			read = memloc(XParam, j, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, j, -1, ib);

			jj = j * 2;
			bb = XBlock.BotLeft[ib];

			ii = memloc(XParam, jj, (XParam.blkwidth - 1), bb);
			//ir = memloc(XParam, jj, (XParam.blkwidth - 2), bb);
			it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), bb);
			//itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), bb);

			z[write] = T(0.5) * (z[ii] + z[it]);
		}
		//now find out aboy botright block
		if (XBlock.BotRight[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.BotRight[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, jj, (XParam.blkwidth - 1), bb);
				//ir = memloc(XParam, jj, (XParam.blkwidth - 2), bb);
				it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), bb);
				//itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), bb);

				z[write] = T(0.5) * (z[ii]  + z[it] );
			}
		}

	}
	else if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, j, -1, ib);

			T w1, w2, w3;


			int jj = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? ceil(j * (T)0.5) : ceil(j * (T)0.5) + XParam.blkwidth / 2;
			

			//ii = memloc(XParam, j, 0, ib);
			ir = memloc(XParam, jj, XParam.blkwidth - 1, XBlock.BotLeft[ib]);
			if(doProlongation)
				z[write] = z[ir];
		}
	}



}

template <class T> void fillTop(Param XParam, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.TopLeft[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam,j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam,j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.TopRight[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, j, XParam.blkwidth, ib);
				jj = (j - (XParam.blkwidth / 2)) * 2;
				ii = memloc(XParam, jj, 0, XBlock.TopRight[ib]);
				ir = memloc(XParam, jj, 1, XBlock.TopRight[ib]);
				it = memloc(XParam, jj + 1, 0, XBlock.TopRight[ib]);
				itr = memloc(XParam, jj + 1, 1, XBlock.TopRight[ib]);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.TopLeft[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, j, XParam.blkwidth, ib);
			read = memloc(XParam, j, 0, XBlock.TopLeft[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, j, XParam.blkwidth, ib);

			jj = j * 2;
			bb = XBlock.TopLeft[ib];

			ii = memloc(XParam,jj, 0, bb);
			ir = memloc(XParam,jj, 1, bb);
			it = memloc(XParam,jj + 1, 0, bb);
			itr = memloc(XParam,jj + 1, 1, bb);

			z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.TopRight[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam,j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - (XParam.blkwidth / 2)) * 2;
				bb = XBlock.TopRight[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j , XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam,jj, 0, bb);
				ir = memloc(XParam,jj, 1, bb);
				it = memloc(XParam,jj + 1, 0, bb);
				itr = memloc(XParam,jj + 1, 1, bb);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam,j, XParam.blkwidth, ib);

			T w1, w2, w3;
			

			int jj = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? ceil(j * (T)0.5) : ceil(j * (T)0.5) + XParam.blkwidth / 2;
			w1 = 1.0 / 3.0;
			w2 = ceil(j * (T)0.5) * 2 > j ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(j * (T)0.5) * 2 > j ? T(0.5) : T(1.0 / 6.0);

			ii = memloc(XParam,j, XParam.blkwidth - 1, ib);
			ir = memloc(XParam,jj, 0,  XBlock.TopLeft[ib]);
			it = memloc(XParam,jj-1, 0, XBlock.TopLeft[ib]);
			//2 scenarios here ib is the leftbot neighbour of the rightbot block or ib is the lefttop neighbour
			if (XBlock.BotLeft[XBlock.TopLeft[ib]] == ib)
			{
				if (j == 0)
				{
					if (XBlock.LeftBot[XBlock.TopLeft[ib]] == XBlock.TopLeft[ib]) // no botom of leftbot block
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						it = ir;

					}
					else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] < XBlock.level[XBlock.TopLeft[ib]]) // exists but is coarser
					{
						w1 = 4.0 / 10.0;
						w2 = 5.0 / 10.0;
						w3 = 1.0 / 10.0;
						it = memloc(XParam, XParam.blkwidth - 1,0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
					}
					else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] == XBlock.level[XBlock.TopLeft[ib]]) // exists with same level
					{
						it = memloc(XParam,  XParam.blkwidth - 1,0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
					}
					else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] > XBlock.level[XBlock.TopLeft[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						it = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
					}


				}


			}
			else//
			{
				if (j == (XParam.blkwidth - 1))
				{
					if (XBlock.RightBot[XBlock.TopRight[ib]] == XBlock.TopRight[ib]) // no botom of leftbot block
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] < XBlock.level[XBlock.TopRight[ib]]) // exists but is coarser
					{
						w1 = 4.0 / 10.0;
						w2 = 1.0 / 10.0;
						w3 = 5.0 / 10.0;
						ir = memloc(XParam, 0, 0, XBlock.RightBot[XBlock.TopRight[ib]]);
					}
					else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] == XBlock.level[XBlock.TopRight[ib]]) // exists with same level
					{
						ir = memloc(XParam, 0, 0, XBlock.RightBot[XBlock.TopRight[ib]]);
					}
					else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] > XBlock.level[XBlock.TopRight[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(XParam, 0,0, XBlock.RightBot[XBlock.TopRight[ib]]);
					}
				}
				//
			}


			z[write] = w1 * z[ii] + w2 * z[ir] + w3 * z[it];
		}
	}



}

template <class T> __global__ void fillTop(int halowidth, int* active, int* level,int * topleft, int * topright,int * botleft, int* leftbot, int* rightbot,  T* a)
{
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = blockDim.x-1;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int TL = topleft[ib];
	int TR = topright[ib];
	int LBTL = leftbot[TL];
	int BLTL = botleft[TL];
	int RBTR = rightbot[TR];


	int lev = level[ib];
	int levTL = level[TL];
	int levTR = level[TR];
	int levLBTL = level[LBTL];
	int levRBTR = level[RBTR];

	int write = memloc(halowidth, blkmemwidth, ix, blockDim.x, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;

	if (TL == ib)
	{
		if (ix < (blockDim.x / 2))
		{
			read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
			a_read = a[read];
		}
		else
		{
			if (TR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
				ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
				it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
				itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levTL == lev)
	{
		read = memloc(halowidth, blkmemwidth, ix, 0, TL);
		a_read = a[read];
	}
	else if (levTL > lev)
	{
		if (ix < (blockDim.x / 2))
		{
			jj = ix * 2;
			

			ii = memloc(halowidth, blkmemwidth, jj, 0, TL);
			ir = memloc(halowidth, blkmemwidth, jj, 1, TL);
			it = memloc(halowidth, blkmemwidth, jj + 1, 0, TL);
			itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TL);
			a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		else
		{
			if (TR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, blockDim.x-1, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
				ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
				it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
				itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);
				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levTL < lev)
	{
		jj = BLTL == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + blockDim.x / 2;
		w1 = 1.0 / 3.0;
		w2 = ceil(ix * (T)0.5) * 2 > ix ? T(1.0 / 6.0) : T(0.5);
		w3 = ceil(ix * (T)0.5) * 2 > ix ? T(0.5) : T(1.0 / 6.0);
		ii = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
		ir = memloc(halowidth, blkmemwidth, jj, 0, TL);
		it = memloc(halowidth, blkmemwidth, jj - 1, 0, TL);
		if (BLTL == ib)
		{
			if (ix == 0)
			{
				if (LBTL == TL)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					it = ir;
				}
				else if (levLBTL < levTL)
				{
					w1 = 4.0 / 10.0;
					w2 = 5.0 / 10.0;
					w3 = 1.0 / 10.0;
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, LBTL);
				}
				else if (levLBTL == levTL)
				{
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, LBTL);
				}
				else if (levLBTL > levTL)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, LBTL);
				}
			}
		}
		else
		{
			if (ix == blockDim.x - 1)
			{
				if (RBTR == TR)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					ir = it;
				}
				else if (levRBTR < levTR)
				{
					w1 = 4.0 / 10.0;
					w2 = 1.0 / 10.0;
					w3 = 5.0 / 10.0;
					ir = memloc(halowidth, blkmemwidth, 0, 0, RBTR);
				}
				else if (levRBTR == levTR)
				{
					ir = memloc(halowidth, blkmemwidth, 0, 0, RBTR);
				}
				else if (levRBTR > levTR)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					ir = memloc(halowidth, blkmemwidth,0, 0, RBTR);
				}
			}
		}
		a_read= w1 * a[ii] + w2 * a[ir] + w3 * a[it];
	}
	a[write] = a_read;
}

template __global__ void fillTop<float>(int halowidth, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, float* a);
template __global__ void fillTop<double>(int halowidth, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, double* a);

template <class T> void fillTopFlux(Param XParam, bool doProlongation, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.TopLeft[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.TopRight[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, j, XParam.blkwidth, ib);
				jj = (j - (XParam.blkwidth / 2)) * 2;
				ii = memloc(XParam, jj, 0, XBlock.TopRight[ib]);
				//ir = memloc(XParam, jj, 1, XBlock.TopRight[ib]);
				it = memloc(XParam, jj + 1, 0, XBlock.TopRight[ib]);
				//itr = memloc(XParam, jj + 1, 1, XBlock.TopRight[ib]);

				z[write] = T(0.5) * (z[ii] + z[it] );

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.TopLeft[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, j, XParam.blkwidth, ib);
			read = memloc(XParam, j, 0, XBlock.TopLeft[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, j, XParam.blkwidth, ib);

			jj = j * 2;
			bb = XBlock.TopLeft[ib];

			ii = memloc(XParam, jj, 0, bb);
			//ir = memloc(XParam, jj, 1, bb);
			it = memloc(XParam, jj + 1, 0, bb);
			//itr = memloc(XParam, jj + 1, 1, bb);

			z[write] = T(0.5) * (z[ii]  + z[it]);
		}
		//now find out aboy lefttop block
		if (XBlock.TopRight[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - (XParam.blkwidth / 2)) * 2;
				bb = XBlock.TopRight[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, jj, 0, bb);
				//ir = memloc(XParam, jj, 1, bb);
				it = memloc(XParam, jj + 1, 0, bb);
				//itr = memloc(XParam, jj + 1, 1, bb);

				z[write] = T(0.5) * (z[ii]  + z[it]);
			}
		}

	}
	else if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, j, XParam.blkwidth, ib);
			int jj = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? floor(j * (T)0.5) : floor(j * (T)0.5) + XParam.blkwidth / 2;
						
			ir = memloc(XParam, jj, 0, XBlock.TopLeft[ib]);
			
			if (doProlongation)
				z[write] = z[ir];

		}
	}



}
template void fillTopFlux<float>(Param XParam, bool doProlongation, int ib, BlockP<float> XBlock, float*& z);
template void fillTopFlux<double>(Param XParam, bool doProlongation, int ib, BlockP<double> XBlock, double*& z);

template <class T> __global__ void fillTopFlux(int halowidth, bool doProlongation, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, T* a)
{
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = blockDim.x - 1;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int TL = topleft[ib];
	int TR = topright[ib];
	int LBTL = leftbot[TL];
	int BLTL = botleft[TL];
	int RBTR = rightbot[TR];


	int lev = level[ib];
	int levTL = level[TL];
	int levTR = level[TR];
	int levLBTL = level[LBTL];
	int levRBTR = level[RBTR];

	int write = memloc(halowidth, blkmemwidth, ix, blockDim.x, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;

	if (TL == ib)
	{
		if (ix < (blockDim.x / 2))
		{
			read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
			a_read = a[read];
		}
		else
		{
			if (TR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
				//ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
				it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
				//itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);

				a_read = T(0.5) * (a[ii]  + a[it] );
			}
		}
	}
	else if (levTL == lev)
	{
		read = memloc(halowidth, blkmemwidth, ix, 0, TL);
		a_read = a[read];
	}
	else if (levTL > lev)
	{
		if (ix < (blockDim.x / 2))
		{
			jj = ix * 2;


			ii = memloc(halowidth, blkmemwidth, jj, 0, TL);
			//ir = memloc(halowidth, blkmemwidth, jj, 1, TL);
			it = memloc(halowidth, blkmemwidth, jj + 1, 0, TL);
			//itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TL);
			a_read = T(0.5) * (a[ii] + a[it]);
		}
		else
		{
			if (TR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
				//ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
				it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
				//itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);
				a_read = T(0.5) * (a[ii] + a[it]);
			}
		}
	}
	else if (levTL < lev)
	{
		jj = BLTL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + blockDim.x / 2;
		
		ir = memloc(halowidth, blkmemwidth, jj, 0, TL);
		if (doProlongation)
			a_read = a[ir];
		else
			a_read =  a[write];
	}
	a[write] = a_read;
}

template __global__ void fillTopFlux<float>(int halowidth, bool doProlongation, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, float* a);
template __global__ void fillTopFlux<double>(int halowidth, bool doProlongation, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, double* a);



template <class T> void fillCorners(Param XParam, BlockP<T> XBlock, T*& z)
{
	int ib;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillCorners(XParam, ib, XBlock, z);
		
	}

}
template void fillCorners<float>(Param XParam, BlockP<float> XBlock, float*& z);
template void fillCorners<double>(Param XParam, BlockP<double> XBlock, double*& z);


template <class T> void fillCorners(Param XParam, BlockP<T> XBlock, EvolvingP<T>& Xev)
{
	int ib;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillCorners(XParam, ib, XBlock, Xev.h);
		fillCorners(XParam, ib, XBlock, Xev.zs);
		fillCorners(XParam, ib, XBlock, Xev.u);
		fillCorners(XParam, ib, XBlock, Xev.v);
	}

}
template void fillCorners<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float>& Xev);
template void fillCorners<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double>& Xev);



template <class T> void fillCorners(Param XParam, int ib, BlockP<T> XBlock, T*& z)
{
	// Run only this function after the filling the other bit of halo (i.e. fctn fillleft...)
	// Most of the time the cormers are not needed. they are when refining a cell! 

	
	T zz;
	int write;
	int ii, ir, it, itr;


	// Bottom left corner
	write = memloc(XParam, -1, -1, ib);
	//check that there is a block there and if there is calculate the value depending on the level of that block
	if (XBlock.LeftTop[XBlock.BotLeft[ib]] == XBlock.BotLeft[ib]) // There is no block
	{
		zz = T(0.5) * (z[memloc(XParam, -1, 0, ib)] + z[memloc(XParam, 0, -1, ib)]);
	}
	else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] == XBlock.level[ib])
	{
		zz = z[memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]])];
	}
	else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] > XBlock.level[ib])
	{
		ii = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
		ir = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 2, XBlock.LeftTop[XBlock.BotLeft[ib]]);
		it = memloc(XParam, XParam.blkwidth - 2, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
		itr = memloc(XParam, XParam.blkwidth - 2, XParam.blkwidth - 2, XBlock.LeftTop[XBlock.BotLeft[ib]]);

		zz = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
	}
	else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] < XBlock.level[ib])
	{
		ii = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
		ir = memloc(XParam, - 1, 0, ib);
		it = memloc(XParam,0, - 1, ib);
		zz = T(0.5) * z[ii] + T(0.25) * (z[ir] + z[it]);
	}

	z[write] = zz;

	// Top Left corner
	write = memloc(XParam, -1, XParam.blkwidth, ib);
	//check that there is a block there and if there is calculate the value depending on the level of that block
	if (XBlock.LeftBot[XBlock.TopLeft[ib]] == XBlock.TopLeft[ib]) // There is no block
	{
		zz = T(0.5) * (z[memloc(XParam, -1, XParam.blkwidth-1, ib)] + z[memloc(XParam, 0, XParam.blkwidth, ib)]);
	}
	else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] == XBlock.level[ib])
	{
		zz = z[memloc(XParam, XParam.blkwidth - 1, 0, XBlock.LeftBot[XBlock.TopLeft[ib]])];
	}
	else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] > XBlock.level[ib])
	{
		ii = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		ir = memloc(XParam, XParam.blkwidth - 1, 1, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		it = memloc(XParam, XParam.blkwidth - 2, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		itr = memloc(XParam, XParam.blkwidth - 2, 1, XBlock.LeftBot[XBlock.TopLeft[ib]]);

		zz = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
	}
	else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] < XBlock.level[ib])
	{
		ii = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		ir = memloc(XParam, -1, XParam.blkwidth - 1, ib);
		it = memloc(XParam, 0, XParam.blkwidth, ib);
		zz = T(0.5) * z[ii] + T(0.25) * (z[ir] + z[it]);
	}

	z[write] = zz;

	//Top Right corner
	write = memloc(XParam, XParam.blkwidth, XParam.blkwidth, ib);
	//check that there is a block there and if there is calculate the value depending on the level of that block
	if (XBlock.RightBot[XBlock.TopRight[ib]] == XBlock.TopRight[ib]) // There is no block
	{
		zz = T(0.5) * (z[memloc(XParam, XParam.blkwidth, XParam.blkwidth - 1, ib)] + z[memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth, ib)]);
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] == XBlock.level[ib])
	{
		zz = z[memloc(XParam, 0, 0, XBlock.RightBot[XBlock.TopRight[ib]])];
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] > XBlock.level[ib])
	{
		ii = memloc(XParam, 0, 0, XBlock.RightBot[XBlock.TopRight[ib]]);
		ir = memloc(XParam, 0, 1, XBlock.RightBot[XBlock.TopRight[ib]]);
		it = memloc(XParam, 1, 0, XBlock.RightBot[XBlock.TopRight[ib]]);
		itr = memloc(XParam, 1, 1, XBlock.RightBot[XBlock.TopRight[ib]]);

		zz = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
	}
	else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] < XBlock.level[ib])
	{
		ii = memloc(XParam, 0, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		ir = memloc(XParam, XParam.blkwidth, XParam.blkwidth - 1, ib);
		it = memloc(XParam, XParam.blkwidth-1, XParam.blkwidth, ib);
		zz = T(0.5) * z[ii] + T(0.25) * ( z[ir] +  z[it]);
	}

	z[write] = zz;

	//Bot Right corner
	write = memloc(XParam, XParam.blkwidth, -1, ib);
	//check that there is a block there and if there is calculate the value depending on the level of that block
	if (XBlock.RightBot[XBlock.BotRight[ib]] == XBlock.BotRight[ib]) // There is no block
	{
		zz = T(0.5) * (z[memloc(XParam, XParam.blkwidth-1, - 1, ib)] + z[memloc(XParam, XParam.blkwidth , 0, ib)]);
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.BotRight[ib]]] == XBlock.level[ib])
	{
		zz = z[memloc(XParam, 0, XParam.blkwidth - 1, XBlock.RightBot[XBlock.BotRight[ib]])];
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.BotRight[ib]]] > XBlock.level[ib])
	{
		ii = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.RightBot[XBlock.BotRight[ib]]);
		ir = memloc(XParam, 0, XParam.blkwidth - 2, XBlock.RightBot[XBlock.BotRight[ib]]);
		it = memloc(XParam, 1, XParam.blkwidth - 1, XBlock.RightBot[XBlock.BotRight[ib]]);
		itr = memloc(XParam, 1, XParam.blkwidth - 2, XBlock.RightBot[XBlock.BotRight[ib]]);

		zz = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.BotRight[ib]]] < XBlock.level[ib])
	{
		ii = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		ir = memloc(XParam, XParam.blkwidth - 1, -1, ib);
		it = memloc(XParam, XParam.blkwidth, 0, ib);
		zz = T(0.5) * z[ii] + T(0.25) * (z[ir] + z[it]);
	}

	z[write] = zz;

}
template void fillCorners<float>(Param XParam, int ib, BlockP<float> XBlock, float*& z);
template void fillCorners<double>(Param XParam, int ib, BlockP<double> XBlock, double*& z);

