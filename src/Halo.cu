#include "hip/hip_runtime.h"
﻿#include "Halo.h"


/*! \fn void void fillHaloD(Param XParam, int ib, BlockP<T> XBlock, T* z)
* \brief Wrapping function for calculating halos on CPU on every side of a block of a single variable
* 
* ## Description
* This fuction is a wraping fuction of the halo functions for CPU. It is called from another wraping function to keep things clean.
* In a sense this is the third (and last) layer of wrapping
* 
*/
template <class T> void fillHaloD(Param XParam, int ib, BlockP<T> XBlock, T* z)
{
	

	fillLeft(XParam, ib, XBlock, z);
	fillRight(XParam, ib, XBlock, z);
	fillTop(XParam, ib, XBlock, z);
	fillBot(XParam, ib, XBlock, z);
	//fill bot
	//fill top
	

}
template void fillHaloD<double>(Param XParam, int ib, BlockP<double> XBlock, double* z);
template void fillHaloD<float>(Param XParam, int ib, BlockP<float> XBlock, float* z);

/*! \fn void fillHaloC(Param XParam, BlockP<T> XBlock, T* z)
* \brief Wrapping function for calculating halos for each block of a single variable on CPU.
*
* ## Description
* This function is a wraping fuction of the halo functions on CPU. It is called from the main Halo CPU function.
* This is layer 2 of 3 wrap so the candy doesn't stick too much.
*
*/
template <class T> void fillHaloC(Param XParam, BlockP<T> XBlock, T* z)
{
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillHaloD(XParam, ib, XBlock, z);
	}
}
template void fillHaloC<float>(Param XParam, BlockP<float> XBlock, float* z);
template void fillHaloC<double>(Param XParam, BlockP<double> XBlock, double* z);

/*! \fn void RecalculateZs(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, T* zb)
* \brief Recalculate water surface after recalculating the values on the halo on the CPU
*
* ## Description
* Recalculate water surface after recalculating the values on the halo on the CPU. zb (bottom elevation) on each halo is calculated
* at the start of the loop or as part of the initial condition. When conserve-elevation is not required, only h is recalculated on the halo at ever 1/2 steps.  
* zs then needs to be recalculated to obtain a mass-conservative solution (if zs is conserved then mass conservation is not garanteed)
* 
* ## Warning
* This function calculate zs everywhere in the block... this is a bit unecessary. Instead it should recalculate only where there is a prolongation or a restiction
*/
template <class T> void RecalculateZs(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, T* zb)
{
	int ib, n;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		/*
		//We only need to recalculate zs on the halo side 
		for (int n = -1; n <= (XParam.blkwidth); n++)
		{
			left = memloc(XParam.halowidth, XParam.blkmemwidth, -1, n, ib);
			right = memloc(XParam.halowidth, XParam.blkmemwidth, XParam.blkwidth, n, ib);
			top = memloc(XParam.halowidth, XParam.blkmemwidth, n, XParam.blkwidth, ib);
			bot = memloc(XParam.halowidth, XParam.blkmemwidth, n, -1, ib);

			Xev.zs[left] = zb[left] + Xev.h[left];
			Xev.zs[right] = zb[right] + Xev.h[right];
			Xev.zs[top] = zb[top] + Xev.h[top];
			Xev.zs[bot] = zb[bot] + Xev.h[bot];

			//printf("n=%d; zsold=%f; zsnew=%f (zb=%f + h=%f)\n",n, Xev.zs[n], zb[n] + Xev.h[n], zb[n] , Xev.h[n]);
		}
		*/
		
		// Recalculate zs everywhere maybe we only need to do that on the halo ?
		for (int j = -1; j < (XParam.blkwidth+1); j++)
		{
			for (int i = -1; i < (XParam.blkwidth+1); i++)
			{
				n = memloc(XParam.halowidth,XParam.blkmemwidth, i, j, ib);
				Xev.zs[n] = zb[n] + Xev.h[n];
			}
		}
		
	}
}
template void RecalculateZs<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev, float* zb);
template void RecalculateZs<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev, double* zb);

template <class T> void Recalculatehh(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, T* zb)
{
	int ib, n;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		/*
		//We only need to recalculate zs on the halo side
		for (int n = -1; n <= (XParam.blkwidth); n++)
		{
			left = memloc(XParam.halowidth, XParam.blkmemwidth, -1, n, ib);
			right = memloc(XParam.halowidth, XParam.blkmemwidth, XParam.blkwidth, n, ib);
			top = memloc(XParam.halowidth, XParam.blkmemwidth, n, XParam.blkwidth, ib);
			bot = memloc(XParam.halowidth, XParam.blkmemwidth, n, -1, ib);

			Xev.zs[left] = zb[left] + Xev.h[left];
			Xev.zs[right] = zb[right] + Xev.h[right];
			Xev.zs[top] = zb[top] + Xev.h[top];
			Xev.zs[bot] = zb[bot] + Xev.h[bot];

			//printf("n=%d; zsold=%f; zsnew=%f (zb=%f + h=%f)\n",n, Xev.zs[n], zb[n] + Xev.h[n], zb[n] , Xev.h[n]);
		}
		*/

		// Recalculate zs everywhere maybe we only need to do that on the halo ?
		for (int j = -1; j < (XParam.blkwidth + 1); j++)
		{
			for (int i = -1; i < (XParam.blkwidth + 1); i++)
			{
				n = memloc(XParam.halowidth, XParam.blkmemwidth, i, j, ib);
				
				Xev.h[n] = max(Xev.zs[n]- zb[n],(T)0.0) ;
			}
		}

	}
}
template void Recalculatehh<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev, float* zb);
template void Recalculatehh<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev, double* zb);


/*! \fn void RecalculateZs(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, T* zb)
* \brief Recalculate water surface after recalculating the values on the halo on the GPU
*
* ## Description
* Recalculate water surface after recalculating the values on the halo on the CPU. zb (bottom elevation) on each halo is calculated
* at the start of the loop or as part of the initial condition. When conserve-elevation is not required, only h is recalculated on the halo at ever 1/2 steps.
* zs then needs to be recalculated to obtain a mass-conservative solution (if zs is conserved then mass conservation is not garanteed)
*
* ## Warning
* This function calculate zs everywhere in the block... this is a bit unecessary. Instead it should recalculate only where there is a prolongation or a restiction
*/
template <class T> __global__ void RecalculateZsGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, T* zb)
{
	unsigned int blkmemwidth = XParam.blkmemwidth;
	
	int ix = threadIdx.x -1;
	int iy = threadIdx.y -1;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];
	
	int  n;
	
	//ib = XBlock.active[ibl];
		
	n = memloc(XParam.halowidth, blkmemwidth, ix, iy, ib);
	Xev.zs[n] = zb[n] + Xev.h[n];
	/*
	if(zb[n] < XParam.eps)
	{
		printf("ix=%d, iy=%d, ib=%d, n=%d; zsold=%f; zsnew=%f (zb=%f + h=%f)\n",ix,iy,ib, n, Xev.zs[n], zb[n] + Xev.h[n], zb[n], Xev.h[n]);
	}
	*/
	
}
template __global__ void RecalculateZsGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev, float* zb);
template __global__ void RecalculateZsGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev, double* zb);

/*! \fn void fillHaloF(Param XParam, bool doProlongation, BlockP<T> XBlock, T* z)
* \brief Wrapping function for calculating flux in the halos for a block and a single variable on CPU.
* ## Depreciated
* This function is was never sucessful and will never be used. It is fundamentally flawed because is doesn't preserve the balance of fluxes on the restiction interface
* It should be deleted soon
* ## Description
* 
* 
*
*/
template <class T> void fillHaloF(Param XParam, bool doProlongation, BlockP<T> XBlock, T* z)
{
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillLeftFlux(XParam, doProlongation, ib, XBlock, z);
		fillBotFlux(XParam, doProlongation, ib, XBlock, z);
		fillRightFlux(XParam, doProlongation, ib, XBlock, z);
		fillTopFlux(XParam, doProlongation, ib, XBlock, z);
	
	}
}
template void fillHaloF<float>(Param XParam, bool doProlongation, BlockP<float> XBlock, float* z);
template void fillHaloF<double>(Param XParam, bool doProlongation, BlockP<double> XBlock, double* z);

/*! \fn void fillHaloGPU(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
* \brief Wrapping function for calculating halos for each block of a single variable on GPU.
*
* ## Description
* This function is a wraping fuction of the halo functions on GPU. It is called from the main Halo GPU function.
* The present imnplementation is naive and slow one that calls the rather complex fillLeft type functions
* 
*/
template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
{

	dim3 blockDimHaloLR(1, XParam.blkwidth, 1);
	dim3 blockDimHaloBT(XParam.blkwidth, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, z);
	//fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, z);
	CUDA_CHECK(hipDeviceSynchronize());
	fillRight << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	//fillRight << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	CUDA_CHECK(hipDeviceSynchronize());
	fillBot << <gridDim, blockDimHaloBT, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, z);
	//fillBot << <gridDim, blockDimHaloBT, 0>> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, z);
	CUDA_CHECK(hipDeviceSynchronize());
	fillTop << <gridDim, blockDimHaloBT, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	//fillTop << <gridDim, blockDimHaloBT, 0>> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	CUDA_CHECK(hipDeviceSynchronize());
	//CUDA_CHECK(hipStreamSynchronize(stream));

}
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, hipStream_t stream, double* z);
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, hipStream_t stream, float* z);

template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock,  T* z)
{

	dim3 blockDimHaloLR(1, XParam.blkwidth, 1);
	dim3 blockDimHaloBT(XParam.blkwidth, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, z);
	//fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, z);
	CUDA_CHECK(hipDeviceSynchronize());
	fillRight << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	//fillRight << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	CUDA_CHECK(hipDeviceSynchronize());
	fillBot << <gridDim, blockDimHaloBT, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, z);
	//fillBot << <gridDim, blockDimHaloBT, 0>> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, z);
	CUDA_CHECK(hipDeviceSynchronize());
	fillTop << <gridDim, blockDimHaloBT, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	//fillTop << <gridDim, blockDimHaloBT, 0>> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	CUDA_CHECK(hipDeviceSynchronize());
	//CUDA_CHECK(hipStreamSynchronize(stream));

}
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock,double* z);
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, float* z);


/*! \fn void fillHaloGPUnew(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
*/
template <class T> void fillHaloGPUnew(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
{
	dim3 blockDimHaloLR(1, XParam.blkwidth, 1);
	dim3 blockDimHaloBT(XParam.blkwidth, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);

	dim3 blockDimHaloLRx2(2, XParam.blkwidth, 1);
	dim3 blockDimHaloBTx2(XParam.blkwidth, 2, 1);
	dim3 gridDimx2(ceil(XParam.nblk/2), 1, 1);

	//fillLeftnew << <gridDimx2, blockDimHaloLRx2, 0>> > (XParam.halowidth, XParam.nblk, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, z);
	fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, z);
	CUDA_CHECK(hipDeviceSynchronize());
	//fillRightnew << <gridDimx2, blockDimHaloLRx2, 0 >> > (XParam.halowidth, XParam.nblk, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	fillRight << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	CUDA_CHECK(hipDeviceSynchronize());
	//fillBotnew << <gridDimx2, blockDimHaloBTx2, 0>> > (XParam.halowidth, XParam.nblk, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, z);
	fillBot << <gridDim, blockDimHaloBT, 0>> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, z);
	CUDA_CHECK(hipDeviceSynchronize());
	//fillTopnew << <gridDimx2, blockDimHaloBTx2, 0 >> > (XParam.halowidth, XParam.nblk, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	fillTop << <gridDim, blockDimHaloBT, 0>> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	CUDA_CHECK(hipDeviceSynchronize());
	//CUDA_CHECK(hipStreamSynchronize(stream));

}
template void fillHaloGPUnew<double>(Param XParam, BlockP<double> XBlock, hipStream_t stream, double* z);
template void fillHaloGPUnew<float>(Param XParam, BlockP<float> XBlock, hipStream_t stream, float* z);


/*! \fn void  fillHaloTopRightC(Param XParam, BlockP<T> XBlock, T* z)
* \brief Wrapping function for calculating flux for halos for each block of a single variable on GPU.
*
* ## Description
* This function is a wraping function of the halo flux functions on GPU. It is called from the main Halo GPU function.
* The present imnplementation is naive and slow one that calls the rather complex fillLeft type functions
*
*/
template <class T> void fillHaloTopRightC(Param XParam, BlockP<T> XBlock, T* z)
{
	// for flux term and actually most terms, only top and right neighbours are needed!

	//fillLeft(XParam, ib, XBlock, z);
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		HaloFluxCPULR(XParam, ib, XBlock, z);
		HaloFluxCPUBT(XParam, ib, XBlock, z);

		//fillRightFlux(XParam,true, ib, XBlock, z);
		//fillTopFlux(XParam,true, ib, XBlock, z);

	}
	


}
template void fillHaloTopRightC<double>(Param XParam, BlockP<double> XBlock, double* z);
template void fillHaloTopRightC<float>(Param XParam, BlockP<float> XBlock, float* z);

template <class T> void fillHaloLRFluxC(Param XParam, BlockP<T> XBlock, T* z)
{
	// for flux term and actually most terms, only top and right neighbours are needed!

	//fillLeft(XParam, ib, XBlock, z);
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		HaloFluxCPULR(XParam, ib, XBlock, z);
		//HaloFluxCPUBT(XParam, ib, XBlock, z);

		//fillRightFlux(XParam,true, ib, XBlock, z);
		//fillTopFlux(XParam,true, ib, XBlock, z);

	}



}
template void fillHaloLRFluxC<double>(Param XParam, BlockP<double> XBlock, double* z);
template void fillHaloLRFluxC<float>(Param XParam, BlockP<float> XBlock, float* z);

template <class T> void fillHaloBTFluxC(Param XParam, BlockP<T> XBlock, T* z)
{
	// for flux term and actually most terms, only top and right neighbours are needed!

	//fillLeft(XParam, ib, XBlock, z);
	int ib;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		//HaloFluxCPULR(XParam, ib, XBlock, z);
		HaloFluxCPUBT(XParam, ib, XBlock, z);

		//fillRightFlux(XParam,true, ib, XBlock, z);
		//fillTopFlux(XParam,true, ib, XBlock, z);

	}



}
template void fillHaloBTFluxC<double>(Param XParam, BlockP<double> XBlock, double* z);
template void fillHaloBTFluxC<float>(Param XParam, BlockP<float> XBlock, float* z);



template <class T> void fillHaloTopRightGPU(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
{

	dim3 blockDimHaloLR(1, 16, 1);
	dim3 blockDimHaloBT(16, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);


	//fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, a);
	//fillRightFlux << <gridDim, blockDimHaloLR, 0, stream >> > (XParam.halowidth,false, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	HaloFluxGPULR << <gridDim, blockDimHaloLR, 0, stream >> > (XParam, XBlock, z);
	
	//fillBot << <gridDim, blockDimHaloBT, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, a);
	//fillTopFlux << <gridDim, blockDimHaloBT, 0, stream >> > (XParam.halowidth,false, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	HaloFluxGPUBT << <gridDim, blockDimHaloBT, 0, stream >> > (XParam, XBlock, z);
	CUDA_CHECK(hipStreamSynchronize(stream));

}
template void fillHaloTopRightGPU<double>(Param XParam, BlockP<double> XBlock, hipStream_t stream, double* z);
template void fillHaloTopRightGPU<float>(Param XParam, BlockP<float> XBlock, hipStream_t stream, float* z);

template <class T> void fillHaloLeftRightGPU(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
{

	dim3 blockDimHaloLR(1, 16, 1);
	//dim3 blockDimHaloBT(16, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);


	//fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, a);
	//fillRightFlux << <gridDim, blockDimHaloLR, 0, stream >> > (XParam.halowidth,false, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	HaloFluxGPULR << <gridDim, blockDimHaloLR, 0, stream >> > (XParam, XBlock, z);

	//fillBot << <gridDim, blockDimHaloBT, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, a);
	//fillTopFlux << <gridDim, blockDimHaloBT, 0, stream >> > (XParam.halowidth,false, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	//HaloFluxGPUBT << <gridDim, blockDimHaloBT, 0, stream >> > (XParam, XBlock, z);
	//CUDA_CHECK(hipStreamSynchronize(stream));

}
template void fillHaloLeftRightGPU<double>(Param XParam, BlockP<double> XBlock, hipStream_t stream, double* z);
template void fillHaloLeftRightGPU<float>(Param XParam, BlockP<float> XBlock, hipStream_t stream, float* z);

template <class T> void fillHaloLeftRightGPUnew(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
{

	dim3 blockDimHaloLR(2, 16, 1);
	//dim3 blockDimHaloBT(16, 1, 1);
	dim3 gridDim(ceil(XParam.nblk/2), 1, 1);

	HaloFluxGPULRnew << <gridDim, blockDimHaloLR, 0, stream >> > (XParam, XBlock, z);

		

}
template void fillHaloLeftRightGPUnew<double>(Param XParam, BlockP<double> XBlock, hipStream_t stream, double* z);
template void fillHaloLeftRightGPUnew<float>(Param XParam, BlockP<float> XBlock, hipStream_t stream, float* z);

template <class T> void fillHaloBotTopGPU(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
{

	//dim3 blockDimHaloLR(1, 16, 1);
	dim3 blockDimHaloBT(16, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);


	//fillLeft << <gridDim, blockDimHaloLR, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.LeftBot, XBlock.LeftTop, XBlock.RightBot, XBlock.BotRight, XBlock.TopRight, a);
	//fillRightFlux << <gridDim, blockDimHaloLR, 0, stream >> > (XParam.halowidth,false, XBlock.active, XBlock.level, XBlock.RightBot, XBlock.RightTop, XBlock.LeftBot, XBlock.BotLeft, XBlock.TopLeft, z);
	//HaloFluxGPULR << <gridDim, blockDimHaloLR, 0, stream >> > (XParam, XBlock, z);

	//fillBot << <gridDim, blockDimHaloBT, 0 >> > (XParam.halowidth, XBlock.active, XBlock.level, XBlock.BotLeft, XBlock.BotRight, XBlock.TopLeft, XBlock.LeftTop, XBlock.RightTop, a);
	//fillTopFlux << <gridDim, blockDimHaloBT, 0, stream >> > (XParam.halowidth,false, XBlock.active, XBlock.level, XBlock.TopLeft, XBlock.TopRight, XBlock.BotLeft, XBlock.LeftBot, XBlock.RightBot, z);
	HaloFluxGPUBT << <gridDim, blockDimHaloBT, 0, stream >> > (XParam, XBlock, z);
	//CUDA_CHECK(hipStreamSynchronize(stream));

}
template void fillHaloBotTopGPU<double>(Param XParam, BlockP<double> XBlock, hipStream_t stream, double* z);
template void fillHaloBotTopGPU<float>(Param XParam, BlockP<float> XBlock, hipStream_t stream, float* z);

template <class T> void fillHaloBotTopGPUnew(Param XParam, BlockP<T> XBlock, hipStream_t stream, T* z)
{

	//dim3 blockDimHaloLR(1, 16, 1);
	dim3 blockDimHaloBT(16, 2, 1);
	dim3 gridDim(ceil(XParam.nblk/2), 1, 1);


	HaloFluxGPUBTnew << <gridDim, blockDimHaloBT, 0, stream >> > (XParam, XBlock, z);
	

}
template void fillHaloBotTopGPUnew<double>(Param XParam, BlockP<double> XBlock, hipStream_t stream, double* z);
template void fillHaloBotTopGPUnew<float>(Param XParam, BlockP<float> XBlock, hipStream_t stream, float* z);


template <class T> void fillHalo(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, T*zb)
{
	
		
		std::thread t0(fillHaloC<T>,XParam, XBlock, Xev.h);
		std::thread t1(fillHaloC<T>,XParam, XBlock, Xev.zs);
		//std::thread t2(fillHaloF<T>,XParam,true, XBlock, Xev.u);
		//std::thread t3(fillHaloF<T>,XParam,true, XBlock, Xev.v);

		std::thread t2(fillHaloC<T>, XParam, XBlock, Xev.u);
		std::thread t3(fillHaloC<T>, XParam, XBlock, Xev.v);

		t0.join();
		t1.join();
		t2.join();
		t3.join();

		if (XParam.conserveElevation)
		{
			conserveElevation(XParam, XBlock, Xev, zb);
		}
		else if (XParam.wetdryfix)
		{
			WetDryRestriction(XParam, XBlock, Xev, zb);
		}

		RecalculateZs(XParam, XBlock, Xev, zb);

		maskbnd(XParam, XBlock, Xev, zb);
	
}
template void fillHalo<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev, float *zb);
template void fillHalo<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev,double * zb);

template <class T> void fillHalo(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev)
{

	std::thread t0(fillHaloC<T>, XParam, XBlock, Xev.h);
	std::thread t1(fillHaloC<T>, XParam, XBlock, Xev.zs);
	std::thread t2(fillHaloF<T>, XParam, true, XBlock, Xev.u);
	std::thread t3(fillHaloF<T>, XParam, true, XBlock, Xev.v);

	t0.join();
	t1.join();
	t2.join();
	t3.join();

	
	//maskbnd(XParam, XBlock, Xev, zb);

}
template void fillHalo<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev);
template void fillHalo<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev);


template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev)
{
	const int num_streams = 4;

	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}


	fillHaloGPU(XParam, XBlock, streams[0], Xev.h);
	fillHaloGPU(XParam, XBlock, streams[1], Xev.zs);
	fillHaloGPU(XParam, XBlock, streams[2], Xev.u);
	fillHaloGPU(XParam, XBlock, streams[3], Xev.v);

	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}
}
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev);
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev);

template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev,T * zb)
{
	const int num_streams = 4;
	dim3 blockDimHalo(XParam.blkwidth,1, 1);

	dim3 gridDim(XBlock.mask.nblk, 1, 1);
	
	dim3 blockDimfull(XParam.blkmemwidth, XParam.blkmemwidth, 1);
	dim3 gridDimfull(XParam.nblk, 1, 1);
	
	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}
	

	fillHaloGPU(XParam, XBlock, streams[0], Xev.h);
	fillHaloGPU(XParam, XBlock, streams[1], Xev.zs);
	fillHaloGPU(XParam, XBlock, streams[2], Xev.u);
	fillHaloGPU(XParam, XBlock, streams[3], Xev.v);
	CUDA_CHECK(hipDeviceSynchronize());

	if (XParam.conserveElevation)
	{
		conserveElevationGPU(XParam, XBlock, Xev, zb);
	}
	else if (XParam.wetdryfix)
	{
		WetDryRestrictionGPU(XParam, XBlock, Xev, zb);
	}

	RecalculateZsGPU << < gridDimfull, blockDimfull, 0 >> > (XParam, XBlock, Xev, zb);
	CUDA_CHECK(hipDeviceSynchronize());

	//if (XBlock.mask.nblk > 0)
	//{
	//	maskbndGPUleft << <gridDim, blockDimHalo, 0, streams[0] >> > (XParam, XBlock, Xev, zb);
	//	maskbndGPUtop << <gridDim, blockDimHalo, 0, streams[1] >> > (XParam, XBlock, Xev, zb);
	//	maskbndGPUright << <gridDim, blockDimHalo, 0, streams[2] >> > (XParam, XBlock, Xev, zb);
	//	maskbndGPUtop << <gridDim, blockDimHalo, 0, streams[3] >> > (XParam, XBlock, Xev, zb);

	//	//CUDA_CHECK(hipDeviceSynchronize());
	//}
	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}

	
}
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev,float *zb);
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev,double* zb);

template <class T> void fillHalo(Param XParam, BlockP<T> XBlock, GradientsP<T> Grad)
{
	
	/*
	std::thread t0(fillHaloF<T>,XParam, true, XBlock, Grad.dhdx);
	std::thread t1(fillHaloF<T>,XParam, true, XBlock, Grad.dudx);
	std::thread t2(fillHaloF<T>,XParam, true, XBlock, Grad.dvdx);
	std::thread t3(fillHaloF<T>,XParam, true, XBlock, Grad.dzsdx);

	std::thread t4(fillHaloF<T>,XParam, true, XBlock, Grad.dhdy);
	std::thread t5(fillHaloF<T>,XParam, true, XBlock, Grad.dudy);
	std::thread t6(fillHaloF<T>,XParam, true, XBlock, Grad.dvdy);
	std::thread t7(fillHaloF<T>,XParam, true, XBlock, Grad.dzsdy);

	*/
	std::thread t0(fillHaloC<T>, XParam, XBlock, Grad.dhdx);
	std::thread t1(fillHaloC<T>, XParam, XBlock, Grad.dudx);
	std::thread t2(fillHaloC<T>, XParam, XBlock, Grad.dvdx);
	std::thread t3(fillHaloC<T>, XParam, XBlock, Grad.dzsdx);

	std::thread t4(fillHaloC<T>, XParam, XBlock, Grad.dhdy);
	std::thread t5(fillHaloC<T>, XParam, XBlock, Grad.dudy);
	std::thread t6(fillHaloC<T>, XParam, XBlock, Grad.dvdy);
	std::thread t7(fillHaloC<T>, XParam, XBlock, Grad.dzsdy);

	t0.join();
	t1.join();
	t2.join();
	t3.join();
	t4.join();
	t5.join();
	t6.join();
	t7.join();

	
}
template void fillHalo<float>(Param XParam, BlockP<float> XBlock, GradientsP<float> Grad);
template void fillHalo<double>(Param XParam, BlockP<double> XBlock, GradientsP<double> Grad);

template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, GradientsP<T> Grad)
{
	const int num_streams = 8;

	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}
		
	fillHaloGPU(XParam, XBlock, streams[0], Grad.dhdx);
	fillHaloGPU(XParam, XBlock, streams[2], Grad.dudx);
	fillHaloGPU(XParam, XBlock, streams[3], Grad.dvdx);
	fillHaloGPU(XParam, XBlock, streams[4], Grad.dzsdx);

	fillHaloGPU(XParam, XBlock, streams[5], Grad.dhdy);
	fillHaloGPU(XParam, XBlock, streams[6], Grad.dudy);
	fillHaloGPU(XParam, XBlock, streams[7], Grad.dvdy);
	fillHaloGPU(XParam, XBlock, streams[1], Grad.dzsdy);

	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}
	
}
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, GradientsP<float> Grad);
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, GradientsP<double> Grad);


template <class T> void fillHalo(Param XParam, BlockP<T> XBlock, FluxP<T> Flux)
{
	
		
	//std::thread t0(fillHaloTopRightC<T>,XParam, XBlock, Flux.Fhu);
	//std::thread t1(fillHaloTopRightC<T>,XParam, XBlock, Flux.Fhv);
	//std::thread t2(fillHaloTopRightC<T>,XParam, XBlock, Flux.Fqux);
	//std::thread t3(fillHaloTopRightC<T>, XParam, XBlock, Flux.Fquy);

	//std::thread t4(fillHaloTopRightC<T>, XParam, XBlock, Flux.Fqvx);
	//std::thread t5(fillHaloTopRightC<T>, XParam, XBlock, Flux.Fqvy);
	//std::thread t6(fillHaloTopRightC<T>, XParam, XBlock, Flux.Su);
	//std::thread t7(fillHaloTopRightC<T>, XParam, XBlock, Flux.Sv);

	std::thread t0(fillHaloLRFluxC<T>, XParam, XBlock, Flux.Fhu);
	std::thread t1(fillHaloLRFluxC<T>, XParam, XBlock, Flux.Fqux);
	std::thread t2(fillHaloLRFluxC<T>, XParam, XBlock, Flux.Su);

	std::thread t6(fillHaloLRFluxC<T>, XParam, XBlock, Flux.Fqvx);

	std::thread t3(fillHaloBTFluxC<T>, XParam, XBlock, Flux.Fhv);
	std::thread t4(fillHaloBTFluxC<T>, XParam, XBlock, Flux.Fqvy);
	std::thread t5(fillHaloBTFluxC<T>, XParam, XBlock, Flux.Sv);

	std::thread t7(fillHaloBTFluxC<T>, XParam, XBlock, Flux.Fquy);

	t0.join();
	t1.join();
	t2.join();
	t3.join();
	t4.join();
	t5.join();
	t6.join();
	t7.join();
	
}
template void fillHalo<float>(Param XParam, BlockP<float> XBlock, FluxP<float> Flux);
template void fillHalo<double>(Param XParam, BlockP<double> XBlock, FluxP<double> Flux);

template <class T> void fillHaloGPU(Param XParam, BlockP<T> XBlock, FluxP<T> Flux)
{
	const int num_streams = 8;

	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}

	dim3 blockDimHalo(XParam.blkwidth, 1, 1);

	dim3 gridDim(XBlock.mask.nblk, 1, 1);


	fillHaloLeftRightGPUnew(XParam, XBlock, streams[0], Flux.Fhu);
	fillHaloLeftRightGPUnew(XParam, XBlock, streams[1], Flux.Su);
	fillHaloLeftRightGPUnew(XParam, XBlock, streams[2], Flux.Fqux);
	fillHaloLeftRightGPUnew(XParam, XBlock, streams[3], Flux.Fqvx);

	

	fillHaloBotTopGPUnew(XParam, XBlock, streams[4], Flux.Fquy);
	fillHaloBotTopGPUnew(XParam, XBlock, streams[5], Flux.Fqvy);
	fillHaloBotTopGPUnew(XParam, XBlock, streams[6], Flux.Fhv);
	fillHaloBotTopGPUnew(XParam, XBlock, streams[7], Flux.Sv);

	
	for (int i = 0; i < num_streams; i++)
	{
		hipStreamSynchronize(streams[i]);
	}
	// Below has now moved to its own function
	//if (XBlock.mask.nblk > 0)
	//{
	//	maskbndGPUFluxleft << <gridDim, blockDimHalo, 0, streams[0] >> > (XParam, XBlock, Flux);
	//	maskbndGPUFluxtop << <gridDim, blockDimHalo, 0, streams[1] >> > (XParam, XBlock, Flux);
	//	maskbndGPUFluxright << <gridDim, blockDimHalo, 0, streams[2] >> > (XParam, XBlock, Flux);
	//	maskbndGPUFluxbot << <gridDim, blockDimHalo, 0, streams[3] >> > (XParam, XBlock, Flux);

	//	//CUDA_CHECK(hipDeviceSynchronize());
	//}
	
	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}

	
}
template void fillHaloGPU<float>(Param XParam, BlockP<float> XBlock, FluxP<float> Flux);
template void fillHaloGPU<double>(Param XParam, BlockP<double> XBlock, FluxP<double> Flux);

template <class T> void bndmaskGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> Xev, FluxP<T> Flux)
{
	const int num_streams = 8;

	hipStream_t streams[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}

	dim3 blockDimHalo(XParam.blkwidth, 1, 1);

	dim3 gridDim(XBlock.mask.nblk, 1, 1);
	if (XBlock.mask.nblk > 0)
	{
		maskbndGPUFluxleft << <gridDim, blockDimHalo, 0, streams[0] >> > (XParam, XBlock, Xev, Flux);
		maskbndGPUFluxtop << <gridDim, blockDimHalo, 0, streams[1] >> > (XParam, XBlock,  Flux);
		maskbndGPUFluxright << <gridDim, blockDimHalo, 0, streams[2] >> > (XParam, XBlock,  Flux);
		maskbndGPUFluxbot << <gridDim, blockDimHalo, 0, streams[3] >> > (XParam, XBlock, Flux);

		//CUDA_CHECK(hipDeviceSynchronize());
	}

	for (int i = 0; i < num_streams; i++)
	{
		hipStreamDestroy(streams[i]);
	}

}
template void bndmaskGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> Xev, FluxP<float> Flux);
template void bndmaskGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> Xev, FluxP<double> Flux);

//template <class T> void refine_linearCPU(Param XParam, int ib, bool isLR, bool isoposit, BlockP<T> XBlock, T* z, T* dzdx, T* dzdy)
//{
//	int Neighblock, Mirrorblock;
//
//	int ir = isoposit ? 0 : XParam.blkwidth - 1;
//	int iw = isoposit ? XParam.blkwidth : -1;
//	if (isLR)
//	{
//		Neighblock = isoposit ? XBlock.RightBot[ib] : XBlock.LeftBot[ib];
//		Mirrorblock = isoposit ? XBlock.LeftBot[Neighblock] : XBlock.RightBot[Neighblock]
//	}
//	else
//	{
//		Neighblock = isoposit ? XBlock.TopLeft[ib] : XBlock.BotLeft[ib];
//		Mirrorblock = isoposit ? XBlock.BotLeft[Neighblock] : XBlock.TopLeft[Neighblock]
//	}
//
//	if (XBlock.level[Neighblock] < XBlock.level[ib])
//	{
//		double ilevdx = calcres(XParam.dx, XBlock.level[ib]) * T(0.25);
//		for (int ix = 0; ix < XParam.blkwidth; ix++)
//		{
//			int jj = Mirrorblock == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + XParam.blkwidth / 2;
//			int il = isLR ? memloc(XParam, ir, jj, Neighblock) : memloc(XParam, jj, ir, Neighblock);
//			int write = isLR ? memloc(XParam, iw, j, ib) : memloc(XParam, j, iw, ib);
//			T faclr = T(-1.0);
//			T facbt = floor(j * (T)0.5) * T(2.0) > j ? 1.0 : -1.0;
//
//			T newz = z[il] + (faclr * dzdx[il] + facbt * dzdy[il]) * ilevdx;
//
//			z[write] = newz;
//
//
//		}
//	}
//}
//template void refine_linearCPU<float>(Param XParam, int ib, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
//template void refine_linearCPU<double>(Param XParam, int ib, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);

template <class T> void refine_linear_Left(Param XParam, int ib, BlockP<T> XBlock, T* z, T * dzdx, T * dzdy)
{
	if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib])
	{

		double ilevdx = calcres(XParam.delta, XBlock.level[ib])*T(0.5);

		for (int j = 0; j < XParam.blkwidth; j++)
		{
			int jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib ? ftoi(floor(j * (T)0.5)) : ftoi(floor(j * (T)0.5) + XParam.blkwidth / 2);
			int il = memloc(XParam, XParam.blkwidth - 1, jj , XBlock.LeftBot[ib]);
			int write = memloc(XParam, -1, j, ib);
			T faclr = T(1.0);
			T facbt = floor(j * (T)0.5) * T(2.0) < (j-T(0.01)) ? 1.0 : -1.0;
			
			T newz = z[il] + (faclr*dzdx[il]+facbt*dzdy[il]) * ilevdx;

			z[write] = newz;


		}
	}
}
template void refine_linear_Left<float>(Param XParam, int ib, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template void refine_linear_Left<double>(Param XParam, int ib, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);

template <class T> __global__ void refine_linear_LeftGPU(Param XParam, BlockP<T> XBlock, T* z, T* dzdx,T*dzdy)
{
	int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	//unsigned int ix = 0;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];


	

	if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib])
	{
		int j = iy;

		double ilevdx = calcres(XParam.delta, XBlock.level[ib]) * T(0.5);

		
		int jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib ? floor(j * (T)0.5) : floor(j * (T)0.5) + XParam.blkwidth / 2;
		int il = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth - 1, jj, XBlock.LeftBot[ib]);
		int write = memloc(XParam.halowidth, blkmemwidth, -1, j, ib);
		T faclr = T(1.0);
		T facbt = floor(j * (T)0.5) * T(2.0) < (j - T(0.01)) ? 1.0 : -1.0;

		T newz = z[il] + (faclr * dzdx[il] + facbt * dzdy[il]) * ilevdx;

		z[write] = newz;


		
	}
}
template __global__ void refine_linear_LeftGPU<float>(Param XParam, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template __global__ void refine_linear_LeftGPU<double>(Param XParam, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);


template <class T> void refine_linear_Right(Param XParam, int ib, BlockP<T> XBlock, T* z, T* dzdx, T* dzdy)
{
	if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib])
	{

		T ilevdx = calcres(T(XParam.delta), XBlock.level[ib] ) * T(0.5);

		for (int j = 0; j < XParam.blkwidth; j++)
		{
			int jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? ftoi(floor(j * (T)0.5)) : ftoi(floor(j * (T)0.5) + XParam.blkwidth / 2);
			int il = memloc(XParam, 0, jj , XBlock.RightBot[ib]);
			int write = memloc(XParam, XParam.blkwidth, j, ib);
			T faclr = T(-1.0);
			T facbt = floor(j * (T)0.5) * T(2.0) < (j - T(0.01)) ? 1.0 : -1.0;

			T newz = z[il] + (faclr * dzdx[il] + facbt * dzdy[il]) * ilevdx;

			z[write] = newz;


		}
	}
}
template void refine_linear_Right<float>(Param XParam, int ib, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template void refine_linear_Right<double>(Param XParam, int ib, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);

template <class T> __global__ void refine_linear_RightGPU(Param XParam, BlockP<T> XBlock, T* z, T* dzdx, T* dzdy)
{
	unsigned int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];



	if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib])
	{

		double ilevdx = calcres(XParam.delta, XBlock.level[ib]) * T(0.5);

		int j = iy;
		int jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? floor(j * (T)0.5) : floor(j * (T)0.5) + XParam.blkwidth / 2;
		int il = memloc(XParam.halowidth, blkmemwidth, 0, jj, XBlock.RightBot[ib]);
		int write = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, j, ib);

		T faclr = T(-1.0);
		T facbt = floor(j * (T)0.5) * T(2.0) < (j - T(0.01)) ? 1.0 : -1.0;

		T newz = z[il] + (faclr * dzdx[il] + facbt * dzdy[il]) * ilevdx;

		z[write] = newz;


		
	}
}
template __global__ void refine_linear_RightGPU<float>(Param XParam, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template __global__ void refine_linear_RightGPU<double>(Param XParam, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);

template <class T> void refine_linear_Bot(Param XParam, int ib, BlockP<T> XBlock, T* z, T* dzdx, T* dzdy)
{
	if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib])
	{

		T ilevdx = calcres(T(XParam.delta), XBlock.level[ib]) * T(0.5);

		for (int i = 0; i < XParam.blkwidth; i++)
		{
			int ii = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? ftoi(floor(i * (T)0.5)) : ftoi(floor(i * (T)0.5) + XParam.blkwidth / 2);
			int jl = memloc(XParam,  ii, XParam.blkwidth - 1, XBlock.BotLeft[ib]);
			int write = memloc(XParam, i, -1, ib);
			
			T facbt = T(1.0);
			T faclr = floor(i * (T)0.5) * T(2.0) < (i - T(0.01)) ? 1.0 : -1.0;

			T newz = z[jl] + (faclr * dzdx[jl] + facbt * dzdy[jl]) * ilevdx;

			z[write] = newz;


		}
	}
}
template void refine_linear_Bot<float>(Param XParam, int ib, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template void refine_linear_Bot<double>(Param XParam, int ib, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);


template <class T> __global__ void refine_linear_BotGPU(Param XParam, BlockP<T> XBlock, T* z, T* dzdx, T* dzdy)
{
	int blkmemwidth = blockDim.x + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;

	int ix = threadIdx.x;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib])
	{

		double ilevdx = calcres(XParam.delta, XBlock.level[ib]) * T(0.5);

		int i = ix;
		int ii = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? floor(i * (T)0.5) : floor(i * (T)0.5) + XParam.blkwidth / 2;
		int jl = memloc(XParam.halowidth, blkmemwidth, ii, XParam.blkwidth - 1, XBlock.BotLeft[ib]);
		int write = memloc(XParam.halowidth, blkmemwidth, i, -1, ib);

		T facbt = T(1.0);
		T faclr = floor(i * (T)0.5) * T(2.0) < (i - T(0.01)) ? 1.0 : -1.0;

		T newz = z[jl] + (faclr * dzdx[jl] + facbt * dzdy[jl]) * ilevdx;

		z[write] = newz;


		
	}
}
template __global__ void refine_linear_BotGPU<float>(Param XParam, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template __global__ void refine_linear_BotGPU<double>(Param XParam, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);

template <class T> void refine_linear_Top(Param XParam, int ib, BlockP<T> XBlock, T* z, T* dzdx, T* dzdy)
{
	if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib])
	{

		double ilevdx = calcres(XParam.delta, XBlock.level[ib]) * T(0.5);

		for (int i = 0; i < XParam.blkwidth; i++)
		{
			int ii = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? ftoi(floor(i * (T)0.5)) : ftoi(floor(i * (T)0.5) + XParam.blkwidth / 2);
			int jl = memloc(XParam, ii , 0, XBlock.TopLeft[ib]);
			int write = memloc(XParam, i, XParam.blkwidth, ib);
			
			T facbt = T(-1.0);
			T faclr = floor(i * (T)0.5) * T(2.0) < (i - T(0.01)) ? 1.0 : -1.0;

			T newz = z[jl] + (faclr * dzdx[jl] + facbt * dzdy[jl]) * ilevdx;

			z[write] = newz;


		}
	}
}
template void refine_linear_Top<float>(Param XParam, int ib, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template void refine_linear_Top<double>(Param XParam, int ib, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);

template <class T> __global__ void refine_linear_TopGPU(Param XParam, BlockP<T> XBlock, T* z, T* dzdx, T* dzdy)
{
	int blkmemwidth = blockDim.x + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;

	int ix = threadIdx.x;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];
	if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib])
	{

		double ilevdx = calcres(XParam.delta, XBlock.level[ib]) * T(0.5);

    int i = ix;
		int ii = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? floor(i * (T)0.5) : floor(i * (T)0.5) + XParam.blkwidth / 2;
		int jl = memloc(XParam.halowidth, blkmemwidth, ii , 0, XBlock.TopLeft[ib]);
		int write = memloc(XParam.halowidth, blkmemwidth, i, XParam.blkwidth, ib);

		T facbt = T(-1.0);
		T faclr = floor(i * (T)0.5) * T(2.0) < (i - T(0.01)) ? 1.0 : -1.0;

		T newz = z[jl] + (faclr * dzdx[jl] + facbt * dzdy[jl]) * ilevdx;

		z[write] = newz;


		
	}
}
template __global__ void refine_linear_TopGPU<float>(Param XParam, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template __global__ void refine_linear_TopGPU<double>(Param XParam, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);

template <class T> void refine_linear(Param XParam, BlockP<T> XBlock, T* z, T* dzdx, T* dzdy)
{
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		int ib = XBlock.active[ibl];
		refine_linear_Left(XParam, ib, XBlock, z, dzdx, dzdy);
		refine_linear_Right(XParam, ib, XBlock, z, dzdx, dzdy);
		refine_linear_Top(XParam, ib, XBlock, z, dzdx, dzdy);
		refine_linear_Bot(XParam, ib, XBlock, z, dzdx, dzdy);
	}
}
template void refine_linear<float>(Param XParam, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template void refine_linear<double>(Param XParam, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);

template <class T> void refine_linearGPU(Param XParam, BlockP<T> XBlock, T* z, T* dzdx, T* dzdy)
{
	dim3 blockDimHaloLR(1, XParam.blkwidth, 1);
	dim3 blockDimHaloBT(XParam.blkwidth, 1, 1);
	dim3 gridDim(XParam.nblk, 1, 1);
		
	refine_linear_LeftGPU<<<gridDim, blockDimHaloLR, 0>>>(XParam, XBlock, z, dzdx, dzdy);
	refine_linear_RightGPU << <gridDim, blockDimHaloLR, 0 >> > (XParam, XBlock, z, dzdx, dzdy);
	refine_linear_TopGPU << <gridDim, blockDimHaloBT, 0 >> > (XParam, XBlock, z, dzdx, dzdy);
	refine_linear_BotGPU << <gridDim, blockDimHaloBT, 0 >> > (XParam, XBlock, z, dzdx, dzdy);
	CUDA_CHECK(hipDeviceSynchronize());
	
}
template void refine_linearGPU<float>(Param XParam, BlockP<float> XBlock, float* z, float* dzdx, float* dzdy);
template void refine_linearGPU<double>(Param XParam, BlockP<double> XBlock, double* z, double* dzdx, double* dzdy);


template <class T> void HaloFluxCPULR(Param XParam, int ib, BlockP<T> XBlock, T *z)
{
	int jj, i,il,itl;
	if (XBlock.level[XBlock.LeftBot[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{
			//
			i = memloc(XParam, 0, j, ib);


			jj = j*2;
			il = memloc(XParam, XParam.blkwidth, jj, XBlock.LeftBot[ib]);
			itl = memloc(XParam, XParam.blkwidth, jj+1, XBlock.LeftBot[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);
			




		}
		//
	}
	if (XBlock.level[XBlock.LeftTop[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			//
			i = memloc(XParam, 0, j, ib);


			jj = (j - XParam.blkwidth / 2) * 2;
			il = memloc(XParam, XParam.blkwidth, jj, XBlock.LeftTop[ib]);
			itl = memloc(XParam, XParam.blkwidth, jj + 1, XBlock.LeftTop[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{
			//
			i = memloc(XParam, XParam.blkwidth, j, ib);


			jj = j * 2;
			il = memloc(XParam, 0, jj, XBlock.RightBot[ib]);
			itl = memloc(XParam, 0, jj + 1, XBlock.RightBot[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.RightTop[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			
			jj = (j - XParam.blkwidth / 2) * 2;
			//
			i = memloc(XParam, XParam.blkwidth, j, ib);
			
			
			il = memloc(XParam, 0, jj, XBlock.RightTop[ib]);
			itl = memloc(XParam, 0, jj + 1, XBlock.RightTop[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
}

template <class T> __global__  void HaloFluxGPULR(Param XParam, BlockP<T> XBlock, T* z)
{
	int jj, i, il, itl;
	int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	//unsigned int ix = 0;
	int iy = threadIdx.y;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];


	int j = iy;


	if (XBlock.level[XBlock.LeftBot[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		
		if (j< (XParam.blkwidth / 2))
		{
			//
			i = memloc(XParam.halowidth, blkmemwidth, 0, j, ib);


			jj = j * 2;
			il = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, jj, XBlock.LeftBot[ib]);
			itl = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, jj + 1, XBlock.LeftBot[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.LeftTop[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		
		if (j >= (XParam.blkwidth / 2))
		{
			//
			i = memloc(XParam.halowidth, blkmemwidth, 0, j, ib);


			jj = (j - XParam.blkwidth / 2) * 2;
			il = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, jj, XBlock.LeftTop[ib]);
			itl = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, jj + 1, XBlock.LeftTop[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		
		if (j < (XParam.blkwidth / 2))
		{
			//
			i = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, j, ib);


			jj = j * 2;
			il = memloc(XParam.halowidth, blkmemwidth, 0, jj, XBlock.RightBot[ib]);
			itl = memloc(XParam.halowidth, blkmemwidth, 0, jj + 1, XBlock.RightBot[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.RightTop[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		if (j >= (XParam.blkwidth / 2))
		{

			jj = (j - XParam.blkwidth / 2) * 2;
			//
			i = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, j, ib);


			il = memloc(XParam.halowidth, blkmemwidth, 0, jj, XBlock.RightTop[ib]);
			itl = memloc(XParam.halowidth, blkmemwidth, 0, jj + 1, XBlock.RightTop[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
}

template <class T> __global__  void HaloFluxGPULRnew(Param XParam, BlockP<T> XBlock, T* z)
{
	int jj, i, il, itl;
	int blkmemwidth = blockDim.y + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	//unsigned int ix = 0;
	int iy = threadIdx.y;
	int ibl = threadIdx.x + blockIdx.x * blockDim.x;
	if (ibl < XParam.nblk)
	{

		int ib = XBlock.active[ibl];


		int j = iy;


		if (XBlock.level[XBlock.LeftBot[ib]] > XBlock.level[ib])//The lower half is a boundary 
		{

			if (j < (XParam.blkwidth / 2))
			{
				//
				i = memloc(XParam.halowidth, blkmemwidth, 0, j, ib);


				jj = j * 2;
				il = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, jj, XBlock.LeftBot[ib]);
				itl = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, jj + 1, XBlock.LeftBot[ib]);

				z[i] = T(0.5) * (z[il] + z[itl]);





			}
			//
		}
		if (XBlock.level[XBlock.LeftTop[ib]] > XBlock.level[ib])//The lower half is a boundary 
		{

			if (j >= (XParam.blkwidth / 2))
			{
				//
				i = memloc(XParam.halowidth, blkmemwidth, 0, j, ib);


				jj = (j - XParam.blkwidth / 2) * 2;
				il = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, jj, XBlock.LeftTop[ib]);
				itl = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, jj + 1, XBlock.LeftTop[ib]);

				z[i] = T(0.5) * (z[il] + z[itl]);





			}
			//
		}
		if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])//The lower half is a boundary 
		{

			if (j < (XParam.blkwidth / 2))
			{
				//
				i = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, j, ib);


				jj = j * 2;
				il = memloc(XParam.halowidth, blkmemwidth, 0, jj, XBlock.RightBot[ib]);
				itl = memloc(XParam.halowidth, blkmemwidth, 0, jj + 1, XBlock.RightBot[ib]);

				z[i] = T(0.5) * (z[il] + z[itl]);





			}
			//
		}
		if (XBlock.level[XBlock.RightTop[ib]] > XBlock.level[ib])//The lower half is a boundary 
		{
			if (j >= (XParam.blkwidth / 2))
			{

				jj = (j - XParam.blkwidth / 2) * 2;
				//
				i = memloc(XParam.halowidth, blkmemwidth, XParam.blkwidth, j, ib);


				il = memloc(XParam.halowidth, blkmemwidth, 0, jj, XBlock.RightTop[ib]);
				itl = memloc(XParam.halowidth, blkmemwidth, 0, jj + 1, XBlock.RightTop[ib]);

				z[i] = T(0.5) * (z[il] + z[itl]);





			}
			//
		}
	}
}

template <class T> void HaloFluxCPUBT(Param XParam, int ib, BlockP<T> XBlock, T* z)
{
	int jj, i, il, itl;
	if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{
			//
			i = memloc(XParam, j, 0, ib);


			jj = j * 2;
			il = memloc(XParam, jj, XParam.blkwidth,  XBlock.BotLeft[ib]);
			itl = memloc(XParam, jj+1, XParam.blkwidth,  XBlock.BotLeft[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.BotRight[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{
			//
			i = memloc(XParam, j, 0, ib);


			jj = (j - XParam.blkwidth / 2) * 2;
			il = memloc(XParam,  jj, XParam.blkwidth, XBlock.BotRight[ib]);
			itl = memloc(XParam,  jj + 1, XParam.blkwidth, XBlock.BotRight[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{
			//
			i = memloc(XParam, j, XParam.blkwidth, ib);


			jj = j * 2;
			il = memloc(XParam, jj, 0,  XBlock.TopLeft[ib]);
			itl = memloc(XParam, jj + 1, 0, XBlock.TopLeft[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.TopRight[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
		{

			jj = (j - XParam.blkwidth / 2) * 2;
			//
			i = memloc(XParam, j, XParam.blkwidth, ib);


			il = memloc(XParam, jj, 0, XBlock.TopRight[ib]);
			itl = memloc(XParam, jj + 1, 0, XBlock.TopRight[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
}

template <class T> __global__ void HaloFluxGPUBT(Param XParam, BlockP<T> XBlock, T* z)
{
	int jj, i, il, itl;
	int blkmemwidth = blockDim.x + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	//unsigned int iy = threadIdx.x;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int j = ix;

	if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		if (j < (XParam.blkwidth / 2))
		{
			//
			i = memloc(XParam.halowidth, blkmemwidth, j, 0, ib);


			jj = j * 2;
			il = memloc(XParam.halowidth, blkmemwidth, jj, XParam.blkwidth, XBlock.BotLeft[ib]);
			itl = memloc(XParam.halowidth, blkmemwidth, jj + 1, XParam.blkwidth, XBlock.BotLeft[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.BotRight[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		if (j >= (XParam.blkwidth / 2))
		{
			//
			i = memloc(XParam.halowidth, blkmemwidth, j, 0, ib);


			jj = (j - XParam.blkwidth / 2) * 2;
			il = memloc(XParam.halowidth, blkmemwidth, jj, XParam.blkwidth, XBlock.BotRight[ib]);
			itl = memloc(XParam.halowidth, blkmemwidth, jj + 1, XParam.blkwidth, XBlock.BotRight[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		if (j < (XParam.blkwidth / 2))
		{
			//
			i = memloc(XParam.halowidth, blkmemwidth, j, XParam.blkwidth, ib);


			jj = j * 2;
			il = memloc(XParam.halowidth, blkmemwidth, jj, 0, XBlock.TopLeft[ib]);
			itl = memloc(XParam.halowidth, blkmemwidth, jj + 1, 0, XBlock.TopLeft[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
	if (XBlock.level[XBlock.TopRight[ib]] > XBlock.level[ib])//The lower half is a boundary 
	{
		if (j >= (XParam.blkwidth / 2))
		{

			jj = (j - XParam.blkwidth / 2) * 2;
			//
			i = memloc(XParam.halowidth, blkmemwidth, j, XParam.blkwidth, ib);


			il = memloc(XParam.halowidth, blkmemwidth, jj, 0, XBlock.TopRight[ib]);
			itl = memloc(XParam.halowidth, blkmemwidth, jj + 1, 0, XBlock.TopRight[ib]);

			z[i] = T(0.5) * (z[il] + z[itl]);





		}
		//
	}
}

template <class T> __global__ void HaloFluxGPUBTnew(Param XParam, BlockP<T> XBlock, T* z)
{
	int jj, i, il, itl;
	int blkmemwidth = blockDim.x + XParam.halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	//unsigned int iy = threadIdx.x;
	int ibl = threadIdx.y + blockIdx.x * blockDim.y;
	if (ibl < XParam.nblk)
	{



		int ib = XBlock.active[ibl];

		int j = ix;

		if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])//The lower half is a boundary 
		{
			if (j < (XParam.blkwidth / 2))
			{
				//
				i = memloc(XParam.halowidth, blkmemwidth, j, 0, ib);


				jj = j * 2;
				il = memloc(XParam.halowidth, blkmemwidth, jj, XParam.blkwidth, XBlock.BotLeft[ib]);
				itl = memloc(XParam.halowidth, blkmemwidth, jj + 1, XParam.blkwidth, XBlock.BotLeft[ib]);

				z[i] = T(0.5) * (z[il] + z[itl]);





			}
			//
		}
		if (XBlock.level[XBlock.BotRight[ib]] > XBlock.level[ib])//The lower half is a boundary 
		{
			if (j >= (XParam.blkwidth / 2))
			{
				//
				i = memloc(XParam.halowidth, blkmemwidth, j, 0, ib);


				jj = (j - XParam.blkwidth / 2) * 2;
				il = memloc(XParam.halowidth, blkmemwidth, jj, XParam.blkwidth, XBlock.BotRight[ib]);
				itl = memloc(XParam.halowidth, blkmemwidth, jj + 1, XParam.blkwidth, XBlock.BotRight[ib]);

				z[i] = T(0.5) * (z[il] + z[itl]);





			}
			//
		}
		if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])//The lower half is a boundary 
		{
			if (j < (XParam.blkwidth / 2))
			{
				//
				i = memloc(XParam.halowidth, blkmemwidth, j, XParam.blkwidth, ib);


				jj = j * 2;
				il = memloc(XParam.halowidth, blkmemwidth, jj, 0, XBlock.TopLeft[ib]);
				itl = memloc(XParam.halowidth, blkmemwidth, jj + 1, 0, XBlock.TopLeft[ib]);

				z[i] = T(0.5) * (z[il] + z[itl]);





			}
			//
		}
		if (XBlock.level[XBlock.TopRight[ib]] > XBlock.level[ib])//The lower half is a boundary 
		{
			if (j >= (XParam.blkwidth / 2))
			{

				jj = (j - XParam.blkwidth / 2) * 2;
				//
				i = memloc(XParam.halowidth, blkmemwidth, j, XParam.blkwidth, ib);


				il = memloc(XParam.halowidth, blkmemwidth, jj, 0, XBlock.TopRight[ib]);
				itl = memloc(XParam.halowidth, blkmemwidth, jj + 1, 0, XBlock.TopRight[ib]);

				z[i] = T(0.5) * (z[il] + z[itl]);





			}
			//
		}
	}
}



template <class T> void fillLeft(Param XParam, int ib, BlockP<T> XBlock, T* &z)
{
	int jj,bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.LeftBot[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.LeftTop[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, -1, j, ib);
				jj = (j - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, XBlock.LeftTop[ib]);
				ir = memloc(XParam, (XParam.blkwidth - 2), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, XBlock.LeftTop[ib]);
				itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, XBlock.LeftTop[ib]);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[ XBlock.LeftBot[ib] ]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, -1, j, ib);
			read = memloc(XParam, (XParam.blkwidth - 1), j, XBlock.LeftBot[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.LeftBot[ib] ]> XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, -1, j, ib);

			jj = j * 2;
			bb = XBlock.LeftBot[ib];

			ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
			ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
			it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
			itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

			z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.LeftTop[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - (XParam.blkwidth / 2)) * 2;
				bb = XBlock.LeftTop[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
				ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
				itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, -1, j, ib);

			T w1, w2, w3;
			

			int jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib? ftoi(ceil(j * (T)0.5)): ftoi(ceil(j * (T)0.5)+ XParam.blkwidth/2);
			w1 = T(1.0 / 3.0);
			w2 = ceil(j * (T)0.5) * 2 > j ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(j * (T)0.5) * 2 > j ? T(0.5) : T(1.0 / 6.0);
						
			ii= memloc(XParam, 0, j, ib);
			ir= memloc(XParam, XParam.blkwidth-1, jj, XBlock.LeftBot[ib]);
			it = memloc(XParam, XParam.blkwidth-1, jj - 1, XBlock.LeftBot[ib]);
			//2 scenarios here ib is the rightbot neighbour of the leftbot block or ib is the righttop neighbour
			if (XBlock.RightBot[XBlock.LeftBot[ib]] == ib)
			{
				if (j == 0)
				{
					if (XBlock.BotRight[XBlock.LeftBot[ib]] == XBlock.LeftBot[ib]) // no botom of leftbot block
					{
						w3 = T(0.5) * (T(1.0) - w1);
						w2 = w3;
						it = ir;

					}
					else if (XBlock.level[XBlock.BotRight[XBlock.LeftBot[ib]]] < XBlock.level[XBlock.LeftBot[ib]]) // exists but is coarser
					{
						w1 = T(4.0 / 10.0);
						w2 = T(5.0 / 10.0);
						w3 = T(1.0 / 10.0);
						it = memloc(XParam, XParam.blkwidth-1, XParam.blkwidth - 1, XBlock.BotRight[XBlock.LeftBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotRight[XBlock.LeftBot[ib]]] == XBlock.level[XBlock.LeftBot[ib]]) // exists with same level
					{
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.BotRight[XBlock.LeftBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotRight[XBlock.LeftBot[ib]]] > XBlock.level[XBlock.LeftBot[ib]]) // exists with higher level
					{
						w1 = T(1.0 / 4.0);
						w2 = T(1.0 / 2.0);
						w3 = T(1.0 / 4.0);
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.BotRight[XBlock.LeftBot[ib]]);
					}
					
					
				}
									
				
			}
			else//righttopleftif == ib
			{
				if (j == (XParam.blkwidth - 1))
				{
					if (XBlock.TopRight[XBlock.LeftTop[ib]] == XBlock.LeftTop[ib]) // no botom of leftbot block
					{
						w3 = T(0.5*(1.0-w1));
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.TopRight[XBlock.LeftTop[ib]]] < XBlock.level[XBlock.LeftTop[ib]]) // exists but is coarser
					{
						w1 = T(4.0 / 10.0);
						w2 = T(1.0 / 10.0);
						w3 = T(5.0 / 10.0);
						ir = memloc(XParam, XParam.blkwidth - 1,0, XBlock.TopRight[XBlock.LeftTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopRight[XBlock.LeftTop[ib]]] == XBlock.level[XBlock.LeftTop[ib]]) // exists with same level
					{
						ir = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.TopRight[XBlock.LeftTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopRight[XBlock.LeftTop[ib]]] > XBlock.level[XBlock.LeftTop[ib]]) // exists with higher level
					{
						w1 = T(1.0 / 4.0);
						w2 = T(1.0 / 2.0);
						w3 = T(1.0 / 4.0);
						ir = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.TopRight[XBlock.LeftTop[ib]]);
					}
				}
				//
			}


			z[write] = w1 * z[ii] + w2 * z[ir] + w3 * z[it];

			

		}
	}
	


}



template <class T> __global__ void fillLeft(int halowidth, int* active, int * level, int* leftbot, int * lefttop, int * rightbot, int* botright,int * topright, T * a)
{
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	//unsigned int ix = 0;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int lev = level[ib];
	int LB = leftbot[ib];
	int LT = lefttop[ib];

	int RBLB = rightbot[LB];
	int BRLB = botright[LB];
	int TRLT = topright[LT];

	int levBRLB = level[BRLB];
	int levTRLT = level[TRLT];
	int levLB = level[LB];
	int levLT = level[LT];
	int write = memloc(halowidth, blkmemwidth, -1, iy, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;

	if (LB == ib)
	{
		if (iy < (blockDim.y / 2))
		{
			read = memloc(halowidth, blkmemwidth, 0, iy, ib);
			a_read = a[read];
		}
		else
		{
			if (LT == ib)
			{
				read = memloc(halowidth, blkmemwidth, 0, iy, ib);
				a_read = a[read];

			}
			else
			{
				
				jj = (iy - (blockDim.y / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj, LT);
				ir = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj, LT);
				it = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj + 1, LT);
				itr = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj + 1, LT);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
		
	}
	else if (levLB == lev )
	{
		read = memloc(halowidth, blkmemwidth, (blockDim.y - 1), iy, LB);
		a_read = a[read];
	}
	else if (levLB > lev)
	{
		if (iy < (blockDim.y / 2))
		{
			jj = iy * 2;
			ii = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj, LB);
			ir = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj, LB);
			it = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj + 1, LB);
			itr = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj + 1, LB);
			a_read= T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		else
		{
			if (LT == ib)
			{
				read = memloc(halowidth, blkmemwidth, 0, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;
								
				ii = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj, LT);
				ir = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj, LT);
				it = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj + 1, LT);
				itr = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj + 1, LT);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levLB < lev)
	{
		jj = RBLB==ib? ceil(iy * (T)0.5): ceil(iy * (T)0.5) + blockDim.y / 2;
		w1 = (T)1.0 / (T)3.0;
		w2 = ceil(iy * (T)0.5) * 2 > iy ? T(1.0 / 6.0) : T(0.5);
		w3 = ceil(iy * (T)0.5) * 2 > iy ? T(0.5) : T(1.0 / 6.0);

		ii = memloc(halowidth, blkmemwidth, 0, iy, ib);
		ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, jj, LB);
		it = memloc(halowidth, blkmemwidth, blockDim.y - 1, jj - 1, LB);
		if (RBLB == ib)
		{
			if (iy == 0)
			{
				if (BRLB == LB)
				{
					w3 = (T)0.5 * (1.0 - w1);
					w2 = w3;
					it = ir;
				}
				else if (levBRLB < levLB)
				{
					w1 = T(4.0 / 10.0);
					w2 = T(5.0 / 10.0);
					w3 = T(1.0 / 10.0);
					it = memloc(halowidth, blkmemwidth, blockDim.y - 1, blockDim.y - 1, BRLB);

				}
				else if (levBRLB == levLB)
				{
					it = memloc(halowidth, blkmemwidth, blockDim.y - 1, blockDim.y - 1, BRLB);
				}
				else if (levBRLB > levLB)
				{
					w1 = T(1.0 / 4.0);
					w2 = T(1.0 / 2.0);
					w3 = T(1.0 / 4.0);
					it = memloc(halowidth, blkmemwidth, blockDim.y - 1, blockDim.y - 1, BRLB);
				}
			}
		}
		else
		{
			if (iy == (blockDim.y - 1))
			{
				if (TRLT == LT)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					ir = it;
				}
				else if (levTRLT < levLT)
				{
					w1 = 4.0 / 10.0;
					w2 = 1.0 / 10.0;
					w3 = 5.0 / 10.0;
					ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, 0, TRLT);
				}
				else if (levTRLT == levLT)
				{
					ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, 0, TRLT);
				}
				else if (levTRLT > levLT)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, 0, TRLT);
					
				}
			}
		}
		a_read = w1 * a[ii] + w2 * a[ir] + w3 * a[it];
	}

	a[write] = a_read;
}
template __global__ void fillLeft<float>(int halowidth, int* active, int* level, int* leftbot, int* lefttop, int* rightbot, int* botright, int* topright, float* a);
template __global__ void fillLeft<double>(int halowidth, int* active, int* level, int* leftbot, int* lefttop, int* rightbot, int* botright, int* topright, double* a);

/*! \fn void void fillLeftnew(...)
* \brief New way of filling the left halo 2 blocks at a time to maximize GPU occupancy
*
* ## Description
* This fuction is a wraping fuction of the halo functions for CPU. It is called from another wraping function to keep things clean.
* In a sense this is the third (and last) layer of wrapping
*
*/
template <class T> __global__ void fillLeftnew(int halowidth, int nblk, int* active, int* level, int* leftbot, int* lefttop, int* rightbot, int* botright, int* topright, T* a)
{
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	//unsigned int ix = 0;
	int iy = threadIdx.y;
	//need to take min of ibl or total number of blks in case nblk is not dividable by 2
	int ibl = threadIdx.x + blockIdx.x * blockDim.x;;
	if (ibl < nblk)
	{


		int ib = active[ibl];

		int lev = level[ib];
		int LB = leftbot[ib];
		int LT = lefttop[ib];

		int RBLB = rightbot[LB];
		int BRLB = botright[LB];
		int TRLT = topright[LT];

		int levBRLB = level[BRLB];
		int levTRLT = level[TRLT];
		int levLB = level[LB];
		int levLT = level[LT];
		int write = memloc(halowidth, blkmemwidth, -1, iy, ib);
		int read;
		int jj, ii, ir, it, itr;
		T a_read;
		T w1, w2, w3;

		if (LB == ib)
		{
			if (iy < (blockDim.y / 2))
			{
				read = memloc(halowidth, blkmemwidth, 0, iy, ib);
				a_read = a[read];
			}
			else
			{
				if (LT == ib)
				{
					read = memloc(halowidth, blkmemwidth, 0, iy, ib);
					a_read = a[read];

				}
				else
				{

					jj = (iy - (blockDim.y / 2)) * 2;
					ii = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj, LT);
					ir = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj, LT);
					it = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj + 1, LT);
					itr = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj + 1, LT);

					a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
				}
			}

		}
		else if (levLB == lev)
		{
			read = memloc(halowidth, blkmemwidth, (blockDim.y - 1), iy, LB);
			a_read = a[read];
		}
		else if (levLB > lev)
		{
			if (iy < (blockDim.y / 2))
			{
				jj = iy * 2;
				ii = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj, LB);
				ir = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj, LB);
				it = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj + 1, LB);
				itr = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj + 1, LB);
				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
			else
			{
				if (LT == ib)
				{
					read = memloc(halowidth, blkmemwidth, 0, iy, ib);
					a_read = a[read];
				}
				else
				{
					jj = (iy - (blockDim.y / 2)) * 2;

					ii = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj, LT);
					ir = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj, LT);
					it = memloc(halowidth, blkmemwidth, (blockDim.y - 1), jj + 1, LT);
					itr = memloc(halowidth, blkmemwidth, (blockDim.y - 2), jj + 1, LT);

					a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
				}
			}
		}
		else if (levLB < lev)
		{
			jj = RBLB == ib ? ceil(iy * (T)0.5) : ceil(iy * (T)0.5) + blockDim.y / 2;
			w1 = (T)1.0 / (T)3.0;
			w2 = ceil(iy * (T)0.5) * 2 > iy ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(iy * (T)0.5) * 2 > iy ? T(0.5) : T(1.0 / 6.0);

			ii = memloc(halowidth, blkmemwidth, 0, iy, ib);
			ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, jj, LB);
			it = memloc(halowidth, blkmemwidth, blockDim.y - 1, jj - 1, LB);
			if (RBLB == ib)
			{
				if (iy == 0)
				{
					if (BRLB == LB)
					{
						w3 = (T)0.5 * (1.0 - w1);
						w2 = w3;
						it = ir;
					}
					else if (levBRLB < levLB)
					{
						w1 = T(4.0 / 10.0);
						w2 = T(5.0 / 10.0);
						w3 = T(1.0 / 10.0);
						it = memloc(halowidth, blkmemwidth, blockDim.y - 1, blockDim.y - 1, BRLB);

					}
					else if (levBRLB == levLB)
					{
						it = memloc(halowidth, blkmemwidth, blockDim.y - 1, blockDim.y - 1, BRLB);
					}
					else if (levBRLB > levLB)
					{
						w1 = T(1.0 / 4.0);
						w2 = T(1.0 / 2.0);
						w3 = T(1.0 / 4.0);
						it = memloc(halowidth, blkmemwidth, blockDim.y - 1, blockDim.y - 1, BRLB);
					}
				}
			}
			else
			{
				if (iy == (blockDim.y - 1))
				{
					if (TRLT == LT)
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						ir = it;
					}
					else if (levTRLT < levLT)
					{
						w1 = 4.0 / 10.0;
						w2 = 1.0 / 10.0;
						w3 = 5.0 / 10.0;
						ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, 0, TRLT);
					}
					else if (levTRLT == levLT)
					{
						ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, 0, TRLT);
					}
					else if (levTRLT > levLT)
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(halowidth, blkmemwidth, blockDim.y - 1, 0, TRLT);

					}
				}
			}
			a_read = w1 * a[ii] + w2 * a[ir] + w3 * a[it];
		}

		a[write] = a_read;
	}
}
template __global__ void fillLeftnew<float>(int halowidth, int nblk, int* active, int* level, int* leftbot, int* lefttop, int* rightbot, int* botright, int* topright, float* a);
template __global__ void fillLeftnew<double>(int halowidth, int nblk, int* active, int* level, int* leftbot, int* lefttop, int* rightbot, int* botright, int* topright, double* a);


template <class T> void fillLeftFlux(Param XParam, bool doProlongation, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii,  it;


	if (XBlock.LeftBot[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.LeftTop[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, -1, j, ib);
				jj = (j - XParam.blkwidth / 2) * 2;
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, XBlock.LeftTop[ib]);
				//ir = memloc(XParam, (XParam.blkwidth - 2), jj, XBlock.LeftTop[ib]);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, XBlock.LeftTop[ib]);
				//itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, XBlock.LeftTop[ib]);

				z[write] = T(0.5) * (z[ii]  + z[it]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.LeftBot[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, -1, j, ib);
			read = memloc(XParam, (XParam.blkwidth - 1), j, XBlock.LeftBot[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.LeftBot[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, -1, j, ib);

			jj = j * 2;
			bb = XBlock.LeftBot[ib];

			ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
			//ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
			it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
			//itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

			z[write] = T(0.5) * (z[ii] + z[it]);
		}
		//now find out aboy lefttop block
		if (XBlock.LeftTop[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.LeftTop[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, -1, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, (XParam.blkwidth - 1), jj, bb);
				//ir = memloc(XParam, (XParam.blkwidth - 2), jj, bb);
				it = memloc(XParam, (XParam.blkwidth - 1), jj + 1, bb);
				//itr = memloc(XParam, (XParam.blkwidth - 2), jj + 1, bb);

				z[write] = T(0.5) * (z[ii] + z[it]);
			}
		}

	}
	else if (XBlock.level[XBlock.LeftBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, -1, j, ib);

			//T w1, w2, w3;


			int jj = XBlock.RightBot[XBlock.LeftBot[ib]] == ib ? ftoi(ceil(j * (T)0.5)) : ftoi(ceil(j * (T)0.5) + XParam.blkwidth / 2);
			

			ii = memloc(XParam, XParam.blkwidth - 1, jj, XBlock.LeftBot[ib]);
			if (doProlongation)
				z[write] = z[ii];


			
		}
	}



}



template <class T> void fillRight(Param XParam, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.RightBot[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, XParam.blkwidth-1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.RightTop[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, XParam.blkwidth - 1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];

				
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, XParam.blkwidth, j, ib);
				jj = (j - (XParam.blkwidth / 2)) * 2;
				ii = memloc(XParam, 0, jj, XBlock.RightTop[ib]);
				ir = memloc(XParam, 1, jj, XBlock.RightTop[ib]);
				it = memloc(XParam, 0, jj + 1, XBlock.RightTop[ib]);
				itr = memloc(XParam, 1, jj + 1, XBlock.RightTop[ib]);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.RightBot[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, XParam.blkwidth, j, ib);
			read = memloc(XParam, 0, j, XBlock.RightBot[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, XParam.blkwidth, j, ib);

			jj = j * 2;
			bb = XBlock.RightBot[ib];

			ii = memloc(XParam, 0, jj, bb);
			ir = memloc(XParam, 1, jj, bb);
			it = memloc(XParam, 0, jj + 1, bb);
			itr = memloc(XParam, 1, jj + 1, bb);

			z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.RightTop[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, XParam.blkwidth-1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - (XParam.blkwidth / 2)) * 2;
				bb = XBlock.RightTop[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, 0, jj, bb);
				ir = memloc(XParam, 1, jj, bb);
				it = memloc(XParam, 0, jj + 1, bb);
				itr = memloc(XParam, 1, jj + 1, bb);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, XParam.blkwidth, j, ib);

			T w1, w2, w3;
			

			int jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? ftoi(ceil(j * (T)0.5)) : ftoi(ceil(j * (T)0.5) + XParam.blkwidth / 2);
			w1 = T(1.0 / 3.0);
			w2 = ceil(j * (T)0.5) * 2 > j ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(j * (T)0.5) * 2 > j ? T(0.5) : T(1.0 / 6.0);

			ii = memloc(XParam, XParam.blkwidth-1, j, ib);
			ir = memloc(XParam, 0, jj, XBlock.RightBot[ib]);
			it = memloc(XParam, 0, jj - 1, XBlock.RightBot[ib]);
			//2 scenarios here ib is the leftbot neighbour of the rightbot block or ib is the lefttop neighbour
			if (XBlock.LeftBot[XBlock.RightBot[ib]] == ib)
			{
				if (j == 0)
				{
					if (XBlock.BotLeft[XBlock.RightBot[ib]] == XBlock.RightBot[ib]) // no botom of leftbot block
					{
						w3 = T(0.5 * (1.0 - w1));
						w2 = w3;
						it = ir;

					}
					else if (XBlock.level[XBlock.BotLeft[XBlock.RightBot[ib]]] < XBlock.level[XBlock.RightBot[ib]]) // exists but is coarser
					{
						w1 = T(4.0 / 10.0);
						w2 = T(5.0 / 10.0);
						w3 = T(1.0 / 10.0);
						it = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.BotLeft[XBlock.RightBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotLeft[XBlock.RightBot[ib]]] == XBlock.level[XBlock.RightBot[ib]]) // exists with same level
					{
						it = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.BotLeft[XBlock.RightBot[ib]]);
					}
					else if (XBlock.level[XBlock.BotLeft[XBlock.RightBot[ib]]] > XBlock.level[XBlock.RightBot[ib]]) // exists with higher level
					{
						w1 = T(1.0 / 4.0);
						w2 = T(1.0 / 2.0);
						w3 = T(1.0 / 4.0);
						it = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.BotLeft[XBlock.RightBot[ib]]);
					}


				}


			}
			else//
			{
				if (j == (XParam.blkwidth - 1))
				{
					if (XBlock.TopLeft[XBlock.RightTop[ib]] == XBlock.RightTop[ib]) // no botom of leftbot block
					{
						w3 = T(0.5 * (1.0 - w1));
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.TopLeft[XBlock.RightTop[ib]]] < XBlock.level[XBlock.RightTop[ib]]) // exists but is coarser
					{
						w1 = T(4.0 / 10.0);
						w2 = T(1.0 / 10.0);
						w3 = T(5.0 / 10.0);
						ir = memloc(XParam, 0, 0, XBlock.TopLeft[XBlock.RightTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopLeft[XBlock.RightTop[ib]]] == XBlock.level[XBlock.RightTop[ib]]) // exists with same level
					{
						ir = memloc(XParam, 0, 0, XBlock.TopLeft[XBlock.RightTop[ib]]);
					}
					else if (XBlock.level[XBlock.TopLeft[XBlock.RightTop[ib]]] > XBlock.level[XBlock.RightTop[ib]]) // exists with higher level
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(XParam, 0, 0, XBlock.TopLeft[XBlock.RightTop[ib]]);
					}
				}
				//
			}


			z[write] = w1 * z[ii] + w2 * z[ir] + w3 * z[it];
		}
	}



}



template <class T> __global__ void fillRight(int halowidth, int* active, int* level, int * rightbot,int* righttop,int * leftbot,int*botleft,int* topleft, T* a)
{
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	//unsigned int ix = blockDim.y - 1;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int RB = rightbot[ib];
	int RT = righttop[ib];
	//int LB = leftbot[ib];
	//int BL = botleft[ib];
	int LBRB = leftbot[RB];
	int TLRT = topleft[RT];
	int BLRB = botleft[RB];


	int lev = level[ib];
	int levRB = level[RB];
	int levRT = level[RT];
	int levBLRB = level[BLRB];
	int levTLRT = level[TLRT];

	int write = memloc(halowidth, blkmemwidth, blockDim.y, iy, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;


	if (RB == ib)
	{
		if (iy < (blockDim.y / 2))
		{
			read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
			a_read = a[read];
		}
		else
		{
			if (RT == ib)
			{
				read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
				ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
				it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
				itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levRB == lev)
	{
		read = memloc(halowidth, blkmemwidth, 0, iy, RB);
		a_read = a[read];
	}
	else if (levRB > lev)
	{
		if (iy < (blockDim.y / 2))
		{
			jj = iy * 2;


			ii = memloc(halowidth, blkmemwidth, 0, jj, RB);
			ir = memloc(halowidth, blkmemwidth, 1, jj, RB);
			it = memloc(halowidth, blkmemwidth, 0, jj + 1, RB);
			itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RB);

			a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		else
		{
			if (RT == ib)
			{
				read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;
				
				ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
				ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
				it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
				itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levRB < lev)
	{
		//
		jj = LBRB == ib ? ceil(iy * (T)0.5) : ceil(iy * (T)0.5) + blockDim.y / 2;
		w1 = 1.0 / 3.0;
		w2 = ceil(iy * (T)0.5) * 2 > iy ? T(1.0 / 6.0) : T(0.5);
		w3 = ceil(iy * (T)0.5) * 2 > iy ? T(0.5) : T(1.0 / 6.0);
		ii = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
		ir = memloc(halowidth, blkmemwidth, 0, jj, RB);
		it = memloc(halowidth, blkmemwidth, 0, jj - 1, RB);
		if (LBRB == ib)
		{
			if (iy == 0)
			{
				if (BLRB == RB)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					it = ir;
				}
				else if (levBLRB < levRB)
				{
					w1 = 4.0 / 10.0;
					w2 = 5.0 / 10.0;
					w3 = 1.0 / 10.0;
					it = memloc(halowidth, blkmemwidth, 0, blockDim.y - 1, BLRB);
				}
				else if (levBLRB == levRB)
				{
					it = memloc(halowidth, blkmemwidth, 0, blockDim.y - 1, BLRB);
				}
				else if (levBLRB > levRB)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					it = memloc(halowidth, blkmemwidth, 0, blockDim.y - 1, BLRB);
				}
			}
		}
		else
		{
			if (iy == (blockDim.y - 1))
			{
				if (TLRT == RT)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					ir = it;
				}
				else if (levTLRT < levRT)
				{
					w1 = 4.0 / 10.0;
					w2 = 1.0 / 10.0;
					w3 = 5.0 / 10.0;
					ir = memloc(halowidth, blkmemwidth, 0, 0, TLRT);
				}
				else if (levTLRT == levRT)
				{
					ir = memloc(halowidth, blkmemwidth, 0, 0, TLRT);
				}
				else if (levTLRT > levRT)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					ir = memloc(halowidth, blkmemwidth, 0, 0, TLRT);
				}
			}
		}

		a_read= w1 * a[ii] + w2 * a[ir] + w3 * a[it];
	}
	a[write] = a_read;
}

template __global__ void fillRight<float>(int halowidth, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, float* a);
template __global__ void fillRight<double>(int halowidth, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, double* a);

template <class T> __global__ void fillRightnew(int halowidth,int nblk, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, T* a)
{
	int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	//unsigned int ix = blockDim.y - 1;
	int iy = threadIdx.y;
	int ibl = threadIdx.x + blockIdx.x * blockDim.x;
	if (ibl < nblk)
	{
		int ib = active[ibl];

		int RB = rightbot[ib];
		int RT = righttop[ib];
		//int LB = leftbot[ib];
		//int BL = botleft[ib];
		int LBRB = leftbot[RB];
		int TLRT = topleft[RT];
		int BLRB = botleft[RB];


		int lev = level[ib];
		int levRB = level[RB];
		int levRT = level[RT];
		int levBLRB = level[BLRB];
		int levTLRT = level[TLRT];

		int write = memloc(halowidth, blkmemwidth, blockDim.y, iy, ib);
		int read;
		int jj, ii, ir, it, itr;
		T a_read;
		T w1, w2, w3;


		if (RB == ib)
		{
			if (iy < (blockDim.y / 2))
			{
				read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
				a_read = a[read];
			}
			else
			{
				if (RT == ib)
				{
					read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
					a_read = a[read];
				}
				else
				{
					jj = (iy - (blockDim.y / 2)) * 2;
					ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
					ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
					it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
					itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

					a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
				}
			}
		}
		else if (levRB == lev)
		{
			read = memloc(halowidth, blkmemwidth, 0, iy, RB);
			a_read = a[read];
		}
		else if (levRB > lev)
		{
			if (iy < (blockDim.y / 2))
			{
				jj = iy * 2;


				ii = memloc(halowidth, blkmemwidth, 0, jj, RB);
				ir = memloc(halowidth, blkmemwidth, 1, jj, RB);
				it = memloc(halowidth, blkmemwidth, 0, jj + 1, RB);
				itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RB);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
			else
			{
				if (RT == ib)
				{
					read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
					a_read = a[read];
				}
				else
				{
					jj = (iy - (blockDim.y / 2)) * 2;

					ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
					ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
					it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
					itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

					a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
				}
			}
		}
		else if (levRB < lev)
		{
			//
			jj = LBRB == ib ? ceil(iy * (T)0.5) : ceil(iy * (T)0.5) + blockDim.y / 2;
			w1 = 1.0 / 3.0;
			w2 = ceil(iy * (T)0.5) * 2 > iy ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(iy * (T)0.5) * 2 > iy ? T(0.5) : T(1.0 / 6.0);
			ii = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
			ir = memloc(halowidth, blkmemwidth, 0, jj, RB);
			it = memloc(halowidth, blkmemwidth, 0, jj - 1, RB);
			if (LBRB == ib)
			{
				if (iy == 0)
				{
					if (BLRB == RB)
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						it = ir;
					}
					else if (levBLRB < levRB)
					{
						w1 = 4.0 / 10.0;
						w2 = 5.0 / 10.0;
						w3 = 1.0 / 10.0;
						it = memloc(halowidth, blkmemwidth, 0, blockDim.y - 1, BLRB);
					}
					else if (levBLRB == levRB)
					{
						it = memloc(halowidth, blkmemwidth, 0, blockDim.y - 1, BLRB);
					}
					else if (levBLRB > levRB)
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						it = memloc(halowidth, blkmemwidth, 0, blockDim.y - 1, BLRB);
					}
				}
			}
			else
			{
				if (iy == (blockDim.y - 1))
				{
					if (TLRT == RT)
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						ir = it;
					}
					else if (levTLRT < levRT)
					{
						w1 = 4.0 / 10.0;
						w2 = 1.0 / 10.0;
						w3 = 5.0 / 10.0;
						ir = memloc(halowidth, blkmemwidth, 0, 0, TLRT);
					}
					else if (levTLRT == levRT)
					{
						ir = memloc(halowidth, blkmemwidth, 0, 0, TLRT);
					}
					else if (levTLRT > levRT)
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(halowidth, blkmemwidth, 0, 0, TLRT);
					}
				}
			}

			a_read = w1 * a[ii] + w2 * a[ir] + w3 * a[it];
		}
		a[write] = a_read;
	}
}

template __global__ void fillRightnew<float>(int halowidth, int nblk, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, float* a);
template __global__ void fillRightnew<double>(int halowidth, int nblk, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, double* a);



template <class T> void fillRightFlux(Param XParam, bool doProlongation, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, it;


	if (XBlock.RightBot[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, XParam.blkwidth - 1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.RightTop[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, XParam.blkwidth - 1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, XParam.blkwidth, j, ib);
				jj = (j - 8) * 2;
				ii = memloc(XParam, 0, jj, XBlock.RightTop[ib]);
				//ir = memloc(XParam, 1, jj, XBlock.RightTop[ib]);
				it = memloc(XParam, 0, jj + 1, XBlock.RightTop[ib]);
				//itr = memloc(XParam, 1, jj + 1, XBlock.RightTop[ib]);

				z[write] = T(0.5) * (z[ii] + z[it]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.RightBot[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, XParam.blkwidth, j, ib);
			read = memloc(XParam, 0, j, XBlock.RightBot[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.RightBot[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, XParam.blkwidth, j, ib);

			jj = j * 2;
			bb = XBlock.RightBot[ib];

			ii = memloc(XParam, 0, jj, bb);
			//ir = memloc(XParam, 1, jj, bb);
			it = memloc(XParam, 0, jj + 1, bb);
			//itr = memloc(XParam, 1, jj + 1, bb);

			//z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			z[write] = T(0.5) * (z[ii] + z[it]);
		}
		//now find out aboy lefttop block
		if (XBlock.RightTop[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, XParam.blkwidth - 1, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.RightTop[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, XParam.blkwidth, j, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, 0, jj, bb);
				//ir = memloc(XParam, 1, jj, bb);
				it = memloc(XParam, 0, jj + 1, bb);
				//itr = memloc(XParam, 1, jj + 1, bb);
				z[write] = T(0.5) * (z[ii] + z[it]);
				//z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.RightBot[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, XParam.blkwidth, j, ib);


			int jj = XBlock.LeftBot[XBlock.RightBot[ib]] == ib ? ftoi(floor(j * (T)0.5)) : ftoi(floor(j * (T)0.5) + XParam.blkwidth / 2);

			ii = memloc(XParam, 0, jj, XBlock.RightBot[ib]);
			if (doProlongation)
				z[write] = z[ii];
		}
	}



}

template void fillRightFlux<float>(Param XParam, bool doProlongation, int ib, BlockP<float> XBlock, float*& z);
template void fillRightFlux<double>(Param XParam, bool doProlongation, int ib, BlockP<double> XBlock, double*& z);



template <class T> __global__ void fillRightFlux(int halowidth, bool doProlongation, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, T* a)
{
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	//unsigned int ix = blockDim.y - 1;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int RB = rightbot[ib];
	int RT = righttop[ib];
	//int LB = leftbot[ib];
	//int BL = botleft[ib];
	int LBRB = leftbot[RB];
	//int TLRT = topleft[RT];
	//int BLRB = botleft[RB];


	int lev = level[ib];
	int levRB = level[RB];
	//int levRT = level[RT];
	//int levBLRB = level[BLRB];
	//int levTLRT = level[TLRT];

	int write = memloc(halowidth, blkmemwidth, blockDim.y, iy, ib);
	int read;
	int jj, ii, ir, it;
	T a_read;
	//T w1, w2;


	if (RB == ib)
	{
		if (iy < (blockDim.y / 2))
		{
			read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
			a_read = a[read];
		}
		else
		{
			if (RT == ib)
			{
				read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
				//ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
				it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
				//itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

				a_read = T(0.5) * (a[ii] + a[it]);
			}
		}
	}
	else if (levRB == lev)
	{
		read = memloc(halowidth, blkmemwidth, 0, iy, RB);
		a_read = a[read];
	}
	else if (levRB > lev)
	{
		if (iy < (blockDim.y / 2))
		{
			jj = iy * 2;


			ii = memloc(halowidth, blkmemwidth, 0, jj, RB);
			//ir = memloc(halowidth, blkmemwidth, 1, jj, RB);
			it = memloc(halowidth, blkmemwidth, 0, jj + 1, RB);
			//itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RB);

			a_read = T(0.5) * (a[ii] + a[it]);
		}
		else
		{
			if (RT == ib)
			{
				read = memloc(halowidth, blkmemwidth, blockDim.y - 1, iy, ib);
				a_read = a[read];
			}
			else
			{
				jj = (iy - (blockDim.y / 2)) * 2;

				ii = memloc(halowidth, blkmemwidth, 0, jj, RT);
				//ir = memloc(halowidth, blkmemwidth, 1, jj, RT);
				it = memloc(halowidth, blkmemwidth, 0, jj + 1, RT);
				//itr = memloc(halowidth, blkmemwidth, 1, jj + 1, RT);

				a_read = T(0.5) * (a[ii] + a[it] );
			}
		}
	}
	else if (levRB < lev)
	{
		//
		jj = LBRB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + blockDim.y / 2;
		
		
		ir = memloc(halowidth, blkmemwidth, 0, jj, RB);
		
		if (doProlongation)
			a_read = a[ir];
		
		else
			a_read = a[write];
	}
	a[write] = a_read;
}
template __global__ void fillRightFlux<float>(int halowidth, bool doProlongation, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, float* a);
template __global__ void fillRightFlux<double>(int halowidth, bool doProlongation, int* active, int* level, int* rightbot, int* righttop, int* leftbot, int* botleft, int* topleft, double* a);



template <class T> void fillBot(Param XParam, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.BotLeft[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.BotRight[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam,j, -1, ib);
				jj = (j - 8) * 2;
				ii = memloc(XParam,jj, (XParam.blkwidth - 1), XBlock.BotRight[ib]);
				ir = memloc(XParam,jj, (XParam.blkwidth - 2), XBlock.BotRight[ib]);
				it = memloc(XParam,jj+1, (XParam.blkwidth - 1), XBlock.BotRight[ib]);
				itr = memloc(XParam,jj+1, (XParam.blkwidth - 2), XBlock.BotRight[ib]);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.BotLeft[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam,j, -1, ib);
			read = memloc(XParam, j, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, j, -1, ib);

			jj = j * 2;
			bb = XBlock.BotLeft[ib];

			ii = memloc(XParam, jj, (XParam.blkwidth - 1), bb);
			ir = memloc(XParam, jj, (XParam.blkwidth - 2), bb);
			it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), bb);
			itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), bb);

			z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
		}
		//now find out aboy botright block
		if (XBlock.BotRight[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.BotRight[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, jj, (XParam.blkwidth - 1), bb);
				ir = memloc(XParam, jj, (XParam.blkwidth - 2), bb);
				it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), bb);
				itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), bb);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, j, -1, ib);

			T w1, w2, w3;
			

			int jj = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? ftoi(ceil(j * (T)0.5)) : ftoi(ceil(j * (T)0.5) + XParam.blkwidth / 2);
			w1 = T(1.0 / 3.0);
			w2 = ceil(j * (T)0.5) * 2 > j ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(j * (T)0.5) * 2 > j ? T(0.5) : T(1.0 / 6.0);

			ii = memloc(XParam, j, 0, ib);
			ir = memloc(XParam, jj, XParam.blkwidth - 1, XBlock.BotLeft[ib]);
			it = memloc(XParam, jj -1, XParam.blkwidth - 1, XBlock.BotLeft[ib]);
			//2 scenarios here ib is the rightbot neighbour of the leftbot block or ib is the righttop neighbour
			if (XBlock.TopLeft[XBlock.BotLeft[ib]] == ib)
			{
				if (j == 0)
				{
					if (XBlock.LeftTop[XBlock.BotLeft[ib]] == XBlock.BotLeft[ib]) // no botom of leftbot block
					{
						w3 = T(0.5 * (1.0 - w1));
						w2 = w3;
						it = ir;

					}
					else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] < XBlock.level[XBlock.BotLeft[ib]]) // exists but is coarser
					{
						w1 = T(4.0 / 10.0);
						w2 = T(5.0 / 10.0);
						w3 = T(1.0 / 10.0);
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
					}
					else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] == XBlock.level[XBlock.BotLeft[ib]]) // exists with same level
					{
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
					}
					else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] > XBlock.level[XBlock.BotLeft[ib]]) // exists with higher level
					{
						w1 = T(1.0 / 4.0);
						w2 = T(1.0 / 2.0);
						w3 = T(1.0 / 4.0);
						it = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
					}


				}


			}
			else//righttopleftif == ib
			{
				if (j == (XParam.blkwidth - 1))
				{
					if (XBlock.RightTop[XBlock.BotRight[ib]] == XBlock.BotRight[ib]) // no botom of leftbot block
					{
						w3 = T(0.5 * (1.0 - w1));
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.RightTop[XBlock.BotRight[ib]]] < XBlock.level[XBlock.BotRight[ib]]) // exists but is coarser
					{
						w1 = T(4.0 / 10.0);
						w2 = T(1.0 / 10.0);
						w3 = T(5.0 / 10.0);
						ir = memloc(XParam, 0,XParam.blkwidth - 1, XBlock.RightTop[XBlock.BotRight[ib]]);
					}
					else if (XBlock.level[XBlock.RightTop[XBlock.BotRight[ib]]] == XBlock.level[XBlock.BotRight[ib]]) // exists with same level
					{
						ir = memloc(XParam,0, XParam.blkwidth - 1, XBlock.RightTop[XBlock.BotRight[ib]]);
					}
					else if (XBlock.level[XBlock.RightTop[XBlock.BotRight[ib]]] > XBlock.level[XBlock.BotRight[ib]]) // exists with higher level
					{
						w1 = T(1.0 / 4.0);
						w2 = T(1.0 / 2.0);
						w3 = T(1.0 / 4.0);
						ir = memloc(XParam,0,XParam.blkwidth - 1, XBlock.RightTop[XBlock.BotRight[ib]]);
					}
				}
				//
			}


			z[write] = w1 * z[ii] + w2 * z[ir] + w3 * z[it];
		}
	}



}


template <class T> __global__ void fillBot(int halowidth, int* active, int* level, int* botleft, int* botright, int* topleft, int* lefttop, int* righttop, T* a)
{
	int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	//unsigned int iy = 0;
	int ibl = blockIdx.x;
	
	int ib = active[ibl];

	int BL = botleft[ib];
	int BR = botright[ib];
	int TLBL = topleft[BL];
	int LTBL = lefttop[BL];
	int RTBR = righttop[BR];

	int lev = level[ib];
	int levBL = level[BL];
	int levBR = level[BR];
	int levLTBL = level[LTBL];
	int levRTBR = level[RTBR];

	int write = memloc(halowidth, blkmemwidth, ix, -1, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;
	if (BL == ib)
	{
		if (ix < (blockDim.x / 2))
		{
			read = memloc(halowidth, blkmemwidth, ix, 0, ib);
			a_read = a[read];
		}
		else
		{
			if (BR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, 0, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x/2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 1), BR);
				ir = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 2), BR);
				it = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 1), BR);
				itr = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 2), BR);
				a_read = T(0.25)* (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levBL == lev)
	{
		read = memloc(halowidth, blkmemwidth, ix, (blockDim.x - 1), BL);
		a_read = a[read];
	}
	else if (levBL > lev)
	{
		if (ix < (blockDim.x / 2))
		{
			jj = ix * 2;
			ii = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 1), BL);
			ir = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 2), BL);
			it = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 1), BL);
			itr = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 2), BL);
			a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		else
		{
			if (BR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, 0, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x/2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 1), BR);
				ir = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 2), BR);
				it = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 1), BR);
				itr = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 2), BR);
				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levBL < lev)
	{
		jj = TLBL == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + blockDim.x / 2;
		w1 = 1.0 / 3.0;
		w2 = ceil(ix * (T)0.5) * 2 > ix ? T(1.0 / 6.0) : T(0.5);
		w3 = ceil(ix * (T)0.5) * 2 > ix ? T(0.5) : T(1.0 / 6.0);

		ii = memloc(halowidth, blkmemwidth, ix, 0, ib);
		ir = memloc(halowidth, blkmemwidth, jj, blockDim.x - 1, BL);
		it = memloc(halowidth, blkmemwidth, jj - 1, blockDim.x - 1, BL);

		if (TLBL == ib)
		{
			if (ix == 0)
			{
				if (LTBL == BL)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					it = ir;
				}
				else if (levLTBL < levBL)
				{
					w1 = 4.0 / 10.0;
					w2 = 5.0 / 10.0;
					w3 = 1.0 / 10.0;
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, LTBL);
				}
				else if (levLTBL == levBL)
				{
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, LTBL);
				}
				else if (levLTBL > levBL)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, LTBL);
				}
			}
		}
		else
		{
			if (ix == (blockDim.x - 1))
			{
				if (RTBR == BR)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					ir = it;
				}
				else if (levRTBR < levBR)
				{
					w1 = 4.0 / 10.0;
					w2 = 1.0 / 10.0;
					w3 = 5.0 / 10.0;
					ir = memloc(halowidth, blkmemwidth,0, blockDim.x - 1, RTBR);
				}
				else if (levRTBR == levBR)
				{
					ir = memloc(halowidth, blkmemwidth,0, blockDim.x - 1, RTBR);
				}
				else if (levRTBR > levBR)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					ir = memloc(halowidth, blkmemwidth, 0, blockDim.x - 1, RTBR);
				}
			}
		}
		a_read = w1 * a[ii] + w2 * a[ir] + w3 * a[it];
	}
	a[write] = a_read;

}
template __global__ void fillBot<float>(int halowidth, int* active, int* level, int* botleft, int* botright, int* topleft, int* lefttop, int* righttop, float* a);
template __global__ void fillBot<double>(int halowidth, int* active, int* level, int* botleft, int* botright, int* topleft, int* lefttop, int* righttop, double* a);

template <class T> __global__ void fillBotnew(int halowidth, int nblk, int* active, int* level, int* botleft, int* botright, int* topleft, int* lefttop, int* righttop, T* a)
{
	int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	//unsigned int iy = 0;
	int ibl = threadIdx.y + blockIdx.x * blockDim.y;
	if (ibl < nblk)
	{
		int ib = active[ibl];

		int BL = botleft[ib];
		int BR = botright[ib];
		int TLBL = topleft[BL];
		int LTBL = lefttop[BL];
		int RTBR = righttop[BR];

		int lev = level[ib];
		int levBL = level[BL];
		int levBR = level[BR];
		int levLTBL = level[LTBL];
		int levRTBR = level[RTBR];

		int write = memloc(halowidth, blkmemwidth, ix, -1, ib);
		int read;
		int jj, ii, ir, it, itr;
		T a_read;
		T w1, w2, w3;
		if (BL == ib)
		{
			if (ix < (blockDim.x / 2))
			{
				read = memloc(halowidth, blkmemwidth, ix, 0, ib);
				a_read = a[read];
			}
			else
			{
				if (BR == ib)
				{
					read = memloc(halowidth, blkmemwidth, ix, 0, ib);
					a_read = a[read];
				}
				else
				{
					jj = (ix - (blockDim.x / 2)) * 2;
					ii = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 1), BR);
					ir = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 2), BR);
					it = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 1), BR);
					itr = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 2), BR);
					a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
				}
			}
		}
		else if (levBL == lev)
		{
			read = memloc(halowidth, blkmemwidth, ix, (blockDim.x - 1), BL);
			a_read = a[read];
		}
		else if (levBL > lev)
		{
			if (ix < (blockDim.x / 2))
			{
				jj = ix * 2;
				ii = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 1), BL);
				ir = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 2), BL);
				it = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 1), BL);
				itr = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 2), BL);
				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
			else
			{
				if (BR == ib)
				{
					read = memloc(halowidth, blkmemwidth, ix, 0, ib);
					a_read = a[read];
				}
				else
				{
					jj = (ix - (blockDim.x / 2)) * 2;
					ii = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 1), BR);
					ir = memloc(halowidth, blkmemwidth, jj, (blockDim.x - 2), BR);
					it = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 1), BR);
					itr = memloc(halowidth, blkmemwidth, jj + 1, (blockDim.x - 2), BR);
					a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
				}
			}
		}
		else if (levBL < lev)
		{
			jj = TLBL == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + blockDim.x / 2;
			w1 = 1.0 / 3.0;
			w2 = ceil(ix * (T)0.5) * 2 > ix ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(ix * (T)0.5) * 2 > ix ? T(0.5) : T(1.0 / 6.0);

			ii = memloc(halowidth, blkmemwidth, ix, 0, ib);
			ir = memloc(halowidth, blkmemwidth, jj, blockDim.x - 1, BL);
			it = memloc(halowidth, blkmemwidth, jj - 1, blockDim.x - 1, BL);

			if (TLBL == ib)
			{
				if (ix == 0)
				{
					if (LTBL == BL)
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						it = ir;
					}
					else if (levLTBL < levBL)
					{
						w1 = 4.0 / 10.0;
						w2 = 5.0 / 10.0;
						w3 = 1.0 / 10.0;
						it = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, LTBL);
					}
					else if (levLTBL == levBL)
					{
						it = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, LTBL);
					}
					else if (levLTBL > levBL)
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						it = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, LTBL);
					}
				}
			}
			else
			{
				if (ix == (blockDim.x - 1))
				{
					if (RTBR == BR)
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						ir = it;
					}
					else if (levRTBR < levBR)
					{
						w1 = 4.0 / 10.0;
						w2 = 1.0 / 10.0;
						w3 = 5.0 / 10.0;
						ir = memloc(halowidth, blkmemwidth, 0, blockDim.x - 1, RTBR);
					}
					else if (levRTBR == levBR)
					{
						ir = memloc(halowidth, blkmemwidth, 0, blockDim.x - 1, RTBR);
					}
					else if (levRTBR > levBR)
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(halowidth, blkmemwidth, 0, blockDim.x - 1, RTBR);
					}
				}
			}
			a_read = w1 * a[ii] + w2 * a[ir] + w3 * a[it];
		}
		a[write] = a_read;
	}

}
template __global__ void fillBotnew<float>(int halowidth, int nblk, int* active, int* level, int* botleft, int* botright, int* topleft, int* lefttop, int* righttop, float* a);
template __global__ void fillBotnew<double>(int halowidth, int nblk, int* active, int* level, int* botleft, int* botright, int* topleft, int* lefttop, int* righttop, double* a);

template <class T> void fillBotFlux(Param XParam, bool doProlongation, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it;


	if (XBlock.BotLeft[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.BotRight[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, j, -1, ib);
				jj = (j - 8) * 2;
				ii = memloc(XParam, jj, (XParam.blkwidth - 1), XBlock.BotRight[ib]);
				//ir = memloc(XParam, jj, (XParam.blkwidth - 2), XBlock.BotRight[ib]);
				it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), XBlock.BotRight[ib]);
				//itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), XBlock.BotRight[ib]);

				z[write] = T(0.5) * (z[ii] + z[it] );

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.BotLeft[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, j, -1, ib);
			read = memloc(XParam, j, (XParam.blkwidth - 1), XBlock.BotLeft[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.BotLeft[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, j, -1, ib);

			jj = j * 2;
			bb = XBlock.BotLeft[ib];

			ii = memloc(XParam, jj, (XParam.blkwidth - 1), bb);
			//ir = memloc(XParam, jj, (XParam.blkwidth - 2), bb);
			it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), bb);
			//itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), bb);

			z[write] = T(0.5) * (z[ii] + z[it]);
		}
		//now find out aboy botright block
		if (XBlock.BotRight[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, 0, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - 8) * 2;
				bb = XBlock.BotRight[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, -1, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, jj, (XParam.blkwidth - 1), bb);
				//ir = memloc(XParam, jj, (XParam.blkwidth - 2), bb);
				it = memloc(XParam, jj + 1, (XParam.blkwidth - 1), bb);
				//itr = memloc(XParam, jj + 1, (XParam.blkwidth - 2), bb);

				z[write] = T(0.5) * (z[ii]  + z[it] );
			}
		}

	}
	else if (XBlock.level[XBlock.BotLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, j, -1, ib);

			//T w1, w2, w3;


			int jj = XBlock.TopLeft[XBlock.BotLeft[ib]] == ib ? ftoi(ceil(j * (T)0.5)) : ftoi(ceil(j * (T)0.5) + XParam.blkwidth / 2);
			

			//ii = memloc(XParam, j, 0, ib);
			ir = memloc(XParam, jj, XParam.blkwidth - 1, XBlock.BotLeft[ib]);
			if(doProlongation)
				z[write] = z[ir];
		}
	}



}

template <class T> void fillTop(Param XParam, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it, itr;


	if (XBlock.TopLeft[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam,j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam,j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.TopRight[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, j, XParam.blkwidth, ib);
				jj = (j - (XParam.blkwidth / 2)) * 2;
				ii = memloc(XParam, jj, 0, XBlock.TopRight[ib]);
				ir = memloc(XParam, jj, 1, XBlock.TopRight[ib]);
				it = memloc(XParam, jj + 1, 0, XBlock.TopRight[ib]);
				itr = memloc(XParam, jj + 1, 1, XBlock.TopRight[ib]);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.TopLeft[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, j, XParam.blkwidth, ib);
			read = memloc(XParam, j, 0, XBlock.TopLeft[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, j, XParam.blkwidth, ib);

			jj = j * 2;
			bb = XBlock.TopLeft[ib];

			ii = memloc(XParam,jj, 0, bb);
			ir = memloc(XParam,jj, 1, bb);
			it = memloc(XParam,jj + 1, 0, bb);
			itr = memloc(XParam,jj + 1, 1, bb);

			z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
		}
		//now find out aboy lefttop block
		if (XBlock.TopRight[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam,j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - (XParam.blkwidth / 2)) * 2;
				bb = XBlock.TopRight[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j , XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam,jj, 0, bb);
				ir = memloc(XParam,jj, 1, bb);
				it = memloc(XParam,jj + 1, 0, bb);
				itr = memloc(XParam,jj + 1, 1, bb);

				z[write] = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
			}
		}

	}
	else if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam,j, XParam.blkwidth, ib);

			T w1, w2, w3;
			

			int jj = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? ftoi(ceil(j * (T)0.5)) : ftoi(ceil(j * (T)0.5) + XParam.blkwidth / 2);
			w1 = T(1.0 / 3.0);
			w2 = ceil(j * (T)0.5) * 2 > j ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(j * (T)0.5) * 2 > j ? T(0.5) : T(1.0 / 6.0);

			ii = memloc(XParam,j, XParam.blkwidth - 1, ib);
			ir = memloc(XParam,jj, 0,  XBlock.TopLeft[ib]);
			it = memloc(XParam,jj-1, 0, XBlock.TopLeft[ib]);
			//2 scenarios here ib is the leftbot neighbour of the rightbot block or ib is the lefttop neighbour
			if (XBlock.BotLeft[XBlock.TopLeft[ib]] == ib)
			{
				if (j == 0)
				{
					if (XBlock.LeftBot[XBlock.TopLeft[ib]] == XBlock.TopLeft[ib]) // no botom of leftbot block
					{
						w3 = T(0.5 * (1.0 - w1));
						w2 = w3;
						it = ir;

					}
					else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] < XBlock.level[XBlock.TopLeft[ib]]) // exists but is coarser
					{
						w1 = T(4.0 / 10.0);
						w2 = T(5.0 / 10.0);
						w3 = T(1.0 / 10.0);
						it = memloc(XParam, XParam.blkwidth - 1,0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
					}
					else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] == XBlock.level[XBlock.TopLeft[ib]]) // exists with same level
					{
						it = memloc(XParam,  XParam.blkwidth - 1,0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
					}
					else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] > XBlock.level[XBlock.TopLeft[ib]]) // exists with higher level
					{
						w1 = T(1.0 / 4.0);
						w2 = T(1.0 / 2.0);
						w3 = T(1.0 / 4.0);
						it = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
					}


				}


			}
			else//
			{
				if (j == (XParam.blkwidth - 1))
				{
					if (XBlock.RightBot[XBlock.TopRight[ib]] == XBlock.TopRight[ib]) // no botom of leftbot block
					{
						w3 = T(0.5 * (1.0 - w1));
						w2 = w3;
						ir = it;

					}
					else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] < XBlock.level[XBlock.TopRight[ib]]) // exists but is coarser
					{
						w1 = T(4.0 / 10.0);
						w2 = T(1.0 / 10.0);
						w3 = T(5.0 / 10.0);
						ir = memloc(XParam, 0, 0, XBlock.RightBot[XBlock.TopRight[ib]]);
					}
					else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] == XBlock.level[XBlock.TopRight[ib]]) // exists with same level
					{
						ir = memloc(XParam, 0, 0, XBlock.RightBot[XBlock.TopRight[ib]]);
					}
					else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] > XBlock.level[XBlock.TopRight[ib]]) // exists with higher level
					{
						w1 = T(1.0 / 4.0);
						w2 = T(1.0 / 2.0);
						w3 = T(1.0 / 4.0);
						ir = memloc(XParam, 0,0, XBlock.RightBot[XBlock.TopRight[ib]]);
					}
				}
				//
			}


			z[write] = w1 * z[ii] + w2 * z[ir] + w3 * z[it];
		}
	}



}

template <class T> __global__ void fillTop(int halowidth, int* active, int* level,int * topleft, int * topright,int * botleft, int* leftbot, int* rightbot,  T* a)
{
	int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	//unsigned int iy = blockDim.x-1;
	int ibl = blockIdx.x;
	int ib = active[ibl];

	int TL = topleft[ib];
	int TR = topright[ib];
	int LBTL = leftbot[TL];
	int BLTL = botleft[TL];
	int RBTR = rightbot[TR];


	int lev = level[ib];
	int levTL = level[TL];
	int levTR = level[TR];
	int levLBTL = level[LBTL];
	int levRBTR = level[RBTR];

	int write = memloc(halowidth, blkmemwidth, ix, blockDim.x, ib);
	int read;
	int jj, ii, ir, it, itr;
	T a_read;
	T w1, w2, w3;

	if (TL == ib)
	{
		if (ix < (blockDim.x / 2))
		{
			read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
			a_read = a[read];
		}
		else
		{
			if (TR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
				ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
				it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
				itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);

				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levTL == lev)
	{
		read = memloc(halowidth, blkmemwidth, ix, 0, TL);
		a_read = a[read];
	}
	else if (levTL > lev)
	{
		if (ix < (blockDim.x / 2))
		{
			jj = ix * 2;
			

			ii = memloc(halowidth, blkmemwidth, jj, 0, TL);
			ir = memloc(halowidth, blkmemwidth, jj, 1, TL);
			it = memloc(halowidth, blkmemwidth, jj + 1, 0, TL);
			itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TL);
			a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
		}
		else
		{
			if (TR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, blockDim.x-1, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
				ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
				it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
				itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);
				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
		}
	}
	else if (levTL < lev)
	{
		jj = BLTL == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + blockDim.x / 2;
		w1 = 1.0 / 3.0;
		w2 = ceil(ix * (T)0.5) * 2 > ix ? T(1.0 / 6.0) : T(0.5);
		w3 = ceil(ix * (T)0.5) * 2 > ix ? T(0.5) : T(1.0 / 6.0);
		ii = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
		ir = memloc(halowidth, blkmemwidth, jj, 0, TL);
		it = memloc(halowidth, blkmemwidth, jj - 1, 0, TL);
		if (BLTL == ib)
		{
			if (ix == 0)
			{
				if (LBTL == TL)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					it = ir;
				}
				else if (levLBTL < levTL)
				{
					w1 = 4.0 / 10.0;
					w2 = 5.0 / 10.0;
					w3 = 1.0 / 10.0;
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, LBTL);
				}
				else if (levLBTL == levTL)
				{
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, LBTL);
				}
				else if (levLBTL > levTL)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					it = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, LBTL);
				}
			}
		}
		else
		{
			if (ix == blockDim.x - 1)
			{
				if (RBTR == TR)
				{
					w3 = 0.5 * (1.0 - w1);
					w2 = w3;
					ir = it;
				}
				else if (levRBTR < levTR)
				{
					w1 = 4.0 / 10.0;
					w2 = 1.0 / 10.0;
					w3 = 5.0 / 10.0;
					ir = memloc(halowidth, blkmemwidth, 0, 0, RBTR);
				}
				else if (levRBTR == levTR)
				{
					ir = memloc(halowidth, blkmemwidth, 0, 0, RBTR);
				}
				else if (levRBTR > levTR)
				{
					w1 = 1.0 / 4.0;
					w2 = 1.0 / 2.0;
					w3 = 1.0 / 4.0;
					ir = memloc(halowidth, blkmemwidth,0, 0, RBTR);
				}
			}
		}
		a_read= w1 * a[ii] + w2 * a[ir] + w3 * a[it];
	}
	a[write] = a_read;
}

template __global__ void fillTop<float>(int halowidth, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, float* a);
template __global__ void fillTop<double>(int halowidth, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, double* a);

template <class T> __global__ void fillTopnew(int halowidth, int nblk, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, T* a)
{
	int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	//unsigned int iy = blockDim.x-1;
	int ibl = threadIdx.y + blockIdx.x * blockDim.y;
	if (ibl < nblk)
	{
		int ib = active[ibl];

		int TL = topleft[ib];
		int TR = topright[ib];
		int LBTL = leftbot[TL];
		int BLTL = botleft[TL];
		int RBTR = rightbot[TR];


		int lev = level[ib];
		int levTL = level[TL];
		int levTR = level[TR];
		int levLBTL = level[LBTL];
		int levRBTR = level[RBTR];

		int write = memloc(halowidth, blkmemwidth, ix, blockDim.x, ib);
		int read;
		int jj, ii, ir, it, itr;
		T a_read;
		T w1, w2, w3;

		if (TL == ib)
		{
			if (ix < (blockDim.x / 2))
			{
				read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
				a_read = a[read];
			}
			else
			{
				if (TR == ib)
				{
					read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
					a_read = a[read];
				}
				else
				{
					jj = (ix - (blockDim.x / 2)) * 2;
					ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
					ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
					it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
					itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);

					a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
				}
			}
		}
		else if (levTL == lev)
		{
			read = memloc(halowidth, blkmemwidth, ix, 0, TL);
			a_read = a[read];
		}
		else if (levTL > lev)
		{
			if (ix < (blockDim.x / 2))
			{
				jj = ix * 2;


				ii = memloc(halowidth, blkmemwidth, jj, 0, TL);
				ir = memloc(halowidth, blkmemwidth, jj, 1, TL);
				it = memloc(halowidth, blkmemwidth, jj + 1, 0, TL);
				itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TL);
				a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
			}
			else
			{
				if (TR == ib)
				{
					read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
					a_read = a[read];
				}
				else
				{
					jj = (ix - (blockDim.x / 2)) * 2;
					ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
					ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
					it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
					itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);
					a_read = T(0.25) * (a[ii] + a[ir] + a[it] + a[itr]);
				}
			}
		}
		else if (levTL < lev)
		{
			jj = BLTL == ib ? ceil(ix * (T)0.5) : ceil(ix * (T)0.5) + blockDim.x / 2;
			w1 = 1.0 / 3.0;
			w2 = ceil(ix * (T)0.5) * 2 > ix ? T(1.0 / 6.0) : T(0.5);
			w3 = ceil(ix * (T)0.5) * 2 > ix ? T(0.5) : T(1.0 / 6.0);
			ii = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
			ir = memloc(halowidth, blkmemwidth, jj, 0, TL);
			it = memloc(halowidth, blkmemwidth, jj - 1, 0, TL);
			if (BLTL == ib)
			{
				if (ix == 0)
				{
					if (LBTL == TL)
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						it = ir;
					}
					else if (levLBTL < levTL)
					{
						w1 = 4.0 / 10.0;
						w2 = 5.0 / 10.0;
						w3 = 1.0 / 10.0;
						it = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, LBTL);
					}
					else if (levLBTL == levTL)
					{
						it = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, LBTL);
					}
					else if (levLBTL > levTL)
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						it = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, LBTL);
					}
				}
			}
			else
			{
				if (ix == blockDim.x - 1)
				{
					if (RBTR == TR)
					{
						w3 = 0.5 * (1.0 - w1);
						w2 = w3;
						ir = it;
					}
					else if (levRBTR < levTR)
					{
						w1 = 4.0 / 10.0;
						w2 = 1.0 / 10.0;
						w3 = 5.0 / 10.0;
						ir = memloc(halowidth, blkmemwidth, 0, 0, RBTR);
					}
					else if (levRBTR == levTR)
					{
						ir = memloc(halowidth, blkmemwidth, 0, 0, RBTR);
					}
					else if (levRBTR > levTR)
					{
						w1 = 1.0 / 4.0;
						w2 = 1.0 / 2.0;
						w3 = 1.0 / 4.0;
						ir = memloc(halowidth, blkmemwidth, 0, 0, RBTR);
					}
				}
			}
			a_read = w1 * a[ii] + w2 * a[ir] + w3 * a[it];
		}
		a[write] = a_read;
	}
}

template __global__ void fillTopnew<float>(int halowidth, int nblk, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, float* a);
template __global__ void fillTopnew<double>(int halowidth, int nblk, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, double* a);


template <class T> void fillTopFlux(Param XParam, bool doProlongation, int ib, BlockP<T> XBlock, T*& z)
{
	int jj, bb;
	int read, write;
	int ii, ir, it;


	if (XBlock.TopLeft[ib] == ib)//The lower half is a boundary 
	{
		for (int j = 0; j < (XParam.blkwidth / 2); j++)
		{

			read = memloc(XParam, j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
			z[write] = z[read];
		}

		if (XBlock.TopRight[ib] == ib) // boundary on the top half too
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else // boundary is only on the bottom half and implicitely level of lefttopib is levelib+1
		{

			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				write = memloc(XParam, j, XParam.blkwidth, ib);
				jj = (j - (XParam.blkwidth / 2)) * 2;
				ii = memloc(XParam, jj, 0, XBlock.TopRight[ib]);
				//ir = memloc(XParam, jj, 1, XBlock.TopRight[ib]);
				it = memloc(XParam, jj + 1, 0, XBlock.TopRight[ib]);
				//itr = memloc(XParam, jj + 1, 1, XBlock.TopRight[ib]);

				z[write] = T(0.5) * (z[ii] + z[it] );

			}
		}
	}
	else if (XBlock.level[ib] == XBlock.level[XBlock.TopLeft[ib]]) // LeftTop block does not exist
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			//

			write = memloc(XParam, j, XParam.blkwidth, ib);
			read = memloc(XParam, j, 0, XBlock.TopLeft[ib]);
			z[write] = z[read];
		}
	}
	else if (XBlock.level[XBlock.TopLeft[ib]] > XBlock.level[ib])
	{

		for (int j = 0; j < XParam.blkwidth / 2; j++)
		{

			write = memloc(XParam, j, XParam.blkwidth, ib);

			jj = j * 2;
			bb = XBlock.TopLeft[ib];

			ii = memloc(XParam, jj, 0, bb);
			//ir = memloc(XParam, jj, 1, bb);
			it = memloc(XParam, jj + 1, 0, bb);
			//itr = memloc(XParam, jj + 1, 1, bb);

			z[write] = T(0.5) * (z[ii]  + z[it]);
		}
		//now find out aboy lefttop block
		if (XBlock.TopRight[ib] == ib)
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//

				read = memloc(XParam, j, XParam.blkwidth - 1, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				z[write] = z[read];
			}
		}
		else
		{
			for (int j = (XParam.blkwidth / 2); j < (XParam.blkwidth); j++)
			{
				//
				jj = (j - (XParam.blkwidth / 2)) * 2;
				bb = XBlock.TopRight[ib];

				//read = memloc(XParam, 0, j, ib);// 1 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				write = memloc(XParam, j, XParam.blkwidth, ib); //0 + (j + XParam.halowidth) * XParam.blkmemwidth + ib * XParam.blksize;
				//z[write] = z[read];
				ii = memloc(XParam, jj, 0, bb);
				//ir = memloc(XParam, jj, 1, bb);
				it = memloc(XParam, jj + 1, 0, bb);
				//itr = memloc(XParam, jj + 1, 1, bb);

				z[write] = T(0.5) * (z[ii]  + z[it]);
			}
		}

	}
	else if (XBlock.level[XBlock.TopLeft[ib]] < XBlock.level[ib]) // Neighbour is coarser; using barycentric interpolation (weights are precalculated) for the Halo 
	{
		for (int j = 0; j < XParam.blkwidth; j++)
		{
			write = memloc(XParam, j, XParam.blkwidth, ib);
			int jj = XBlock.BotLeft[XBlock.TopLeft[ib]] == ib ? ftoi(floor(j * (T)0.5)) : ftoi(floor(j * (T)0.5) + XParam.blkwidth / 2);
						
			ir = memloc(XParam, jj, 0, XBlock.TopLeft[ib]);
			
			if (doProlongation)
				z[write] = z[ir];

		}
	}



}
template void fillTopFlux<float>(Param XParam, bool doProlongation, int ib, BlockP<float> XBlock, float*& z);
template void fillTopFlux<double>(Param XParam, bool doProlongation, int ib, BlockP<double> XBlock, double*& z);

template <class T> __global__ void fillTopFlux(int halowidth, bool doProlongation, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, T* a)
{
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	//unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	//unsigned int iy = blockDim.x - 1;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = active[ibl];

	int TL = topleft[ib];
	int TR = topright[ib];
	//int LBTL = leftbot[TL];
	int BLTL = botleft[TL];
	//int RBTR = rightbot[TR];


	int lev = level[ib];
	int levTL = level[TL];
	//int levTR = level[TR];
	//int levLBTL = level[LBTL];
	//int levRBTR = level[RBTR];

	int write = memloc(halowidth, blkmemwidth, ix, blockDim.x, ib);
	int read;
	int jj, ii, ir, it;
	T a_read;
	//T w1, w2, w3;

	if (TL == ib)
	{
		if (ix < (blockDim.x / 2))
		{
			read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
			a_read = a[read];
		}
		else
		{
			if (TR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
				//ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
				it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
				//itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);

				a_read = T(0.5) * (a[ii]  + a[it] );
			}
		}
	}
	else if (levTL == lev)
	{
		read = memloc(halowidth, blkmemwidth, ix, 0, TL);
		a_read = a[read];
	}
	else if (levTL > lev)
	{
		if (ix < (blockDim.x / 2))
		{
			jj = ix * 2;


			ii = memloc(halowidth, blkmemwidth, jj, 0, TL);
			//ir = memloc(halowidth, blkmemwidth, jj, 1, TL);
			it = memloc(halowidth, blkmemwidth, jj + 1, 0, TL);
			//itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TL);
			a_read = T(0.5) * (a[ii] + a[it]);
		}
		else
		{
			if (TR == ib)
			{
				read = memloc(halowidth, blkmemwidth, ix, blockDim.x - 1, ib);
				a_read = a[read];
			}
			else
			{
				jj = (ix - (blockDim.x / 2)) * 2;
				ii = memloc(halowidth, blkmemwidth, jj, 0, TR);
				//ir = memloc(halowidth, blkmemwidth, jj, 1, TR);
				it = memloc(halowidth, blkmemwidth, jj + 1, 0, TR);
				//itr = memloc(halowidth, blkmemwidth, jj + 1, 1, TR);
				a_read = T(0.5) * (a[ii] + a[it]);
			}
		}
	}
	else if (levTL < lev)
	{
		jj = BLTL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + blockDim.x / 2;
		
		ir = memloc(halowidth, blkmemwidth, jj, 0, TL);
		if (doProlongation)
			a_read = a[ir];
		else
			a_read =  a[write];
	}
	a[write] = a_read;
}

template __global__ void fillTopFlux<float>(int halowidth, bool doProlongation, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, float* a);
template __global__ void fillTopFlux<double>(int halowidth, bool doProlongation, int* active, int* level, int* topleft, int* topright, int* botleft, int* leftbot, int* rightbot, double* a);



template <class T> void fillCorners(Param XParam, BlockP<T> XBlock, T*& z)
{
	int ib;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillCorners(XParam, ib, XBlock, z);
		
	}

}
template void fillCorners<float>(Param XParam, BlockP<float> XBlock, float*& z);
template void fillCorners<double>(Param XParam, BlockP<double> XBlock, double*& z);


template <class T> void fillCorners(Param XParam, BlockP<T> XBlock, EvolvingP<T>& Xev)
{
	int ib;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		fillCorners(XParam, ib, XBlock, Xev.h);
		fillCorners(XParam, ib, XBlock, Xev.zs);
		fillCorners(XParam, ib, XBlock, Xev.u);
		fillCorners(XParam, ib, XBlock, Xev.v);
	}

}
template void fillCorners<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float>& Xev);
template void fillCorners<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double>& Xev);



template <class T> void fillCorners(Param XParam, int ib, BlockP<T> XBlock, T*& z)
{
	// Run only this function after the filling the other bit of halo (i.e. fctn fillleft...)
	// Most of the time the cormers are not needed. they are when refining a cell! 

	
	T zz;
	int write;
	int ii, ir, it, itr;


	// Bottom left corner
	write = memloc(XParam, -1, -1, ib);
	//check that there is a block there and if there is calculate the value depending on the level of that block
	if (XBlock.LeftTop[XBlock.BotLeft[ib]] == XBlock.BotLeft[ib]) // There is no block
	{
		zz = T(0.5) * (z[memloc(XParam, -1, 0, ib)] + z[memloc(XParam, 0, -1, ib)]);
	}
	else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] == XBlock.level[ib])
	{
		zz = z[memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]])];
	}
	else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] > XBlock.level[ib])
	{
		ii = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
		ir = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 2, XBlock.LeftTop[XBlock.BotLeft[ib]]);
		it = memloc(XParam, XParam.blkwidth - 2, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
		itr = memloc(XParam, XParam.blkwidth - 2, XParam.blkwidth - 2, XBlock.LeftTop[XBlock.BotLeft[ib]]);

		zz = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
	}
	else if (XBlock.level[XBlock.LeftTop[XBlock.BotLeft[ib]]] < XBlock.level[ib])
	{
		ii = memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth - 1, XBlock.LeftTop[XBlock.BotLeft[ib]]);
		ir = memloc(XParam, - 1, 0, ib);
		it = memloc(XParam,0, - 1, ib);
		zz = T(0.5) * z[ii] + T(0.25) * (z[ir] + z[it]);
	}

	z[write] = zz;

	// Top Left corner
	write = memloc(XParam, -1, XParam.blkwidth, ib);
	//check that there is a block there and if there is calculate the value depending on the level of that block
	if (XBlock.LeftBot[XBlock.TopLeft[ib]] == XBlock.TopLeft[ib]) // There is no block
	{
		zz = T(0.5) * (z[memloc(XParam, -1, XParam.blkwidth-1, ib)] + z[memloc(XParam, 0, XParam.blkwidth, ib)]);
	}
	else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] == XBlock.level[ib])
	{
		zz = z[memloc(XParam, XParam.blkwidth - 1, 0, XBlock.LeftBot[XBlock.TopLeft[ib]])];
	}
	else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] > XBlock.level[ib])
	{
		ii = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		ir = memloc(XParam, XParam.blkwidth - 1, 1, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		it = memloc(XParam, XParam.blkwidth - 2, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		itr = memloc(XParam, XParam.blkwidth - 2, 1, XBlock.LeftBot[XBlock.TopLeft[ib]]);

		zz = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
	}
	else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] < XBlock.level[ib])
	{
		ii = memloc(XParam, XParam.blkwidth - 1, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		ir = memloc(XParam, -1, XParam.blkwidth - 1, ib);
		it = memloc(XParam, 0, XParam.blkwidth, ib);
		zz = T(0.5) * z[ii] + T(0.25) * (z[ir] + z[it]);
	}

	z[write] = zz;

	//Top Right corner
	write = memloc(XParam, XParam.blkwidth, XParam.blkwidth, ib);
	//check that there is a block there and if there is calculate the value depending on the level of that block
	if (XBlock.RightBot[XBlock.TopRight[ib]] == XBlock.TopRight[ib]) // There is no block
	{
		zz = T(0.5) * (z[memloc(XParam, XParam.blkwidth, XParam.blkwidth - 1, ib)] + z[memloc(XParam, XParam.blkwidth - 1, XParam.blkwidth, ib)]);
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] == XBlock.level[ib])
	{
		zz = z[memloc(XParam, 0, 0, XBlock.RightBot[XBlock.TopRight[ib]])];
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.TopRight[ib]]] > XBlock.level[ib])
	{
		ii = memloc(XParam, 0, 0, XBlock.RightBot[XBlock.TopRight[ib]]);
		ir = memloc(XParam, 0, 1, XBlock.RightBot[XBlock.TopRight[ib]]);
		it = memloc(XParam, 1, 0, XBlock.RightBot[XBlock.TopRight[ib]]);
		itr = memloc(XParam, 1, 1, XBlock.RightBot[XBlock.TopRight[ib]]);

		zz = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
	}
	else if (XBlock.level[XBlock.LeftBot[XBlock.TopLeft[ib]]] < XBlock.level[ib])
	{
		ii = memloc(XParam, 0, 0, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		ir = memloc(XParam, XParam.blkwidth, XParam.blkwidth - 1, ib);
		it = memloc(XParam, XParam.blkwidth-1, XParam.blkwidth, ib);
		zz = T(0.5) * z[ii] + T(0.25) * ( z[ir] +  z[it]);
	}

	z[write] = zz;

	//Bot Right corner
	write = memloc(XParam, XParam.blkwidth, -1, ib);
	//check that there is a block there and if there is calculate the value depending on the level of that block
	if (XBlock.RightBot[XBlock.BotRight[ib]] == XBlock.BotRight[ib]) // There is no block
	{
		zz = T(0.5) * (z[memloc(XParam, XParam.blkwidth-1, - 1, ib)] + z[memloc(XParam, XParam.blkwidth , 0, ib)]);
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.BotRight[ib]]] == XBlock.level[ib])
	{
		zz = z[memloc(XParam, 0, XParam.blkwidth - 1, XBlock.RightBot[XBlock.BotRight[ib]])];
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.BotRight[ib]]] > XBlock.level[ib])
	{
		ii = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.RightBot[XBlock.BotRight[ib]]);
		ir = memloc(XParam, 0, XParam.blkwidth - 2, XBlock.RightBot[XBlock.BotRight[ib]]);
		it = memloc(XParam, 1, XParam.blkwidth - 1, XBlock.RightBot[XBlock.BotRight[ib]]);
		itr = memloc(XParam, 1, XParam.blkwidth - 2, XBlock.RightBot[XBlock.BotRight[ib]]);

		zz = T(0.25) * (z[ii] + z[ir] + z[it] + z[itr]);
	}
	else if (XBlock.level[XBlock.RightBot[XBlock.BotRight[ib]]] < XBlock.level[ib])
	{
		ii = memloc(XParam, 0, XParam.blkwidth - 1, XBlock.LeftBot[XBlock.TopLeft[ib]]);
		ir = memloc(XParam, XParam.blkwidth - 1, -1, ib);
		it = memloc(XParam, XParam.blkwidth, 0, ib);
		zz = T(0.5) * z[ii] + T(0.25) * (z[ir] + z[it]);
	}

	z[write] = zz;

}
template void fillCorners<float>(Param XParam, int ib, BlockP<float> XBlock, float*& z);
template void fillCorners<double>(Param XParam, int ib, BlockP<double> XBlock, double*& z);


template <class T> __global__ void fillCornersGPU(Param XParam, BlockP<T> XBlock, T* z)
{
	int blkmemwidth = blockDim.x + XParam.halowidth * 2;
	int halowidth = XParam.halowidth;
 	//unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	//unsigned int iy = blockDim.x-1;
	int ibl = blockIdx.x;
	int ib = XBlock.active[ibl];

	int TL = XBlock.TopLeft[ib];
	int TR = XBlock.TopRight[ib];
	int LB = XBlock.LeftBot[ib];
	int LT = XBlock.LeftTop[ib];
	int BL = XBlock.BotLeft[ib];
	int BR = XBlock.BotRight[ib];
	int RB = XBlock.RightBot[ib];
	int RT = XBlock.RightTop[ib];

	//int LBTL = XBlock.leftbot[TL];
	//int BLTL = XBlock.botleft[TL];
	//int RBTR = XBlock.rightbot[TR];

	int iout, ii;
	

	if (ix == 0 && iy == 0)
	{
		// Bot left corner

		iout = memloc(halowidth, blkmemwidth, -1, -1, ib);


		if (BL == ib && LB == ib)//
		{
			ii = memloc(halowidth, blkmemwidth, 0, 0, ib);
		}
		else
		{
			if (BL != ib)
			{
				ii = memloc(halowidth, blkmemwidth, -1, blockDim.x - 1, BL);
			}
			else
			{
				ii = memloc(halowidth, blkmemwidth, -1, -1, LB);
			}

		}
		z[iout] = z[ii];
	}
	if (ix == 0 && iy == blockDim.x)
	{
		// Top left corner
		iout = memloc(halowidth, blkmemwidth, -1, blockDim.x, ib);
		if (TL == ib && LT == ib)//
		{
			ii = memloc(halowidth, blkmemwidth, 0, blockDim.x - 1, ib);
		}
		else
		{
			if (TL != ib)
			{
				ii = memloc(halowidth, blkmemwidth, 0, -1, TL);
			}
			else
			{
				ii = memloc(halowidth, blkmemwidth, blockDim.x, blockDim.x - 1, LT);
			}

		}
		z[iout] = z[ii];
	}

	if (ix == blockDim.x && iy == blockDim.x)
	{
		
		// Top right corner
		iout = memloc(halowidth, blkmemwidth, blockDim.x, blockDim.x, ib);
		if (TR == ib && RT == ib)//
		{
			ii = memloc(halowidth, blkmemwidth, blockDim.x - 1, blockDim.x - 1, ib);
		}
		else
		{
			if (TR != ib)
			{
				ii = memloc(halowidth, blkmemwidth, blockDim.x, 0, TR);
			}
			else
			{
				ii = memloc(halowidth, blkmemwidth, 0, blockDim.x, RT);
			}

		}
		z[iout] = z[ii];

	}

	if (ix == blockDim.x && iy == 0)
	{
		// Bot right corner
		iout = memloc(halowidth, blkmemwidth, blockDim.x, -1, ib);
		if (BR == ib && RB == ib)//
		{
			ii = memloc(halowidth, blkmemwidth, blockDim.x - 1, 0, ib);
		}
		else
		{
			if (BR != ib)
			{
				ii = memloc(halowidth, blkmemwidth, blockDim.x, blockDim.x - 1, BR);
			}
			else
			{
				ii = memloc(halowidth, blkmemwidth, 0, blockDim.x, RB);
			}

		}
		z[iout] = z[ii];
	}
	
}
template __global__ void fillCornersGPU<float>(Param XParam, BlockP<float> XBlock, float* z);
template __global__ void fillCornersGPU<double>(Param XParam, BlockP<double> XBlock, double* z);
