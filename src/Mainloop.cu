#include "hip/hip_runtime.h"
#include "Mainloop.h"



template <class T> void MainLoop(Param &XParam, Forcing<float> XForcing, Model<T>& XModel, Model<T> &XModel_g)
{
	
	log("Initialising model main loop");
	
	Loop<T> XLoop = InitLoop(XParam, XModel);

	//Define some useful variables 
	Initmeanmax(XParam, XLoop, XModel, XModel_g);

	// fill halo for zb
	// only need to do that once 
	fillHaloC(XParam, XModel.blocks, XModel.zb);
	if (XParam.GPUDEVICE >= 0)
	{
		CUDA_CHECK(hipStreamCreate(&XLoop.streams[0]));
		fillHaloGPU(XParam, XModel_g.blocks, XLoop.streams[0], XModel_g.zb);

		hipStreamDestroy(XLoop.streams[0]);
	}



	log("\t\tCompleted");
	log("Model Running...");
	while (XLoop.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		updateBnd(XParam, XLoop, XForcing, XModel, XModel_g);
		

		// Calculate Forcing at this step
		updateforcing(XParam, XLoop, XForcing);

		// Core engine
		if (XParam.GPUDEVICE >= 0)
		{
			FlowGPU(XParam, XLoop, XForcing, XModel_g);
		}
		else
		{
			FlowCPU(XParam, XLoop, XForcing, XModel);
		}
				
		// Time keeping
		XLoop.totaltime = XLoop.totaltime + XLoop.dt;

		// Apply tsunami deformation if any (this needs to happen after totaltime has been incremented)
		deformstep(XParam, XLoop, XForcing.deform, XModel, XModel_g);

		// Do Sum & Max variables Here
		Calcmeanmax(XParam, XLoop, XModel, XModel_g);

		// Check & collect TSoutput
		pointoutputstep(XParam, XLoop, XModel, XModel_g);

		// Check for map output
		mapoutput(XParam, XLoop, XModel, XModel_g);

		// Reset mean/Max if needed
		resetmeanmax(XParam, XLoop, XModel, XModel_g);

		printstatus(XLoop.totaltime, XLoop.dt);
	}
	

	

}
template void MainLoop<float>(Param& XParam, Forcing<float> XForcing, Model<float>& XModel, Model<float>& XModel_g);
template void MainLoop<double>(Param& XParam, Forcing<float> XForcing, Model<double>& XModel, Model<double>& XModel_g);




 
template <class T> Loop<T> InitLoop(Param &XParam, Model<T> &XModel)
{
	Loop<T> XLoop;
	XLoop.atmpuni = XParam.Paref;
	XLoop.totaltime = XParam.totaltime;
	XLoop.nextoutputtime = XParam.totaltime + XParam.outputtimestep;
	
	// Prepare output files
	InitSave2Netcdf(XParam, XModel);
	InitTSOutput(XParam);
	// Add empty row for each output point
	// This will allow for the loop to each point to work later
	for (int o = 0; o < XParam.TSnodesout.size(); o++)
	{
		XLoop.TSAllout.push_back(std::vector<Pointout>());
	}

	// GPU stuff
	if (XParam.GPUDEVICE >= 0)
	{
		XLoop.blockDim = (16, 16, 1);
		XLoop.gridDim = (XParam.nblk, 1, 1);
	}

	//XLoop.hugenegval = std::numeric_limits<T>::min();

	XLoop.hugeposval = std::numeric_limits<T>::max();
	XLoop.hugenegval = T(-1.0)* XLoop.hugeposval;
	XLoop.epsilon = std::numeric_limits<T>::epsilon();


	XLoop.dtmax = initdt(XParam, XLoop, XModel);

	return XLoop;

}

template <class T> void updateBnd(Param XParam, Loop<T> XLoop, Forcing<float> XForcing, Model<T> XModel, Model<T> XModel_g)
{
	if (XParam.GPUDEVICE >= 0)
	{
		Flowbnd(XParam, XLoop, XModel_g.blocks, XForcing.left, XModel_g.evolv);
		Flowbnd(XParam, XLoop, XModel_g.blocks, XForcing.right, XModel_g.evolv);
		Flowbnd(XParam, XLoop, XModel_g.blocks, XForcing.top, XModel_g.evolv);
		Flowbnd(XParam, XLoop, XModel_g.blocks, XForcing.bot, XModel_g.evolv);
	}
	else
	{
		Flowbnd(XParam, XLoop, XModel.blocks, XForcing.left, XModel.evolv);
		Flowbnd(XParam, XLoop, XModel.blocks, XForcing.right, XModel.evolv);
		Flowbnd(XParam, XLoop, XModel.blocks, XForcing.top, XModel.evolv);
		Flowbnd(XParam, XLoop, XModel.blocks, XForcing.bot, XModel.evolv);
	}
}




template <class T> void mapoutput(Param XParam, Loop<T> &XLoop,Model<T> XModel, Model<T> XModel_g)
{
	XLoop.nstepout++;

	if (XLoop.nextoutputtime - XLoop.totaltime <= XLoop.dt * T(0.00001) && XParam.outputtimestep > 0.0)
	{
		char buffer[256];
		sprintf(buffer, "%e", XParam.outputtimestep / XLoop.nstepout);
		std::string str(buffer);

		log("Output to map. Totaltime = "+ std::to_string(XLoop.totaltime) +" s; Mean dt = " + str + " s");
		if (XParam.GPUDEVICE >= 0)
		{
			for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
			{
				CUDA_CHECK(hipMemcpy(XModel.OutputVarMap[XParam.outvars[ivar]], XModel_g.OutputVarMap[XParam.outvars[ivar]], XParam.nblkmem * XParam.blksize * sizeof(T), hipMemcpyDeviceToHost));
			}
		}
		
		Save2Netcdf(XParam, XLoop, XModel);


		XLoop.nextoutputtime = min(XLoop.nextoutputtime + XParam.outputtimestep, XParam.endtime);

		XLoop.nstepout = 0;
	}
}

template <class T> void pointoutputstep(Param XParam, Loop<T> &XLoop, Model<T> XModel, Model<T> XModel_g)
{
	//
	dim3 blockDim(XParam.blkwidth, XParam.blkwidth, 1);
	dim3 gridDim(XModel.bndblk.nblkTs, 1, 1);
	FILE* fsSLTS;
	if (XParam.GPUDEVICE>=0)
	{

		for (int o = 0; o < XParam.TSnodesout.size(); o++)
		{
			//
			Pointout stepread;
		
			stepread.time = XLoop.totaltime;
			stepread.zs = 0.0;// That is a bit useless
			stepread.h = 0.0;
			stepread.u = 0.0;
			stepread.v = 0.0;
			XLoop.TSAllout[o].push_back(stepread);
					
			
			storeTSout << <gridDim, blockDim, 0 >> > (XParam,(int)XParam.TSnodesout.size(), o, XLoop.nTSsteps, XParam.TSnodesout[o].block, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XModel_g.bndblk.Tsout, XModel_g.evolv, XModel_g.TSstore);
			CUDA_CHECK(hipDeviceSynchronize());
		}
		//CUDA_CHECK(hipDeviceSynchronize());
	}
	else
	{
		for (int o = 0; o < XParam.TSnodesout.size(); o++)
		{
			//
			Pointout stepread;

			int i = memloc(XParam.halowidth, XParam.blkmemwidth, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSnodesout[o].block);

			stepread.time = XLoop.totaltime;
			stepread.zs = XModel.evolv.zs[i];
			stepread.h = XModel.evolv.h[i];;
			stepread.u = XModel.evolv.u[i];;
			stepread.v = XModel.evolv.v[i];;
			XLoop.TSAllout[o].push_back(stepread);

		}
	}
	XLoop.nTSsteps++;

	// if the buffer is full or if the model is complete
	if ((XLoop.nTSsteps + 1) * XParam.TSnodesout.size() * 4 > XParam.maxTSstorage || XParam.endtime - XLoop.totaltime <= XLoop.dt * 0.00001f)
	{

		//Flush to disk
		if (XParam.GPUDEVICE >= 0 && XParam.TSnodesout.size() > 0)
		{
			CUDA_CHECK(hipMemcpy(XModel.TSstore, XModel_g.TSstore, XParam.maxTSstorage * sizeof(T), hipMemcpyDeviceToHost));
			int oo;
			
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				for (int istep = 0; istep < XLoop.TSAllout[o].size(); istep++)
				{
					oo = o * 4 + istep * XParam.TSnodesout.size() * 4;
					//
					XLoop.TSAllout[o][istep].h = XModel.TSstore[0 + oo];
					XLoop.TSAllout[o][istep].zs = XModel.TSstore[1 + oo];
					XLoop.TSAllout[o][istep].u = XModel.TSstore[2 + oo];
					XLoop.TSAllout[o][istep].v = XModel.TSstore[3 + oo];
				}
			}

		}
		for (int o = 0; o < XParam.TSnodesout.size(); o++)
		{
			fsSLTS = fopen(XParam.TSnodesout[o].outname.c_str(), "a");


			for (int n = 0; n < XLoop.nTSsteps; n++)
			{
				//


				fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", XLoop.TSAllout[o][n].time, XLoop.TSAllout[o][n].zs, XLoop.TSAllout[o][n].h, XLoop.TSAllout[o][n].u, XLoop.TSAllout[o][n].v);


			}
			fclose(fsSLTS);
			//reset output buffer
			XLoop.TSAllout[o].clear();
		}
		// Reset buffer counter
		XLoop.nTSsteps = 0;




	}
}


template <class T> __global__ void storeTSout(Param XParam,int noutnodes, int outnode, int istep,int blknode, int inode,int jnode, int * blkTS, EvolvingP<T> XEv, T* store)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = blkTS[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);

	//printf("ib=%d; ix=%d; iy=%d; blknode=%d; inode=%d; jnode=%d\n", ib, ix,iy,blknode,inode,jnode);

	if (ib == blknode && ix == inode && iy == jnode)
	{
		store[0 + outnode * 4 + istep * noutnodes * 4] = XEv.h[i];
		store[1 + outnode * 4 + istep * noutnodes * 4] = XEv.zs[i];
		store[2 + outnode * 4 + istep * noutnodes * 4] = XEv.u[i];
		store[3 + outnode * 4 + istep * noutnodes * 4] = XEv.v[i];



		//printf("XEv.h[i]=%f; store[h]=%f\n", XEv.h[i], store[0 + outnode * 4 + istep * noutnodes * 4]);
	}
}


template <class T> __host__ double initdt(Param XParam, Loop<T> XLoop, Model<T> XModel)
{
	//dim3 blockDim = (XParam.blkwidth, XParam.blkwidth, 1);
	//dim3 gridDim = (XParam.nblk, 1, 1);

	double initdt;

	XLoop.dtmax = XLoop.hugeposval;


	BlockP<T> XBlock = XModel.blocks;

	/*
	if (XParam.GPUDEVICE >= 0)
	{
		CalcInitdtGPU <<< gridDim, blockDim, 0 >>> (XParam, XModel.blocks, XModel.evolv, XModel.time.dtmax);
		initdt = double(CalctimestepGPU(XParam, XLoop, XModel.blocks, XModel.time));
	}
	else
	{
	*/
		CalcInitdtCPU(XParam, XModel.blocks, XModel.evolv, XModel.time.dtmax);
		initdt = double(CalctimestepCPU(XParam, XLoop, XModel.blocks, XModel.time));

	//}

	
	return initdt;
}
template __host__ double initdt<float>(Param XParam, Loop<float> XLoop, Model<float> XModel);
template __host__ double initdt<double>(Param XParam, Loop<double> XLoop, Model<double> XModel);

template <class T> __host__ void CalcInitdtCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEvolv, T* dtmax)
{
	int ib;
	T delta;
	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];

		delta = calcres(XParam.dx, XBlock.level[ib]);

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(XParam.halowidth, XParam.blkmemwidth, ix, iy, ib);

				dtmax[i] = delta / sqrt(XParam.g * std::max(XEvolv.h[i],T(XParam.eps)));
			}
		}
	}
}

template <class T> __global__ void CalcInitdtGPU(Param XParam, BlockP<T> XBlock,EvolvingP<T> XEvolv, T* dtmax)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;

	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	T delta = calcres(XParam.dx, XBlock.level[ib]);

	dtmax[i] = delta / sqrt(XParam.g * max(XEvolv.h[i],T(XParam.eps)));
}


template <class T> void printstatus(T totaltime, T dt)
{
	std::cout << "\r\e[K" << std::flush;
	std::cout << "\rtotaltime = "<< std::to_string(totaltime) << "   dt = " << std::to_string(dt) << std::flush;
	std::cout << "\r" << std::flush;
	//std::cout << std::endl; // all done
}
