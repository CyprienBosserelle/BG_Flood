#include "hip/hip_runtime.h"
#include "Kurganov.h"


template <class T> __global__ void updateKurgXGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T*zb)
{
	
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	int ix = threadIdx.x;
	int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];
	int RB, LBRB, LB, RBLB, levRB, levLB;
	RB = XBlock.RightBot[ib];
	levRB = XBlock.level[RB];
	LBRB = XBlock.LeftBot[RB];

	LB = XBlock.LeftBot[ib];
	levLB = XBlock.level[LB];
	RBLB = XBlock.RightBot[LB];

	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps)+epsi;
	T delta = calcres(T(XParam.dx), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	// This is based on kurganov and Petrova 2007


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix-1, iy, ib);

	

	
	T dhdxi = XGrad.dhdx[i];
	T dhdxmin = XGrad.dhdx[ileft];
	T cm = T(1.0);
	T fmu = T(1.0);

	T hi = XEv.h[i];

	T hn = XEv.h[ileft];
	

	if (hi > eps || hn > eps)
	{
		T dx, zi, zl, zn, zr, zlr, hl, up, hp, hr, um, hm, sl, sr,ga;

		// along X
		dx = delta * T(0.5);
		zi = XEv.zs[i] - hi;

		//printf("%f\n", zi);


		//zl = zi - dx*(dzsdx[i] - dhdx[i]);
		zl = zi - dx * (XGrad.dzsdx[i] - dhdxi);
		//printf("%f\n", zl);

		zn = XEv.zs[ileft] - hn;

		//printf("%f\n", zn);
		zr = zn + dx * (XGrad.dzsdx[ileft] - dhdxmin);


		zlr = max(zl, zr);

		//hl = hi - dx*dhdx[i];
		hl = hi - dx * dhdxi;
		up = XEv.u[i] - dx * XGrad.dudx[i];
		hp = max(T(0.0), hl + zl - zlr);

		hr = hn + dx * dhdxmin;
		um = XEv.u[ileft] + dx * XGrad.dudx[ileft];
		hm = max(T(0.0), hr + zr - zlr);

		ga = g * T(0.5);

		T fh, fu, fv, dt;

		
		//solver below also modifies fh and fu
		dt = KurgSolver(g, delta, epsi, CFL, cm, fmu, hp, hm, up, um, fh, fu);

		if (dt < dtmax[i])
		{
			dtmax[i] = dt;
		}
		else
		{
			dtmax[i] = T(1.0) / epsi;
		}
		


		if (fh > T(0.0))
		{
			fv = (XEv.v[ileft] + dx * XGrad.dvdx[ileft]) * fh;// Eq 3.7 third term? (X direction)
		}
		else
		{
			fv = (XEv.v[i] - dx * XGrad.dvdx[i]) * fh;
		}
		//fv = (fh > 0.f ? vv[xminus + iy*nx] + dx*dvdx[xminus + iy*nx] : vv[i] - dx*dvdx[i])*fh;
		//dtmax needs to be stored in an array and reduced at the end
		//dtmax = dtmaxf;
		//dtmaxtmp = min(dtmax, dtmaxtmp);
		/*if (ix == 11 && iy == 0)
		{
			printf("a=%f\t b=%f\t c=%f\t d=%f\n", ap*(qm*um + ga*hm2), -am*(qp*up + ga*hp2),( ap*(qm*um + g*sq(hm) / 2.0f) - am*(qp*up + g*sq(hp) / 2.0f) + ap*am*(qp - qm) ) *ad/100.0f, ad);
		}
		*/
		/*
		#### Topographic source term
		
		In the case of adaptive refinement, care must be taken to ensure
		well-balancing at coarse/fine faces (see [notes/balanced.tm]()). */
		if ((ix == blockDim.y) && levRB < lev)//(ix==16) i.e. in the right halo
		{
			int jj = LBRB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + blockDim.y / 2;
			int iright = memloc(halowidth, blkmemwidth, 0, jj, RB);;
			hi = XEv.h[iright];
			zi = zb[iright];
		}
		if ((ix == 0) && levLB < lev)//(ix==16) i.e. in the right halo
		{
			int jj = RBLB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + blockDim.y / 2;
			int ilc = memloc(halowidth, blkmemwidth, blockDim.y - 1, jj, LB);
			hn = XEv.h[ilc];
			zn = zb[ilc];
		}

		sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
		sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));
		
		////Flux update

		XFlux.Fhu[i] = fmu * fh;
		XFlux.Fqux[i] = fmu * (fu - sl);
		XFlux.Su[i] = fmu * (fu - sr);
		XFlux.Fqvx[i] = fmu * fv;
	}
	else
	{
		dtmax[i] = T(1.0) / epsi;
		XFlux.Fhu[i] = T(0.0);
		XFlux.Fqux[i] = T(0.0);
		XFlux.Su[i] = T(0.0);
		XFlux.Fqvx[i] = T(0.0);
	}

	


}
template __global__ void updateKurgXGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax, float* zb);
template __global__ void updateKurgXGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax, double *zb);


template <class T> __global__ void AddSlopeSourceXGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T * zb)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.y + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];

	// neighbours for source term
	int RB, LBRB, LB, RBLB, levRB, levLB;
	RB = XBlock.RightBot[ib];
	levRB = XBlock.level[RB];
	LBRB = XBlock.LeftBot[RB];

	LB = XBlock.LeftBot[ib];
	levLB = XBlock.level[LB];
	RBLB = XBlock.RightBot[LB];



	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;
	T delta = calcres(T(XParam.dx), lev);
	T g = T(XParam.g);

	T ga = T(0.5) * g;

	

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);


	T dhdxi = XGrad.dhdx[i];
	T dhdxmin = XGrad.dhdx[ileft];
	T cm = T(1.0);
	T fmu = T(1.0);

	T dx, zi, zl, zn, zr, zlr, hl, hp, hr, hm;

	T hi = XEv.h[i];

	T hn = XEv.h[ileft];

	if (hi > eps || hn > eps)
	{

		// along X these are same as in Kurgannov
		dx = delta * T(0.5);
		zi = XEv.zs[i] - hi;

		zl = zi - dx * (XGrad.dzsdx[i] - dhdxi);

		zn = XEv.zs[ileft] - hn;

		zr = zn + dx * (XGrad.dzsdx[ileft] - dhdxmin);

		zlr = max(zl, zr);

		hl = hi - dx * dhdxi;
		hp = max(T(0.0), hl + zl - zlr);

		hr = hn + dx * dhdxmin;
		hm = max(T(0.0), hr + zr - zlr);


		//#### Topographic source term
		//In the case of adaptive refinement, care must be taken to ensure
		//	well - balancing at coarse / fine faces(see[notes / balanced.tm]()). * /

		if ((ix == blockDim.y) && levRB < lev)//(ix==16) i.e. in the right halo
		{
			int jj = LBRB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + blockDim.y / 2;
			int iright = memloc(halowidth, blkmemwidth, 0, jj, RB);;
			hi = XEv.h[iright];
			zi = zb[iright];
		}
		if ((ix == 0) && levLB < lev)//(ix==16) i.e. in the right halo
		{
			int jj = RBLB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + blockDim.y / 2;
			int ilc = memloc(halowidth, blkmemwidth, blockDim.y - 1, jj, LB);
			hn = XEv.h[ilc];
			zn = zb[ilc];
		}

		T sl, sr;
		sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
		sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));


		XFlux.Fqux[i] = XFlux.Fqux[i] - fmu * sl;
		XFlux.Su[i] = XFlux.Su[i] - fmu * sr;
	}
}
template __global__ void AddSlopeSourceXGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* zb);
template __global__ void AddSlopeSourceXGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* zb);

template <class T> __host__ void updateKurgXCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T*zb)
{

	
	T delta;
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps)+epsi;

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	int RB, LBRB, LB, RBLB, levRB, levLB;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		int lev = XBlock.level[ib];
		delta = calcres(T(XParam.dx), lev);

		// neighbours for source term
		
		RB = XBlock.RightBot[ib];
		levRB = XBlock.level[RB];
		LBRB = XBlock.LeftBot[RB];

		LB = XBlock.LeftBot[ib];
		levLB = XBlock.level[LB];
		RBLB = XBlock.RightBot[LB];
		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth + XParam.halowidth); ix++)
			{




				// This is based on kurganov and Petrova 2007


				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
				int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);


				T dhdxi = XGrad.dhdx[i];
				T dhdxmin = XGrad.dhdx[ileft];
				T cm = T(1.0);
				T fmu = T(1.0);

				T hi = XEv.h[i];

				T hn = XEv.h[ileft];
				

				if (hi > eps || hn > eps)
				{
					T dx, zi, zl, zn, zr, zlr, hl, up, hp, hr, um, hm,ga;

					// along X
					dx = delta * T(0.5);
					zi = XEv.zs[i] - hi;

					//printf("%f\n", zi);


					//zl = zi - dx*(dzsdx[i] - dhdx[i]);
					zl = zi - dx * (XGrad.dzsdx[i] - dhdxi);
					//printf("%f\n", zl);

					zn = XEv.zs[ileft] - hn;

					//printf("%f\n", zn);
					zr = zn + dx * (XGrad.dzsdx[ileft] - dhdxmin);


					zlr = max(zl, zr);

					//hl = hi - dx*dhdx[i];
					hl = hi - dx * dhdxi;
					up = XEv.u[i] - dx * XGrad.dudx[i];
					hp = max(T(0.0), hl + zl - zlr);

					hr = hn + dx * dhdxmin;
					um = XEv.u[ileft] + dx * XGrad.dudx[ileft];
					hm = max(T(0.0), hr + zr - zlr);

					ga = g * T(0.5);
					///// Reimann solver
					T fh, fu, fv, sl, sr, dt;

					//solver below also modifies fh and fu
					dt = KurgSolver(g, delta, epsi, CFL, cm, fmu, hp, hm, up, um, fh, fu);

					if (dt < dtmax[i])
					{
						dtmax[i] = dt;
					}
					else
					{
						dtmax[i] = T(1.0) / epsi;
					}



					if (fh > T(0.0))
					{
						fv = (XEv.v[ileft] + dx * XGrad.dvdx[ileft]) * fh;// Eq 3.7 third term? (X direction)
					}
					else
					{
						fv = (XEv.v[i] - dx * XGrad.dvdx[i]) * fh;
					}
					//fv = (fh > 0.f ? vv[xminus + iy*nx] + dx*dvdx[xminus + iy*nx] : vv[i] - dx*dvdx[i])*fh;
					//dtmax needs to be stored in an array and reduced at the end
					//dtmax = dtmaxf;
					//dtmaxtmp = min(dtmax, dtmaxtmp);
					/*if (ix == 11 && iy == 0)
					{
						printf("a=%f\t b=%f\t c=%f\t d=%f\n", ap*(qm*um + ga*hm2), -am*(qp*up + ga*hp2),( ap*(qm*um + g*sq(hm) / 2.0f) - am*(qp*up + g*sq(hp) / 2.0f) + ap*am*(qp - qm) ) *ad/100.0f, ad);
					}
					*/
					/*
					#### Topographic source term

					In the case of adaptive refinement, care must be taken to ensure
					well-balancing at coarse/fine faces (see [notes/balanced.tm]()). */

					if ((ix == XParam.blkwidth) && levRB < lev)//(ix==16) i.e. in the right halo
					{
						int jj = LBRB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + XParam.blkwidth / 2;
						int iright = memloc(halowidth, blkmemwidth, 0, jj, RB);;
						hi = XEv.h[iright];
						zi = zb[iright];
					}
					if ((ix == 0) && levLB < lev)//(ix==16) i.e. in the right halo if you 
					{
						int jj = RBLB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + XParam.blkwidth / 2;
						int ilc = memloc(halowidth, blkmemwidth, XParam.blkwidth - 1, jj, LB);
						hn = XEv.h[ilc];
						zn = zb[ilc];
					}

					sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
					sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));

					////Flux update
					XFlux.Fhu[i] = fmu * fh;
					XFlux.Fqux[i] = fmu * (fu - sl);
					XFlux.Su[i] = fmu * (fu - sr);
					XFlux.Fqvx[i] = fmu * fv;
				}
				else
				{
					dtmax[i] = T(1.0) / epsi;
					XFlux.Fhu[i] = T(0.0);
					XFlux.Fqux[i] = T(0.0);
					XFlux.Su[i] = T(0.0);
					XFlux.Fqvx[i] = T(0.0);
				}

			}
		}
	}


}
template __host__ void updateKurgXCPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax, float *zb);
template __host__ void updateKurgXCPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax, double *zb);


template <class T> __host__ void AddSlopeSourceXCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* zb)
{
	T delta;
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		int lev = XBlock.level[ib];
		delta = calcres(XParam.dx, lev);

		// neighbours for source term
		int RB, LBRB, LB, RBLB, levRB, levLB;
		RB = XBlock.RightBot[ib];
		levRB = XBlock.level[RB];
		LBRB = XBlock.LeftBot[RB];

		LB = XBlock.LeftBot[ib];
		levLB = XBlock.level[LB];
		RBLB = XBlock.RightBot[LB];



		T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
		T eps = T(XParam.eps) + epsi;

		T g = T(XParam.g);
		T ga = T(0.5) * g;

		for (int iy = 0; iy < XParam.blkwidth; iy++)
		{
			for (int ix = 0; ix < (XParam.blkwidth + XParam.halowidth); ix++)
			{

				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
				int ileft = memloc(halowidth, blkmemwidth, ix - 1, iy, ib);


				T dhdxi = XGrad.dhdx[i];
				T dhdxmin = XGrad.dhdx[ileft];
				T cm = T(1.0);
				T fmu = T(1.0);

				T dx, zi, zl, zn, zr, zlr, hl, hp, hr, hm;

				T hi = XEv.h[i];

				T hn = XEv.h[ileft];

				if (hi > eps || hn > eps)
				{

					// along X these are same as in Kurgannov
					dx = delta * T(0.5);
					zi = XEv.zs[i] - hi;

					zl = zi - dx * (XGrad.dzsdx[i] - dhdxi);

					zn = XEv.zs[ileft] - hn;

					zr = zn + dx * (XGrad.dzsdx[ileft] - dhdxmin);

					zlr = max(zl, zr);

					hl = hi - dx * dhdxi;
					hp = max(T(0.0), hl + zl - zlr);

					hr = hn + dx * dhdxmin;
					hm = max(T(0.0), hr + zr - zlr);


					//#### Topographic source term
					//In the case of adaptive refinement, care must be taken to ensure
					//	well - balancing at coarse / fine faces(see[notes / balanced.tm]()). * /

					if ((ix == XParam.blkwidth) && levRB < lev)//(ix==16) i.e. in the right halo
					{
						int jj = LBRB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + XParam.blkwidth / 2;
						int iright = memloc(halowidth, blkmemwidth, 0, jj, RB);;
						hi = XEv.h[iright];
						zi = zb[iright];
					}
					if ((ix == 0) && levLB < lev)//(ix==16) i.e. in the right halo if you 
					{
						int jj = RBLB == ib ? floor(iy * (T)0.5) : floor(iy * (T)0.5) + XParam.blkwidth / 2;
						int ilc = memloc(halowidth, blkmemwidth, XParam.blkwidth - 1, jj, LB);
						hn = XEv.h[ilc];
						zn = zb[ilc];
					}

					T sl, sr;
					sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
					sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));


					XFlux.Fqux[i] = XFlux.Fqux[i] - fmu * sl;
					XFlux.Su[i] = XFlux.Su[i] - fmu * sr;
				}
			}
		}
	}
}
template __host__ void AddSlopeSourceXCPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* zb);
template __host__ void AddSlopeSourceXCPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* zb);




template <class T> __global__ void updateKurgYGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax, T* zb)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];

	int TL, BLTL, BL, TLBL, levTL, levBL;
	TL = XBlock.TopLeft[ib];
	levTL = XBlock.level[TL];
	BLTL = XBlock.BotLeft[TL];

	BL = XBlock.BotLeft[ib];
	levBL = XBlock.level[BL];
	TLBL = XBlock.TopLeft[BL];

	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps)+epsi;
	T delta = calcres(T(XParam.dx), lev);
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	

	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ibot = memloc(halowidth, blkmemwidth, ix , iy-1, ib);

	T cm = T(1.0);
	T fmv = T(1.0);
		
	T dhdyi = XGrad.dhdy[i];
	T dhdymin = XGrad.dhdy[ibot];
	T hi = XEv.h[i];
	T hn = XEv.h[ibot];
	T dx, zi, zl, zn, zr, zlr, hl, up, hp, hr, um, hm,ga;



	if (hi > eps || hn > eps)
	{
		hn = XEv.h[ibot];
		dx = delta * T(0.5);
		zi = XEv.zs[i] - hi;
		zl = zi - dx * (XGrad.dzsdy[i] - dhdyi);
		zn = XEv.zs[ibot] - hn;
		zr = zn + dx * (XGrad.dzsdy[ibot] - dhdymin);
		zlr = max(zl, zr);

		hl = hi - dx * dhdyi;
		up = XEv.v[i] - dx * XGrad.dvdy[i];
		hp = max(T(0.0), hl + zl - zlr);

		hr = hn + dx * dhdymin;
		um = XEv.v[ibot] + dx * XGrad.dvdy[ibot];
		hm = max(T(0.0), hr + zr - zlr);


		ga = g * T(0.5);

		//// Reimann solver
		T fh, fu, fv, sl, sr, dt;

		//solver below also modifies fh and fu
		dt = KurgSolver(g, delta, epsi, CFL, cm, fmv, hp, hm, up, um, fh, fu);

		if (dt < dtmax[i])
		{
			dtmax[i] = dt;
		}
		else
		{
			dtmax[i] = T(1.0) / epsi;
		}

		
		if (fh > T(0.0))
		{
			fv = (XEv.u[ibot] + dx * XGrad.dudy[ibot]) * fh;
		}
		else
		{
			fv = (XEv.u[i] - dx * XGrad.dudy[i]) * fh;
		}
		//fv = (fh > 0.f ? uu[ix + yminus*nx] + dx*dudy[ix + yminus*nx] : uu[i] - dx*dudy[i])*fh;
		/**
		#### Topographic source term

		In the case of adaptive refinement, care must be taken to ensure
		well-balancing at coarse/fine faces (see [notes/balanced.tm]()). */
		//sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
		//sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));

		//#### Topographic source term
		//In the case of adaptive refinement, care must be taken to ensure
		//	well - balancing at coarse / fine faces(see[notes / balanced.tm]()). * /

		if ((iy == blockDim.x) && levTL < lev)//(ix==16) i.e. in the right halo
		{
			int jj = BLTL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + blockDim.x / 2;
			int itop = memloc(halowidth, blkmemwidth, jj, 0, TL);;
			hi = XEv.h[itop];
			zi = zb[itop];
		}
		if ((iy == 0) && levBL < lev)//(ix==16) i.e. in the right halo
		{
			int jj = TLBL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + blockDim.x / 2;
			int ibc = memloc(halowidth, blkmemwidth, jj, blockDim.x - 1, BL);
			hn = XEv.h[ibc];
			zn = zb[ibc];
		}

		sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
		sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));

		////Flux update

		XFlux.Fhv[i] = fmv * fh;
		XFlux.Fqvy[i] = fmv * (fu - sl);
		XFlux.Sv[i] = fmv * (fu - sr);
		XFlux.Fquy[i] = fmv * fv;
	}
	else
	{
		dtmax[i] = T(1.0) / epsi;
		XFlux.Fhv[i] = T(0.0);
		XFlux.Fqvy[i] = T(0.0);
		XFlux.Sv[i] = T(0.0);
		XFlux.Fquy[i] = T(0.0);
	}

}
template __global__ void updateKurgYGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax, float* zb);
template __global__ void updateKurgYGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax, double *zb);


template <class T> __global__ void AddSlopeSourceYGPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* zb)
{
	unsigned int halowidth = XParam.halowidth;
	unsigned int blkmemwidth = blockDim.x + halowidth * 2;
	unsigned int blksize = blkmemwidth * blkmemwidth;
	unsigned int ix = threadIdx.x;
	unsigned int iy = threadIdx.y;
	unsigned int ibl = blockIdx.x;
	unsigned int ib = XBlock.active[ibl];

	int lev = XBlock.level[ib];

	// neighbours for source term
	int TL, BLTL, BL, TLBL, levTL, levBL;
	TL = XBlock.TopLeft[ib];
	levTL = XBlock.level[TL];
	BLTL = XBlock.BotLeft[TL];

	BL = XBlock.BotLeft[ib];
	levBL = XBlock.level[BL];
	TLBL = XBlock.TopLeft[BL];



	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;
	T delta = calcres(T(XParam.dx), lev);
	T g = T(XParam.g);
	T ga = T(0.5) * g;


	int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
	int ibot = memloc(halowidth, blkmemwidth, ix, iy - 1, ib);


	
	T cm = T(1.0);
	T fmv = T(1.0);

	T dx, zi, zl, zn, zr, zlr, hl, hp, hr, hm;

	T dhdyi = XGrad.dhdy[i];
	T dhdymin = XGrad.dhdy[ibot];
	T hi = XEv.h[i];
	T hn = XEv.h[ibot];


	if (hi > eps || hn > eps)
	{

		// along X these are same as in Kurgannov
		dx = delta * T(0.5);
		zi = XEv.zs[i] - hi;

		zl = zi - dx * (XGrad.dzsdy[i] - dhdyi);
		zn = XEv.zs[ibot] - hn;
		zr = zn + dx * (XGrad.dzsdy[ibot] - dhdymin);
		zlr = max(zl, zr);

		hl = hi - dx * dhdyi;
		hp = max(T(0.0), hl + zl - zlr);

		hr = hn + dx * dhdymin;
		hm = max(T(0.0), hr + zr - zlr);


		//#### Topographic source term
		//In the case of adaptive refinement, care must be taken to ensure
		//	well - balancing at coarse / fine faces(see[notes / balanced.tm]()). * /

		if ((iy == blockDim.x) && levTL < lev)//(ix==16) i.e. in the right halo
		{
			int jj = BLTL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + blockDim.x / 2;
			int itop = memloc(halowidth, blkmemwidth, jj, 0, TL);;
			hi = XEv.h[itop];
			zi = zb[itop];
		}
		if ((iy == 0) && levBL < lev)//(ix==16) i.e. in the right halo
		{
			int jj = TLBL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + blockDim.x / 2;
			int ibc = memloc(halowidth, blkmemwidth, jj, blockDim.x - 1, BL);
			hn = XEv.h[ibc];
			zn = zb[ibc];
		}

		T sl, sr;
		sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
		sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));


		XFlux.Fqvy[i] = XFlux.Fqvy[i] - fmv * sl;
		XFlux.Sv[i] = XFlux.Sv[i] - fmv * sr;
	}
}
template __global__ void AddSlopeSourceYGPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* zb);
template __global__ void AddSlopeSourceYGPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* zb);




template <class T> __host__ void updateKurgYCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* dtmax,T*zb)
{

	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps)+epsi;
	T delta;
	T g = T(XParam.g);
	T CFL = T(XParam.CFL);
	

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	int TL, BLTL, BL, TLBL, levTL, levBL, lev;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		


		
		TL = XBlock.TopLeft[ib];
		levTL = XBlock.level[TL];
		BLTL = XBlock.BotLeft[TL];

		BL = XBlock.BotLeft[ib];
		levBL = XBlock.level[BL];
		TLBL = XBlock.TopLeft[BL];

		lev = XBlock.level[ib];

		delta = calcres(XParam.dx, lev);

		for (int iy = 0; iy < (XParam.blkwidth + XParam.halowidth); iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{
				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
				int ibot = memloc(halowidth, blkmemwidth, ix, iy - 1, ib);

				T cm = T(1.0);
				T fmv = T(1.0);

				T dhdyi = XGrad.dhdy[i];
				T dhdymin = XGrad.dhdy[ibot];
				T hi = XEv.h[i];
				T hn = XEv.h[ibot];
				T dx, zi, zl, zn, zr, zlr, hl, up, hp, hr, um, hm, ga;


				

				if (hi > eps || hn > eps)
				{
					hn = XEv.h[ibot];
					dx = delta * T(0.5);
					zi = XEv.zs[i] - hi;
					zl = zi - dx * (XGrad.dzsdy[i] - dhdyi);
					zn = XEv.zs[ibot] - hn;
					zr = zn + dx * (XGrad.dzsdy[ibot] - dhdymin);
					zlr = max(zl, zr);

					hl = hi - dx * dhdyi;
					up = XEv.v[i] - dx * XGrad.dvdy[i];
					hp = max(T(0.0), hl + zl - zlr);

					hr = hn + dx * dhdymin;
					um = XEv.v[ibot] + dx * XGrad.dvdy[ibot];
					hm = max(T(0.0), hr + zr - zlr);


					ga = g * T(0.5);

					//// Reimann solver
					T fh, fu, fv, sl, sr, dt;

					//solver below also modifies fh and fu
					dt = KurgSolver(g, delta, epsi, CFL, cm, fmv, hp, hm, up, um, fh, fu);

					if (dt < dtmax[i])
					{
						dtmax[i] = dt;
					}
					else
					{
						dtmax[i] = T(1.0) / epsi;
					}


					if (fh > T(0.0))
					{
						fv = (XEv.u[ibot] + dx * XGrad.dudy[ibot]) * fh;
					}
					else
					{
						fv = (XEv.u[i] - dx * XGrad.dudy[i]) * fh;
					}
					//fv = (fh > 0.f ? uu[ix + yminus*nx] + dx*dudy[ix + yminus*nx] : uu[i] - dx*dudy[i])*fh;
					/**
					#### Topographic source term

					In the case of adaptive refinement, care must be taken to ensure
					well-balancing at coarse/fine faces */

					if ((iy == XParam.blkwidth) && levTL < lev)//(ix==16) i.e. in the top halo
					{
						int jj = BLTL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + XParam.blkwidth / 2;
						int itop = memloc(halowidth, blkmemwidth, jj, 0, TL);
						hi = XEv.h[itop];
						zi = zb[itop];
					}
					if ((iy == 0) && levBL < lev)//(ix==16) i.e. in the bot halo
					{
						int jj = TLBL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + XParam.blkwidth / 2;
						int ibc = memloc(halowidth, blkmemwidth, jj, XParam.blkwidth - 1, BL);
						hn = XEv.h[ibc];
						zn = zb[ibc];
					}

					sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
					sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));

					////Flux update

					XFlux.Fhv[i] = fmv * fh;
					XFlux.Fqvy[i] = fmv *  (fu - sl);
					XFlux.Sv[i] = fmv * (fu - sr);
					XFlux.Fquy[i] = fmv * fv;
				}
				else
				{
					dtmax[i] = T(1.0) / epsi;
					XFlux.Fhv[i] = T(0.0);
					XFlux.Fqvy[i] = T(0.0);
					XFlux.Sv[i] = T(0.0);
					XFlux.Fquy[i] = T(0.0);
				}
			}
		}
	}
}
template __host__ void updateKurgYCPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* dtmax, float *zb);
template __host__ void updateKurgYCPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* dtmax, double *zb);

template <class T> __host__ void AddSlopeSourceYCPU(Param XParam, BlockP<T> XBlock, EvolvingP<T> XEv, GradientsP<T> XGrad, FluxP<T> XFlux, T* zb)
{
	T delta;
	T g = T(XParam.g);
	T ga = T(0.5) * g;
	T epsi = nextafter(T(1.0), T(2.0)) - T(1.0);
	T eps = T(XParam.eps) + epsi;

	int ib;
	int halowidth = XParam.halowidth;
	int blkmemwidth = XParam.blkmemwidth;

	for (int ibl = 0; ibl < XParam.nblk; ibl++)
	{
		ib = XBlock.active[ibl];
		

		int lev = XBlock.level[ib];
		delta = calcres(XParam.dx, lev);
		// neighbours for source term
		int TL, BLTL, BL, TLBL, levTL, levBL;
		TL = XBlock.TopLeft[ib];
		levTL = XBlock.level[TL];
		BLTL = XBlock.BotLeft[TL];

		BL = XBlock.BotLeft[ib];
		levBL = XBlock.level[BL];
		TLBL = XBlock.TopLeft[BL];





		for (int iy = 0; iy < (XParam.blkwidth + XParam.halowidth); iy++)
		{
			for (int ix = 0; ix < XParam.blkwidth; ix++)
			{

				int i = memloc(halowidth, blkmemwidth, ix, iy, ib);
				int ibot = memloc(halowidth, blkmemwidth, ix, iy - 1, ib);



				T cm = T(1.0);
				T fmv = T(1.0);

				T dx, zi, zl, zn, zr, zlr, hl, hp, hr, hm;

				T dhdyi = XGrad.dhdy[i];
				T dhdymin = XGrad.dhdy[ibot];
				T hi = XEv.h[i];
				T hn = XEv.h[ibot];


				if (hi > eps || hn > eps)
				{

					// along X these are same as in Kurgannov
					dx = delta * T(0.5);
					zi = XEv.zs[i] - hi;

					zl = zi - dx * (XGrad.dzsdy[i] - dhdyi);
					zn = XEv.zs[ibot] - hn;
					zr = zn + dx * (XGrad.dzsdy[ibot] - dhdymin);
					zlr = max(zl, zr);

					hl = hi - dx * dhdyi;
					hp = max(T(0.0), hl + zl - zlr);

					hr = hn + dx * dhdymin;
					hm = max(T(0.0), hr + zr - zlr);


					//#### Topographic source term
					//In the case of adaptive refinement, care must be taken to ensure
					//	well - balancing at coarse / fine faces(see[notes / balanced.tm]()). * /

					if ((iy == XParam.blkwidth) && levTL < lev)//(ix==16) i.e. in the right halo
					{
						int jj = BLTL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + XParam.blkwidth / 2;
						int itop = memloc(halowidth, blkmemwidth, jj, 0, TL);;
						hi = XEv.h[itop];
						zi = zb[itop];
					}
					if ((iy == 0) && levBL < lev)//(ix==16) i.e. in the right halo
					{
						int jj = TLBL == ib ? floor(ix * (T)0.5) : floor(ix * (T)0.5) + XParam.blkwidth / 2;
						int ibc = memloc(halowidth, blkmemwidth, jj, XParam.blkwidth - 1, BL);
						hn = XEv.h[ibc];
						zn = zb[ibc];
					}

					T sl, sr;
					sl = ga * (utils::sq(hp) - utils::sq(hl) + (hl + hi) * (zi - zl));
					sr = ga * (utils::sq(hm) - utils::sq(hr) + (hr + hn) * (zn - zr));


					XFlux.Fqvy[i] = XFlux.Fqvy[i] - fmv * sl;
					XFlux.Sv[i] = XFlux.Sv[i] - fmv * sr;
				}
			}
		}
	}
}
template __host__ void AddSlopeSourceYCPU<float>(Param XParam, BlockP<float> XBlock, EvolvingP<float> XEv, GradientsP<float> XGrad, FluxP<float> XFlux, float* zb);
template __host__ void AddSlopeSourceYCPU<double>(Param XParam, BlockP<double> XBlock, EvolvingP<double> XEv, GradientsP<double> XGrad, FluxP<double> XFlux, double* zb);



template <class T> __host__ __device__ T KurgSolver(T g, T delta,T epsi, T CFL, T cm, T fm,  T hp, T hm, T up,T um, T &fh, T &fu)
{
	//// Reimann solver
	T dt;

	//We can now call one of the approximate Riemann solvers to get the fluxes.
	T cp, cmo, ap, am, qm, qp, a, dlt, ad, hm2, hp2, ga, apm;


	cp = sqrt(g * hp);
	cmo = sqrt(g * hm);

	ap = max(max(up + cp, um + cmo), T(0.0));
	//ap = max(ap, 0.0f);

	am = min(min(up - cp, um - cmo), T(0.0));
	//am = min(am, 0.0f);
	ad = T(1.0) / (ap - am);
	//Correct for spurious currents in really shallow depth
	qm = hm * um;
	qp = hp * up;
	//qm = hm*um*(sqrtf(2.0f) / sqrtf(1.0f + max(1.0f, powf(epsc / hm, 4.0f))));
	//qp = hp*up*(sqrtf(2.0f) / sqrtf(1.0f + max(1.0f, powf(epsc / hp, 4.0f))));

	hm2 = hm * hm;
	hp2 = hp * hp;
	a = max(ap, -am);
	ga = g * T(0.5);
	apm = ap * am;
	dlt = delta * cm / fm;

	if (a > epsi)
	{
		fh = (ap * qm - am * qp + apm * (hp - hm)) * ad;// H  in eq. 2.24 or eq 3.7 for F(h)
		fu = (ap * (qm * um + ga * hm2) - am * (qp * up + ga * hp2) + apm * (qp - qm)) * ad;// Eq 3.7 second term (Y direction)
		dt = CFL * dlt / a;
		

	}
	else
	{
		fh = T(0.0);
		fu = T(0.0);
		dt = T(1.0) / epsi;
	}
	return dt;
}
